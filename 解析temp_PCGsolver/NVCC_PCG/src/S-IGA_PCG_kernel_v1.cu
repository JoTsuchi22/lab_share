#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// cuda
#include "hip/hip_runtime.h"
#include ""

// header
#include "S_IGA_header.h"
#include "S_IGA_sub.h"

using namespace std;

// kernel
int RoundUpFunc(int a, int b)
{
	return ((a + b - 1) / b);
}


void Preprocessing_IGA_pararel(information *info)
{
	// GPU側でメモリを割り当てる
	hipMalloc((void**)&dev_a, N * N * sizeof(double));
	hipMalloc((void**)&dev_b, N * N * sizeof(double));
	hipMalloc((void**)&dev_c, N * N * sizeof(double));

    // 配列aと配列bをGPUにコピーする
	hipMemcpy(dev_a, a, N * N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * N * sizeof(double), hipMemcpyHostToDevice);

	int size_of_parallel_process = N * N;
	int block_dim = 128;
	int grid_dim = RoundUpFunc(size_of_parallel_process, block_dim);					// グリッド内のブロック数 (1次元，各ブロックで同じ値，切り上げ)
	pararel <<<grid_dim, block_dim>>>(dev_a, dev_b, dev_c, N, N, N * N);	// カーネル スレッドあたりの処理・演算が小さくなるように書く (できるだけforループ使わない)

	// 配列cをGPUからCPUにコピーする
	hipMemcpy(c, dev_c, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // GPU側で割り当てたメモリを開放する
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

__global__ void pararel()
{

}