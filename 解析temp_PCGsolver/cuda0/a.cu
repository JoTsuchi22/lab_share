#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define PP 10

static int c[10][10];
static double d[PP][PP];

void MM();

int main()
{
    int i, j;

    for (i = 0; i < 10; i++)
    {
        for (j = 0; j < 10; j++)
        {
            c[i][j] = i * 10 + j;
            d[i][j] = i * 10 + j;
        }
    }
    MM();

    for (i = 0; i < 2; i++)
    printf("%le\n", sin(0));

    printf("%d\n", PP);
    printf("%le\n", d[2][2]);
}

void MM()
{
    d[2][1] = 99;
    printf("%d\n", c[2][1]);
}