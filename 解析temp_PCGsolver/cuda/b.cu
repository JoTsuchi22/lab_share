#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <time.h>

// #include <stdlib.h>
// #include <iostream.h>

// #include "host.h"
// #include "device.h"
#include "hip/hip_runtime.h"


#define SKIP_S_IGA 2 // 重ね合わせとJ積分を行う 0, 重ね合わせをスキップしてJ積分を行う 1, J積分を行わない 2

#define ERROR -999
#define PI 3.14159265359

#define MAX_NO_CCpoint_ON_ELEMENT 16					   //分割節点数
#define DIMENSION 2										   //次元数
#define MAX_KIEL_SIZE MAX_NO_CCpoint_ON_ELEMENT *DIMENSION //要素分割マトリックスの大きさ
#define Ng 4											   // Gauss-Legendreの足す回数
#define POW_Ng Ng *Ng									   // NgのDIMENSION乗の計算
#define Ng_extended 10									   // Gauss-Legendreの足す回数
#define POW_Ng_extended Ng_extended *Ng_extended		   // NgのDIMENSION乗の計算
#define D_MATRIX_SIZE 3									   //応力歪マトリックスの大きさ（2次元:3 3次元:6）

#define K_DIVISION_LENGE 10 //全体剛性マトリックスのcol&ptrを制作時に分ける節点数
#define EPS 1.0e-10			//連立1次方程式の残差
#define N_STRAIN 4
#define N_STRESS 4
//各種最大配置可能数
#define MAX_N_KNOT 10
#define MAX_N_ELEMENT 14
// #define MAX_N_ELEMENT 1000
#define MAX_N_NODE 110
#define MAX_N_LOAD 100
#define MAX_N_CONSTRAINT 100
#define MAX_K_WHOLE_SIZE MAX_N_NODE *DIMENSION
#define MAX_NON_ZERO 100
#define MAX_N_PATCH 10
#define MAX_N_Controlpoint_in_Patch 120
#define MAX_N_ORDER 5

#define MAX_N_DISTRIBUTE_FORCE 100
// #define DISTRIBUTE_FORCE_Ng 3

// for s-IGA
#define GAUSS_1DIR Ng_extended												  //重なり判定のための一方向ガウス点数
#define NO_GAUSS_PT GAUSS_1DIR *GAUSS_1DIR									  //重なり判定のためのガウス点総数
#define MAX_N_POINT_OVER GAUSS_1DIR *GAUSS_1DIR								  //要素重なり判定に用いるローカルメッシュ上1要素内の点数
#define MAX_N_MESH 10														  //重合IGAを行うモデルの総数（ローカルメッシュ+1）
#define MAX_N_ELEMENT_OVER 10												  //グローバルメッシュ内の1要素に重なる最大要素数
#define MAX_N_ELEMENT_OVER_POINT 5											  //ローカル要素内の1点に重なるグローバル要素
#define MAX_N_ELEMENT_OVER_ELEMENT MAX_N_ELEMENT_OVER_POINT *MAX_N_POINT_OVER //ローカルメッシュ内の1要素に重なる最大要素数

//重ね合わせの結果

#define DBL_MAX 1.7976931348623158e+308 // max value

#define DIVISION_ELE_XI 10
#define DIVISION_ELE_ETA 10

/////変更後はmake cleanしてからmakeする/////////////////////////////////////////////////////
///////最大値////////////////////////////////////////////////////////////////////////////////
#define MAX_PATCHES MAX_N_PATCH //最大パッチ数
// #define MAX_PATCHES 15								//最大パッチ数
#define MAX_ORDER MAX_N_ORDER //最大次数(p)
// #define MAX_ORDER 3									//最大次数(p)
#define MAX_CNRL_P MAX_N_Controlpoint_in_Patch //最大コントロールポイント数(n)
// #define MAX_CNRL_P 2500								//最大コントロールポイント数(n)
///////各パッチでの最大値/////////////////////////////////////////////////////////////////////
#define MAX_KNOTS (MAX_CNRL_P + MAX_ORDER + 1) //ノットベクトルの最大長さ(n+p+1)
// #define MAX_KNOTS MAX_N_KNOT							//ノットベクトルの最大長さ(n+p+1)
///////各パッチ、各方向での最大値//////////////////////////////////////////////////////////////
// #define MAX_ELEMENTS MAX_N_ELEMENT					//最大要素数
#define MAX_ELEMENTS 10								 //最大要素数
#define MAX_DIVISION 10								 //一要素あたりの最大分割数
#define MAX_POINTS (MAX_ELEMENTS * MAX_DIVISION + 1) //最大点数
/////////////////////////////////////////////////////////////////////////////////////////////

// gauss array
static int GP_1dir;							   // 1方向のガウス点数
static int GP_2D;							   // 2次元のガウス点数
static double Gxi[POW_Ng_extended][DIMENSION]; //ガウス点
static double w[POW_Ng_extended];			   //ガウス点での重み

// static int DIMENSION;
static int KIEL_SIZE; //要素分割マトリックスの大きさ

static int Controlpoint_of_Element[MAX_N_ELEMENT][MAX_NO_CCpoint_ON_ELEMENT];
static double Node_Coordinate[MAX_N_NODE][DIMENSION + 1];
static double Equivalent_Nodal_Force[MAX_N_NODE][DIMENSION]; // Equivalent nodal forces arising from the distributed load
static int K_Whole_Ptr[MAX_K_WHOLE_SIZE + 1], K_Whole_Col[MAX_NON_ZERO];
// static int K_Whole_Ptr[MAX_N_MESH][MAX_K_WHOLE_SIZE + 1], K_Whole_Col[MAX_N_MESH][MAX_NON_ZERO];
static double K_Whole_Val[MAX_NON_ZERO];
// static double K_Whole_Val[MAX_N_MESH][MAX_NON_ZERO];
static int Index_Dof[MAX_K_WHOLE_SIZE];
static int INC[MAX_N_PATCH][MAX_N_ELEMENT][DIMENSION];
static int Adress_Controlpoint[MAX_N_PATCH][1000][1000]; // INCの配列をいじったものAdress_Controlpoint[ξ][η]；コントールポイント番号、任意のパッチ上でξ方向[]番目、η方向[]番目のコントロールポイント番号を示す
static int Order[MAX_N_PATCH][DIMENSION];
static int No_knot[MAX_N_PATCH][DIMENSION];
static int No_Control_point[MAX_N_PATCH][DIMENSION];
static double element_coordinate_Nopoint[MAX_N_ELEMENT][DIMENSION];
static double Gausspoint_coordinates[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION];
// static int same_point[100];
static int same_point_in_Element[MAX_N_NODE];
static int Patch_controlpoint[MAX_N_PATCH][MAX_N_Controlpoint_in_Patch]; //バッチとコントロールポイント番号の要素コネクティビティ
static int Element_patch[MAX_N_ELEMENT];								 //要素がどのパッチに属しているか示す配列(要素番号は1つのモデルで通し番号)
static int No_Controlpoint_in_patch[MAX_N_PATCH];
static int No_Control_point_ON_ELEMENT[10000];

static int Node_To_Node[K_DIVISION_LENGE][10000], Total_Control_Point_To_Node[K_DIVISION_LENGE]; //ある節点に関係する節点番号s
// static int col_N[10][1000];

// static double b_grad_glo[MAX_ELEMENTS][POW_Ng][DIMENSION * DIMENSION][2 * POW_Ng];

static double sol_vec[MAX_K_WHOLE_SIZE];
static double rhs_vec[MAX_K_WHOLE_SIZE];
static double diag_scaling[MAX_K_WHOLE_SIZE];

static double Shape[DIMENSION][MAX_N_NODE][10];
static double shape_func[MAX_N_NODE];
static double dShape_func1[MAX_N_NODE];
static double dShape_func2[MAX_N_NODE];
static double dShape[DIMENSION][MAX_N_NODE];
static double Position_Knots[MAX_N_PATCH][DIMENSION][MAX_N_KNOT];
static double Position_Data_param[DIMENSION];

static double Displacement[MAX_K_WHOLE_SIZE];
// static double Disp_overlay[MAX_K_WHOLE_SIZE];
static double Strain[MAX_N_ELEMENT][POW_Ng_extended][N_STRAIN];
static double Strain_glo[MAX_N_ELEMENT][POW_Ng_extended][N_STRAIN];
static double Strain_overlay[MAX_N_ELEMENT][POW_Ng_extended][N_STRAIN];
static double Strain_aux_mode1[MAX_N_ELEMENT][POW_Ng_extended][N_STRAIN];
static double Strain_aux_mode2[MAX_N_ELEMENT][POW_Ng_extended][N_STRAIN];
static double Stress[MAX_N_ELEMENT][POW_Ng_extended][N_STRESS];
static double Stress_glo[MAX_N_ELEMENT][POW_Ng_extended][N_STRESS];
static double Stress_overlay[MAX_N_ELEMENT][POW_Ng_extended][N_STRESS];
static double Stress_aux_mode1_local[MAX_N_ELEMENT][POW_Ng_extended][N_STRESS];
static double Stress_aux_mode1[MAX_N_ELEMENT][POW_Ng_extended][N_STRESS];
static double Stress_aux_mode2_local[MAX_N_ELEMENT][POW_Ng_extended][N_STRESS];
static double Stress_aux_mode2[MAX_N_ELEMENT][POW_Ng_extended][N_STRESS];
static double StrainEnergyDensity[MAX_N_ELEMENT][POW_Ng_extended];
static double StrainEnergyDensity_overlay[MAX_N_ELEMENT][POW_Ng_extended];
static double StrainEnergyDensity_aux_mode1[MAX_N_ELEMENT][POW_Ng_extended];
static double StrainEnergyDensity_aux_mode2[MAX_N_ELEMENT][POW_Ng_extended];
static double Disp_grad[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION * DIMENSION]; // Disp_grad[MAX_N_ELEMENT][POW_Ng_extended][0] = 𝜕u1/𝜕x1  Disp_grad[MAX_N_ELEMENT][POW_Ng_extended][1] = 𝜕u1/𝜕x2  Disp_grad[MAX_N_ELEMENT][POW_Ng_extended][2] = 𝜕u2/𝜕x1 Disp_grad[MAX_N_ELEMENT][POW_Ng_extended][3] = 𝜕u2/𝜕x2
static double Disp_grad_glo[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION * DIMENSION];
static double Disp_grad_overlay[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION * DIMENSION];
static double Disp_grad_aux_mode1_local[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION * DIMENSION];
static double Disp_grad_aux_mode1[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION * DIMENSION];
static double Disp_grad_aux_mode2_local[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION * DIMENSION];
static double Disp_grad_aux_mode2[MAX_N_ELEMENT][POW_Ng_extended][DIMENSION * DIMENSION];
static double ReactionForce[MAX_K_WHOLE_SIZE];

static double difference[MAX_N_PATCH][MAX_N_KNOT][DIMENSION]; /*隣り合うノットベクトルの差*/
static int ENC[MAX_N_PATCH][MAX_N_ELEMENT][DIMENSION];		  /*ENC[パッチ][全ての要素][0,1]=x,y方向の何番目の要素か*/
static int real_Total_Element;								  /*ゼロエレメントを除いた要素数*/
static int real_element[MAX_N_ELEMENT];						  /*ゼロエレメントではない要素の番号*/
static int Total_element_all_ID[MAX_N_ELEMENT];				  /*ゼロエレメントではない要素＝1、ゼロエレメント＝0*/

static int line_No_Total_element[MAX_N_PATCH][DIMENSION];			 /*ゼロエレメントを含むすべての要素列の数*/
static int line_No_real_element[MAX_N_PATCH][DIMENSION];			 /*ゼロエレメントではない要素列の数*/
static int real_element_line[MAX_N_PATCH][MAX_N_ELEMENT][DIMENSION]; /*ゼロエレメントではない要素列*/

static int No_points_for_colored_points; /*zarusobaで点に色付ける時の全ての点の数*/
/*
static double data_result_shape_x[10000];
static double data_result_shape_y[10000];
static double data_result_disp_x[10000];
static double data_result_disp_y[10000];
*/

static int No_points_for_new_zarusoba; /*zarusobaで点に色付ける時の全ての点の数*/
/*
static double data_result_shape_x_for_new_zarusoba[10000];
static double data_result_shape_y_for_new_zarusoba[10000];
static double data_result_disp_x_for_new_zarusoba[10000];
static double data_result_disp_y_for_new_zarusoba[10000];
*/

// static double Strain_x[MAX_N_ELEMENT][POW_Ng_extended][N_STRAIN];

// for s-IGA
static int Total_mesh;

static int Element_mesh[MAX_N_ELEMENT]; //要素がどのメッシュ内にあるかを示す配列
static int Patch_mesh[MAX_N_PATCH];		//パッチがどのメッシュ内にあるかを示す配列

static int Total_Patch_on_mesh[MAX_N_MESH];		//各メッシュ上のパッチ数
static int Total_Patch_to_mesh[MAX_N_MESH + 1]; //メッシュ[]までのパッチ数（メッシュ[]内のパッチ数は含まない）
static int Total_Patch_to_Now;					//現メッシュまでのパッチ数（現メッシュのパッチ数は含まない）

static int Total_Control_Point_on_mesh[MAX_N_MESH];		//各メッシュ上のコントロールポイント数
static int Total_Control_Point_to_mesh[MAX_N_MESH + 1]; //メッシュ[]までのコントロールポイント数（メッシュ[]内のコントロールポイント数は含まない）
static int Total_Control_Point_to_Now;					//現メッシュまでのコントロールポイント数（現メッシュのコントロールポイント数は含まない）

static int Total_Element_on_mesh[MAX_N_MESH];
static int Total_Element_to_mesh[MAX_N_MESH + 1];
static int Total_Element_to_Now;
// static int Total_Element_to_patch[MAX_N_PATCH];
static int El_No_on_mesh[MAX_N_MESH][MAX_N_ELEMENT]; //メッシュ内でのコントロールポイント配列

static int Total_Constraint_all_mesh;
static int Total_Constraint_on_mesh[MAX_N_MESH];
static int Total_Constraint_to_mesh[MAX_N_MESH + 1];
static int Total_Load_on_mesh[MAX_N_MESH];
static int Total_Load_to_mesh[MAX_N_MESH + 1];
static int Total_DistributeForce_on_mesh[MAX_N_MESH];
static int Total_DistributeForce_to_mesh[MAX_N_MESH + 1];
static int Constraint_ID[MAX_N_NODE * DIMENSION];
static int Constraint_Node_Dir_on_mesh[MAX_N_MESH][MAX_N_CONSTRAINT][2];
static double Value_of_Constraint_on_mesh[MAX_N_MESH][MAX_N_CONSTRAINT];

static double Control_Coord[DIMENSION][MAX_N_NODE];
static double Control_Weight[MAX_N_NODE];

static int real_Total_Element_on_mesh[MAX_N_MESH];
static int real_Total_Element_to_mesh[MAX_N_MESH + 1];
static int real_Total_Element_to_Now;
static int real_El_No_on_mesh[MAX_N_MESH][MAX_N_ELEMENT];

static int temp_element_n[MAX_N_ELEMENT_OVER_POINT];
static int element_n_point[MAX_N_ELEMENT_OVER_ELEMENT];
static int NNLOVER[MAX_N_ELEMENT];
// static int NELOVER[MAX_N_ELEMENT][MAX_N_ELEMENT_OVER_ELEMENT];
static int NELOVER[MAX_N_ELEMENT][MAX_N_ELEMENT_OVER];
static int Check_BDBJ_flag[MAX_N_ELEMENT];
static int Total_BDBJ_flag;
static int Same_BDBJ_flag[POW_Ng_extended];

static int iPatch_array[MAX_N_DISTRIBUTE_FORCE], iCoord_array[MAX_N_DISTRIBUTE_FORCE], type_load_array[MAX_N_DISTRIBUTE_FORCE];
static double val_Coord_array[MAX_N_DISTRIBUTE_FORCE], Range_Coord_array[MAX_N_DISTRIBUTE_FORCE][2], Coeff_Dist_Load_array[MAX_N_DISTRIBUTE_FORCE][3];

// for test
// static int shape_check_frag;

// for Interaction integral
static double T[DIMENSION][DIMENSION];
static double K_mode1;
static double K_mode2;
static double J_integral_value_aux_mode1;
static double J_integral_value_aux_mode2;

//重ね合わせの結果
double E;					   //ヤング率(GPa)
double nu;					   //ポアソン比(-)
int patch_n;				   //パッチ数
int cntl_p_n;				   //コントロールポイント数
int order_xi[MAX_PATCHES];	   //ξ基底関数の次数(p)
int order_eta[MAX_PATCHES];	   //η基底関数の次数(p)
int knot_n_xi[MAX_PATCHES];	   //ξノットベクトルの数(n+p+1)
int knot_n_eta[MAX_PATCHES];   //ηノットベクトルの数(n+p+1)
int cntl_p_n_xi[MAX_PATCHES];  //ξ方向コントロールポイント数(n)
int cntl_p_n_eta[MAX_PATCHES]; //η方向コントロールポイント数(n)

double knot_vec_xi[MAX_PATCHES][MAX_KNOTS];	  //ξノットベクトル
double knot_vec_eta[MAX_PATCHES][MAX_KNOTS];  //ηノットベクトル
double cntl_px[MAX_PATCHES][MAX_CNRL_P];	  //コントロールポイントx座標
double cntl_py[MAX_PATCHES][MAX_CNRL_P];	  //コントロールポイントy座標
double disp_cntl_px[MAX_PATCHES][MAX_CNRL_P]; //コントロールポイント上のx方向変位
double disp_cntl_py[MAX_PATCHES][MAX_CNRL_P]; //コントロールポイント上のy方向変位
double weight[MAX_PATCHES][MAX_CNRL_P];		  //重み

double output_xi_loc[MAX_ELEMENTS][Ng];
double output_eta_loc[MAX_ELEMENTS][Ng];
double coord_x[MAX_POINTS][MAX_POINTS];												 //メッシュx座標
double coord_y[MAX_POINTS][MAX_POINTS];												 //メッシュy座標
double coord_x_gauss[MAX_POINTS][MAX_POINTS];										 //メッシュx座標 for gauss
double coord_y_gauss[MAX_POINTS][MAX_POINTS];										 //メッシュy座標 for gauss
double dxi_x[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		 // ∂x/∂ξ
double dxi_y[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		 // ∂y/∂ξ
double deta_x[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		 // ∂x/∂η
double deta_y[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		 // ∂y/∂η
double dxi_x_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];	 // ∂x/∂ξ for Gauss
double dxi_y_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];	 // ∂y/∂ξ for Gauss
double deta_x_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // ∂x/∂η for Gauss
double deta_y_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // ∂y/∂η for Gauss

double disp_px_glo[MAX_ELEMENTS][MAX_N_Controlpoint_in_Patch];
double disp_py_glo[MAX_ELEMENTS][MAX_N_Controlpoint_in_Patch];
double disp_x[MAX_POINTS][MAX_POINTS];												// x方向変位
double disp_y[MAX_POINTS][MAX_POINTS];												// y方向変位
double disp_x_glo_gauss[MAX_ELEMENTS * Ng][MAX_ELEMENTS * Ng];						// x方向変位 for Gauss
double disp_y_glo_gauss[MAX_ELEMENTS * Ng][MAX_ELEMENTS * Ng];						// y方向変位 for Gauss
double dxi_disp_x[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];	// ∂u/∂ξ
double dxi_disp_y[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];	// ∂v/∂ξ
double deta_disp_x[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // ∂u/∂η
double deta_disp_y[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // ∂v/∂η

double strain_xx[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		// x方向ひずみ
double strain_yy[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		// y方向ひずみ
double strain_xy[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		//剪断ひずみ
double strain_xx_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // x方向ひずみ for Gauss
double strain_yy_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // y方向ひずみ for Gauss
double strain_xy_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; //剪断ひずみ for Gauss

double stress_xx[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		// x方向垂直応力
double stress_yy[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		// y方向垂直応力
double stress_xy[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1];		//剪断応力
double stress_xx_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // x方向垂直応力
double stress_yy_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; // y方向垂直応力
double stress_xy_gauss[MAX_ELEMENTS][MAX_ELEMENTS][MAX_DIVISION + 1][MAX_DIVISION + 1]; //剪断応力

int fields_flag = 1;  // s-IGAのためのNURBS_inputでは変位データは必ず読み込ませる
int division_ele_xi;  //ξ方向の一要素あたりの分割数
int division_ele_eta; //η方向の一要素あたりの分割数
int division_n_xi;	  //ξ方向の表示する点の数
int division_n_eta;	  //η方向の表示する点の数
int element_n_xi;	  //ξ方向要素数
int element_n_eta;	  //η方向要素数

int temp_index[MAX_PATCHES][MAX_CNRL_P];
double temp_cntl_px[MAX_CNRL_P];
double temp_cntl_py[MAX_CNRL_P];
double temp_weight[MAX_CNRL_P];
double temp_disp_x[MAX_CNRL_P];
double temp_disp_y[MAX_CNRL_P];

// for s-IGA
int n_patch_glo;  //グローバルメッシュ上のパッチ数
int n_patch_loc;  //ローカルメッシュ上のパッチ数
int glo_cntl_p_n; //グローバルメッシュ上のコントロールポイント数
int loc_cntl_p_n; //ローカルメッシュ上のコントロールポイント数

// for graph
int graph_patch_n; //グラフ作成用出力ファイル内のパッチ番号

// for GP info
//  static double coordinate_GP[MAX_ELEMENTS*MAX_ELEMENTS][POW_Ng_extended][DIMENSION];
//  static double strain_GP[MAX_ELEMENTS*MAX_ELEMENTS][POW_Ng_extended][3];
//  static double stress_GP[MAX_ELEMENTS*MAX_ELEMENTS][POW_Ng_extended][3];
//  static double stress_r_theta_GP[MAX_ELEMENTS*MAX_ELEMENTS][POW_Ng_extended][3];
//  static double stress_theory_r_theta[MAX_ELEMENTS*MAX_ELEMENTS][POW_Ng_extended][3];
//  static double Jac[MAX_ELEMENTS*MAX_ELEMENTS][POW_Ng_extended];

//解析条件パラメータの設定
static int DM = 1;					 //平面応力状態:DM=0	平面ひずみ状態:DM=1
static int check_over_parameter = 1; //要素の重なりの判定(要素の物体上の端点:0 ガウス点:1)

// void Force_Dis( int Total_DistributeForce, int DistributeForce[MAX_N_DISTRIBUTE_FORCE][3], double Val_DistributeForce[MAX_N_DISTRIBUTE_FORCE],int *Total_Load,int Load_Node_Dir[MAX_N_LOAD][2],double Value_of_Load[MAX_N_LOAD],int Total_Control_Point, int El_No, int *Total_Element );
__host__ void Make_gauss_array(int select_GP);
__host__ int Make_K_EL(int El_No, double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE], double E, double nu, int DM);
__host__ int Make_coupled_K_EL(int El_No_loc, int El_No_glo,
							   double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION],
							   double XG[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION],
							   double K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE],
							   double E, double nu, int DM);
__host__ int Make_Displacement_grad_glo(int El_No_loc, int El_No_glo,
										double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION],
										double XG[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION]);
__host__ void Get_InputData(int tm,
							double *E, double *nu, int *Total_Element, int *Total_Control_Point,
							int *Total_Load, int *No_Patch, int Load_Node_Dir[MAX_N_LOAD][2], double Value_of_Load[MAX_N_LOAD],
							int *Total_Constraint, int Constraint_Node_Dir[MAX_N_CONSTRAINT][2], double Value_of_Constraint[MAX_N_CONSTRAINT],
							int *Total_DistributeForce, char *argv[]);

//全体剛性マトリックス
__host__ int Make_Index_Dof(int Total_Control_Point, int Total_Constraint, int Constraint_Node_Dir[MAX_N_CONSTRAINT][2]);
__host__ void Make_K_Whole_Ptr_Col(int Total_Element, int Total_Control_Point, int K_Whole_Size);
__host__ void Make_K_Whole_Val(double E, double nu, int Total_Element, int DM);
__host__ void Make_Displacement_grad_glo_check(int Total_Element);
// for s-IGA
__host__ void Check_coupled_Glo_Loc_element_for_end(double element_loc[DIMENSION], int mesh_n_over, int mesh_n_org);
__host__ void Check_coupled_Glo_Loc_element_for_Gauss(double element_loc[DIMENSION], int mesh_n_over, int mesh_n_org);
__host__ void Make_Loc_Glo();
// void Make_K_Whole_overlaid(int K_Whole_overlaid_Size);
//連立1次方程式
__host__ void Make_F_Vec(int Total_Load, int Load_Node_Dir[MAX_N_LOAD][2], double Value_of_Load[MAX_N_LOAD], int K_Whole_Size);
__host__ void Make_F_Vec_disp_const(int Mesh_No, int Total_Constraint, int Constraint_Node_Dir[MAX_N_CONSTRAINT][2], double Value_of_Constraint[MAX_N_CONSTRAINT], double E, double nu, int DM);
__host__ void mat_vec_crs(double vec_result[], double vec[], const int ndof);
__host__ double inner_product(int ndof, double vec1[], double vec2[]);
__host__ int check_conv_CG(int ndof, double alphak, double pp[], double eps, int itr);
__host__ void Diag_Scaling_CG_pre(int ndof, int flag_operation);
__host__ void CG_Solver(int ndof, int max_itr, double eps, int flag_ini_val);
// PCG solver
__host__ int RowCol_to_icount(int row, int col);
__host__ void PCG_Solver(int ndof, int max_itr, double eps);
__host__ void Make_M(double *M, int *M_Ptr, int *M_Col, int ndof);
__host__ void M_mat_vec_crs(double *M, int *M_Ptr, int *M_Col, double vec_result[], double vec[], const int ndof);
__host__ int M_check_conv_CG(int ndof, double alphak, double pp[], double eps, double *solution_vec);
__host__ void CG(int ndof, double *solution_vec, double *M, int *M_Ptr, int *M_Col, double *right_vec);
//各種値
__host__ void Make_Strain(int Total_Element);
__host__ void Make_Stress_2D(double E, double nu, int Total_Element, int DM);
__host__ void Make_Stress_2D_glo(double E, double nu, int Total_Element, int DM);
__host__ void Make_StrainEnergyDensity_2D();
__host__ void Make_Displacement_grad(int El_No);
__host__ void Make_StrainEnergyDensity_2D_overlay();
// void Make_Displacement_grad_overlay(double E, double nu, int Total_Element , int El_No, int Total_Control_Point);
__host__ void Make_ReactionForce(int Total_Control_Point);
__host__ void Make_Parameter_z(int Total_Element, double E, double nu, int DM);
__host__ void Make_Parameter_z_overlay(int Total_Element, double E, double nu, int DM);
//分布荷重
__host__ void Force_dis(int Distriction_Force[DIMENSION][3], double Val_Distribute_Force[DIMENSION], double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double Fe[DIMENSION]);
// void Make_Output( int Total_Control_Point, int Total_Element );
// NURBSの計算
__host__ void element_coordinate(int Total_Element);
__host__ void calculate_Controlpoint_using_NURBS(double element[DIMENSION], int Total_Element);
__host__ void calculate_extendmesh_using_NURBS(double element_emsh[DIMENSION]);
__host__ void Gausspoint_coordinate(int Total_Element);
__host__ int Jacobian(int El_No, double a[DIMENSION][DIMENSION], double Local_coord[DIMENSION], double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION]);
__host__ int Make_B_Matrix(int El_No, double B[D_MATRIX_SIZE][MAX_KIEL_SIZE], double Local_coord[DIMENSION], double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double *J);

/*///J積分
int Make_B_x_Matrix_Quad_4(double B_x[DIMENSION][KIEL_SIZE], double Local_coord[DIMENSION], double X[No_Control_point_ON_ELEMENT][DIMENSION], double *J );
void Make_Strain_x_Quad_4(double E, double nu, int Total_Element);
void Make_EMT(double E, double nu, int Total_Element);*/

/* Shape Function */
__host__ double Shape_func(int I_No, double Local_coord[DIMENSION], int El_No);

/* Distributed Load */

__host__ int SerchForElement(int mesh_n, int iPatch, int Total_Element, int iX, int iY);

__host__ void Setting_Dist_Load_2D(int mesh_n, int iPatch, int Total_Element, int iCoord, double val_Coord,
								   double Range_Coord[2], int type_load, double Coeff_Dist_Load[3]);

__host__ void Add_Equivalent_Nodal_Forec_to_F_Vec(int Total_Control_Point);

//重ね合わせの結果
__host__ void GetLocData();
__host__ void ReadFile();
__host__ int CalcXiEtaByNR(double px, double py,
						   double *input_knot_vec_xi, double *input_knot_vec_eta,
						   double *cntl_px, double *cntl_py,
						   double *disp_cntl_px, double *disp_cntl_py,
						   int cntl_p_n_xi, int cntl_p_n_eta,
						   double *weight, int order_xi, int order_eta,
						   double *output_xi, double *output_eta,
						   double *disp_x_glo, double *disp_y_glo,
						   double *strain_xx_glo, double *strain_yy_glo, double *strain_xy_glo);
__host__ static void Calculation(int order_xi, int order_eta,
								 int knot_n_xi, int knot_n_eta,
								 int cntl_p_n_xi, int cntl_p_n_eta,
								 double *input_knot_vec_xi, double *input_knot_vec_eta,
								 double *cntl_px, double *cntl_py,
								 double *disp_cntl_px, double *disp_cntl_py,
								 double *weight);
__host__ static void Calculation_overlay(int order_xi_loc, int order_eta_loc,
										 int knot_n_xi_loc, int knot_n_eta_loc,
										 int cntl_p_n_xi_loc, int cntl_p_n_eta_loc,
										 double *knot_vec_xi_loc, double *knot_vec_eta_loc,
										 double *cntl_px_loc, double *cntl_py_loc,
										 double *weight_loc,
										 int order_xi_glo, int order_eta_glo,
										 int cntl_p_n_xi_glo, int cntl_p_n_eta_glo,
										 double *knot_vec_xi_glo, double *knot_vec_eta_glo,
										 double *cntl_px_glo, double *cntl_py_glo,
										 double *disp_cntl_px_glo, double *disp_cntl_py_glo,
										 double *weight_glo);
// void Calculation_overlay_at_GP(double E, double nu,
// 							   int order_xi_glo, int order_eta_glo,
// 							   int knot_n_xi_glo, int knot_n_eta_glo,
// 							   int cntl_p_n_xi_glo, int cntl_p_n_eta_glo,
// 							   double *knot_vec_xi_glo, double *knot_vec_eta_glo,
// 							   double *cntl_px_glo, double *cntl_py_glo,
// 							   double *disp_cntl_px_glo, double *disp_cntl_py_glo,
// 							   double *weight_glo);
// static void Calculation_at_GP(double E, double nu);
__host__ void K_output_svg(int ndof);

//要素合成マトリックス
__host__ int Make_B_Matrix(int El_No, double B[D_MATRIX_SIZE][MAX_KIEL_SIZE], double Local_coord[DIMENSION], double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double *J);
__host__ int Make_b_grad_Matrix(int El_No, double b_grad[DIMENSION * DIMENSION][2 * MAX_NO_CCpoint_ON_ELEMENT], double Local_coord[DIMENSION], double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double *J);
__host__ int Make_D_Matrix_2D(double D[D_MATRIX_SIZE][D_MATRIX_SIZE], double E, double nu, int DM);

// Interaction integral
__host__ void Make_auxiliary_mode1(int e, double E, double nu, int DM, double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double crack_front_coordinates_x, double crack_front_coordinates_y);
__host__ void Make_auxiliary_mode2(int e, double E, double nu, int DM, double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double crack_front_coordinates_x, double crack_front_coordinates_y);

FILE *fp;

int main(int argc, char *argv[])
{
	clock_t start, end, t1;

	int i, j, k;
	// int l;
	int re;
	// int p;
	// int q, r;
	int Total_Element;
	int Total_Control_Point;
	int No_Patch = 0;
	// int Total_net = 0;
	static int Total_Load = 0, Load_Node_Dir[MAX_N_LOAD][2];
	static double Value_of_Load[MAX_N_LOAD];
	static int Total_Constraint = 0, Constraint_Node_Dir[MAX_N_CONSTRAINT][2];
	static double Value_of_Constraint[MAX_N_CONSTRAINT];
	static int Total_DistributeForce = 0;
	int K_Whole_Size = 0;
	int El_No = 0;
	// static double element[DIMENSION];
	// static double element_emsh[DIMENSION];
	static double element_loc[DIMENSION];
	static double E, nu;
	static int max_itr;

	// for s-IGA
	int tm;
	// int K_Whole_overlaid_Size = 0;

	// 複数ファイル読み込みのためコメントアウト
	// for s-IGA
	//引数の個数確認
	if (argc <= 1)
	{
		printf("Argument is missing\n");
	}
	if (argc == 2) /*通常IGA：input file 1つ*/
	{
		printf("IGA carried out.(No local mesh)\n");
	}
	if (argc >= 3) /*s-IGA：input file 複数*/
	{
		printf("s-IGA carried out.(%d local meshes)\n", argc - 2);
	}
	Total_mesh = argc - 1;

	start = clock();
	//ローカルメッシュのためにinput file読み込みのループ
	for (tm = 0; tm < Total_mesh; tm++)
	{
		Get_InputData(tm, &E, &nu, &Total_Element, &Total_Control_Point, &Total_Load, &No_Patch, Load_Node_Dir, Value_of_Load, &Total_Constraint, Constraint_Node_Dir, Value_of_Constraint, &Total_DistributeForce, argv);
		// printf("Finish Get_InputData\n");
		if (tm == 0 && Total_mesh > 1)
		{
			printf("Finish Get_InputData(Global mesh:%s)\n", argv[1]);
		}
		if (tm > 0)
		{
			printf("Finish Get_inputData(Local mesh No.[%d]:%s)\n", tm, argv[tm + 1]);
		}
		printf("Total Element=%d Node=%d Constraint=%d Load=%d\n", Total_Element, Total_Control_Point, Total_Constraint, Total_Load);
		printf("E;%le nu;%le\n", E, nu);
		// printf("Finish Get_InputData\n");
	} //ローカルメッシュのためにinput fileの読み込みのループ終了
	//グローバルパッチとローカルパッチの重なり
	//ローカルメッシュ同士は被りなしと仮定

	return 0;
}

//ファイルからデータをもらう
__host__ void Get_InputData(int tm,
							double *E, double *nu, int *Total_Element, int *Total_Control_Point,
							int *Total_Load, int *No_Patch, int Load_Node_Dir[MAX_N_LOAD][2], double Value_of_Load[MAX_N_LOAD],
							int *Total_Constraint, int Constraint_Node_Dir[MAX_N_CONSTRAINT][2], double Value_of_Constraint[MAX_N_CONSTRAINT],
							int *Total_DistributeForce, char *argv[])
{
	int i, j, k, l, iii;
	int n, p, q, h, x, y;
	char s[256];
	int ii, jj, kk, kkk;
	int e, b, B;
	int iiloc, jjloc, kkloc;
	int r = 0;
	/* for the distributed loads*/

	if ((fp = fopen(argv[tm + 1], "r")) == NULL)
		printf("file open error!!\n");
	//次元数
	/*fscanf(fp,"%d",&DIMENSION );
	printf("DIMENSION=%d\n",DIMENSION);
	fgets(s, 256, fp);*/
	//材料定数
	fscanf(fp, "%le %le", &*E, &*nu);
	fgets(s, 256, fp);
	printf("E:%le nu:%le\n", *E, *nu);
	// E[tm]=*E;
	// nu[tm]=*nu;

	//パッチ数
	fscanf(fp, "%d", &*No_Patch);
	fgets(s, 256, fp);
	printf("No_Patch:%d\n", *No_Patch);
	Total_Patch_on_mesh[tm] = *No_Patch;
	Total_Patch_to_mesh[tm + 1] = Total_Patch_to_mesh[tm] + *No_Patch;
	// Total_Patch_to_mesh[tm+1] += *No_Patch;
	printf("Total_Patch_to_mesh[%d]=%d\n",
		   tm, Total_Patch_to_mesh[tm]);
	Total_Patch_to_Now = Total_Patch_to_mesh[tm];
	printf("Total_Patch_to_Now=%d\n", Total_Patch_to_Now);

	//コントロールポイント数
	fscanf(fp, "%d", &*Total_Control_Point);
	fgets(s, 256, fp);
	printf("Total_Control_Point:%d\n", *Total_Control_Point);
	Total_Control_Point_on_mesh[tm] = *Total_Control_Point;
	Total_Control_Point_to_mesh[tm + 1] = Total_Control_Point_to_mesh[tm] + *Total_Control_Point;
	// Total_Control_Point_to_mesh[tm+1] += *Total_Control_Point;
	Total_Control_Point_to_Now = Total_Control_Point_to_mesh[tm];
	printf("Total_Control_Point_to_mesh[%d]=%d\n",
		   tm, Total_Control_Point_to_mesh[tm]);
	printf("Total_Control_Point_to_mesh[%d]=%d\n",
		   tm + 1, Total_Control_Point_to_mesh[tm + 1]);
	printf("Total_Control_Point_to_Now=%d\n", Total_Control_Point_to_Now);

	//ξη方向の各次数
	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &Order[l + Total_Patch_to_mesh[tm]][j]);
			printf("Order[%d][%d]=%d\n", l + Total_Patch_to_mesh[tm], j, Order[l + Total_Patch_to_mesh[tm]][j]);
		}
	}

	fgets(s, 256, fp);
	//ノット数
	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &No_knot[l + Total_Patch_to_mesh[tm]][j]);
			printf("No_knot[%d][%d]=%d\n", l + Total_Patch_to_mesh[tm], j, No_knot[l + Total_Patch_to_mesh[tm]][j]);
		}
	}

	fgets(s, 256, fp);
	//各パッチ各方向のコントロールポイント数
	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &No_Control_point[l + Total_Patch_to_mesh[tm]][j]);
			printf("No_Control_point[%d][%d]:%d\n", l + Total_Patch_to_mesh[tm], j, No_Control_point[l + Total_Patch_to_mesh[tm]][j]);
		}
	}

	fgets(s, 256, fp);

	for (l = 0; l < *No_Patch; l++)
	{
		No_Controlpoint_in_patch[l + Total_Patch_to_mesh[tm]] = 1.0;
	}

	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			No_Controlpoint_in_patch[l + Total_Patch_to_mesh[tm]] *= No_Control_point[l + Total_Patch_to_mesh[tm]][j];
		}
	}

	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			if (No_knot[l + Total_Patch_to_mesh[tm]][j] != No_Control_point[l + Total_Patch_to_mesh[tm]][j] + Order[l + Total_Patch_to_mesh[tm]][j] + 1)
			{
				printf("wrong relationship between the number of knot vector and the number of control_point \n");
				printf("in mesh_No.%d in patch_No.%d direction:%d\n", tm, l, j);
			}
		}
	}

	for (l = 0; l < *No_Patch; l++)
	{
		printf("No_Controlpoint_in_patch[%d]:%d\t", l + Total_Patch_to_mesh[tm], No_Controlpoint_in_patch[l + Total_Patch_to_mesh[tm]]);
	}
	printf("\n");

	for (l = 0; l < *No_Patch; l++)
	{
		// printf("l;%d\n",l);
		for (i = 0; i < No_Controlpoint_in_patch[l + Total_Patch_to_mesh[tm]]; i++)
		{
			// printf("i:%d\n",i );
			fscanf(fp, "%d", &Patch_controlpoint[l + Total_Patch_to_mesh[tm]][i]);
			if (tm > 0)
			{
				Patch_controlpoint[l + Total_Patch_to_mesh[tm]][i] += Total_Control_Point_to_mesh[tm];
			}
			// printf("Patch_controlpoint[%d][%d]=%d\n",
			//		l+Total_Patch_to_mesh[tm], i,
			//		Patch_controlpoint[l+Total_Patch_to_mesh[tm]][i]);
		}
	}

	/*for (l = 0; l < *No_Patch; l++) {
			//printf("l:%d\n",l);
			for ( i = 0; i < No_Controlpoint_in_patch[l]; i++) {
				//printf("i:%d\n", i);
				printf("Patch_controlpoint[%d][%d]:%d\n",l,i,Patch_controlpoint[l][i]);
			}printf("\n");
		}*/

	fscanf(fp, "%d %d %d", Total_Constraint, Total_Load, Total_DistributeForce);
	Total_Constraint_on_mesh[tm] = *Total_Constraint;
	Total_Constraint_to_mesh[tm + 1] = Total_Constraint_to_mesh[tm] + *Total_Constraint;
	if (tm == Total_mesh - 1)
	{
		for (i = 0; i < Total_mesh; i++)
		{
			Total_Constraint_all_mesh += Total_Constraint_on_mesh[i];
		}
		printf("Total_Constraint_all_mesh;%d\n", Total_Constraint_all_mesh);
	}
	printf("Total_Constraint;%d\n", *Total_Constraint);
	printf("Total_Constraint_on_mesh[%d];%d\n", tm, Total_Constraint_on_mesh[tm]);
	printf("Total_Constraint_to_mesh[%d];%d\n", tm, Total_Constraint_to_mesh[tm]);
	Total_Load_on_mesh[tm] = *Total_Load;
	Total_Load_to_mesh[tm + 1] = Total_Load_to_mesh[tm] + *Total_Load;
	printf("Total_Load;%d\n", *Total_Load);
	printf("Total_DistributedForce;%d\n", *Total_DistributeForce);
	Total_DistributeForce_on_mesh[tm] = *Total_DistributeForce;
	Total_DistributeForce_to_mesh[tm + 1] = Total_DistributeForce_to_mesh[tm] + *Total_DistributeForce;
	fgets(s, 256, fp);

	//ノットベクトルの読み込み
	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			for (k = 0; k < No_knot[l + Total_Patch_to_mesh[tm]][j]; k++)
			{
				fscanf(fp, "%le", &Position_Knots[l + Total_Patch_to_mesh[tm]][j][k]);
				printf("%le\t", Position_Knots[l + Total_Patch_to_mesh[tm]][j][k]);
				// printf("Position_Knots[%d][%d][%d]=%le\n",
				//         l+Total_Patch_to_mesh[tm], j, k,
				//         Position_Knots[l+Total_Patch_to_mesh[tm]][j][k]);
			}
			printf("\n");
		}
	}

	for (l = 0; l < *No_Patch; l++)
	{
		No_Control_point_ON_ELEMENT[l + Total_Patch_to_mesh[tm]] = 1.0;
	}

	*Total_Element = 0.0;

	for (l = 0; l < *No_Patch; l++)
	{
		if (DIMENSION == 2)
		{
			*Total_Element += (No_Control_point[l + Total_Patch_to_mesh[tm]][0] - Order[l + Total_Patch_to_mesh[tm]][0]) * (No_Control_point[l + Total_Patch_to_mesh[tm]][1] - Order[l + Total_Patch_to_mesh[tm]][1]);
			No_Control_point_ON_ELEMENT[l + Total_Patch_to_mesh[tm]] = (Order[l + Total_Patch_to_mesh[tm]][0] + 1) * (Order[l + Total_Patch_to_mesh[tm]][1] + 1);
		}
		else
		{
			*Total_Element += (No_Control_point[l + Total_Patch_to_mesh[tm]][0] - Order[l + Total_Patch_to_mesh[tm]][0]) * (No_Control_point[l + Total_Patch_to_mesh[tm]][1] - Order[l + Total_Patch_to_mesh[tm]][1]) * (No_Control_point[l + Total_Patch_to_mesh[tm]][2] - Order[l + Total_Patch_to_mesh[tm]][2]);
			No_Control_point_ON_ELEMENT[l + Total_Patch_to_mesh[tm]] = (Order[l + Total_Patch_to_mesh[tm]][0] + 1) * (Order[l + Total_Patch_to_mesh[tm]][1] + 1) * (Order[l + Total_Patch_to_mesh[tm]][2] + 1);
		}
	}
	printf("Total_Element=%d\n", *Total_Element);
	Total_Element_on_mesh[tm] = *Total_Element;
	Total_Element_to_mesh[tm + 1] = Total_Element_to_mesh[tm] + *Total_Element;
	// Total_Element_to_mesh[tm] += *Total_Element;
	Total_Element_to_Now = Total_Element_to_mesh[tm];
	printf("Total_Element_on_mesh[%d]=%d\n",
		   tm, Total_Element_on_mesh[tm]);
	printf("Total_Element_to_Now=%d\n",
		   Total_Element_to_Now);

	for (l = 0; l < *No_Patch; l++)
	{
		printf("No_Control_point_ON_ELEMENT[%d]=%d\n",
			   l + Total_Patch_to_mesh[tm],
			   No_Control_point_ON_ELEMENT[l + Total_Patch_to_mesh[tm]]);
	}

	//節点座標
	for (i = 0; i < *Total_Control_Point; i++)
	{
		fscanf(fp, "%d", &ii);
		for (j = 0; j < DIMENSION + 1; j++)
			fscanf(fp, "%le", &Node_Coordinate[ii + Total_Control_Point_to_mesh[tm]][j]); // Node_Coordinate[i][2]:重み
	}
	for (i = 0; i < *Total_Control_Point; i++)
	{
		for (j = 0; j < DIMENSION + 1; j++)
		{
			//コントロールポイント座標・重みの新たな配列（for s-IGA/NewtonLaphson）
			if (j < DIMENSION)
			{
				Control_Coord[j][i + Total_Control_Point_to_mesh[tm]] = Node_Coordinate[i + Total_Control_Point_to_mesh[tm]][j];
			}
			else if (j == DIMENSION)
			{
				Control_Weight[i + Total_Control_Point_to_mesh[tm]] = Node_Coordinate[i + Total_Control_Point_to_mesh[tm]][DIMENSION];
			}
			printf("Node_Coordinate[%d][%d]=%e\n", i + Total_Control_Point_to_mesh[tm], j, Node_Coordinate[i + Total_Control_Point_to_mesh[tm]][j]);
		}
	}
	fgets(s, 256, fp);

	//拘束
	for (i = 0; i < *Total_Constraint; i++)
		fscanf(fp, "%d %d %le",
			   &Constraint_Node_Dir[i + Total_Constraint_to_mesh[tm]][0],
			   &Constraint_Node_Dir[i + Total_Constraint_to_mesh[tm]][1],
			   &Value_of_Constraint[i + Total_Constraint_to_mesh[tm]]);
	for (i = 0; i < *Total_Constraint; i++)
	{
		Constraint_Node_Dir[i + Total_Constraint_to_mesh[tm]][0] = Constraint_Node_Dir[i + Total_Constraint_to_mesh[tm]][0] + Total_Control_Point_to_mesh[tm];
		Constraint_Node_Dir_on_mesh[tm][i][0] = Constraint_Node_Dir[i][0];
		Constraint_Node_Dir_on_mesh[tm][i][1] = Constraint_Node_Dir[i][1];
		Value_of_Constraint_on_mesh[tm][i] = Value_of_Constraint[i];
		printf("Constraint_Node_Dir[%d][0]= %d Constraint_Node_Dir[%d][1]=%d Value_of_Constraint[%d]= %e \n",
			   i + Total_Constraint_to_mesh[tm],
			   Constraint_Node_Dir[i + Total_Constraint_to_mesh[tm]][0],
			   i + Total_Constraint_to_mesh[tm],
			   Constraint_Node_Dir[i + Total_Constraint_to_mesh[tm]][1],
			   i + Total_Constraint_to_mesh[tm],
			   Value_of_Constraint[i + Total_Constraint_to_mesh[tm]]);
	}
	fgets(s, 256, fp);

	//荷重
	for (i = 0; i < *Total_Load; i++)
	{
		fscanf(fp, "%d %d %le",
			   &Load_Node_Dir[i + Total_Load_to_mesh[tm]][0],
			   &Load_Node_Dir[i + Total_Load_to_mesh[tm]][1],
			   &Value_of_Load[i + Total_Load_to_mesh[tm]]);
		Load_Node_Dir[i + Total_Load_to_mesh[tm]][0] = Load_Node_Dir[i + Total_Load_to_mesh[tm]][0] + Total_Control_Point_to_mesh[tm];

		printf("Load_Node_Dir[%d][0]= %d Load_Node_Dir[%d][1]= %d Value_of_Load[%d]= %e\n",
			   i + Total_Load_to_mesh[tm],
			   Load_Node_Dir[i + Total_Load_to_mesh[tm]][0],
			   i + Total_Load_to_mesh[tm],
			   Load_Node_Dir[i + Total_Load_to_mesh[tm]][1],
			   i + Total_Load_to_mesh[tm],
			   Value_of_Load[i + Total_Load_to_mesh[tm]]);
	}

	int iPatch, iCoord, type_load;
	double Range_Coord[2], val_Coord, Coeff_Dist_Load[3];
	// int iPatch_array[MAX_N_DISTRIBUTE_FORCE], iCoord_array[MAX_N_DISTRIBUTE_FORCE], type_load_array[MAX_N_DISTRIBUTE_FORCE];
	// double val_Coord_array[MAX_N_DISTRIBUTE_FORCE], Range_Coord_array[MAX_N_DISTRIBUTE_FORCE][2], Coeff_Dist_Load_array[MAX_N_DISTRIBUTE_FORCE][3];

	fgets(s, 256, fp);
	for (i = 0; i < *Total_DistributeForce; i++)
	{
		fscanf(fp, "%d %d %d %lf %lf %lf %lf %lf %lf", &type_load, &iPatch, &iCoord, &val_Coord, &Range_Coord[0], &Range_Coord[1], &Coeff_Dist_Load[0], &Coeff_Dist_Load[1], &Coeff_Dist_Load[2]);
		printf("Distibuted load nober: %d\n", i);
		printf("type_load: %d  iPatch: %d iCoord: %d  val_Coord: %.15e  Range_Coord: %.15e  %.15e\n Coef_Dist_Load: %.15e %.15e %.15e\n",
			   type_load, iPatch, iCoord,
			   val_Coord, Range_Coord[0], Range_Coord[1], Coeff_Dist_Load[0], Coeff_Dist_Load[1], Coeff_Dist_Load[2]);
		/*
		type_load: Direction of distributed load: 0-x direction, 1-y direction, 2-normal to the segemet/surface
		iPatch: Patch number to which the distributed load is assigned., 0, 1, ...
		iCoord: 0: Distributed load is applied to line along Xi axis.
						1: Distributed load is applied to line along Eta axis
		val_Coord: その時のもう片方の座標
		Range_Coord[0]: Local coordinate value at which the distributed load starts.
		Range_Coord[1]: Local coordinate value at which the distributed load ends.
		Coeff_Dist_Load[0], &Coeff_Dist_Load[1], &Coeff_Dist_Load[2]: The coefficients of distributed load value:
			Coeff_Dist_Load[0]*Xi + Coeff_Dist_Load[1]*Xi + Coeff_Dist_Load[2]*Xi^2
		or
			Coeff_Dist_Load[0]*Xi + Coeff_Dist_Load[1]*Eta + Coeff_Dist_Load[2]*Eta^2
		*/
		// for s-IGA
		type_load_array[i + Total_DistributeForce_to_mesh[tm]] = type_load;
		iPatch_array[i + Total_DistributeForce_to_mesh[tm]] = iPatch;
		iCoord_array[i + Total_DistributeForce_to_mesh[tm]] = iCoord;
		val_Coord_array[i + Total_DistributeForce_to_mesh[tm]] = val_Coord;
		Range_Coord_array[i + Total_DistributeForce_to_mesh[tm]][0] = Range_Coord[0];
		Range_Coord_array[i + Total_DistributeForce_to_mesh[tm]][1] = Range_Coord[1];
		Coeff_Dist_Load_array[i + Total_DistributeForce_to_mesh[tm]][0] = Coeff_Dist_Load[0];
		Coeff_Dist_Load_array[i + Total_DistributeForce_to_mesh[tm]][1] = Coeff_Dist_Load[1];
		Coeff_Dist_Load_array[i + Total_DistributeForce_to_mesh[tm]][2] = Coeff_Dist_Load[2];
	}

	/* Setting_Dist_Load_2D(&*Total_Control_Point,&iPatch, &*Total_Element, iCoord, val_Coord,
			 Range_Coord[2],  norm_dir, type_load, Coeff_Dist_Load); */
	//自由度共有の計算
	//(同じ座標を計算して要素コネクティビィティのコントロールポイント番号を入れ替える)
	/*(2018_01_31)for (ii = 0; ii < *Total_Control_Point; ii++) {
			same_point[ii]=ii;
		}

		for (ii = 0 ; ii < *Total_Control_Point; ii++) {
			for ( jj = ii-1; jj >= 0 ; jj--) {
				if (Node_Coordinate[ii][0]== Node_Coordinate[jj][0] && Node_Coordinate[ii][1]==Node_Coordinate[jj][1]) {
					printf("同じ座標の番号ii:%d jj:%d\n",ii,jj);
					same_point[ii]=jj;
					//printf("same_point_1[%d]:%d\n",ii,same_point[ii]);
				}
			}
		}*/

	/*	for (ii = 0; ii < *Total_Control_Point; ii++) {
			printf("same_point[%d]:%d\n",ii,same_point[ii]);
		}*/
	// INC\の計算（節点番号をξ、ηの番号で表す為の配列）
	if (DIMENSION == 2) // for s-IGA
	{
		e = 0;
		for (l = 0; l < *No_Patch; l++)
		{
			i = 0;
			for (jj = 0; jj < No_Control_point[l + Total_Patch_to_Now][1]; jj++)
			{
				for (ii = 0; ii < No_Control_point[l + Total_Patch_to_Now][0]; ii++)
				{

					INC[l + Total_Patch_to_Now][Patch_controlpoint[l + Total_Patch_to_Now][i]][0] = ii;
					INC[l + Total_Patch_to_Now][Patch_controlpoint[l + Total_Patch_to_Now][i]][1] = jj;
					// printf("INC[%d][%d][0]=%d INC[%d][%d][1]=%d\n"
					//         ,l+Total_Patch_to_Now, Patch_controlpoint[l+Total_Patch_to_Now][i]
					//         ,INC[l+Total_Patch_to_Now][Patch_controlpoint[l+Total_Patch_to_Now][i]][0]
					//         ,l+Total_Patch_to_Now, Patch_controlpoint[l+Total_Patch_to_Now][i]
					//         ,INC[l+Total_Patch_to_Now][Patch_controlpoint[l+Total_Patch_to_Now][i]][1]);
					// printf("INC[%d][0]=ii;%d INC[%d][1]=jj;%d\n",i, ii, i,jj);
					// printf("INC[%d][0]=%d INC[%d][1]=%d\n",i, INC[i][0], i,INC[i][1] );
					Adress_Controlpoint[l + Total_Patch_to_Now][ii][jj] = Patch_controlpoint[l + Total_Patch_to_Now][i];
					// printf("Adress_con[%d][%d][%d]=%d\n",
					//         l+Total_Patch_to_Now,ii,jj,Adress_Controlpoint[l+Total_Patch_to_Now][ii][jj]);

					// if (ii >= Order[l][0] && jj >= Order[l+Total_Patch_to_Now][1])
					if (ii >= Order[l + Total_Patch_to_Now][0] && jj >= Order[l + Total_Patch_to_Now][1])
					{

						for (jjloc = 0; jjloc <= Order[l + Total_Patch_to_Now][1]; jjloc++)
						{
							for (iiloc = 0; iiloc <= Order[l + Total_Patch_to_Now][0]; iiloc++)
							{
								// printf("jjloc:%d iiloc:%d\n",jjloc,iiloc);
								B = Patch_controlpoint[l + Total_Patch_to_Now][i - jjloc * No_Control_point[l + Total_Patch_to_Now][0] - iiloc];
								b = jjloc * (Order[l + Total_Patch_to_Now][0] + 1) + iiloc;
								// printf("B=%d b=%d e=%d\n",B,b,e);
								// Controlpoint_of_Element[e][b] = B;
								Controlpoint_of_Element[e + Total_Element_to_Now][b] //= B;
									= B;
								//	= B + Total_Control_Point_to_Now;
								// printf("Controlpoint_of_Element[%d][%d]=%d\n",
								//        e+Total_Element_to_Now,b,
								// 		Controlpoint_of_Element[e+Total_Element_to_Now][b]);
							}
						}
						// Element_patch[e] = l+Total_Patch_to_Now;
						Element_patch[e + Total_Element_to_Now] = l + Total_Patch_to_Now;
						// printf("Element_patch[%d]=%d\n",
						//		e+Total_Element_to_Now,
						//		Element_patch[e+Total_Element_to_Now]);

						Element_mesh[e + Total_Element_to_Now] = tm;
						// printf("Element_mesh[%d]=%d\n",
						//         e+Total_Element_to_Now,
						//         Element_mesh[e+Total_Element_to_Now]);

						El_No_on_mesh[tm][e] = e + Total_Element_to_Now;
						// printf("El_No_on_mesh[%d][%d]=%d\n",
						//         tm,e,El_No_on_mesh[tm][e]);
						e++;
					}
					i++;
				}
			}
			Patch_mesh[l + Total_Patch_to_Now] = tm;
			// printf("Patch_mesh[%d]=%d\n",
			//         l+Total_Patch_to_Now,
			//         Patch_mesh[l+Total_Patch_to_Now]);
		}
		/*for ( i = 0; i < *Total_Control_Point; i++){
				for ( l = 0; l < *No_Patch; l++) {
					printf("INC[%d][%d][0]=%d INC[%d][%d][1]=%d\n",l,i, INC[l][i][0], l,i,INC[l][i][1] );
				}
		}*/

		/*for ( l = 0; l < *No_Patch; l++) {
			for (i = 0; i < *Total_Element; i++) {
					for (b = 0; b < No_Control_point_ON_ELEMENT[l]; b++) {
					printf("Controlpoint_of_Element[%d][%d]=%d\n",i,b,Controlpoint_of_Element[i][b]);
				}
			}
		}*/

		/*for ( i = 0; i < *Total_Element; i++) {
			printf("Element_patch[%d]:%d\n",i,Element_patch[i]);
		}*/
		/*for (i = 0; i < *Total_Element; i++) {
				for (b = 0; b < No_Control_point_ON_ELEMENT; b++) {
					printf("Controlpoint_of_Element_before[%d][%d]=%d\n",i,b,Controlpoint_of_Element[i][b]);
					Controlpoint_of_Element[i][b]=same_point[Controlpoint_of_Element[i][b]];
					printf("Controlpoint_of_Element_after[%d][%d]=%d\n",i,b,Controlpoint_of_Element[i][b]);
				}
			}*/
	}

	if (DIMENSION == 3)
	{
		e = 0;
		for (l = 0; l < *No_Patch; l++)
		{
			i = 0;
			for (kk = 0; kk < No_Control_point[l][2]; kk++)
			{
				for (jj = 0; jj < No_Control_point[l][1]; jj++)
				{
					for (ii = 0; ii < No_Control_point[l][0]; ii++)
					{

						// printf("kk=%d\n",kk );
						INC[l][Patch_controlpoint[l][i]][0] = ii;
						INC[l][Patch_controlpoint[l][i]][1] = jj;
						INC[l][Patch_controlpoint[l][i]][2] = kk;
						// printf("INC[%d][0]=%d INC[%d][1]=%d\n",i, INC[i][0], i,INC[i][1] );
						if (ii >= Order[l][0] && jj >= Order[l][1] && kk >= Order[l][2])
						{
							for (kkloc = 0; kkloc < Order[l][2]; kkloc++)
							{
								for (jjloc = 0; jjloc <= Order[l][1]; jjloc++)
								{
									for (iiloc = 0; iiloc <= Order[l][0]; iiloc++)
									{
										// printf("jjloc:%d iiloc:%d\n",jjloc,iiloc);
										B = Patch_controlpoint[l][i - jjloc * No_Control_point[l][0] - iiloc];
										b = jjloc * (Order[l][0] + 1) + iiloc;
										// printf("B=%d b=%d e=%d\n",B,b,e);
										Controlpoint_of_Element[e][b] = B;
									}
								}
							}
							Element_patch[e] = l + Total_Patch_to_Now;
							e++;
						}
						i++;
					}
				}
			}
		}
	}

	// for s-IGA line_No_real_elementの初期化

	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			line_No_real_element[l + Total_Patch_to_Now][j] = 0;
		}
	}

	for (l = 0; l < *No_Patch; l++)
	{
		for (j = 0; j < DIMENSION; j++)
		{

			// printf("Order[%d]= ",j);
			// scanf("%d",&Order[j]);

			// printf("No_knot[%d]= ",j);
			// scanf("%d",&No_knot[j]);

			line_No_Total_element[l + Total_Patch_to_Now][j] = No_knot[l + Total_Patch_to_Now][j] - 2 * Order[l + Total_Patch_to_Now][j] - 1;

			// for(i=0;i<No_knot[j];i++){
			// printf("Position_Knots[%d][%d]= ",j,i);
			// scanf("%lf",&Position_Knots[j][i]);
			// }

			for (kkk = Order[l + Total_Patch_to_Now][j]; kkk < No_knot[l + Total_Patch_to_Now][j] - Order[l + Total_Patch_to_Now][j] - 1; kkk++)
			{
				difference[l + Total_Patch_to_Now][kkk - Order[l + Total_Patch_to_Now][j]][j] = Position_Knots[l + Total_Patch_to_Now][j][kkk + 1] - Position_Knots[l + Total_Patch_to_Now][j][kkk];
				// printf("[[[%d]]] ξ[%d]-ξ[%d]=%lf\n",kkk-Order[l][j],kkk+1,kkk,difference[kkk-Order[l][j]][j]);

				if (difference[l + Total_Patch_to_Now][kkk - Order[l + Total_Patch_to_Now][j]][j] != 0)
				{
					line_No_real_element[l + Total_Patch_to_Now][j]++;
				}
			}
			//  printf("line_No_real_element[%d][%d]=%d\n",l,j,line_No_real_element[l][j]);
		}
	}

	/*要素に行番号、列番号をつける*/

	if (DIMENSION == 2)
	{
		// printf("Total_element_all= ");
		// scanf("%d",&Total_element_all);

		for (h = 0; h < *Total_Element; h++)
		{
			Total_element_all_ID[h] = 0;
			// Total_element_all_ID[h]=h;
		}

		i = 0;
		for (l = 0; l < *No_Patch; l++)
		{
			for (y = 0; y < line_No_Total_element[l + Total_Patch_to_Now][1]; y++)
			{
				for (x = 0; x < line_No_Total_element[l + Total_Patch_to_Now][0]; x++)
				{
					ENC[l + Total_Patch_to_mesh[tm]][i + Total_Element_to_mesh[tm]][0] = x;
					ENC[l + Total_Patch_to_mesh[tm]][i + Total_Element_to_mesh[tm]][1] = y;
					// printf("ENC[%d][%d][0]=%d\tENC[%d][%d][1]=%d\n",
					//		l+Total_Patch_to_mesh[tm],
					//		i+Total_Element_to_mesh[tm],
					//		ENC[l+Total_Patch_to_mesh[tm]][i+Total_Element_to_mesh[tm]][0],
					//		l+Total_Patch_to_mesh[tm],
					//		i+Total_Element_to_mesh[tm],
					//		ENC[l+Total_Patch_to_mesh[tm]][i+Total_Element_to_mesh[tm]][1]);
					i++;
				}
			}
		}
	}

	/*必要な要素の行と列の番号を求める*/

	for (j = 0; j < DIMENSION; j++)
	{
		for (l = 0; l < *No_Patch; l++)
		{
			e = 0;
			for (k = 0; k < line_No_Total_element[l + Total_Patch_to_Now][j]; k++)
			{
				// printf("//%d,%d//\n",j,line_No_Total_element[j]);
				if (difference[l + Total_Patch_to_Now][k][j] != 0)
				{
					// printf("k=%d\n",k);
					real_element_line[l + Total_Patch_to_Now][e][j] = k;
					// printf("real_element_line[%d][%d]=%d\n",e,j,real_element_line[e][j]);
					e++;
					// printf("e=%d\n",e);
				}
			}
		}
	}

	/*必要な要素列上の要素のIDを1にする*/

	if (DIMENSION == 2)
	{
		for (n = 0; n < *Total_Element; n++)
		{
			// for(j=0;j<DIMENSION;j++){
			for (p = 0; p < line_No_real_element[Element_patch[n + Total_Element_to_Now]][0]; p++)
			{
				if (ENC[Element_patch[n + Total_Element_to_mesh[tm]]][n + Total_Element_to_mesh[tm]][0] == real_element_line[Element_patch[n + Total_Element_to_Now]][p][0])
				{
					for (q = 0; q < line_No_real_element[Element_patch[n + Total_Element_to_Now]][1]; q++)
					{
						if (ENC[Element_patch[n + Total_Element_to_mesh[tm]]][n + Total_Element_to_mesh[tm]][1] == real_element_line[Element_patch[n + Total_Element_to_Now]][q][1])
						{
							// Total_element_all_ID[n]++;
							Total_element_all_ID[n]++;
							// printf("Total_element_all_ID[%d]=%d\n"
							//         ,n,Total_element_all_ID[n]);
							// break;
						}
					}
				}
			}
			//}

			// printf("Total_element_all_ID[%d]=%d\n",n,Total_element_all_ID[n]);

			/*IDが1の要素に番号を振る*/

			if (Total_element_all_ID[n] == 1)
			{
				real_element[r + real_Total_Element_to_mesh[tm]] = n + Total_Element_to_Now;
				// printf("real_element[%d]=%d\n",
				//        r + real_Total_Element_to_mesh[tm],
				//        real_element[r + real_Total_Element_to_mesh[tm]]);
				real_El_No_on_mesh[tm][r] = n + Total_Element_to_Now;
				// printf("real_El_No_on_mesh[%d][%d]=%d\n",
				//         tm,r,real_El_No_on_mesh[tm][r]);
				r++;
			}
		}
		// for s-IGA real_Total_Elementの初期化
		real_Total_Element = 0;

		for (l = 0; l < *No_Patch; l++)
		{
			real_Total_Element += line_No_real_element[l + Total_Patch_to_Now][0] * line_No_real_element[l + Total_Patch_to_Now][1];
		}
		real_Total_Element_on_mesh[tm] = real_Total_Element;
		real_Total_Element_to_mesh[tm + 1] = real_Total_Element_to_mesh[tm] + real_Total_Element;
		// real_Total_Element_to_mesh[tm] += real_Total_Element;
		real_Total_Element_to_Now = real_Total_Element_to_mesh[tm];
		// printf("real_Total_Element_on_mesh[%d]=%d\n",
		//		tm, real_Total_Element_on_mesh[tm]);
		// printf("real_Total_Element_to_mesh[%d]=%d\n",
		//		tm, real_Total_Element_to_mesh[tm]);
	}

	/*
	for(rr=0;rr<real_Total_Element;rr++){
		printf("real_element[%d]=%d\n",rr,real_element[rr]);
	}
	*/
	//}
	//
	/* For distributed load 2D */

	for (iii = 0; iii < *Total_Control_Point; iii++)
	{
		Equivalent_Nodal_Force[iii + Total_Control_Point_to_mesh[tm]][0] = 0.0;
		Equivalent_Nodal_Force[iii + Total_Control_Point_to_mesh[tm]][1] = 0.0;
	}

	for (i = 0; i < *Total_DistributeForce; i++)
	{

		type_load = type_load_array[i + Total_DistributeForce_to_mesh[tm]];
		iPatch = iPatch_array[i + Total_DistributeForce_to_mesh[tm]];
		iCoord = iCoord_array[i + Total_DistributeForce_to_mesh[tm]];
		val_Coord = val_Coord_array[i + Total_DistributeForce_to_mesh[tm]];
		Range_Coord[0] = Range_Coord_array[i + Total_DistributeForce_to_mesh[tm]][0];
		Range_Coord[1] = Range_Coord_array[i + Total_DistributeForce_to_mesh[tm]][1];
		Coeff_Dist_Load[0] = Coeff_Dist_Load_array[i + Total_DistributeForce_to_mesh[tm]][0];
		Coeff_Dist_Load[1] = Coeff_Dist_Load_array[i + Total_DistributeForce_to_mesh[tm]][1];
		Coeff_Dist_Load[2] = Coeff_Dist_Load_array[i + Total_DistributeForce_to_mesh[tm]][2];

		printf("type_load:%d\tiPatch:%d\tiCoord:%d\tval_coord:%lf\t",
			   type_load, iPatch, iCoord, val_Coord);
		printf("Range0:%lf\tRange1:%lf\t",
			   Range_Coord[0], Range_Coord[1]);
		printf("Coeff0:%lf\n", Coeff_Dist_Load[0]);
		Setting_Dist_Load_2D(tm, iPatch, Total_Element_to_mesh[tm + 1], iCoord, val_Coord,
							 Range_Coord, type_load, Coeff_Dist_Load);
	}
	/*-------------------------------------------------------------------------------------*/
}

//拘束されている行数を省いた行列の番号の制作
__host__ int Make_Index_Dof(int Total_Control_Point,
							int Total_Constraint,
							int Constraint_Node_Dir[MAX_N_CONSTRAINT][2])
{
	int i, k = 0;

	// printf("Total_Constraint=%d\n",Total_Constraint);
	// Index_Dofの初期化(複数メッシュ読み込みのため)
	for (i = 0; i < Total_Control_Point * 2; i++)
	{
		Index_Dof[i] = 0;
		// printf("test_DOF[%d]:%d\n",i,Index_Dof[i]);
	}
	//拘束されている自由度(Degree Of free)をERRORにする
	for (i = 0; i < Total_Constraint; i++)
	{
		// printf("%d\n",Constraint_Node_Dir[i][0] * DIMENSION + Constraint_Node_Dir[i][1]);
		Index_Dof[Constraint_Node_Dir[i][0] * DIMENSION + Constraint_Node_Dir[i][1]] = ERROR;
	}
	// ERROR以外に番号を付ける
	for (i = 0; i < Total_Control_Point * DIMENSION; i++)
	{
		if (Index_Dof[i] != ERROR)
		{
			Index_Dof[i] = k;
			k++;
		}
		// printf("Index_Dof[%d]=%d\n",i,Index_Dof[i]);
	}
	printf("Max_Index_Dof=%d\n", k);
	return k;
}

__host__ void Make_K_Whole_Ptr_Col(int Total_Element,
								   int Total_Control_Point,
								   int K_Whole_Size)
{
	int i, ii, j, jj, k;
	int NE;
	int N, i_index, j_index;

	//初期化
	// for (i = 0; i < Total_Control_Point * DIMENSION; i++)
	// Total_Control_Point_To_Node[i] = 0;
	for (i = 0; i < K_Whole_Size + 1; i++)
		K_Whole_Ptr[i] = 0;

	for (N = 0; N < Total_Control_Point; N += K_DIVISION_LENGE)
	{ //大きく分割するためのループ
		//各節点に接する節点を取得
		for (i = 0; i < K_DIVISION_LENGE; i++)
		{
			Total_Control_Point_To_Node[i] = 0;
		}
		for (i = 0; i < Total_Element; i++)
		{
			for (ii = 0; ii < No_Control_point_ON_ELEMENT[Element_patch[i]]; ii++)
			{
				NE = Controlpoint_of_Element[i][ii] - N;
				// printf("NE=%d\n",NE);
				// printf("K_DIVISION_LENGE=%d,N=%d,NE=%d\n",K_DIVISION_LENGE,N,NE);    //K_DIVISION_LENGE=0,N=0,NE=コネクティビティ的な
				if (0 <= NE && NE < K_DIVISION_LENGE)
				{
					for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[i]]; j++) //ローカル要素
					{
						// printf("j=%d\n",j);
						//数字がない時
						if (Total_Control_Point_To_Node[NE] == 0)
						{
							//節点番号を取得
							Node_To_Node[NE][0] = Controlpoint_of_Element[i][j];
							Total_Control_Point_To_Node[NE]++;
							// printf("Node_To_Node[%d][0]=%d\n",NE,Node_To_Node[NE][0]);
						}
						// printf("②Total_Control_Point_To_Node[%d]=%d\n",NE,Total_Control_Point_To_Node[NE]);
						//同じものがあったら
						// k > 0 以降の取得
						// kのカウント
						for (k = 0; k < Total_Control_Point_To_Node[NE]; k++)
						{
							// printf("Node_To_Node[%d][%d]=%d\n",NE,k,Node_To_Node[NE][k]);
							//
							// printf("k_1=%d\t",k);
							if (Node_To_Node[NE][k] == Controlpoint_of_Element[i][j])
							{
								// printf("break\t");
								break;
							}
						}
						// printf("\nk_2=%d\n",k);
						//未設定のNode_To_Node取得
						if (k == Total_Control_Point_To_Node[NE])
						{
							Node_To_Node[NE][k] = Controlpoint_of_Element[i][j];
							// printf("Node_To_Node[%d][%d]=%d\n",NE,k,Node_To_Node[NE][k]);
							Total_Control_Point_To_Node[NE]++;
							// printf("③Total_Control_Point_To_Node[%d]=%d\n",NE,Total_Control_Point_To_Node[NE]);
						}
					}
					//別メッシュとの重なりを考慮
					if (NNLOVER[i] > 0)
					{
						for (jj = 0; jj < NNLOVER[i]; jj++)
						{
							for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[NELOVER[i][jj]]]; j++) //ローカル要素
							{
								// printf("j=%d\n",j);
								//数字がない時
								if (Total_Control_Point_To_Node[NE] == 0)
								{
									//節点番号を取得
									Node_To_Node[NE][0] = Controlpoint_of_Element[NELOVER[i][jj]][j];
									Total_Control_Point_To_Node[NE]++;
									// printf("Node_To_Node[%d][0]=%d\n",NE,Node_To_Node[NE][0]);
								}
								// printf("②Total_Control_Point_To_Node[%d]=%d\n",NE,Total_Control_Point_To_Node[NE]);
								//同じものがあったら
								// k > 0 以降の取得
								// kのカウント
								for (k = 0; k < Total_Control_Point_To_Node[NE]; k++)
								{
									// printf("Node_To_Node[%d][%d]=%d\n",NE,k,Node_To_Node[NE][k]);
									//
									// printf("k_1=%d\t",k);
									if (Node_To_Node[NE][k] == Controlpoint_of_Element[NELOVER[i][jj]][j])
									{
										// printf("break\t");
										break;
									}
								}
								// printf("\nk_2=%d\n",k);
								//未設定のNode_To_Node取得
								if (k == Total_Control_Point_To_Node[NE])
								{
									Node_To_Node[NE][k] = Controlpoint_of_Element[NELOVER[i][jj]][j];
									// printf("Node_To_Node[%d][%d]=%d\n",NE,k,Node_To_Node[NE][k]);
									Total_Control_Point_To_Node[NE]++;
									// printf("③Total_Control_Point_To_Node[%d]=%d\n",NE,Total_Control_Point_To_Node[NE]);
								}
							}
						}
					}
				}
				// printf("\n");
			}
			// printf("\n");
		}
		//順番に並び替える
		for (i = 0; i < K_DIVISION_LENGE; i++)
		{
			if (N + i < Total_Control_Point)
			{
				// printf("Node[%d] T=%d; \n",N+i, Total_Control_Point_To_Node[ i ]);
				for (j = 0; j < Total_Control_Point_To_Node[i]; j++)
				{
					int Min = Node_To_Node[i][j], No = j;
					for (k = j; k < Total_Control_Point_To_Node[i]; k++)
					{
						if (Min > Node_To_Node[i][k])
						{
							Min = Node_To_Node[i][k];
							No = k;
						}
					}
					for (k = No; k > j; k--)
					{
						Node_To_Node[i][k] = Node_To_Node[i][k - 1];
					}
					Node_To_Node[i][j] = Min;
					//				printf("%d ",Node_To_Node[i][j]);
				}
				//			printf("\n");
			}

			//並べ替えたNode_To_Node確認
			// for (j = 0; j < Total_Control_Point_To_Node[i]; j++)
			// {
			// 	printf("sort_Node_To_Node[%d][%d]=%d\n",i,j,Node_To_Node[i][j]);
			// }
		}

		//節点からcol ptrを求める
		ii = 0;
		k = 0;
		for (i = 0; i < K_DIVISION_LENGE; i++)
		{
			for (ii = 0; ii < DIMENSION; ii++)
			{
				if (N + i < Total_Control_Point)
				{
					i_index = Index_Dof[(N + i) * DIMENSION + ii];
					// printf("i = %d\n", i);
					// printf("N = %d\n", N);
					// printf("i_index = %d\n", i_index);
					k = 0;
					if (i_index >= 0)
					{
						// K_Whole_Ptr[i_index + 1] = K_Whole_Ptr[i_index];
						K_Whole_Ptr[i_index + 1] = K_Whole_Ptr[i_index];
						// printf("K_Whole_Ptr[%d][%d]=%d\n",tm,i_index,K_Whole_Ptr[tm][i_index+1]);
						for (j = 0; j < Total_Control_Point_To_Node[i]; j++)
						{
							// printf("Total_Control_Point_To_Node[%d] = %d\n", i, Total_Control_Point_To_Node[i]);
							for (jj = 0; jj < DIMENSION; jj++)
							{
								j_index = Index_Dof[Node_To_Node[i][j] * DIMENSION + jj];
								if (j_index >= 0 && j_index >= i_index)
								{
									K_Whole_Ptr[i_index + 1]++;
									// col_N[N/K_DIVISION_LENGE][k] = j_index;
									K_Whole_Col[K_Whole_Ptr[i_index] + k] = j_index;
									// printf("K_Whole_Col[%d]=%d\n"
									//        ,K_Whole_Ptr[i_index]+k
									//        ,K_Whole_Col[K_Whole_Ptr[i_index]+k]);
									k++;
									// printf("ptr[%d]=%d,col[%d]=%d\n",i_index+1,K_Whole_Ptr[i_index+1],K_Whole_Ptr[i_index]+k,K_Whole_Col[K_Whole_Ptr[i_index]+k]);
								}
							}
						}
					}
				}
			}
		}
		// col_N[N/K_DIVISION_LENGE][ k ] = -1;
	}
	// for(i=0;i<K_Whole_Size_array[tm]+1;i++)
	/*
	for(i=0;i<K_Whole_Size+1;i++)
	{
		printf("K_Whole_Ptr[%d]=%d\n",
				i,K_Whole_Ptr[i]);
	}*/
	/*
	for( i = 0; i < K_Whole_Size+1; i++ )//printf("K_Whole_Ptr[%d]= %d\n",i,K_Whole_Ptr[i]);
	//col合成
	k = 0;
	for( N = 0; N < Total_Control_Point ; N +=K_DIVISION_LENGE ){
		for(i = 0; col_N[ N/K_DIVISION_LENGE ][i] != -1; i++ ){
			K_Whole_Col[k] = col_N[ N/K_DIVISION_LENGE ][i];
			k++;
		}
	}
	*/
}

// valを求める
__host__ void Make_K_Whole_Val(double E, double nu, int Total_Element, int DM)
{
	int i, j, j1, j2, k1, k2, l;
	int a, b, re;

	for (i = 0; i < MAX_NON_ZERO; i++)
	{
		K_Whole_Val[i] = 0.0;
	}

	for (re = 0; re < Total_Element; re++)
	{
		i = real_element[re];
		Check_BDBJ_flag[i] = 0;
	}

	/*for(rr=0;rr<line_No_real_element[0]*line_No_real_element[1];rr++){
			printf("real_element[%d]=%d\n",rr,real_element[rr]);
		}
		printf("real_Total_element=%d\n",real_Total_Element);*/
	// re=0;
	for (re = 0; re < Total_Element; re++)
	{
		// if(i==real_element[re]){
		// printf("re=%d\n",re);
		i = real_element[re];
		// printf("El_No;i=%d\n", real_element[re]);

		if (Element_mesh[i] == 0 && re == 0) /*2つめの条件は効率化のため*/
		{
			Make_gauss_array(0);
		}
		else if (Element_mesh[i] > 0)
		{
			printf("NNLOVER[%d]:%d\tNNLOVER[%d]:%d\tElement_mesh[%d]:%d\n", i, NNLOVER[i], real_element[re - 1], NNLOVER[real_element[re - 1]], real_element[re - 1], Element_mesh[real_element[re - 1]]);
			if (NNLOVER[i] == 1 && (NNLOVER[real_element[re - 1]] != 1 || Element_mesh[real_element[re - 1]] == 0)) /*2つめ以降の条件は効率化のため*/
			{
				Make_gauss_array(0);
			}
			else if (NNLOVER[i] >= 2 && (NNLOVER[real_element[re - 1]] == 1 || Element_mesh[real_element[re - 1]] == 0)) /*2つめ以降の条件は効率化のため*/
			{
				Make_gauss_array(1);
			}
		}
		// printf("i= %d\tGaussPt_3D=%d\n",i ,GaussPt_3D);

		for (j = 0; j < GP_2D; j++)
		{
			Same_BDBJ_flag[j] = 0;
		}

		KIEL_SIZE = No_Control_point_ON_ELEMENT[Element_patch[i]] * DIMENSION;
		double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE];
		// printf("Total_Element=%d\tre=%d\tEl_No=%d\n", Total_Element, re, i);
		//各要素のKelを求める
		for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; j1++)
		{
			for (j2 = 0; j2 < DIMENSION; j2++)
			{
				X[j1][j2] = Node_Coordinate[Controlpoint_of_Element[i][j1]][j2];
			}
		}

		Make_K_EL(i, X, K_EL, E, nu, DM);

		// Valを求める
		// for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[El_No_on_mesh[tm][i]]]; j1++)
		for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; j1++)
		{
			for (j2 = 0; j2 < DIMENSION; j2++)
			{
				a = Index_Dof[Controlpoint_of_Element[i][j1] * DIMENSION + j2];
				if (a >= 0)
				{
					// for (k1 = 0; k1 < No_Control_point_ON_ELEMENT[Element_patch[El_No_on_mesh[tm][i]]]; k1++)
					for (k1 = 0; k1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; k1++)
					{
						for (k2 = 0; k2 < DIMENSION; k2++)
						{
							b = Index_Dof[Controlpoint_of_Element[i][k1] * DIMENSION + k2];
							if (b >= 0 && b >= a)
							{
								for (l = K_Whole_Ptr[a]; l < K_Whole_Ptr[a + 1]; l++)
								{
									if (K_Whole_Col[l] == b)
									{
										// printf("real_element[%d] = %d\n", re, real_element[re]);
										// printf("Controlpoint_of_Element[%d][%d] = %d\n", i, k1, Controlpoint_of_Element[i][k1]);
										// printf("l = %d\n", l);
										// printf("K_Whole_Col[%d] = %d\n", l, K_Whole_Col[l]);
										// printf("b = %d\n", b);
										K_Whole_Val[l] += K_EL[j1 * DIMENSION + j2][k1 * DIMENSION + k2];
										// printf("K_Whole_Val[%d] = %.20e\n", l, K_Whole_Val[l]);
										// printf("K_Whole_Val[%d]=%le\n",l,K_Whole_Val[l]);
										break;
									}
								}
							}
						}
					}
				}
			}
		}

		if (Element_mesh[i] > 0) //ローカルメッシュ上の要素について
		{
			if (NNLOVER[i] > 0) //重なっている要素が存在するとき
			{
				for (j = 0; j < NNLOVER[i]; j++)
				{
					double XG[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION];
					KIEL_SIZE = No_Control_point_ON_ELEMENT[Element_patch[NELOVER[i][j]]] * DIMENSION;
					double coupled_K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE];
					for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[NELOVER[i][j]]]; j1++)
					{
						for (j2 = 0; j2 < DIMENSION; j2++)
						{
							XG[j1][j2] = Node_Coordinate[Controlpoint_of_Element[NELOVER[i][j]][j1]][j2];
							//重なっている要素の物理座標取得
						}
					}
					Make_coupled_K_EL(i, NELOVER[i][j],
									  X,
									  XG,
									  coupled_K_EL,
									  E, nu, DM);

					Check_BDBJ_flag[i] += Total_BDBJ_flag;
					if (j == NNLOVER[i] - 1)
					{
						for (j1 = 0; j1 < GP_2D; j1++)
						{
							// printf("Same_BDBJ_flag[%d]=%d\n",j1,Same_BDBJ_flag[j1]);
							if (Same_BDBJ_flag[j1] != 1)
							{
								printf("ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR\n");
							}
						}
						printf("-------------------------Check_BDBJ_flag[%d]=%d-------------------------\n", i, Check_BDBJ_flag[i]);
						if (Check_BDBJ_flag[i] != GP_2D)
						{
							printf("ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR-ERROR\n");
						}
					}

					// Valを求める
					// for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[El_No_on_mesh[tm][i]]]; j1++)
					for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[NELOVER[i][j]]]; j1++)
					{
						for (j2 = 0; j2 < DIMENSION; j2++)
						{
							a = Index_Dof[Controlpoint_of_Element[NELOVER[i][j]][j1] * DIMENSION + j2];
							if (a >= 0)
							{
								// for (k1 = 0; k1 < No_Control_point_ON_ELEMENT[Element_patch[El_No_on_mesh[tm][i]]]; k1++)
								for (k1 = 0; k1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; k1++)
								{
									for (k2 = 0; k2 < DIMENSION; k2++)
									{
										b = Index_Dof[Controlpoint_of_Element[i][k1] * DIMENSION + k2];
										if (b >= 0 && b >= a)
										{
											for (l = K_Whole_Ptr[a]; l < K_Whole_Ptr[a + 1]; l++)
											{
												if (K_Whole_Col[l] == b)
												{
													K_Whole_Val[l] += coupled_K_EL[j1 * DIMENSION + j2][k1 * DIMENSION + k2];
													// printf("coupled_K_Whole_Val[%d]=%le\n",l,K_Whole_Val[l]);
													break;
												}
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
		// re++;
		// }
	}
}

///////////////////////////////////////////////////////////////////////////
/////////////////////連立1次方程式の解法
/////////////////////////////////////////////////////////////////////
//分布荷重の等価節点力を足す
__host__ void Add_Equivalent_Nodal_Forec_to_F_Vec(int Total_Control_Point)
{
	int i, j, index;
	for (j = 0; j < DIMENSION; j++)
	{
		for (i = 0; i < Total_Control_Point; i++)
		{
			index = Index_Dof[i * DIMENSION + j];
			if (index >= 0)
			{
				rhs_vec[index] += Equivalent_Nodal_Force[i][j];
				// printf("i = %d index = %d rhs_vec[index] = %f\n", i, index, rhs_vec[index]);

				// for s-IGA
				/*
				//結合剛性行列の右辺の組み立て
				rhs_overlaid_vec[index+K_Whole_Size_array_to_Now]=rhs_vec[index];
				printf("i = %d index = %d rhs_overlaid_vec[index] = %f\n"
					   , i, index+K_Whole_Size_array_to_Now
					   , rhs_overlaid_vec[index+K_Whole_Size_array_to_Now]);
				*/
			}
			// printf("i = %d  j = %d  Equivalent_Nodal_Force[i][j] = %f\n",i, j, Equivalent_Nodal_Force[i][j]);
		}
	}
}

//荷重の行列を作る
__host__ void Make_F_Vec(int Total_Load, int Load_Node_Dir[MAX_N_LOAD][2], double Value_of_Load[MAX_N_LOAD], int K_Whole_Size)
{
	int i, index;
	for (i = 0; i < K_Whole_Size; i++)
		rhs_vec[i] = 0.0;
	for (i = 0; i < Total_Load; i++)
	{
		index = Index_Dof[Load_Node_Dir[i][0] * DIMENSION + Load_Node_Dir[i][1]];
		if (index >= 0)
			rhs_vec[index] += Value_of_Load[i];
	}
}

//強制変位対策
__host__ void Make_F_Vec_disp_const(int Mesh_No, int Total_Constraint,
									int Constraint_Node_Dir[MAX_N_CONSTRAINT][2],
									double Value_of_Constraint[MAX_N_CONSTRAINT],
									double E, double nu, int DM)
{
	int ie, idir, inode, jdir, jnode, kk_const;
	int ii, iii, b, bb, jj, j1, j2, ii_local, jj_local;
	int iee;

	int i;

	Make_gauss_array(0);

	// for (ie = 0; ie < real_Total_Element; ie++)
	for (ie = 0; ie < real_Total_Element_to_mesh[Total_mesh]; ie++)
	{
		i = real_element[ie];

		// double X[No_Control_point_ON_ELEMENT[Element_patch[real_El_No_on_mesh[Mesh_No][ie]]]][DIMENSION];

		KIEL_SIZE = No_Control_point_ON_ELEMENT[Element_patch[i]] * DIMENSION;
		double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE];

		iii = 0;
		for (idir = 0; idir < DIMENSION; idir++)
		{
			for (inode = 0; inode < No_Control_point_ON_ELEMENT[Element_patch[i]]; inode++)
			{
				// b = Index_Dof[Controlpoint_of_Element[ie ][inode] * DIMENSION + idir];
				b = Index_Dof[Controlpoint_of_Element[i][inode] * DIMENSION + idir];
				if (b < 0)
					iii++;
			}
		}
		// printf("iii;%d\n",iii);
		if (iii > 0)
		{
			for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; j1++)
			{
				for (j2 = 0; j2 < DIMENSION; j2++)
				{
					// X[j1][j2] = Node_Coordinate[Controlpoint_of_Element[ie ][j1]][j2];
					X[j1][j2] = Node_Coordinate[Controlpoint_of_Element[i][j1]][j2];
				} // end for j2
			}	  // end for j1
			iee = i;
			Make_K_EL(iee, X, K_EL, E, nu, DM);
			for (idir = 0; idir < DIMENSION; idir++)
			{
				for (inode = 0; inode < No_Control_point_ON_ELEMENT[Element_patch[i]]; inode++)
				{
					// ii = Controlpoint_of_Element[ie ][inode] * DIMENSION + idir;
					ii = Controlpoint_of_Element[real_El_No_on_mesh[Mesh_No][ie]][inode] * DIMENSION + idir;
					b = Index_Dof[ii];
					if (b >= 0)
					{
						ii_local = inode * DIMENSION + idir;
						for (jdir = 0; jdir < DIMENSION; jdir++)
						{
							for (jnode = 0; jnode < No_Control_point_ON_ELEMENT[Element_patch[i]]; jnode++)
							{
								// jj = Controlpoint_of_Element[ie ][jnode] * DIMENSION + jdir;
								jj = Controlpoint_of_Element[i][jnode] * DIMENSION + jdir;
								bb = Index_Dof[jj];
								if (bb < 0)
								{
									jj_local = jnode * DIMENSION + jdir; // printf("%d,%d\n",ie,jnode);
									for (kk_const = 0; kk_const < Total_Constraint; kk_const++)
									{
										// if (Controlpoint_of_Element[ie ][jnode] == Constraint_Node_Dir[kk_const][0] && jdir == Constraint_Node_Dir[kk_const][1])
										if (Controlpoint_of_Element[i][jnode] == Constraint_Node_Dir[kk_const][0] && jdir == Constraint_Node_Dir[kk_const][1])
										{
											rhs_vec[b] -= K_EL[ii_local][jj_local] * Value_of_Constraint[kk_const]; // if(kk_const >= 28){printf("%d , %d ,%16.15e\n",ii_local, jj_local ,  K_EL[ii_local][jj_local]);}
										}																			// end if Controlpoint_of_Element[ie][jnode]
									}																				// end for kk_const
								}																					// end if bb
							}																						// end for jnode
						}																							// end for jdir
					}																								// end if b>=0
				}																									// end for inode
			}																										// end for idir
		}																											// end if iii>0
	}																												// end for ie
} // end

__host__ void mat_vec_crs(double vec_result[], double vec[], const int ndof)
{
	int i, j, icount = 0;
	/* zero clear */

	for (i = 0; i < ndof; i++)
		vec_result[i] = 0;
	for (i = 0; i < ndof; i++)
	{
		// for (j = K_Whole_Ptr[i]; j < K_Whole_Ptr[i + 1]; j++)
		for (j = K_Whole_Ptr[i]; j < K_Whole_Ptr[i + 1]; j++)
		{
			vec_result[i] += K_Whole_Val[icount] * vec[K_Whole_Col[j]];
			// vec_result[i] += K_Whole_overlaid_Val[icount] * vec[K_Whole_overlaid_Col[j]];
			// if (i != K_Whole_overlaid_Col[j])
			if (i != K_Whole_Col[j])
				vec_result[K_Whole_Col[j]] += K_Whole_Val[icount] * vec[i];
			// vec_result[K_Whole_overlaid_Col[j]] += K_Whole_overlaid_Val[icount] * vec[i];
			icount++;
		}
	}
}

__host__ double inner_product(int ndof, double vec1[], double vec2[])
{
	double rrr = 0.0;
	int i;
	for (i = 0; i < ndof; i++)
	{
		rrr += vec1[i] * vec2[i];
		// printf("vec1[%d]=%f vec2[%d]=%f\n",i,vec1[i],i,vec2[i]); /*-nan 10/23*/
	}
	return (rrr);
}

__host__ int check_conv_CG(int ndof, double alphak, double pp[], double eps, int itr)
{
	double rrr1 = 0.0, rrr2 = 0.0, rrr3;
	int i, istop = 0;
	/* Checking the convergence of the CG solver */
	/* istop =0; Not converged, istop = 1; converged */
	printf("ndof=%d alphak= %15e\t", ndof, alphak);
	for (i = 0; i < ndof; i++)
	{
		rrr1 += pp[i] * pp[i];
		rrr2 += sol_vec[i] * sol_vec[i];
		// printf("pp[%d]=%f sol_vec[%d]=%f\n",i,pp[i],i,sol_vec[i]); /*-nan 10/23*/
	}
	rrr3 = fabs(alphak) * sqrt(rrr1 / rrr2);
	printf("Iteration# = %d  residual = %15e (%15e)\n", itr, rrr3, eps);
	if (rrr3 < eps)
		istop = 1;
	/* Temporaty Oct. 10, 2017 by H.Okada */
	// if(itr < 100) istop=0;
	// printf("Iteration# = %d  residual = %15e (%15e)\n",itr, rrr3, eps);
	return (istop);
}

__host__ void Diag_Scaling_CG_pre(int ndof, int flag_operation)
{
	int i, j;
	int icount = 0;
	/* flag_opertion = 0: Preprocess to the CG solver
			A <-- Dt A D  and b <-- Dt b */
	/* flag_operation = 1: Post process to the CG solver
			b <-- Dt b  */
	printf("ndof=%d\n", ndof);
	if (flag_operation == 0)
	{
		diag_scaling[0] = 1.0 / sqrt(K_Whole_Val[0]);
		// diag_scaling[0] = 1.0 / sqrt(K_Whole_overlaid_Val[0]);
		/* diag_scaling[0] = 1.0; */
		for (i = 1; i < ndof; i++)
		{
			// printf("%d %le\n",K_Whole_Ptr[i], K_Whole_Val[K_Whole_Ptr[i]]);
			diag_scaling[i] = 1.0 / sqrt(K_Whole_Val[K_Whole_Ptr[i]]);
			printf("diag=%le\n", diag_scaling[i]);
			printf("K_Whole_Val[%d] = %.16e\n", K_Whole_Ptr[i], K_Whole_Val[K_Whole_Ptr[i]]);
			printf("sqrt=%le\n", sqrt(K_Whole_Val[K_Whole_Ptr[i]]));
			// diag_scaling[i]
			//     = 1.0 / sqrt(K_Whole_overlaid_Val[K_Whole_overlaid_Ptr[i]]);
			/* diag_scaling[i] = 1.0; */
		}
		for (i = 0; i < ndof; i++)
		{
			for (j = K_Whole_Ptr[i]; j < K_Whole_Ptr[i + 1]; j++)
			// for (j = K_Whole_overlaid_Ptr[i]; j < K_Whole_overlaid_Ptr[i + 1]; j++)
			{
				// printf("Check scling icount=%d i=%d K_Whole_Col[%d] = %d\n",icount,i,j,K_Whole_Col[j]);
				K_Whole_Val[icount] = K_Whole_Val[icount] * diag_scaling[i] * diag_scaling[K_Whole_Col[j]];
				// K_Whole_overlaid_Val[icount]
				//     = K_Whole_overlaid_Val[icount] * diag_scaling[i]
				//     * diag_scaling[K_Whole_overlaid_Col[j]];
				// printf("K_Whole_Val = %f\n",K_Whole_Val[icount]);
				icount++;
			}
			printf("rhs_vec_before[%d]:%le diag_scaling[%d]:%le\n", i, rhs_vec[i], i, diag_scaling[i]);
			// rhs_overlaid_vec[i] = rhs_overlaid_vec[i] * diag_scaling[i];
			rhs_vec[i] = rhs_vec[i] * diag_scaling[i];
			// printf("rhs_vec[%d]:%le\n",i,rhs_vec[i]);
		}
	}
	if (flag_operation == 1)
		for (i = 0; i < ndof; i++)
		{
			// printf("solvec[%d] = %f\n",i, sol_vec[i]);
			sol_vec[i] = sol_vec[i] * diag_scaling[i];
		}

	printf("\nqq\n");
}

__host__ void CG_Solver(int ndof, int max_itr, double eps, int flag_ini_val)
{
	static double gg[MAX_K_WHOLE_SIZE], dd[MAX_K_WHOLE_SIZE], pp[MAX_K_WHOLE_SIZE];
	static double qqq, ppp, rrr;
	static double alphak, betak;
	int i;
	int itr;
	int ii, istop;

	/* Program to solve linear equations by using the CG method */
	if (flag_ini_val == 0)
		for (i = 0; i < ndof; i++)
			sol_vec[i] = 0.0;
	/* Initializing the solution vector if it were not given */
	mat_vec_crs(dd, sol_vec, ndof);
	for (i = 0; i < ndof; i++)
	{
		// gg[i] = rhs_overlaid_vec[i] - dd[i];
		gg[i] = rhs_vec[i] - dd[i];
		// printf("rhs_vec[%d]=%f dd[%d]=%f\n", i, rhs_overlaid_vec[i], i, dd[i]);
		printf("rhs_vec[%d]=%f dd[%d]=%f\n", i, rhs_vec[i], i, dd[i]);
		pp[i] = gg[i];
	}

	printf("\nrr");

	for (itr = 0; itr < max_itr; itr++)
	{
		ppp = inner_product(ndof, gg, gg);
		mat_vec_crs(dd, pp, ndof);
		rrr = inner_product(ndof, dd, pp);
		alphak = ppp / rrr;
		// printf("ppp=%f rrr=%f\n", ppp, rrr); /*ppp,rrrも-nan,10/22*/
		// printf("i=%d",i);
		for (ii = 0; ii < ndof; ii++)
		{
			sol_vec[ii] += alphak * pp[ii];
			gg[ii] -= alphak * dd[ii];
		}
		qqq = inner_product(ndof, gg, dd);
		betak = qqq / rrr;
		for (ii = 0; ii < ndof; ii++)
			pp[ii] = gg[ii] - betak * pp[ii];
		istop = check_conv_CG(ndof, alphak, pp, eps, itr);
		if (istop == 1)
			break;
	}

	printf("\nss");
}

__host__ void Make_M(double *M, int *M_Ptr, int *M_Col, int ndof)
{
	int i, j;
	int ndof_glo = 0;

	// グローバルパッチのdofを求める
	for (i = 0; i < Total_Control_Point_on_mesh[0] * DIMENSION; i++)
	{
		if (Index_Dof[i] != ERROR)
		{
			ndof_glo++;
		}
	}
	printf("ndof		%d\n", ndof);
	printf("ndof_glo	%d\n", ndof_glo);

	int counter = 0;

	// M = [[K^G, 0], [0, K^L]] を作成
	M_Ptr[0] = 0;
	for (i = 0; i < ndof; i++)
	{
		M_Ptr[i + 1] = M_Ptr[i];

		for (j = K_Whole_Ptr[i]; j < K_Whole_Ptr[i + 1]; j++)
		{
			if (i < ndof_glo && K_Whole_Col[j] < ndof_glo)
			{
				M[counter] = K_Whole_Val[j];
				M_Col[counter] = K_Whole_Col[j];
				counter++;
				M_Ptr[i + 1]++;
			}
			else if (i >= ndof_glo)
			{
				M[counter] = K_Whole_Val[j];
				M_Col[counter] = K_Whole_Col[j];
				counter++;
				M_Ptr[i + 1]++;
			}
		}
	}
}

__host__ void M_mat_vec_crs(double *M, int *M_Ptr, int *M_Col, double *vec_result, double *vec, const int ndof)
{
	int i, j, icount = 0;

	for (i = 0; i < ndof; i++)
		vec_result[i] = 0;
	for (i = 0; i < ndof; i++)
	{
		for (j = M_Ptr[i]; j < M_Ptr[i + 1]; j++)
		{
			vec_result[i] += M[icount] * vec[M_Col[j]];
			if (i != M_Col[j])
				vec_result[M_Col[j]] += M[icount] * vec[i];
			icount++;
		}
	}
}

__host__ int M_check_conv_CG(int ndof, double alphak, double *pp, double eps, double *solution_vec)
{
	double rrr1 = 0.0, rrr2 = 0.0, rrr3;
	int i, istop = 0;
	for (i = 0; i < ndof; i++)
	{
		rrr1 += pp[i] * pp[i];
		rrr2 += solution_vec[i] * solution_vec[i];
	}
	rrr3 = fabs(alphak) * sqrt(rrr1 / rrr2);
	if (rrr3 < eps)
		istop = 1;
	return (istop);
}

__host__ void CG(int ndof, double *solution_vec, double *M, int *M_Ptr, int *M_Col, double *right_vec)
{
	int i /*, j*/;
	// int icount = 0;

	// 対角スケーリング 前処理
	// diag_scaling[0] = 1.0 / sqrt(M[0]);
	// for (i = 1; i < ndof; i++)
	// {
	// 	diag_scaling[i] = 1.0 / sqrt(M[M_Ptr[i]]);
	// }
	// for (i = 0; i < ndof; i++)
	// {
	// 	for (j = M_Ptr[i]; j < M_Ptr[i + 1]; j++)
	// 	{
	// 		M[icount] = M[icount] * diag_scaling[i] * diag_scaling[M_Col[j]];
	// 		icount++;
	// 	}
	// 	right_vec[i] = right_vec[i] * diag_scaling[i];
	// }

	// CG solver
	static double gg[MAX_K_WHOLE_SIZE], dd[MAX_K_WHOLE_SIZE], pp[MAX_K_WHOLE_SIZE];
	static double qqq, ppp, rrr;
	static double alphak, betak;
	int itr;
	int ii, istop;
	int max_itr = ndof;
	double eps = 1.0e-13;

	for (i = 0; i < ndof; i++)
	{
		solution_vec[i] = 0.0;
	}
	M_mat_vec_crs(M, M_Ptr, M_Col, dd, solution_vec, ndof);
	for (i = 0; i < ndof; i++)
	{
		gg[i] = right_vec[i] - dd[i];
		pp[i] = gg[i];
	}
	for (itr = 0; itr < max_itr; itr++)
	{
		ppp = inner_product(ndof, gg, gg);
		M_mat_vec_crs(M, M_Ptr, M_Col, dd, pp, ndof);
		rrr = inner_product(ndof, dd, pp);
		alphak = ppp / rrr;
		for (ii = 0; ii < ndof; ii++)
		{
			solution_vec[ii] += alphak * pp[ii];
			gg[ii] -= alphak * dd[ii];
		}
		qqq = inner_product(ndof, gg, dd);
		betak = qqq / rrr;
		for (ii = 0; ii < ndof; ii++)
			pp[ii] = gg[ii] - betak * pp[ii];
		istop = M_check_conv_CG(ndof, alphak, pp, eps, solution_vec);
		if (istop == 1)
			break;
	}
	printf("\titr %d\n", itr);

	// 対角スケーリング 後処理
	// for (i = 0; i < ndof; i++)
	// {
	// 	solution_vec[i] = solution_vec[i] * diag_scaling[i];
	// }
}

__host__ int RowCol_to_icount(int row, int col)
{
	for (int j = K_Whole_Ptr[row]; j < K_Whole_Ptr[row + 1]; j++)
	{
		if (K_Whole_Col[j] == col)
		{
			return j;
		}
		else if (K_Whole_Col[j] > col)
		{
			return -1;
		}
	}
	return -1;
}

// 前処理付共役勾配法により[K]{d}={f}を解く
__host__ void PCG_Solver(int ndof, int max_itetarion, double eps)
{
	int i, j, k;

	double *r = (double *)malloc(sizeof(double) * ndof);
	double *p = (double *)calloc(ndof, sizeof(double));
	double *y = (double *)malloc(sizeof(double) * ndof);
	double *r2 = (double *)calloc(ndof, sizeof(double));

	// 初期化
	for (i = 0; i < ndof; i++)
		sol_vec[i] = 0.0;

	// 前処理行列作成
	double *M = (double *)malloc(sizeof(double) * MAX_NON_ZERO);
	int *M_Ptr = (int *)malloc(sizeof(int) * MAX_K_WHOLE_SIZE + 1);
	int *M_Col = (int *)malloc(sizeof(int) * MAX_NON_ZERO);
	Make_M(M, M_Ptr, M_Col, ndof);

	// 第0近似解に対する残差の計算
	double *ax = (double *)calloc(ndof, sizeof(double));
	for (i = 0; i < ndof; i++)
	{
		for (j = K_Whole_Ptr[i]; j < K_Whole_Ptr[i + 1]; j++)
		{
			ax[i] += K_Whole_Val[j] * sol_vec[K_Whole_Col[j]];
			if (i != K_Whole_Col[j])
			{
				ax[K_Whole_Col[j]] += K_Whole_Val[j] * sol_vec[i];
			}
		}
	}
	for (i = 0; i < ndof; i++)
	{
		r[i] = rhs_vec[i] - ax[i];
	}
	free(ax);

	// 第0近似解に対する残差の計算
	// for (i = 0; i < ndof; i++)
	// {
	// 	r[i] = rhs_vec[i];
	// }

	// p_0 = (LDL^T)^-1 r_0 の計算 <- CG法で M = [[K^G, 0], [0, K^L]] とし，p_0 = (LDL^T)^-1 r_0 = M^-1 r_0
	CG(ndof, p, M, M_Ptr, M_Col, r);

	// double rr0 = inner_product(ndof, r, p), rr1;
	double rr0;
	double alpha, beta;

	double e = 0.0;
	for (k = 0; k < max_itetarion; k++)
	{
		// rr0 の計算
		rr0 = inner_product(ndof, r, p);

		// y = AP の計算
		for (i = 0; i < ndof; i++)
		{
			double *temp_array_K = (double *)calloc(ndof, sizeof(double));
			for (j = 0; j < ndof; j++)
			{
				int temp1;
				if (i <= j)
				{
					temp1 = RowCol_to_icount(i, j); // temp_array_K[i][j]
				}
				else if (i > j)
				{
					temp1 = RowCol_to_icount(j, i); // temp_array_K[i][j] = temp_array_K[j][i]
				}

				if (temp1 != -1)
				{
					temp_array_K[j] = K_Whole_Val[temp1];
				}
			}
			y[i] = inner_product(ndof, temp_array_K, p);
			free(temp_array_K);
		}

		// alpha = r*r/(P*AP)の計算
		double temp_scaler = inner_product(ndof, p, y);
		alpha = rr0 / temp_scaler;
		// printf("alpha %le\n", alpha);

		// 解x、残差rの更新
		for (i = 0; i < ndof; i++)
		{
			sol_vec[i] += alpha * p[i];
			r[i] -= alpha * y[i];
		}

		// (r*r)_(k+1)の計算
		CG(ndof, r2, M, M_Ptr, M_Col, r);

		// rr1 = inner_product(ndof, r, r2); // 旧
		// rr1 = inner_product(ndof, y, r2); // 新
		// printf("rr1 %le\n", rr1);

		// 収束判定 (||r||<=eps)
		// double rr1 = inner_product(ndof, y, r2);
		// e = sqrt(fabs(rr1));
		// if(e < eps)
		// {
		//     k++;
		//     break;
		// }

		// 収束判定 (CG法と同じ)
		double e1 = 0.0, e2 = 0.0;
		for (i = 0; i < ndof; i++)
		{
			e1 += p[i] * p[i];
			e2 += sol_vec[i] * sol_vec[i];
		}
		e = fabs(alpha) * sqrt(e1 / e2);
		if (e < eps)
		{
			k++;
			break;
		}

		// βの計算とPの更新
		// beta = rr1 / rr0; //旧
		// beta = - rr1 / temp_scaler; // 新
		beta = -inner_product(ndof, y, r2) / temp_scaler;

		for (i = 0; i < ndof; i++)
		{
			// p[i] = r2[i] - beta * p[i];
			p[i] = r2[i] + beta * p[i];
		}
		// printf("beta %le\n", beta);

		// (r*r)_(k+1)を次のステップのために確保しておく
		// rr0 = rr1;

		printf("itr %d\t", k);
		printf("eps %.15e", e);
		// if (rr1 < 0)
		// {
		// 	printf("\t rr1 < 0");
		// }
		printf("\n");
	}

	int max_itr_result = k;
	double eps_result = e;

	printf("\nndof = %d\n", ndof);
	printf("itr_result = %d\n", max_itr_result);
	printf("eps_result = %.15e\n", eps_result);

	free(r), free(p), free(y), free(r2);
	free(M), free(M_Ptr), free(M_Col);
}

////////////////////////////////////////////////////////////////////////
/////////////////基底関数
////////////////////////////////////////////////////////////////////////
// IGAの基底関数
__host__ void ShapeFunction1D(double Position_Data_param[DIMENSION], int j, int e)
{

	int ii;
	int p;

	// printf("shapefuc_Position_Data_param[%d]:%le\n", j,Position_Data_param[j]);

	for (ii = 0; ii < No_knot[Element_patch[e]][j]; ii++)
	{
		if (Position_Knots[Element_patch[e]][j][ii] == Position_Knots[Element_patch[e]][j][ii + 1])
		{
			Shape[j][ii][0] = 0.0;
		}
		else if (Position_Knots[Element_patch[e]][j][ii] != Position_Knots[Element_patch[e]][j][ii + 1] && Position_Knots[Element_patch[e]][j][ii] <= Position_Data_param[j] && Position_Data_param[j] < Position_Knots[Element_patch[e]][j][ii + 1])
		{
			Shape[j][ii][0] = 1.0;
		}
		else if (Position_Knots[Element_patch[e]][j][ii] != Position_Knots[Element_patch[e]][j][ii + 1] && Position_Knots[Element_patch[e]][j][ii + 1] == Position_Knots[Element_patch[e]][j][(No_knot[Element_patch[e]][j] - 1)] && Position_Knots[Element_patch[e]][j][ii] <= Position_Data_param[j] && Position_Data_param[j] <= Position_Knots[Element_patch[e]][j][ii + 1])
		{
			Shape[j][ii][0] = 1.0;
		}
		else
			Shape[j][ii][0] = 0.0;
		// printf("Shape[%d][%d][0]=%le   ",j,ii,Shape[j][ii][0]);
	}

	for (ii = 0; ii < No_knot[Element_patch[e]][j]; ii++)
	{
		for (p = 1; p <= Order[Element_patch[e]][j]; p++)
		{
			Shape[j][ii][p] = 0.0;
		}
	}
	double left_term, right_term;
	for (p = 1; p <= Order[Element_patch[e]][j]; p++)
	{
		for (ii = 0; ii < No_knot[Element_patch[e]][j]; ii++)
		{
			left_term = 0.0;
			right_term = 0.0;

			if ((Position_Data_param[j] - Position_Knots[Element_patch[e]][j][ii]) * Shape[j][ii][p - 1] == 0 && Position_Knots[Element_patch[e]][j][ii + p] - Position_Knots[Element_patch[e]][j][ii] == 0)
				left_term = 0.0;
			else
			{
				left_term = (Position_Data_param[j] - Position_Knots[Element_patch[e]][j][ii]) / (Position_Knots[Element_patch[e]][j][ii + p] - Position_Knots[Element_patch[e]][j][ii]) * Shape[j][ii][p - 1];
				// printf("else left\tleft_term;%le\n",left_term);
			}
			if ((Position_Knots[Element_patch[e]][j][ii + p + 1] - Position_Data_param[j]) * Shape[j][ii + 1][p - 1] == 0 && Position_Knots[Element_patch[e]][j][ii + p + 1] - Position_Knots[Element_patch[e]][j][ii + 1] == 0)
				right_term = 0.0;
			else
			{
				right_term = (Position_Knots[Element_patch[e]][j][ii + p + 1] - Position_Data_param[j]) / (Position_Knots[Element_patch[e]][j][ii + p + 1] - Position_Knots[Element_patch[e]][j][ii + 1]) * Shape[j][ii + 1][p - 1];
				// printf("else right\tright_term;%le\n",right_term);
			}
			Shape[j][ii][p] = left_term + right_term;
			// printf("Shape[%d][%d][%d]=%le\n",j,ii,p,Shape[j][ii][p]);
		}
	}
	// printf("order[%d]:%d\n",j,Order[Element_patch[e]][j] );
	double dleft_term, dright_term;
	for (ii = 0; ii < No_Control_point[Element_patch[e]][j] + 1; ii++)
	{
		// printf("No_Control_point[%d]=%d\n",j,No_Control_point[j] );
		dleft_term = 0.0;
		dright_term = 0.0;

		if (Order[Element_patch[e]][j] * Shape[j][ii][Order[Element_patch[e]][j] - 1] == 0 && Position_Knots[Element_patch[e]][j][ii + Order[Element_patch[e]][j]] - Position_Knots[Element_patch[e]][j][ii] == 0)
			dleft_term = 0.0;
		else
			dleft_term = Order[Element_patch[e]][j] / (Position_Knots[Element_patch[e]][j][ii + Order[Element_patch[e]][j]] - Position_Knots[Element_patch[e]][j][ii]) * Shape[j][ii][Order[Element_patch[e]][j] - 1];
		/*printf("test_Shape_left[%d][%d][%d]=%le\n", j,ii,Order[Element_patch[e]][j]-1,Shape[j][ii][Order[Element_patch[e]][j]-1]);
		printf("Position_Knots[Element_patch[e]][%d][%d]:%le\n", j,ii+Order[Element_patch[e]][j],Position_Knots[Element_patch[e]][j][ii+Order[Element_patch[e]][j]]);
		printf("Position_Knots[Element_patch[e]][%d][%d]:%le\n", j,ii,Position_Knots[Element_patch[e]][j][ii]);
		printf("dleft_term=%f\n",dleft_term );*/

		if (Order[Element_patch[e]][j] * Shape[j][ii + 1][Order[Element_patch[e]][j] - 1] == 0 && Position_Knots[Element_patch[e]][j][ii + Order[Element_patch[e]][j] + 1] - Position_Knots[Element_patch[e]][j][ii + 1] == 0)
			dright_term = 0.0;
		else
			dright_term = Order[Element_patch[e]][j] / (Position_Knots[Element_patch[e]][j][ii + Order[Element_patch[e]][j] + 1] - Position_Knots[Element_patch[e]][j][ii + 1]) * Shape[j][ii + 1][Order[Element_patch[e]][j] - 1];
		/*printf("test_Shape_right[%d][%d][%d]=%le\n", j,ii+1,Order[Element_patch[e]][j]-1,Shape[j][ii+1][Order[Element_patch[e]][j]-1]);
		printf("Position_Knots[%d][%d]:%le\n", j,ii+Order[Element_patch[e]][j]+1,Position_Knots[j][ii+Order[Element_patch[e]][j]+1]);
		printf("Position_Knots[%d][%d]:%le\n", j,ii+1,Position_Knots[j][ii+1]);
		printf("dright_term=%f\n",dright_term );*/

		dShape[j][ii] = dleft_term - dright_term;

		// printf("PP=%d\n",PP );

		// printf("dShape[%d][%d]= %f\n",j,ii,dShape[j][ii]);
	}
}

__host__ void ShapeFunc_from_paren(double Local_coord[DIMENSION], int j, int e)
{
	int i = 0;
	// printf("Local_coord[%d]:%le\n",j,Local_coord[j]);
	i = INC[Element_patch[e]][Controlpoint_of_Element[e][0]][j];
	// printf("El_No:%d\n",e );
	// printf("i:%d\n",i);
	// printf("Position_Knots[%d][%d][%d]:%le Position_Knots[%d][%d][%d]:%le\n"
	//         ,Element_patch[e],j,i+1,Position_Knots[Element_patch[e]][j][i+1]
	//         ,Element_patch[e],j,i,Position_Knots[Element_patch[e]][j][i]);
	Position_Data_param[j] = ((Position_Knots[Element_patch[e]][j][i + 1] - Position_Knots[Element_patch[e]][j][i]) * Local_coord[j] + (Position_Knots[Element_patch[e]][j][i + 1] + Position_Knots[Element_patch[e]][j][i])) / 2;
	// Position_Data_param[j] = ((Position_Knots[Element_patch[e]][j][i + 1] - Position_Knots[Element_patch[e]][j][i]) * Local_coord[j] + (Position_Knots[Element_patch[e]][j][i + 1] + Position_Knots[Element_patch[e]][j][i])) / 2;
	// printf("Position_Data_param[%d]:%le\n", j, Position_Data_param[j]);
}

__host__ double dShapeFunc_from_paren(int j, int e)
{
	int i;
	double dPosition_Data_param;

	i = INC[Element_patch[e]][Controlpoint_of_Element[e][0]][j];
	// printf("El_No:%d\n",e );
	// printf("i:%d\n",i);
	// printf("Position_Knots[Element_patch[e]][%d][%d]:%le Position_Knots[%d][%d]:%le\n", j,i+1,Position_Knots[j][i+1],j,i,Position_Knots[j][i]);
	dPosition_Data_param = (Position_Knots[Element_patch[e]][j][i + 1] - Position_Knots[Element_patch[e]][j][i]) / 2;
	// printf("dPosition_Data_param:%le\t",dPosition_Data_param);
	return dPosition_Data_param;
}

__host__ double Shape_func(int I_No, double Local_coord[DIMENSION], int El_No)
{

	int i, j;
	double R;
	double weight_func;
	weight_func = 0.0;
	// shape_func[]={0.0};

	// MAX_N_NODEでshape_funcを初期化していたところを修正
	// // for (i = 0; i < MAX_N_NODE; i++)
	// {
	// 	shape_func[i] = 1.0;
	// }

	// for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	// {
	// 	for (j = 0; j < DIMENSION; j++)
	// 	{
	// 		ShapeFunc_from_paren(Local_coord, j, El_No);
	// 		ShapeFunction1D(Position_Data_param, j, El_No);
	// 		shape_func[Controlpoint_of_Element[El_No][i]] *= Shape[j][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j]][Order[Element_patch[El_No]][j]];
	// 		//shape_func[Controlpoint_of_Element[El_No][i]] *= Shape[j][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j]][Order[Element_patch[El_No]][j]]; /*基底関数*/
	// 	    //printf("%d",shape_func[0][0]);
	//         //printf("Shape[%d][%d][%d]=%le\n",
	//         //        j,
	//         //        INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j],
	//         //        Order[Element_patch[El_No]][j],
	//         //        Shape[j][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j]][Order[Element_patch[El_No]][j]]);
	// 	}
	//     //printf("Controlpoint_of_element[%d][%d]=%d\n",
	//     //        El_No,i,Controlpoint_of_Element[El_No][i]);
	//     //printf("shape_func=%le\n",shape_func[Controlpoint_of_Element[El_No][i]]);
	// 	//printf("weight[%d];%f\n",Controlpoint_of_Element[El_No][i],Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION]);
	// 	weight_func += shape_func[Controlpoint_of_Element[El_No][i]] * Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION];
	// }

	int Total_CP = 0;

	for (i = 0; i < Total_mesh; i++)
	{
		Total_CP += Total_Control_Point_on_mesh[i];
	}
	for (i = 0; i < Total_CP; i++)
	{
		shape_func[i] = 1.0;
	}

	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			ShapeFunc_from_paren(Local_coord, j, El_No);
			ShapeFunction1D(Position_Data_param, j, El_No);
			shape_func[Controlpoint_of_Element[El_No][i]] *= Shape[j][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j]][Order[Element_patch[El_No]][j]];
		}
		weight_func += shape_func[Controlpoint_of_Element[El_No][i]] * Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION];
	}
	// printf("weight_func_inShapefunc=%le\n",weight_func);
	if (I_No < No_Control_point_ON_ELEMENT[Element_patch[El_No]])
		R = shape_func[Controlpoint_of_Element[El_No][I_No]] * Node_Coordinate[Controlpoint_of_Element[El_No][I_No]][DIMENSION] / weight_func;

	else
		R = ERROR;
	// printf("R=%le\n",R);
	return R;
}

__host__ void NURBS_deriv(double Local_coord[DIMENSION], int El_No)
{
	double weight_func;
	// double shape_func[100][50];

	double dWeight_func1;
	double dWeight_func2;

	int i, j;
	// int ii;

	// for(ii = 0; ii < NN+1; ii++){
	// printf("NdShape3[%d]= %f\n",ii,dShape3[ii]);
	// }

	// for (ii = 0; ii < NN+1; ii++)printf("NdShape1[%d]= %f\n",ii,dShape1[ii]);
	// for (jj = 0; jj < MM+1; jj++)printf("NdShape2[%d]= %f\n",jj,dShape2[jj]);
	// printf("\n");

	// //for (i = 0; i < Total_Control_Point_to_mesh[Total_mesh]; i++)	//"Total_Control_Point_to_mesh[Total_mesh]"仮
	//"Total_Control_Point_to_mesh[Total_mesh]"仮
	// MAX_N_NODEでshape_funcを初期化していたところを修正
	// for (i = 0; i < MAX_N_NODE; i++)
	// {
	// 	shape_func[i] = 1.0;
	// }

	int Total_CP = 0;

	for (i = 0; i < Total_mesh; i++)
	{
		Total_CP += Total_Control_Point_on_mesh[i];
	}
	for (i = 0; i < Total_CP; i++)
	{
		shape_func[i] = 1.0;
	}

	weight_func = 0.0;

	dWeight_func1 = 0.0;
	dWeight_func2 = 0.0;

	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			// printf("b_before_shape_func:%lf\n",
			//		shape_func[Controlpoint_of_Element[El_No][i]]);
			ShapeFunc_from_paren(Local_coord, j, El_No);
			ShapeFunction1D(Position_Data_param, j, El_No);
			// printf("before_shape_func:%lf\n",
			//		shape_func[Controlpoint_of_Element[El_No][i]]);
			shape_func[Controlpoint_of_Element[El_No][i]] *= Shape[j][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j]][Order[Element_patch[El_No]][j]];
			// printf("Shape[%d][%d][%d]:%le\n",
			//         j,
			//         INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j],
			//         Order[Element_patch[El_No]][j],
			//         Shape[j][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][j]][Order[Element_patch[El_No]][j]]);
			// printf("after_shape_func:%lf\n",
			//		shape_func[Controlpoint_of_Element[El_No][i]]);
		}
		weight_func += shape_func[Controlpoint_of_Element[El_No][i]] * Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION];
		// printf("Node_Coordinate[%d][%d]:%le\n", Controlpoint_of_Element[El_No][i],DIMENSION,Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION]);
		// printf("shape_func[%d]:%le\n",Controlpoint_of_Element[El_No][i],shape_func[Controlpoint_of_Element[El_No][i]]);
		// printf("weight_func:%le\n", weight_func);
	}
	// printf("weight_func:%le\n", weight_func);
	// for(jj=0;jj<NN;jj++) for(kk=0;kk<MM;kk++)shape_func[jj][kk] = Shape1[jj][PP]*Shape2[kk][QQ]*Node_Coordinate[MM*jj+kk][2]/weight_func;
	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		dWeight_func1 += dShape[0][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][0]] * Shape[1][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][1]][Order[Element_patch[El_No]][1]] * Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION];
		dWeight_func2 += Shape[0][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][0]][Order[Element_patch[El_No]][0]] * dShape[1][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][1]] * Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION];
	}
	// printf("dWeight_func1:%le dWeight_func2:%le\n",dWeight_func1,dWeight_func2);
	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		dShape_func1[Controlpoint_of_Element[El_No][i]] = Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION] * (weight_func * dShape[0][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][0]] * Shape[1][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][1]][Order[Element_patch[El_No]][1]] - dWeight_func1 * shape_func[Controlpoint_of_Element[El_No][i]]) / (weight_func * weight_func);
		dShape_func2[Controlpoint_of_Element[El_No][i]] = Node_Coordinate[Controlpoint_of_Element[El_No][i]][DIMENSION] * (weight_func * Shape[0][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][0]][Order[Element_patch[El_No]][0]] * dShape[1][INC[Element_patch[El_No]][Controlpoint_of_Element[El_No][i]][1]] - dWeight_func2 * shape_func[Controlpoint_of_Element[El_No][i]]) / (weight_func * weight_func);
		// printf("NURBS_deriv;Controlpoint_of_Element[%d][%d]:%d\n",El_No,i,Controlpoint_of_Element[El_No][i]);
		// printf("dShape_func1[%d]:%le\n",Controlpoint_of_Element[El_No][i],dShape_func1[Controlpoint_of_Element[El_No][i]]);
		// printf("dShape_func2[%d]:%le\n",Controlpoint_of_Element[El_No][i],dShape_func2[Controlpoint_of_Element[El_No][i]]);
	}
}

__host__ double dShape_func(int I_No, int xez, double Local_coord[DIMENSION], int El_No)
{
	double dR;

	// printf("El_No=%d\n",El_No);

	NURBS_deriv(Local_coord, El_No);

	if (xez != 0 && xez != 1)
		dR = ERROR;

	else if (I_No < No_Control_point_ON_ELEMENT[Element_patch[El_No]])
	{
		if (xez == 0)
		{
			// dR = dShape_func1[Controlpoint_of_Element[El_No][I_No]] * dShapeFunc_from_paren(xez, El_No);
			dR =
				dShape_func1[Controlpoint_of_Element[El_No][I_No]] * dShapeFunc_from_paren(xez, El_No);
			// printf("dShape_func1[%d]:%le\n",Controlpoint_of_Element[El_No][I_No],dShape_func1[Controlpoint_of_Element[El_No][I_No]]);
		}
		else if (xez == 1)
		{
			// dR = dShape_func2[Controlpoint_of_Element[El_No][I_No]] * dShapeFunc_from_paren(xez, El_No);
			dR =
				dShape_func2[Controlpoint_of_Element[El_No][I_No]] * dShapeFunc_from_paren(xez, El_No);
			// printf("dShape_func2[%d]:%le\n",Controlpoint_of_Element[El_No][I_No],dShape_func2[Controlpoint_of_Element[El_No][I_No]]);
		}
		// printf("dR%lf\n",dR);
	}

	else
		dR = ERROR;
	// printf("dR:%le\n",dR);

	// printf("I_No=%d xez=%d dR=%le\t", I_No, xez, dR );
	// printf("Controlpoint_of_Element[%d][%d]:%d\t",El_No,I_No,Controlpoint_of_Element[El_No][I_No]);
	// printf("dShape_func1:%le\t",dShape_func1[Controlpoint_of_Element[El_No][I_No]]);
	// printf("dShape_func2:%le\n",dShape_func2[Controlpoint_of_Element[El_No][I_No]]);

	/*for (i = 0; i < DIMENSION; i++) {
	printf("Local_coord[%d]=%lf\n",i,Local_coord[i] );
}*/
	// printf("dR:%le\t", dR);
	return dR;
}
/*
//形状関数
double N_Quad_4(int I_No, double Local_coord[DIMENSION] )
{
	double N;
	if(I_No==0) N= (1.0+Local_coord[0])*(1.0-Local_coord[1])/4.0;
	else if(I_No==1) N = (1.0+Local_coord[0])*(1.0+Local_coord[1])/4.0;
	else if(I_No==2) N = (1.0-Local_coord[0])*(1.0+Local_coord[1])/4.0;
	else if(I_No==3) N = (1.0-Local_coord[0])*(1.0-Local_coord[1])/4.0;
	else N = ERROR;
	return N;
}

//形状関数の偏微分（I_No:節点番号 xez:偏微分の分母部分0ξ1η2ζ）
double dN_Quad_4(int I_No, double Local_coord[DIMENSION], int xez)
{
	double dN;
	if(xez!=0 && xez!=1) dN = ERROR;

	else if(I_No==0)
		{
		if( xez == 0 )	    dN = (1.0-Local_coord[1])/4.0;
		else if( xez == 1 ) dN = (1.0+Local_coord[0])*(-1)/4.0;
		}

	else if(I_No==1)
		{
		if( xez == 0 )      dN = (1.0+Local_coord[1])/4.0;
		else if( xez == 1 ) dN = (1.0+Local_coord[0])/4.0;
		}

	else if(I_No==2)
		{
		if( xez == 0 )      dN = (1.0+Local_coord[1])*(-1)/4.0;
		else if( xez == 1 ) dN = (1.0-Local_coord[0])/4.0;
		}

	else if(I_No==3)
		{
		if( xez == 0 )      dN = (1.0-Local_coord[1])*(-1)/4.0;
		else if( xez == 1 ) dN = (1.0-Local_coord[0])*(-1)/4.0;
		}

	else dN = ERROR;

	return dN;
}
*/

//逆行列を元の行列に代入
__host__ double InverseMatrix_2D(double M[2][2])
{
	int i, j;
	double a[2][2];
	double det = M[0][0] * M[1][1] - M[0][1] * M[1][0];

	if (det == 0)
		return ERROR;

	for (i = 0; i < 2; i++)
	{
		for (j = 0; j < 2; j++)
			a[i][j] = M[i][j];
	}
	M[0][0] = a[1][1] / det;
	M[0][1] = a[0][1] * (-1) / det;
	M[1][0] = a[1][0] * (-1) / det;
	M[1][1] = a[0][0] / det;
	// printf("det;%le\n", det);
	return det;
}

__host__ double InverseMatrix_3X3(double M[3][3])
{
	int i, j;
	double a[3][3];
	double det = M[0][0] * M[1][1] * M[2][2] + M[1][0] * M[2][1] * M[0][2] + M[2][0] * M[0][1] * M[1][2] - M[0][0] * M[2][1] * M[1][2] - M[2][0] * M[1][1] * M[0][2] - M[1][0] * M[0][1] * M[2][2];

	if (det == 0)
		return ERROR;

	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
			a[i][j] = M[i][j];
	}
	M[0][0] = (a[1][1] * a[2][2] - a[1][2] * a[2][1]) / det;
	M[0][1] = (a[0][2] * a[2][1] - a[0][1] * a[2][2]) / det;
	M[0][2] = (a[0][1] * a[1][2] - a[0][2] * a[1][1]) / det;
	M[1][0] = (a[1][2] * a[2][0] - a[1][0] * a[2][2]) / det;
	M[1][1] = (a[0][0] * a[2][2] - a[0][2] * a[2][0]) / det;
	M[1][2] = (a[0][2] * a[1][0] - a[0][0] * a[1][2]) / det;
	M[2][0] = (a[1][0] * a[2][1] - a[1][1] * a[2][0]) / det;
	M[2][1] = (a[0][1] * a[2][0] - a[0][0] * a[2][1]) / det;
	M[2][2] = (a[0][0] * a[1][1] - a[0][1] * a[1][0]) / det;
	// printf("det;%le\n", det);
	return det;
}

////////////////////////////////////////////////////////////////////////
/////////////////Newton-Raphson法
////////////////////////////////////////////////////////////////////////
// from NURBSviewer
__host__ double BasisFunc(double *knot_vec, int knot_index, int order, double xi,
						  double *output, double *d_output)
{
	int p, j;
	double sum1 = 0.0;
	double sum2 = 0.0;
	double temp_basis[order][order];
	(*output) = 0.0;
	(*d_output) = 0.0;

	if (knot_vec[knot_index] <= xi && knot_vec[knot_index + order + 1] >= xi)
	{
		for (j = 0; j <= order; j++)
		{
			if ((knot_vec[knot_index + j] <= xi) && (xi <= knot_vec[knot_index + j + 1]))
			{
				temp_basis[j][0] = 1.0;
			}
			else
			{
				temp_basis[j][0] = 0.0;
			}
		}

		if (order > 0)
		{
			for (p = 1; p <= order; p++)
			{
				for (j = 0; j <= order - p; j++)
				{
					sum1 = 0.0;
					sum2 = 0.0;
					if ((knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) != 0.0)
					{
						sum1 = (xi - knot_vec[knot_index + j]) / (knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) * temp_basis[j][p - 1];
					}
					if ((knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) != 0.0)
					{
						sum2 = (knot_vec[knot_index + j + p + 1] - xi) / (knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) * temp_basis[j + 1][p - 1];
					}
					temp_basis[j][p] = sum1 + sum2;
				}
			}
			sum1 = 0.0;
			sum2 = 0.0;
			if ((knot_vec[knot_index + order] - knot_vec[knot_index]) != 0.0)
			{
				sum1 = order / (knot_vec[knot_index + order] - knot_vec[knot_index]) * temp_basis[0][order - 1];
			}
			if ((knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) != 0.0)
			{
				sum2 = order / (knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) * temp_basis[1][order - 1];
			}
		}
		(*output) = temp_basis[0][order];
		(*d_output) = sum1 - sum2;
	}
	return (*output);
}

__host__ double rBasisFunc(double *knot_vec, int knot_index,
						   int order, double xi,
						   double *output, double *d_output)
{
	int p, j;
	double sum1 = 0.0;
	double sum2 = 0.0;
	double temp_basis[order][order];
	(*output) = 0.0;
	(*d_output) = 0.0;

	if (knot_vec[knot_index] <= xi && xi <= knot_vec[knot_index + order + 1])
	{
		if (knot_index == 0)
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[j] <= xi) && (xi <= knot_vec[j + 1]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}
		else
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[knot_index + j] < xi) && (xi <= knot_vec[knot_index + j + 1]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}

		if (order > 0)
		{
			for (p = 1; p <= order; p++)
			{
				for (j = 0; j <= order - p; j++)
				{
					sum1 = 0.0;
					sum2 = 0.0;
					if ((knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) != 0.0)
					{
						sum1 = (xi - knot_vec[knot_index + j]) / (knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) * temp_basis[j][p - 1];
					}
					if ((knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) != 0.0)
					{
						sum2 = (knot_vec[knot_index + j + p + 1] - xi) / (knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) * temp_basis[j + 1][p - 1];
					}
					temp_basis[j][p] = sum1 + sum2;
				}
			}
			sum1 = 0.0;
			sum2 = 0.0;

			// for (int temp_i = 0; temp_i < No_knot[0][0]; temp_i++)
			// {
			// 	printf("knot_vec[%d] = %f\n", temp_i, knot_vec[temp_i]);
			// }

			if ((knot_vec[knot_index + order] - knot_vec[knot_index]) != 0.0)
			{
				sum1 = order / (knot_vec[knot_index + order] - knot_vec[knot_index]) * temp_basis[0][order - 1];
			}
			if ((knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) != 0.0)
			{
				sum2 = order / (knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) * temp_basis[1][order - 1];
			}
		}
		(*output) = temp_basis[0][order];
		(*d_output) = sum1 - sum2;
	}
	return (*output);
}

__host__ double lBasisFunc(double *knot_vec, int knot_index,
						   int cntl_p_n, int order, double xi,
						   double *output, double *d_output)
{
	int p, j;
	double sum1 = 0.0;
	double sum2 = 0.0;
	double temp_basis[order][order];
	(*output) = 0.0;
	(*d_output) = 0.0;

	if (knot_vec[knot_index] <= xi && xi <= knot_vec[knot_index + order + 1])
	{
		if (knot_index == cntl_p_n - 1)
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[cntl_p_n - 1 + j] <= xi) && (xi <= knot_vec[cntl_p_n + j]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}
		else
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[knot_index + j] <= xi) && (xi < knot_vec[knot_index + j + 1]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}

		if (order > 0)
		{
			for (p = 1; p <= order; p++)
			{
				for (j = 0; j <= order - p; j++)
				{
					sum1 = 0.0;
					sum2 = 0.0;
					if ((knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) != 0.0)
					{
						sum1 = (xi - knot_vec[knot_index + j]) / (knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) * temp_basis[j][p - 1];
					}
					if ((knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) != 0.0)
					{
						sum2 = (knot_vec[knot_index + j + p + 1] - xi) / (knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) * temp_basis[j + 1][p - 1];
					}
					temp_basis[j][p] = sum1 + sum2;
				}
			}
			sum1 = 0.0;
			sum2 = 0.0;
			if ((knot_vec[knot_index + order] - knot_vec[knot_index]) != 0.0)
			{
				sum1 = order / (knot_vec[knot_index + order] - knot_vec[knot_index]) * temp_basis[0][order - 1];
			}
			if ((knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) != 0.0)
			{
				sum2 = order / (knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) * temp_basis[1][order - 1];
			}
		}
		(*output) = temp_basis[0][order];
		(*d_output) = sum1 - sum2;
	}
	return (*output);
}

__host__ double NURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
							  double *cntl_px, double *cntl_py,
							  int cntl_p_n_xi, int cntl_p_n_eta,
							  double *weight, int order_xi, int order_eta,
							  double xi, double eta,
							  double *output_x, double *output_y,
							  double *output_dxi_x, double *output_deta_x,
							  double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	// int index_max_xi = cntl_p_n_xi; //2020_09_12
	int index_max_xi = cntl_p_n_xi - 1; // 2020_09_12
	int index_min_eta = 0;
	// int index_max_eta = cntl_p_n_eta; //2020_09_12
	int index_max_eta = cntl_p_n_eta - 1; // 2020_09_12

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] > xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi; // 2020_09_12

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] > eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta; // 2020_09_12

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		BasisFunc(input_knot_vec_xi, i, order_xi, xi,
				  &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			BasisFunc(input_knot_vec_eta, j, order_eta, eta,
					  &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}

__host__ double rNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
							   double *cntl_px, double *cntl_py,
							   int cntl_p_n_xi, int cntl_p_n_eta,
							   double *weight, int order_xi, int order_eta,
							   double xi, double eta,
							   double *output_x, double *output_y,
							   double *output_dxi_x, double *output_deta_x,
							   double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		rBasisFunc(input_knot_vec_xi, i, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			rBasisFunc(input_knot_vec_eta, j, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}

__host__ double lNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
							   double *cntl_px, double *cntl_py,
							   int cntl_p_n_xi, int cntl_p_n_eta,
							   double *weight, int order_xi, int order_eta,
							   double xi, double eta,
							   double *output_x, double *output_y,
							   double *output_dxi_x, double *output_deta_x,
							   double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		lBasisFunc(input_knot_vec_xi, i,
				   cntl_p_n_xi, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			lBasisFunc(input_knot_vec_eta, j,
					   cntl_p_n_eta, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}

__host__ double rlNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
								double *cntl_px, double *cntl_py,
								int cntl_p_n_xi, int cntl_p_n_eta,
								double *weight, int order_xi, int order_eta,
								double xi, double eta,
								double *output_x, double *output_y,
								double *output_dxi_x, double *output_deta_x,
								double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		rBasisFunc(input_knot_vec_xi, i, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			lBasisFunc(input_knot_vec_eta, j,
					   cntl_p_n_eta, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}

__host__ double lrNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
								double *cntl_px, double *cntl_py,
								int cntl_p_n_xi, int cntl_p_n_eta,
								double *weight, int order_xi, int order_eta,
								double xi, double eta,
								double *output_x, double *output_y,
								double *output_dxi_x, double *output_deta_x,
								double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		lBasisFunc(input_knot_vec_xi, i,
				   cntl_p_n_xi, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			rBasisFunc(input_knot_vec_eta, j, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}

//算出したローカルパッチ各要素の頂点の物理座標のグローバルパッチでの(xi,eta)算出
__host__ int Calc_xi_eta(double px, double py,
						 double *input_knot_vec_xi, double *input_knot_vec_eta,
						 int cntl_p_n_xi, int cntl_p_n_eta, int order_xi, int order_eta,
						 double *output_xi, double *output_eta)
{
	double temp_xi, temp_eta;
	double temp_x, temp_y;
	double temp_matrix[2][2];
	double temp_dxi, temp_deta;
	// double temp_tol_x = DBL_MAX;
	// double temp_tol_y = DBL_MAX;
	double temp_tol_x, temp_tol_y;

	(*output_xi) = 0;
	(*output_eta) = 0;

	int i;
	// int repeat = 1000;
	// double tol = 10e-8;
	// int repeat = 10000;
	int repeat = 100;
	double tol = 10e-14;

	//初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	// printf("r_temp_xi_eta % 1.8e % 1.8e\n", temp_xi, temp_eta);

	for (i = 0; i < repeat; i++)
	{
		rNURBS_surface(Position_Knots[0][0], Position_Knots[0][1],
					   Control_Coord[0], Control_Coord[1],
					   No_Control_point[0][0], No_Control_point[0][1],
					   Control_Weight, Order[0][0], Order[0][1],
					   temp_xi, temp_eta,
					   &temp_x, &temp_y,
					   &temp_matrix[0][0], &temp_matrix[0][1],
					   &temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		// printf("xi_0:  % 1.8e\n", temp_xi);
		// printf("eta_0: % 1.8e\n", temp_eta);
		// printf("px: % 1.8e\n",px);
		// printf("temp_x: % 1.8e\n",temp_x);
		// printf("py: % 1.8e\n",py);
		// printf("temp_y: % 1.8e\n",temp_y);
		// printf("temp_tol_x:  % 1.8e\n", temp_tol_x);
		// printf("temp_tol_y:  % 1.8e\n", temp_tol_y);

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol)
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			// printf("r_xi:  % 1.8e\n", temp_xi);
			// printf("r_eta: % 1.8e\n", temp_eta);

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// printf("r_xi:  % 1.8e\n", temp_xi);
		// printf("r_eta: % 1.8e\n", temp_eta);
		// printf("i=%d\n",i);

		// double temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}

	//初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	// printf("l_temp_xi_eta % 1.8e % 1.8e\n", temp_xi, temp_eta);

	for (i = 0; i < repeat; i++)
	{
		lNURBS_surface(Position_Knots[0][0], Position_Knots[0][1],
					   Control_Coord[0], Control_Coord[1],
					   No_Control_point[0][0], No_Control_point[0][1],
					   Control_Weight, Order[0][0], Order[0][1],
					   temp_xi, temp_eta,
					   &temp_x, &temp_y,
					   &temp_matrix[0][0], &temp_matrix[0][1],
					   &temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol)
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			// printf("l_xi:  % 1.8e\n", temp_xi);
			// printf("l_eta: % 1.8e\n", temp_eta);
			// printf("i=%d\n",i);

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// double temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}

	//初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	// printf("rl_temp_xi_eta % 1.8e % 1.8e\n", temp_xi, temp_eta);

	for (i = 0; i < repeat; i++)
	{
		rlNURBS_surface(Position_Knots[0][0], Position_Knots[0][1],
						Control_Coord[0], Control_Coord[1],
						No_Control_point[0][0], No_Control_point[0][1],
						Control_Weight, Order[0][0], Order[0][1],
						temp_xi, temp_eta,
						&temp_x, &temp_y,
						&temp_matrix[0][0], &temp_matrix[0][1],
						&temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol)
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			// printf("rl_xi:  % 1.8e\n", temp_xi);
			// printf("rl_eta: % 1.8e\n", temp_eta);
			// printf("i=%d\n",i);

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// double temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}

	//初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	// printf("lr_temp_xi_eta % 1.8e % 1.8e\n", temp_xi, temp_eta);

	for (i = 0; i < repeat; i++)
	{
		lrNURBS_surface(Position_Knots[0][0], Position_Knots[0][1],
						Control_Coord[0], Control_Coord[1],
						No_Control_point[0][0], No_Control_point[0][1],
						Control_Weight, Order[0][0], Order[0][1],
						temp_xi, temp_eta,
						&temp_x, &temp_y,
						&temp_matrix[0][0], &temp_matrix[0][1],
						&temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol)
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			// printf("lr_xi:  % 1.8e\n", temp_xi);
			// printf("lr_eta: % 1.8e\n", temp_eta);
			// printf("i=%d\n",i);

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// double temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}

	// printf("% 1.8e % 1.8e\n", temp_x, temp_y);
	return 0;
}

////////////////////////////////////////////////////////////////////////
/////////////////要素剛性マトリックス
////////////////////////////////////////////////////////////////////////
__host__ int Jacobian(int El_No, double a[DIMENSION][DIMENSION], double Local_coord[DIMENSION], double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION])
{
	int i, j, k;
	// printf("El_No_jacobi:%d\n",El_No);
	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			a[i][j] = 0.0;
			for (k = 0; k < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; k++)
			{
				// printf("Local_coord[%d]:%le\n",j,Local_coord[j] );
				a[i][j] += dShape_func(k, j, Local_coord, El_No) * X[k][i];
				// printf(" X[%d][%d]=%lf\t",k,i, X[k][i] );
				// printf("k=%d a[%d][%d]:%le\n",k,i,j,a[i][j]);
			}
			// printf("<<<最終a[%d][%d]:%le>>>\n",i,j,a[i][j]);
		}
	}
	/*for (i = 0; i < DIMENSION; i++) {
		for (j = 0; j < DIMENSION; j++) {
			printf("a[%d][%d]:%le\n",i,j,a[i][j]);
		}
	}*/
	return 0;
}

// Bマトリックスを求める関数
__host__ int Make_B_Matrix(int El_No,
						   double B[D_MATRIX_SIZE][MAX_KIEL_SIZE],
						   double Local_coord[DIMENSION],
						   double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION],
						   double *J)
{
	double a[DIMENSION][DIMENSION], b[DIMENSION][MAX_NO_CCpoint_ON_ELEMENT];

	int i, j, k;

	Jacobian(El_No, a, Local_coord, X);

	*J = InverseMatrix_2D(a);
	// printf("B_Matri_J:%le\n",*J);
	if (*J <= 0)
		return -999;

	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++)
		{
			b[i][j] = 0.0;
			for (k = 0; k < DIMENSION; k++)
			{
				b[i][j] += a[k][i] * dShape_func(j, k, Local_coord, El_No);
			}
		}
	}

	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		B[0][2 * i] = b[0][i];
		B[0][2 * i + 1] = 0.0;
		B[1][2 * i] = 0.0;
		B[1][2 * i + 1] = b[1][i];
		B[2][2 * i] = b[1][i];
		B[2][2 * i + 1] = b[0][i];
	}
	/*for (i = 0; i < D_MATRIX_SIZE; i++)for (j = 0; j < KIEL_SIZE; j++) {
		printf("B[%d][%d]_B_mat:%le\n",i,j,B[i][j]);
	}*/
	return 0;
}

//変位勾配を求めるためのマトリクスb_gradを求める関数
__host__ int Make_b_grad_Matrix(int El_No, double b_grad[DIMENSION * DIMENSION][2 * MAX_NO_CCpoint_ON_ELEMENT], double Local_coord[DIMENSION],
								double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double *J)
{
	double a[DIMENSION][DIMENSION], b[DIMENSION][MAX_NO_CCpoint_ON_ELEMENT];
	int i, j, k;

	// printf("El_No : %d\n", El_No);
	// printf("No_Control_point_ON_ELEMENT[Element_patch[El_No]] : %d\n", No_Control_point_ON_ELEMENT[Element_patch[El_No]]);

	for (i = 0; i < DIMENSION * DIMENSION; i++)
	{
		for (j = 0; j < 2 * No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++)
		{
			b_grad[i][j] = 0.0;
		}
	}

	Jacobian(El_No, a, Local_coord, X);

	*J = InverseMatrix_2D(a);

	if (*J <= 0)
		return -999;

	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++)
		{
			b[i][j] = 0.0;
			for (k = 0; k < DIMENSION; k++)
			{
				b[i][j] += a[k][i] * dShape_func(j, k, Local_coord, El_No);
			}
		}
	}

	/*３次元にする場合は修正が必要*/
	for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++)
	{
		b_grad[0][2 * j] = b[0][j];
		b_grad[0][2 * j + 1] = 0.0; //   𝜕N1/𝜕x1    0.0      𝜕N2/𝜕x1    0.0      𝜕N3/𝜕x1      0.0    𝜕N4/𝜕x1    0.0   ...
		b_grad[1][2 * j] = b[1][j];
		b_grad[1][2 * j + 1] = 0.0; //   𝜕N1/𝜕x2    0.0      𝜕N2/𝜕x2    0.0      𝜕N3/𝜕x2      0.0    𝜕N4/𝜕x2    0.0   ...
		b_grad[2][2 * j] = 0.0;
		b_grad[2][2 * j + 1] = b[0][j]; //     0.0      𝜕N1/𝜕x1    0.0      𝜕N2/𝜕x1    0.0      𝜕N3/𝜕x1    0.0    𝜕N4/𝜕x1 ...
		b_grad[3][2 * j] = 0.0;
		b_grad[3][2 * j + 1] = b[1][j]; //     0.0      𝜕N1/𝜕x2    0.0      𝜕N2/𝜕x2    0.0      𝜕N3/𝜕x2    0.0    𝜕N4/𝜕x2 ...
	}

	/*for (i = 0; i < DIMENSION * DIMENSION; i++){
		for (j = 0; j < 2 * No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++) {
			printf("b_grad[%d][%d] =  %le\t",i, j, b_grad[i][j]);
		}
		printf("\n");
	}*/
	return 0;
}

//応力歪マトリックス
__host__ int Make_D_Matrix_2D(double D[D_MATRIX_SIZE][D_MATRIX_SIZE], double E, double nu, int DM)
{
	int i, j;

	if (DM == 0) //平面応力状態
	{
		// printf("E:%le nu:%le\n",E,nu);
		double Eone = E / (1.0 - nu * nu);
		double D1[D_MATRIX_SIZE][D_MATRIX_SIZE] = {{Eone, nu * Eone, 0}, {nu * Eone, Eone, 0}, {0, 0, (1 - nu) / 2 * Eone}};

		for (i = 0; i < D_MATRIX_SIZE; i++)
			for (j = 0; j < D_MATRIX_SIZE; j++)
				D[i][j] = D1[i][j];
	}

	else if (DM == 1) //平面ひずみ状態(2Dの場合はこっち)
	{
		// printf("E:%le nu:%le\n",E,nu);
		double Eone = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
		double D1[D_MATRIX_SIZE][D_MATRIX_SIZE] = {{Eone, nu / (1.0 - nu) * Eone, 0}, {nu / (1.0 - nu) * Eone, Eone, 0}, {0, 0, (1 - 2 * nu) / 2 / (1.0 - nu) * Eone}};

		for (i = 0; i < D_MATRIX_SIZE; i++)
			for (j = 0; j < D_MATRIX_SIZE; j++)
				D[i][j] = D1[i][j];
	}

	else
		return ERROR;

	return 0;
}

//ガウスの数値積分法の中身
__host__ int BDBJ(double B[D_MATRIX_SIZE][MAX_KIEL_SIZE], double D[D_MATRIX_SIZE][D_MATRIX_SIZE], double J, double K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE])
{
	int i, j, k;
	double BD[MAX_KIEL_SIZE][D_MATRIX_SIZE];

	//[B]T[D][B]Jの計算
	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < D_MATRIX_SIZE; j++)
		{
			BD[i][j] = 0;
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				// printf("B[%d][%d]=%le D[%d][%d]=%le\n", k,i,B[k][i],k,j,D[k][j] );
				// printf("B[%d][%d]=%le\n", k,i,B[k][i]);
				BD[i][j] += B[k][i] * D[k][j];
				// printf("BD[%d][%d]=%e\n",i,j,BD[i][j] );
			}
		}
	}
	// for( j = 0; j < D_MATRIX_SIZE; j++ )for( i = 0; i < KIEL_SIZE; i++ )printf("B[%d][%d]=%le\n",j,i,B[j][i] );
	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			K_EL[i][j] = 0;
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				K_EL[i][j] += BD[i][k] * B[k][j];
			}
			K_EL[i][j] *= J;
		}
	}
	return 0;
}

//結合ガウスの数値積分法の中身
__host__ int coupled_BDBJ(double B[D_MATRIX_SIZE][MAX_KIEL_SIZE],
						  double D[D_MATRIX_SIZE][D_MATRIX_SIZE],
						  double BG[D_MATRIX_SIZE][MAX_KIEL_SIZE],
						  double J, double K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE])
{
	int i, j, k;
	double BD[MAX_KIEL_SIZE][D_MATRIX_SIZE];

	//[B]T[D][B]Jの計算
	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < D_MATRIX_SIZE; j++)
		{
			BD[i][j] = 0;
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				// printf("B[%d][%d]=%le D[%d][%d]=%le\n", k,i,B[k][i],k,j,D[k][j] );
				// printf("B[%d][%d]=%le\n", k,i,B[k][i]);
				BD[i][j] += BG[k][i] * D[k][j];
				// printf("BD[%d][%d]=%e\n",i,j,BD[i][j] );
			}
		}
	}
	// for( j = 0; j < D_MATRIX_SIZE; j++ )for( i = 0; i < KIEL_SIZE; i++ )printf("B[%d][%d]=%le\n",j,i,B[j][i] );
	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			K_EL[i][j] = 0;
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				K_EL[i][j] += BD[i][k] * B[k][j];
			}
			K_EL[i][j] *= J;
		}
	}
	return 0;
}

//要素合成マトリックス
__host__ int Make_K_EL(int El_No, double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE], double E, double nu, int DM)
{
	int i, j, k, l;

	double K1[MAX_KIEL_SIZE][MAX_KIEL_SIZE], B[D_MATRIX_SIZE][MAX_KIEL_SIZE];
	double D[D_MATRIX_SIZE][D_MATRIX_SIZE];
	double J = 0.0;
	double J_test = 0.0;

	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			K_EL[i][j] = 0.0;
		}
	}

	Make_D_Matrix_2D(D, E, nu, DM);

	// for (i = 0; i < POW_Ng; i++)
	for (i = 0; i < GP_2D; i++)
	{

		// printf("i=%d\n",i );
		Make_B_Matrix(El_No, B, Gxi[i], X, &J);

		BDBJ(B, D, J, K1);
		J_test += J;
		for (k = 0; k < KIEL_SIZE; k++)
		{
			for (l = 0; l < KIEL_SIZE; l++)
			{
				K_EL[k][l] += w[i] * K1[k][l];
			}
		} // printf("w[%d]=%f\n",i,w[i]);
	}
	// printf("El_No:%d J_test=%e\n", El_No, J_test);
	// printf("G=%f\n",G );
	/*for ( k = 0; k < KIEL_SIZE; k++) {
		for ( l = 0; l < KIEL_SIZE; l++) {
			printf("K_EL[%d][%d]:%le\n",k,l,K_EL[k][l]);
		}
	}*/

	return 0;
}

//結合要素剛性マトリックス
__host__ int Make_coupled_K_EL(int El_No_loc, int El_No_glo,
							   double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION],
							   double XG[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION],
							   double K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE],
							   double E, double nu, int DM)
{
	int i, j, jj, k, l;
	int BDBJ_flag;

	double K1[MAX_KIEL_SIZE][MAX_KIEL_SIZE];
	double B[D_MATRIX_SIZE][MAX_KIEL_SIZE], BG[D_MATRIX_SIZE][MAX_KIEL_SIZE];
	double D[D_MATRIX_SIZE][D_MATRIX_SIZE];
	double J = 0.0;
	double J_test = 0.0;
	double G_Gxi[GP_2D][DIMENSION]; //グローバルパッチ上での親要素内座標xi_bar,eta_bar

	Total_BDBJ_flag = 0;

	// printf("El_No;%d\tcoupled_El_No;%d\n",El_No_loc, El_No_glo);
	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			K_EL[i][j] = 0.0;
		}
	}

	Make_D_Matrix_2D(D, E, nu, DM);

	for (i = 0; i < GP_2D; i++) //ガウス点のループ(local)
	{
		// printf("gauss point number:%d\n", i);

		////ローカルガウス点がグローバル要素に含まれているかの判定
		//ローカル要素ガウス点の物理座標算出
		double data_result_shape[2] = {0.0};
		double output_xi, output_eta;
		int patch_n = 0;

		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No_loc]]; j++)
		{
			double R_shape_func = Shape_func(j, Gxi[i], El_No_loc);

			for (jj = 0; jj < DIMENSION; jj++)
			{
				data_result_shape[jj] += R_shape_func * X[j][jj];
			}
		}

		//ローカル要素ガウス点のグローバルパッチ上のパラメータ空間座標算出
		for (j = 0; j < Total_Patch_on_mesh[0]; j++) //グローバルメッシュ[0]上
		{
			Calc_xi_eta(data_result_shape[0], data_result_shape[1],
						Position_Knots[j][0], Position_Knots[j][1],
						No_Control_point[j][0], No_Control_point[j][1], Order[j][0], Order[j][1],
						&output_xi, &output_eta);
			// printf("  x: % 1.8e\n", data_result_shape[0]);
			// printf("  y: % 1.8e\n", data_result_shape[1]);
			// printf(" xi: % 1.8e\n", output_xi);
			// printf("eta: % 1.8e\n", output_eta);
			// printf("patch_n: %d\n", j);
			patch_n = j;
		}
		//要素内外判定

		if (output_xi >= Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0]] &&
			output_xi < Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0] + 1] &&
			output_eta >= Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1]] &&
			output_eta < Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1] + 1]) //要素内であるとき
		{
			BDBJ_flag = 1;
			// printf("BDBJ_flag\n");

			//親要素座標の算出
			G_Gxi[i][0] = -1.0 + 2.0 * (output_xi - Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0]]) /
									 (Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0] + 1] - Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0]]);
			G_Gxi[i][1] = -1.0 + 2.0 * (output_eta - Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1]]) /
									 (Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1] + 1] - Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1]]);
			// printf("G_Gxi[][]=\n");
		}
		else //要素外であるとき
		{
			BDBJ_flag = 0;
		}

		// printf("i=%d\n",i );

		////結合要素剛性マトリックス計算
		//要素内であるとき、次を計算
		if (BDBJ_flag)
		{
			Total_BDBJ_flag++;
			Same_BDBJ_flag[i]++;
			// printf("BDBJ_flag\ti=%d\n",i );
			//重なるグローバル要素のBマトリックス
			Make_B_Matrix(El_No_glo, BG, G_Gxi[i], XG, &J);
			//ローカル要素のBマトリックス
			Make_B_Matrix(El_No_loc, B, Gxi[i], X, &J);
			// BGTDBLの計算
			coupled_BDBJ(B, D, BG, J, K1);
			J_test += J;
			for (k = 0; k < KIEL_SIZE; k++)
			{
				for (l = 0; l < KIEL_SIZE; l++)
				{
					K_EL[k][l] += w[i] * K1[k][l];
				}
			} // printf("w[%d]=%f\n",i,w[i]);
		}
	}
	// printf("El_No:%d J_test=%e\n", El_No_loc, J_test);
	// printf("G=%f\n",G );
	/*for ( k = 0; k < KIEL_SIZE; k++) {
		for ( l = 0; l < KIEL_SIZE; l++) {
			printf("K_EL[%d][%d]:%le\n",k,l,K_EL[k][l]);
		}
	}*/

	// if (i == GP_2D - 1)
	// {
	// 	printf("-------------------Total_BDBJ_flag=%d-------------------\n", Total_BDBJ_flag);
	// }

	return 0;
}

///////////////////////////////////////////////////
//////////////歪と応力, ひずみエネルギ密度, 変位勾配
///////////////////////////////////////////////////
__host__ void Make_Strain(int Total_Element)
{
	static double U[MAX_KIEL_SIZE];
	double B[D_MATRIX_SIZE][MAX_KIEL_SIZE], X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], J;

	int N, e, i, j;
	// printf("Strain\n");

	Make_gauss_array(0);

	for (e = 0; e < Total_Element; e++)
	{
		// printf("\nElementNo.:%d\n",e);
		for (N = 0; N < GP_2D; N++)
			for (i = 0; i < N_STRAIN; i++)
				Strain[e][N][i] = 0.0;
		// Bマトリックスと各要素の変位を取得
		// printf("El_No:%d,No_Control_point_ON_ELEMENT[%d]:%d\n", El_No,Element_patch[El_No],No_Control_point_ON_ELEMENT[Element_patch[El_No]]);
		for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
		{
			for (j = 0; j < DIMENSION; j++)
			{
				U[i * DIMENSION + j] = Displacement[Controlpoint_of_Element[e][i] * DIMENSION + j];
				X[i][j] = Node_Coordinate[Controlpoint_of_Element[e][i]][j];
			}
		}
		//歪
		for (N = 0; N < GP_2D; N++)
		{
			// printf("N:%d\n",N);
			Make_B_Matrix(e, B, Gxi[N], X, &J);
			for (i = 0; i < D_MATRIX_SIZE; i++)
				for (j = 0; j < KIEL_SIZE; j++)
				{
					Strain[e][N][i] += B[i][j] * U[j];
					// printf("B[%d][%d]_in_strain:%le * ",i,j,B[i][j]);
					// if(e==1){
					// printf("U[%d]=%le = %le\n",j,U[j],B[i][j]*U[j]);
					// }
				}
		}
	}
}

//応力
__host__ void Make_Stress_2D(double E, double nu, int Total_Element, int DM)
{

	static double D[D_MATRIX_SIZE][D_MATRIX_SIZE];
	int e, i, j, k;
	Make_gauss_array(0);
	Make_D_Matrix_2D(D, E, nu, DM);

	for (e = 0; e < Total_Element; e++)
	{
		for (k = 0; k < GP_2D; k++)
			for (i = 0; i < N_STRESS; i++)
				Stress[e][k][i] = 0.0;
		for (k = 0; k < GP_2D; k++)
			for (i = 0; i < D_MATRIX_SIZE; i++)
				for (j = 0; j < D_MATRIX_SIZE; j++)
					Stress[e][k][i] += D[i][j] * Strain[e][k][j];
	}
}

__host__ void Make_Stress_2D_glo(double E, double nu, int Total_Element, int DM)
{
	Make_gauss_array(0);

	static double D[D_MATRIX_SIZE][D_MATRIX_SIZE];
	int e, i, j, k;
	Make_D_Matrix_2D(D, E, nu, DM);

	for (e = 0; e < Total_Element; e++)
	{
		for (k = 0; k < GP_2D; k++)
			for (i = 0; i < N_STRESS; i++)
				Stress_glo[e][k][i] = 0.0;
		for (k = 0; k < GP_2D; k++)
			for (i = 0; i < D_MATRIX_SIZE; i++)
				for (j = 0; j < D_MATRIX_SIZE; j++)
					Stress_glo[e][k][i] += D[i][j] * Strain_glo[e][k][j];
	}
}

__host__ void Make_Displacement_grad(int El_No)
{
	static double U[MAX_KIEL_SIZE];
	double b_grad[DIMENSION * DIMENSION][2 * MAX_NO_CCpoint_ON_ELEMENT];
	double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], J;

	Make_gauss_array(0);

	int N, e, re, i, j;

	// printf("変位勾配\n");
	for (re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++)
	{
		e = real_element[re];
		// printf("\nElementNo.:%d\n",e);
		for (N = 0; N < GP_2D; N++)
			for (i = 0; i < DIMENSION * DIMENSION; i++)
				Disp_grad[e][N][i] = 0.0;
		// Bマトリックスと各要素の変位を取得
		// printf("El_No:%d,No_Control_point_ON_ELEMENT[%d]:%d\n", El_No,Element_patch[El_No],No_Control_point_ON_ELEMENT[Element_patch[El_No]]);
		for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
		{
			for (j = 0; j < DIMENSION; j++)
			{
				U[i * DIMENSION + j] = Displacement[Controlpoint_of_Element[e][i] * DIMENSION + j];
				X[i][j] = Node_Coordinate[Controlpoint_of_Element[e][i]][j];
			}
		}
		//変位勾配
		for (N = 0; N < GP_2D; N++)
		{
			// printf("N:%d\n",N);
			Make_b_grad_Matrix(e, b_grad, Gxi[N], X, &J);
			for (i = 0; i < DIMENSION * DIMENSION; i++)
				for (j = 0; j < 2 * No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++)
				{
					Disp_grad[e][N][i] += b_grad[i][j] * U[j];
					// printf("b_grad[%d][%d] = %lf  また　U = %lf",i,j,b_grad[i][j],U[j]);
				}
		}
	}
}

__host__ void Make_StrainEnergyDensity_2D()
{
	int re, e, k;
	// Make_D_Matrix_2D( D, E, nu ,DM);
	for (re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++)
	{
		e = real_element[re];
		for (k = 0; k < POW_Ng; k++)
		{
			StrainEnergyDensity[e][k] = 0.0;
		}
	}

	for (re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++)
	{
		e = real_element[re];
		for (k = 0; k < POW_Ng; k++)
		{
			StrainEnergyDensity[e][k] = (Stress[e][k][0] * Strain[e][k][0] + Stress[e][k][1] * Strain[e][k][1] + Stress[e][k][2] * Strain[e][k][2]) / 2.0;
		}
	}
}

// void Make_Displacement_grad_overlay(double E, double nu, int Total_Element , int El_No, int Total_Control_Point){
// 	static double U[MAX_KIEL_SIZE];
// 	double b_grad[DIMENSION * DIMENSION][2 * No_Control_point_ON_ELEMENT[Element_patch[El_No]]];
// 	double X[No_Control_point_ON_ELEMENT[Element_patch[El_No]]][DIMENSION],J;
// 	double G = pow(0.6,0.5);
// 	double Gxi[POW_Ng][DIMENSION] = { {-G,-G},{0.0,-G},{G,-G},{-G,0.0},{0.0,0.0},{G,0.0},{-G,G},{0.0,G},{G,G} };

// 	int N, e, re, i, j;
// 	//printf("変位勾配\n");
//     	for(re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++){
//         	e = real_element[re];
// 		//printf("\nElementNo.:%d\n",e);
// 		for(N = 0; N < POW_Ng; N++)
// 			for(i = 0; i <  DIMENSION * DIMENSION; i++)
// 				Disp_grad_overlay[e][N][i] = 0.0;
// 		//Bマトリックスと各要素の変位を取得
// 		//printf("El_No:%d,No_Control_point_ON_ELEMENT[%d]:%d\n", El_No,Element_patch[El_No],No_Control_point_ON_ELEMENT[Element_patch[El_No]]);
// 		for(i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++){
// 			for(j = 0; j < DIMENSION; j++){
// 				U[i * DIMENSION + j] = Disp_overlay[Controlpoint_of_Element[e][i] * DIMENSION + j];
// 				X[i][j] = Node_Coordinate[Controlpoint_of_Element[e][i]][j];
// 			}
// 		}
// 		//変位勾配
// 		for(N = 0; N < POW_Ng; N++){
// 			//printf("N:%d\n",N);
// 			Make_b_grad_Matrix(e, b_grad, Gxi[N], X ,&J , Total_Control_Point);
// 			for(i = 0; i < DIMENSION * DIMENSION; i++)
// 				for(j = 0; j < 2 * No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++){
// 				Disp_grad_overlay[e][N][i] += b_grad[i][j] * U[j];
// 				//printf("b_grad[%d][%d] = %lf  また　U = %lf",i,j,b_grad[i][j],U[j]);
// 			}
// 		}
// 	}
// }

__host__ void Make_StrainEnergyDensity_2D_overlay()
{
	int re, e, k;
	for (re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++)
	{
		e = real_element[re];
		for (k = 0; k < POW_Ng; k++)
		{
			StrainEnergyDensity_overlay[e][k] = 0.0;
		}
	}

	for (re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++)
	{
		e = real_element[re];
		for (k = 0; k < POW_Ng; k++)
		{
			StrainEnergyDensity_overlay[e][k] = (Stress_overlay[e][k][0] * Strain_overlay[e][k][0] + Stress_overlay[e][k][1] * Strain_overlay[e][k][1] + Stress_overlay[e][k][2] * Strain_overlay[e][k][2]) / 2.0;
		}
	}
}

__host__ void Make_ReactionForce(int Total_Control_Point)
{
	int e, i, j, k, l, re;
	double B[D_MATRIX_SIZE][MAX_KIEL_SIZE], X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], J;

	Make_gauss_array(0);

	for (i = 0; i < Total_Control_Point * DIMENSION; i++)
		ReactionForce[i] = 0.0;
	// printf("ReactionForce\n");
	for (re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++)
	{
		e = real_element[re];
		// printf("%d,No_Control_point_ON_ELEMENT[%d]:%d\n",e,Element_patch[e], No_Control_point_ON_ELEMENT[Element_patch[e]]);
		// Bマトリックスを取得
		for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
		{
			for (j = 0; j < DIMENSION; j++)
			{
				X[i][j] = Node_Coordinate[Controlpoint_of_Element[e][i]][j];
				// printf("Node_Coordinate[%d][%d]:%le\n",Controlpoint_of_Element[e][i],j, Node_Coordinate[ Controlpoint_of_Element[e][i] ][j]);
				// printf("X[%d][%d]:%le\n",Controlpoint_of_Element[e][i],j,X[i][j] );
			}
		}
		for (k = 0; k < GP_2D; k++)
		{
			Make_B_Matrix(e, B, Gxi[k], X, &J);
			for (j = 0; j < D_MATRIX_SIZE; j++)
				for (l = 0; l < DIMENSION; l++)
					for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
						ReactionForce[Controlpoint_of_Element[e][i] * DIMENSION + l] += B[j][i * DIMENSION + l] * Stress[e][k][j] * w[k] * J;
			// printf("J:%le\n", J);
		}
	}
}

////////////////////////////////////////////////////////////////
//////////////////分布荷重//////////////////////////////////////
////////////////////////////////////////////////////////////////

//面番号ごとの節点番号の取り方の指定
/*void Force_Dis_NodeOfElement( int Number, int DistributeForce[MAX_N_DISTRIBUTE_FORCE][3], int ForceDis_NoE[No_Control_point_ON_ELEMENT[Element_patch[El_No]] ){
	int i;
	if( DistributeForce[Number][1] == 0 )
		for(i = 0; i < No_Control_point_ON_ELEMENT[iPatch]; i++ )
			ForceDis_NoE[i] = Controlpoint_of_Element[ DistributeForce[Number][0] ][i];
	else if( DistributeForce[Number][1] == 1){
		ForceDis_NoE[0] = Controlpoint_of_Element[ DistributeForce[Number][0] ][1];		ForceDis_NoE[1] = Controlpoint_of_Element[ DistributeForce[Number][0] ][2];
		ForceDis_NoE[2] = Controlpoint_of_Element[ DistributeForce[Number][0] ][3];		ForceDis_NoE[3] = Controlpoint_of_Element[ DistributeForce[Number][0] ][0];
	}
	else if( DistributeForce[Number][1] == 2){
		ForceDis_NoE[0] = Controlpoint_of_Element[ DistributeForce[Number][0] ][2];		ForceDis_NoE[1] = Controlpoint_of_Element[ DistributeForce[Number][0] ][3];
		ForceDis_NoE[2] = Controlpoint_of_Element[ DistributeForce[Number][0] ][0];		ForceDis_NoE[3] = Controlpoint_of_Element[ DistributeForce[Number][0] ][1];
	}
	else if( DistributeForce[Number][1] == 3){
		ForceDis_NoE[0] = Controlpoint_of_Element[ DistributeForce[Number][0] ][3];		ForceDis_NoE[1] = Controlpoint_of_Element[ DistributeForce[Number][0] ][0];
		ForceDis_NoE[2] = Controlpoint_of_Element[ DistributeForce[Number][0] ][1];		ForceDis_NoE[3] = Controlpoint_of_Element[ DistributeForce[Number][0] ][2];
	}

}
void Force_Dis( int Total_DistributeForce, int DistributeForce[MAX_N_DISTRIBUTE_FORCE][3], double Val_DistributeForce[MAX_N_DISTRIBUTE_FORCE],
					 int *Total_Load,int Load_Node_Dir[MAX_N_LOAD][2],double Value_of_Load[MAX_N_LOAD],int Total_Control_Point, int El_No, int *Total_Element ){


	int i,j, DF;
	static int ForceDistribute_Controlpoint_of_Element[MAX_NO_CCpoint_ON_ELEMENT];
	static double Out_Force_Distribute[MAX_NO_CCpoint_ON_ELEMENT];
	double a[DIMENSION][DIMENSION];
	static double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION];
	double G = pow( 0.6 , 0.5 );
	double J = 0.0,Sum=0.0;
	double w[DISTRIBUTE_FORCE_Ng] = {(5.0/9.0),(8.0/9.0),(5.0/9.0)};
	double Gxi[DISTRIBUTE_FORCE_Ng][DIMENSION] = { { 1.0, G },{1.0,0.0},{ 1.0, (-1.0)*G } };



	for( DF = 0; DF < Total_DistributeForce; DF++){
		//回転させた要素の節点座標の取得
		Force_Dis_NodeOfElement(DF, DistributeForce, ForceDistribute_Controlpoint_of_Element,El_No);

		for( i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++ ){
			for(j = 0; j < DIMENSION; j++ )
				X[i][j] = Node_Coordinate[ ForceDistribute_Controlpoint_of_Element[i] ][j];
			Out_Force_Distribute[i] = 0.0;
		}
		//各節点ごとの力の計算
		for( j = 0; j < DISTRIBUTE_FORCE_Ng; j++ ){
			Jacobian( a, Gxi[j], X , El_No,Total_Control_Point);
			J = pow(a[0][1],2)+pow(a[1][1],2);
			J = sqrt(J);
			for(i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++ )
				Out_Force_Distribute[i] +=Shape_func(i,Total_Control_Point,Gxi[j],El_No) * Val_DistributeForce[DF] * J * w[j];
		}
		//求めた値を荷重(Load)に追加
		for( i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++ ){
			if( Out_Force_Distribute[i] != 0 ){
				Load_Node_Dir[ *Total_Load ][0] = ForceDistribute_Controlpoint_of_Element[i];
				Load_Node_Dir[ *Total_Load ][1] = DistributeForce[DF][2];
				Value_of_Load[ *Total_Load ]  = Out_Force_Distribute[i];
				*Total_Load += 1;
				Sum += Out_Force_Distribute[i];
			}
		}
		printf("DistributeForce :Element= %d Factor= %d Direction= %d SumForce= %le\n",
			DistributeForce[DF][0],DistributeForce[DF][1],DistributeForce[DF][2],Sum);
		Sum=0.0;

	}

}*/

__host__ void Make_Parameter_z(int Total_Element, double E, double nu, int DM)
{
	int e, k;
	Make_gauss_array(0);

	if (DM == 0)
	{
		// Make_strain_z
		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < GP_2D; k++)
				Strain[e][k][3] = 0.0;

		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < GP_2D; k++)
				Strain[e][k][3] = -1.0 * nu / E * (Stress[e][k][0] + Stress[e][k][1]);
	}

	if (DM == 1)
	{
		// Make_stree_z
		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < GP_2D; k++)
				Stress[e][k][3] = 0.0;

		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < GP_2D; k++)
				Stress[e][k][3] = E * nu / (1.0 + nu) / (1 - 2.0 * nu) * (Strain[e][k][0] + Strain[e][k][1]);
	}
}

__host__ void Make_Parameter_z_overlay(int Total_Element, double E, double nu, int DM)
{
	int e, k;

	if (DM == 0)
	{
		// Make_strain_z
		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < POW_Ng; k++)
				Strain_overlay[e][k][3] = 0.0;

		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < POW_Ng; k++)
				Strain_overlay[e][k][3] = -1.0 * nu / E * (Stress_overlay[e][k][0] + Stress_overlay[e][k][1]);
	}

	if (DM == 1)
	{
		// Make_stree_z
		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < POW_Ng; k++)
				Stress_overlay[e][k][3] = 0.0;

		for (e = 0; e < Total_Element; e++)
			for (k = 0; k < POW_Ng; k++)
				Stress_overlay[e][k][3] = E * nu / (1.0 + nu) / (1 - 2.0 * nu) * (Strain_overlay[e][k][0] + Strain_overlay[e][k][1]);
	}
}

__host__ void element_coordinate(int Total_Element)
{
	int i, j, k, e, l = 0;
	double element_edge[9][DIMENSION] = {{-1.0, -1.0}, {1.0, -1.0}, {1.0, 1.0}, {-1.0, 1.0}, {0.0, -1.0}, {1.0, 0.0}, {0.0, 1.0}, {-1.0, 0.0}, {0.0, 0.0}};

	for (e = 0; e < Total_Element; e++)
	{
		for (k = 0; k < 9; k++)
		{
			double data_result_shape[2] = {0.0};
			for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
			{
				double R_shape_func = Shape_func(i, element_edge[k], e);
				for (j = 0; j < DIMENSION; j++)
				{
					data_result_shape[j] += R_shape_func * Node_Coordinate[Controlpoint_of_Element[e][i]][j];
				}
			}
			element_coordinate_Nopoint[l][0] = data_result_shape[0];
			element_coordinate_Nopoint[l][1] = data_result_shape[1];
			l++;
		}
	}
	for (l = 0; l < 9 * Total_Element; l++)
		same_point_in_Element[l] = l;
	for (l = 0; l < 9 * Total_Element; l++)
	{
		for (i = l - 1; i >= 0; i--)
		{
			if (element_coordinate_Nopoint[l][0] == element_coordinate_Nopoint[i][0] && element_coordinate_Nopoint[l][1] == element_coordinate_Nopoint[i][1])
			{
				// printf("同じ座標の番号l:%d i:%d\n",l,i);
				same_point_in_Element[l] = i;
			}
		}
	}
}

/*
void ourput_graph_2D(FILE *fp, int e, double element_gg, double element_ee, double data_result_shape_x, double data_result_shape_y, double data_result_disp_x, double data_result_disp_y)
{
	fp = fopen("NURBS/NURBS_disp.dat", "a");
	fprintf(fp, "%d %20.13e  %20.13e %20.13e %20.13e %20.13e %20.13e\n", e, element_gg, element_ee, data_result_shape_x, data_result_shape_y, data_result_disp_x, data_result_disp_y);
	fclose(fp);
}
*/

__host__ void calculate_Controlpoint_using_NURBS(double element[DIMENSION], int Total_Element)
{
	int e, b, j, re, i;
	// int p = 0;
	// for(e=0; e < Total_Element; e++){

	double R_shape_func;

	for (re = 0; re < Total_Element; re++)
	{
		e = real_element[re];
		// printf("\n");
		//  printf("Element_No:%d\n",e );
		double element_gg = 0.0, element_ee = 0.0, element_delta;

		int i_gg, i_ee;
		// int element_ndiv = 10; 分割数
		int element_ndiv = 1;

		No_points_for_colored_points = (element_ndiv + 1) * (element_ndiv + 1) * real_Total_Element;

		element_delta = 2.0 / element_ndiv;

		for (i_ee = 0; i_ee < element_ndiv + 1; i_ee++)
			for (i_gg = 0; i_gg < element_ndiv + 1; i_gg++)
			{
				double data_result_shape[3] = {0.0};
				double data_result_disp[3] = {0.0};

				element_gg = -1.0 + element_delta * i_gg;
				element_ee = -1.0 + element_delta * i_ee;
				element[0] = element_gg;
				element[1] = element_ee;

				// printf("element_gg:%le\n",element_gg);
				// printf("element_ee:%le\n",element_ee);

				for (b = 0; b < No_Control_point_ON_ELEMENT[Element_patch[e]]; b++)
				{
					R_shape_func = Shape_func(b, element, e);
					for (j = 0; j < DIMENSION; j++)
					{
						data_result_disp[j] += R_shape_func * Displacement[Controlpoint_of_Element[e][b] * DIMENSION + j];
						data_result_shape[j] += R_shape_func * Node_Coordinate[Controlpoint_of_Element[e][b]][j];
					}
				}

				fp = fopen("NURBS/NURBS_points.dat", "a");
				fprintf(fp, "%lf	%lf\n",
						data_result_shape[0],
						data_result_shape[1]);
				fclose(fp);

				fp = fopen("shapefunc/shape_func_xi.dat", "a");
				fprintf(fp, "%lf	%lf\t", element[0], Position_Data_param[0]);
				for (i = 0; i < No_Control_point[Element_patch[e]][0]; i++)
				{
					fprintf(fp, "%lf\t", Shape[0][i][Order[Element_patch[e]][0]]);
				}
				fprintf(fp, "\n");
				fclose(fp);

				fp = fopen("shapefunc/shape_func_eta.dat", "a");
				fprintf(fp, "%lf	%lf\t", element[1], Position_Data_param[1]);
				for (i = 0; i < No_Control_point[Element_patch[e]][1]; i++)
				{
					fprintf(fp, "%lf\t", Shape[1][i][Order[Element_patch[e]][1]]);
				}
				fprintf(fp, "\n");
				fclose(fp);

				// data_result_shape_x[p]=data_result_shape[0];
				// data_result_shape_y[p]=data_result_shape[1];
				// p++;

				// printf("\n");

				// ourput_graph_2D(fp, e, element_gg, element_ee, data_result_shape[0], data_result_shape[1], data_result_disp[0], data_result_disp[1]);

				/*
				data_result_shape_x[p] = data_result_shape[0];
				data_result_shape_y[p] = data_result_shape[1];
				data_result_disp_x[p] = data_result_disp[0];
				data_result_disp_y[p] = data_result_disp[1];
				p++;
				*/

				// NURBS_points(fp,No_points_for_colored_points e,data_result_shape[0],data_result_shape[1])
			}
	}
}

__host__ void Gausspoint_coordinate(int Total_Element)
{
	int i, j, k, e;

	Make_gauss_array(0);

	for (e = 0; e < Total_Element; e++)
	{
		for (k = 0; k < GP_2D; k++)
		{
			double data_result_shape[2] = {0.0};
			for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
			{
				double R_shape_func = Shape_func(i, Gxi[k], e);
				for (j = 0; j < DIMENSION; j++)
				{
					data_result_shape[j] += R_shape_func * Node_Coordinate[Controlpoint_of_Element[e][i]][j];
				}
			}
			Gausspoint_coordinates[e][k][0] = data_result_shape[0];
			Gausspoint_coordinates[e][k][1] = data_result_shape[1];
		}
	}
}

__host__ void calculate_extendmesh_using_NURBS(double element_emsh[DIMENSION])
{
	int e, b, j, re;

	for (re = 0; re < real_Total_Element; re++)
	{
		e = real_element[re];
		// printf("\n");
		// printf("Element_No:%d\n",e );
		double element_gg = 0.0, element_ee = 0.0, element_delta_ee, element_delta_gg;

		int i_gg, i_ee, element_ndiv_ee = 10, element_ndiv_gg = 1;

		No_points_for_new_zarusoba = (element_ndiv_ee + 1) * (element_ndiv_gg + 1) * real_Total_Element;

		element_delta_ee = 2.0 / element_ndiv_ee;
		element_delta_gg = 2.0 / element_ndiv_gg;

		for (i_ee = 0; i_ee < element_ndiv_ee + 1; i_ee++)
			for (i_gg = 0; i_gg < element_ndiv_gg + 1; i_gg++)
			{
				double data_result_shape[3] = {0.0};
				double data_result_disp[3] = {0.0};

				element_gg = -1.0 + element_delta_gg * i_gg;
				element_ee = -1.0 + element_delta_ee * i_ee;
				element_emsh[0] = element_gg;
				element_emsh[1] = element_ee;

				// printf("element_gg:%le\n",element_gg);
				// printf("element_ee:%le\n",element_ee);

				for (b = 0; b < No_Control_point_ON_ELEMENT[Element_patch[e]]; b++)
				{
					double R_shape_func = Shape_func(b, element_emsh, e);
					for (j = 0; j < DIMENSION; j++)
					{
						data_result_disp[j] += R_shape_func * Displacement[Controlpoint_of_Element[e][b] * DIMENSION + j];
						data_result_shape[j] += R_shape_func * Node_Coordinate[Controlpoint_of_Element[e][b]][j];
					}
				}

				// data_result_shape_x[p]=data_result_shape[0];
				// data_result_shape_y[p]=data_result_shape[1];
				// p++;

				// printf("\n");
				/*
				data_result_shape_x_for_new_zarusoba[p] = data_result_shape[0];
				data_result_shape_y_for_new_zarusoba[p] = data_result_shape[1];
				data_result_disp_x_for_new_zarusoba[p] = data_result_disp[0];
				data_result_disp_y_for_new_zarusoba[p] = data_result_disp[1];
				p++;
				*/

				// NURBS_points(fp,No_points_for_colored_points e,data_result_shape[0],data_result_shape[1])
			}
	}
}

// Newton Raphsonによって出力されたxi,etaから重なる要素を求める
__host__ int ele_check(int patch_n, double para_coord[DIMENSION])
{
	int i;
	int j;
	int k, kk;
	int l, ll;
	int RangeCheck_flag;					 //要素を求め終えたら立てるフラグ
	int temp_ad[DIMENSION][MAX_N_ORDER + 1]; //要素の位置を求めるための値
	int No_line[DIMENSION];					 // xi,etaが含まれている要素の列数
	int n = 1;

	for (j = 0; j < DIMENSION; j++)
	{
		//初期化
		l = 0;
		No_line[j] = 0;
		// printf("No_line[%d]=%d\n",j,No_line[j]);
		//  temp_ad[j][MAX_N_ORDER + 1] =  0;
		for (i = 0; i < MAX_N_ORDER + 1; i++)
		{
			temp_ad[j][i] = 0;
		}
		RangeCheck_flag = 0;

		for (k = 0; k < No_Control_point[patch_n][j] - 1; k++)
		{
			if (RangeCheck_flag == 1)
				break;
			// Local要素の頂点がGlobalパッチ内にない場合
			if (para_coord[j] < Position_Knots[patch_n][j][0] || para_coord[j] > Position_Knots[patch_n][j][No_knot[patch_n][j] - 1])
			{
				// printf("no over element\n");
				RangeCheck_flag++;
			}
			// Local要素の頂点がGlobal要素内部にある場合
			if (para_coord[j] < Position_Knots[patch_n][j][Order[patch_n][j] + k])
			{
				// printf("if\nPosition_Knots[%d][%d][%d]=%le\n",
				//         patch_n,j,Order[patch_n][j]+k,Position_Knots[patch_n][j][Order[patch_n][j]+k]);
				int kk = 0;
				for (kk = 0; kk < k + 1; kk++)
				{
					// if ( RangeCheck_flag == 1 ) break;
					if (para_coord[j] > Position_Knots[patch_n][j][Order[patch_n][j] + k - kk])
					{
						// printf("ifif\nPosition_Knots[%d][%d][%d]=%le\n",
						//         patch_n,j,Order[patch_n][j]+k-kk,
						//		Position_Knots[patch_n][j][Order[patch_n][j]+k-kk]);
						temp_ad[j][l] = k - kk;
						// printf("temp_ad[%d][%d]=%d\n",j,l,temp_ad[j][l]);
						l++;
						RangeCheck_flag++;
						break;
					}
				}
			}
			// Local要素の頂点がGlobal要素境界上にある場合
			if (para_coord[j] == Position_Knots[patch_n][j][Order[patch_n][j] + k])
			{
				//頂点の座標がGlobalパッチの始点上にある場合
				if (para_coord[j] == Position_Knots[patch_n][j][0])
				{
					temp_ad[j][l] = k;
					// printf("start point\n");
					// printf("temp_ad[%d][%d]=%d\n",j,l,temp_ad[j][l]);
					l++;
					break;
				}
				//頂点の座標がGlobalパッチの終点上にある場合
				if (para_coord[j] == Position_Knots[patch_n][j][No_knot[patch_n][j] - 1])
				{
					temp_ad[j][l] = k - 1;
					// printf("finish point\n");
					// printf("temp_ad[%d][%d]=%d\n",j,l,temp_ad[j][l]);
					l++;
					break;
				}
				//頂点の座標がGlobal要素境界上にある場合
				else
				{
					temp_ad[j][l] = k - 1;
					// printf("temp_ad[%d][%d]=%d\n",j,l,temp_ad[j][l]);
					// printf("data_result_shape[%d]=%le in element_line[%d] on patch[%d] on mesh[0]\n",
					//		j,phys_coord[j],k-1,patch_n);
					l++;
					temp_ad[j][l] = k;
					// printf("temp_ad[%d][%d]=%d\n",j,l,temp_ad[j][l]);
					l++;
					// break;
				}
				for (kk = 0; kk < Order[patch_n][j]; kk++)
				{
					if (para_coord[j] == Position_Knots[patch_n][j][Order[patch_n][j] + k + kk + 1])
					//多重ノット（次数分ループ）
					{
						printf("C0 continuity\n");
						temp_ad[j][l] = k + kk;
						// printf("temp_ad[%d][%d]=%d\n",j,l,temp_ad[j][l]);
						l++;
					}
					if (para_coord[j] != Position_Knots[patch_n][j][Order[patch_n][j] + k + kk + 1])
						break;
				}
				RangeCheck_flag++;
			}
		}
		No_line[j] = l;
		// printf("No_line[%d]=%d\n", j, No_line[j]);
		n *= l; //各方向のNo_lineを掛け合わせる
	}
	/*
	for (j = 0; j < DIMENSION; j++)
	{
		printf("No_line[%d];%d\n",j,No_line[j]);
	}*/
	if (DIMENSION == 2)
	{
		for (l = 0; l < No_line[1]; l++)
		{
			for (ll = 0; ll < No_line[0]; ll++)
			{
				temp_element_n[l * No_line[0] + ll] = temp_ad[0][ll] + temp_ad[1][l] * line_No_Total_element[patch_n][0];
				// printf("el[%d](x,y)=(%le,%le) in element[%d] on patch[%d] on mesh[0]\n"
				//		,element_n_over
				//		,phys_coord[0],phys_coord[1]
				//		,temp_element_n[l*No_line[0]+ll],patch_n);
				// printf("temp_element_n[%d];%d\n",
				//		l*No_line[0]+ll,temp_element_n[l*No_line[0]+ll]);
			}
		}
	}
	// printf("n_element_over;%d\n", n);	//重なっている要素の総数
	return n;
}

//昇順ソート
__host__ void sort(int total)
{
	int i, j;
	int tmp;

	for (i = 0; i < total; i++)
	{
		for (j = i + 1; j < total; j++)
		{
			if (element_n_point[i] > element_n_point[j])
			{
				tmp = element_n_point[i];
				element_n_point[i] = element_n_point[j];
				element_n_point[j] = tmp;
			}
		}
	}
}

//重複削除
__host__ int duplicate_delete(int total, int element_n)
{
	int i, j;

	j = 0;
	NELOVER[element_n][j] = element_n_point[0];
	j++;
	for (i = 1; i < total; i++)
	{
		if (element_n_point[i] != element_n_point[i - 1])
		{
			NELOVER[element_n][j] = element_n_point[i];
			// printf("NELOVER\n");
			j++;
		}
	}
	// j;要素element_nに重なる要素の総数
	return j;
}

// coupled matrix求める
//要素の重なりを求める(要素の端点から求める)
__host__ void Check_coupled_Glo_Loc_element_for_end(double element_loc[DIMENSION], int mesh_n_over, int mesh_n_org)
{
	int re;
	int e;
	int b, j;
	int i;
	int k;
	int l, ll;
	int n_elements_over_point[MAX_N_POINT_OVER];
	// int n_points;
	// int n_elements; // = 0;
	//  int patch_n, itr_n;
	int patch_n = 0, itr_n = 0;

	int element_ndiv = 1;
	/*n_points = (element_ndiv + 1) * (element_ndiv + 1)
	 * real_Total_Element_on_mesh[mesh_n_over];*/

	/*for (i = 0; i < MAX_N_ELEMENT_OVER; i++)
	{
		temp_element_n[i] = 0;
		printf("i;%d\n",i);
	}*/
	// printf("i;%d\n",i);
	//ローカルパッチ(mesh_n_over)各要素の頂点の物理座標算出
	// from Func.:calculate_Controlpoint_using_NURBS
	for (re = 0; re < real_Total_Element_on_mesh[mesh_n_over]; re++)
	{
		e = real_element[re + real_Total_Element_to_mesh[mesh_n_over]];
		// printf("in_array_real_element:%d\n",
		//		re + real_Total_Element_to_mesh[mesh_n_over+1]);
		// printf("\n");
		// printf("Element_No:%d\n",e );
		double element_gg = 0.0, element_ee = 0.0, element_delta;

		int i_gg, i_ee;

		double output_para[DIMENSION];
		int Total_n_elements = 0;

		k = 0;
		ll = 0;
		element_delta = 2.0 / element_ndiv;

		for (i_ee = 0; i_ee < element_ndiv + 1; i_ee++)
		{
			for (i_gg = 0; i_gg < element_ndiv + 1; i_gg++)
			{
				double data_result_shape[3] = {0.0};

				element_gg = -1.0 + element_delta * i_gg;
				element_ee = -1.0 + element_delta * i_ee;
				element_loc[0] = element_gg;
				element_loc[1] = element_ee;

				// printf("element_gg:%le\n",element_gg);
				// printf("element_ee:%le\n",element_ee);

				// printf("e=%d\n",e);
				// printf("element_patch[e]=%d\n",Element_patch[e]);

				for (b = 0; b < No_Control_point_ON_ELEMENT[Element_patch[e]]; b++)
				{
					double R_shape_func = Shape_func(b, element_loc, e);
					for (j = 0; j < DIMENSION; j++)
					{
						data_result_shape[j] += R_shape_func * Node_Coordinate[Controlpoint_of_Element[e][b]][j];
					}
				}
				// printf("data_result_shape[0]=%le\n",data_result_shape[0]);
				// printf("data_result_shape[1]=%le\n",data_result_shape[1]);
				//算出したローカルパッチ各要素の頂点の物理座標のグローバルパッチでの(xi,eta)算出
				// from NURBSviewer/NURBS_view/clickcalc.c/func.:calcXiEtaByNR
				for (i = 0; i < Total_Patch_on_mesh[mesh_n_org]; i++)
				{
					int ii = Calc_xi_eta(data_result_shape[0], data_result_shape[1],
										 Position_Knots[i][0], Position_Knots[i][1],
										 No_Control_point[i][0], No_Control_point[i][1], Order[i][0], Order[i][1],
										 &output_para[0], &output_para[1]);
					// printf("Newton_iteration;%d\n",ii);
					// printf("patch: %d\n", i);
					// printf("  x: % 1.8e\n", data_result_shape[0]);
					// printf("  y: % 1.8e\n", data_result_shape[1]);
					// printf(" xi: % 1.8e\n", output_para[0]);
					// printf("eta: % 1.8e\n", output_para[1]);
					patch_n = i;
					itr_n = ii;
				}
				// Newton Laphsonによって出力されたxi,etaから重なる要素を求める
				n_elements_over_point[k] = ele_check(patch_n, output_para);
				// printf("itr_n;%d\n",itr_n);
				if (itr_n == 0) // data_result_shapeがグローバルメッシュ上にないとき
				{
					n_elements_over_point[k] = 0;
				}
				// printf("n_elements_over_point[%d];%d\n",
				//		k,n_elements_over_point[k]);
				Total_n_elements += n_elements_over_point[k];
				// printf("Total_n_elements;%d\n",Total_n_elements);
				for (l = 0; l < n_elements_over_point[k]; l++)
				{
					element_n_point[ll] = temp_element_n[l];
					// printf("element_n_point[%d]=%d\n",
					//		ll,element_n_point[ll]);
					ll++;
				}
				k++;
				// printf("(x,y)=(%le,%le) in element[%d] on patch[%d] on mesh[0]\n",
				//         data_result_shape[0],data_result_shape[1],
				//         temp_ad[0]+temp_ad[1]*line_No_Total_element[i][0],i);
				//}
			}
		}
		// printf("Total_n_elements;%d\n",Total_n_elements);
		if (Total_n_elements > 0)
		{
			//昇順ソート
			sort(Total_n_elements);
			//重複削除
			NNLOVER[e] = duplicate_delete(Total_n_elements, e);
		}
		else
		{
			NNLOVER[e] = 0;
		}
		/*
		for (i = 0; i < Total_n_elements; i++)
		{
			printf("element_n_point[%d]=%d\n",i,element_n_point[i]);
		}
		*/
		printf("NNLOVER[%d]=%d\n", e, NNLOVER[e]);

		for (i = 0; i < NNLOVER[e]; i++)
		{
			printf("NELOVER[%d][%d]=%d\n", e, i, NELOVER[e][i]);
		}
	}
}

//要素の重なりを求める(要素のガウス点から求める)
__host__ void Check_coupled_Glo_Loc_element_for_Gauss(double element_loc[DIMENSION], int mesh_n_over, int mesh_n_org)
{
	int re;
	int e;
	int i, j, k, m;
	int b;
	int l, ll;
	int n_elements_over_point[MAX_N_POINT_OVER];
	// int patch_n, itr_n;
	int patch_n = 0, itr_n = 0;

	int Check_coupled_No[MAX_N_ELEMENT_OVER];
	double Percent_Check_coupled_No;
	int MAX_NNLOVER = 0;

	// int gauss_1dir = 3;	//重なり判定のための一方向ガウス点数
	// int no_gauss_pt = gauss_1dir * gauss_1dir;	//重なり判定のためのガウス点総数

	for (i = 0; i < MAX_N_ELEMENT_OVER; i++)
	{
		Check_coupled_No[i] = 0;
	}

	for (m = 0; m < 2; m++) //最初Ng個のガウス点で重なりを求め，NNLOVER[e]>=2のeに対して，再度10個のガウス点で重なりを求める
	{
		Make_gauss_array(m);

		//ローカルパッチ(mesh_n_over)各要素の頂点の物理座標算出
		// from Func.:calculate_Controlpoint_using_NURBS
		for (re = 0; re < real_Total_Element_on_mesh[mesh_n_over]; re++)
		{
			e = real_element[re + real_Total_Element_to_mesh[mesh_n_over]];
			// printf("Element_No:%d\n",e );

			int i_gg, i_ee;
			int g_n;

			double output_para[DIMENSION];
			int Total_n_elements;

			if (m == 0 || (m == 1 && NNLOVER[e] >= 2))
			{
				// printf("-----------------------------------------m:%d Element_No:%d-----------------------------------------\n",m,e );
				if (m == 1)
				{
					NNLOVER[e] = 0;
					for (i = 0; i < NNLOVER[e]; i++)
					{
						NELOVER[e][i] = 0;
					}
				}

				Total_n_elements = 0;
				k = 0;
				ll = 0;

				for (i_ee = 0; i_ee < GP_1dir; i_ee++)
				{
					for (i_gg = 0; i_gg < GP_1dir; i_gg++)
					{
						double data_result_shape[3] = {0.0};

						g_n = i_ee * GP_1dir + i_gg;
						element_loc[0] = Gxi[g_n][0];
						element_loc[1] = Gxi[g_n][1];

						// printf("e=%d\n",e);
						// printf("element_patch[e]=%d\n",Element_patch[e]);

						for (b = 0; b < No_Control_point_ON_ELEMENT[Element_patch[e]]; b++)
						{
							double R_shape_func = Shape_func(b, element_loc, e);
							for (j = 0; j < DIMENSION; j++)
							{
								data_result_shape[j] += R_shape_func * Node_Coordinate[Controlpoint_of_Element[e][b]][j];
							}
						}

						//算出したローカルパッチ各要素の頂点の物理座標のグローバルパッチでの(xi,eta)算出
						// from NURBSviewer/NURBS_view/clickcalc.c/func.:calcXiEtaByNR
						for (i = 0; i < Total_Patch_on_mesh[mesh_n_org]; i++)
						{
							int ii = Calc_xi_eta(data_result_shape[0], data_result_shape[1],
												 Position_Knots[i][0], Position_Knots[i][1],
												 No_Control_point[i][0], No_Control_point[i][1], Order[i][0], Order[i][1],
												 &output_para[0], &output_para[1]);
							// printf("Newton_iteration;%d\n",ii);
							// printf("patch: %d\n", i);
							// printf("  x: % 1.8e\n", data_result_shape[0]);
							// printf("  y: % 1.8e\n", data_result_shape[1]);
							// printf(" xi: % 1.8e\n", output_para[0]);
							// printf("eta: % 1.8e\n", output_para[1]);
							patch_n = i;
							itr_n = ii;
						}
						// Newton Laphsonによって出力されたxi,etaから重なる要素を求める
						n_elements_over_point[k] = ele_check(patch_n, output_para);
						// printf("itr_n;%d\n",itr_n);
						if (itr_n == 0) // data_result_shapeがグローバルメッシュ上にないとき
						{
							n_elements_over_point[k] = 0;
						}
						// printf("n_elements_over_point[%d];%d\n",
						// 		k,n_elements_over_point[k]);
						Total_n_elements += n_elements_over_point[k];
						// printf("Total_n_elements;%d\n",Total_n_elements);
						for (l = 0; l < n_elements_over_point[k]; l++)
						{
							element_n_point[ll] = temp_element_n[l];
							// printf("element_n_point[%d]=%d\n",
							//		ll,element_n_point[ll]);
							ll++;
						}
						k++;
						// printf("(x,y)=(%le,%le) in element[%d] on patch[%d] on mesh[0]\n",
						//         data_result_shape[0],data_result_shape[1],
						//         temp_ad[0]+temp_ad[1]*line_No_Total_element[i][0],i);
						//}
					}
				}
				// printf("Total_n_elements;%d\n",Total_n_elements);

				//昇順ソート
				sort(Total_n_elements);
				//重複削除
				NNLOVER[e] = duplicate_delete(Total_n_elements, e); // NNLOVER:要素eに重なる要素の総数

				/*
				for (i = 0; i < Total_n_elements; i++)
				{
					printf("element_n_point[%d]=%d\n",i,element_n_point[i]);
				}
				*/
				// printf("NNLOVER[%d]=%d\n",e,NNLOVER[e]);

				// for (i = 0; i < NNLOVER[e]; i++)
				// {
				// 	// printf("NELOVER[%d][%d]=%d\n",e,i,NELOVER[e][i]);
				// }
			}
		}
	}

	for (re = 0; re < real_Total_Element_on_mesh[mesh_n_over]; re++)
	{
		e = real_element[re + real_Total_Element_to_mesh[mesh_n_over]];
		printf("-----------------------------------------Element_No:%d-----------------------------------------\n", e);

		Check_coupled_No[NNLOVER[e]]++;

		if (MAX_NNLOVER < NNLOVER[e])
		{
			MAX_NNLOVER = NNLOVER[e];
		}

		printf("NNLOVER[%d] = %d\n", e, NNLOVER[e]);

		for (i = 0; i < NNLOVER[e]; i++)
		{
			printf("\tNELOVER[%d][%d] = %d\n", e, i, NELOVER[e][i]); //要素eに重なるi番目の要素番号
		}
	}

	printf("MAX_NNLOVER = %d\n", MAX_NNLOVER);
	for (i = 0; i <= MAX_NNLOVER; i++)
	{
		Percent_Check_coupled_No = (double)Check_coupled_No[i] * 100.0 / (double)real_Total_Element_on_mesh[mesh_n_over];
		printf("Check_coupled_No[%d] = %d\t\t%3.1lf %%\n", i, Check_coupled_No[i], Percent_Check_coupled_No);
	}
	printf("---------------------------------------------------------------------------------------------------------------------------\n");
}

__host__ void Make_Loc_Glo()
{
	int i, j, k;
	int jj;
	int e;
	int j_n;
	int count;

	j_n = real_Total_Element_to_mesh[Total_mesh] - real_Total_Element_on_mesh[0];
	// printf("j_n=%d\n",j_n);
	// printf("%d\t%d\n",)
	for (i = 0; i < real_Total_Element_on_mesh[0]; i++)
	{
		e = real_element[i];
		count = 0;

		for (j = 0; j < j_n; j++)
		{
			jj = real_element[real_Total_Element_to_mesh[1] + j]; //ローカルメッシュ上のreal element番号
			// printf("jj=%d\n",jj);
			if (NNLOVER[jj] > 0)
			{
				// printf("jj=%d\n",jj);
				for (k = 0; k < NNLOVER[jj]; k++)
				{
					if (NELOVER[jj][k] == e)
					{
						NELOVER[e][count] = jj;
						// printf("NELOVER[%d][%d]=%d\n",e,count,
						//							  NELOVER[e][count]);
						count++;
					}
				}
			}
		}
		NNLOVER[e] = count;
		// printf("NNLOVER[%d]=%d\n",e,NNLOVER[e]);
	}
}

/*
////////////////////////////////////////////////////////////////
//////////////////AVS出力///////////////////////////////////////
////////////////////////////////////////////////////////////////

//節点座標と要素の節点番号の書き込み
void AVS_inputInp_Quad_4( int Total_Element, int Total_Control_Point ){
	int i;

	fprintf(fp,"%d	%d\n", Total_Control_Point, Total_Element);	//総節点数、総要素数
	for(i = 0; i < Total_Control_Point; i++ ){
		fprintf(fp,"%d	%e	%e	%e\n", i+1,Node_Coordinate[i][0],Node_Coordinate[i][1],0.0);
														//節点番号(1〜)、ＸＹＺ座標（２次元だとZ=0と記述
	}
	for( i = 0; i < Total_Element; i++ ){
		fprintf(fp,"%d	0	quad	", i+1);					//要素番号(1〜)、材料番号(0)、要素の形
		//要素タイプごとに順番を確認のとこ
		fprintf(fp,"%d	", Controlpoint_of_Element[i][2]+1);
		fprintf(fp,"%d	", Controlpoint_of_Element[i][3]+1);
		fprintf(fp,"%d	", Controlpoint_of_Element[i][0]+1);
		fprintf(fp,"%d	", Controlpoint_of_Element[i][1]+1);
		fprintf(fp,"\n");
	}

}
//計算結果のデータの書き込み
void AVS_inputAns_2D( int Total_Control_Point, int Total_Element ){

	int i,j,k, NodeDataCom = 6;
	int ElementDataCom = 12;	//N_STRAIN + N_STRESS + 4
	double Str;

	fprintf(fp,"%d	%d\n", NodeDataCom, ElementDataCom);	//各節点に存在するデータ数、各要素に存在するデータ数

	//節点のデータの書き込み（変位ＸＹＺ方向＋自由利用分XYZ）
	fprintf(fp,"%d	", NodeDataCom);						//節点データ成分数
	for( i = 0; i < NodeDataCom; i++ )
		fprintf(fp,"1	");								//各成分の構成数
	fprintf(fp,"\n");

	fprintf(fp,"DisX,\nDisY,\nDisZ,\nOpX,\nOpY,\nOpZ,\n");		//各節点データ成分のラベル
	for( i = 0; i < Total_Control_Point; i++ ){
		fprintf(fp,"%d	%e	%le	", i+1, Displacement[i*DIMENSION], Displacement[i*DIMENSION+1]);
		fprintf(fp,"%e	%le	%le	%le\n", 0.0, 0.0, 0.0, 0.0 );
	}

	//要素データの書き込み（応力＋自由利用２、歪＋自由利用２）
	fprintf(fp,"%d	", ElementDataCom);						//要素データ成分数
	for( i = 0; i < ElementDataCom; i++ )
		fprintf(fp,"1	");									//各成分の構成数
	fprintf(fp,"\n");

	fprintf(fp,"SigXX,\nSigYY,\nSigXY,\nSigZZ,\nSigOp1,\nSigOp2,\n");		//各節点データ成分のラベル
	fprintf(fp,"IpuXX,\nIpuYY,\nIpuXY,\nIpuZZ,\nIpuOp1,\nIpuOp2,\n");		//各節点データ成分のラベル

	for( i = 0; i < Total_Element; i++ ){
		fprintf(fp,"%d	", i+1);
		for( j = 0; j < N_STRESS; j++ ){
			Str = 0.0;
			for( k = 0; k < POW_Ng; k++ )		Str += Stress[i][k][j];
			fprintf(fp,"	%e",Str / (double)(POW_Ng) );
		}
		for( ; j < 6; j++ )				fprintf(fp,"	%e", 0.0 );

		for( j = 0; j < N_STRAIN; j++ ){
			Str = 0.0;
			for( k = 0; k < POW_Ng; k++ )
				Str += Strain[i][k][j];
			fprintf(fp,"	%e",Str / (double)(POW_Ng) );
		}
		for( ; j < 6; j++ )				fprintf(fp,"	%e", 0.0 );

		fprintf(fp,"\n");
	}
}

void Make_Output( int Total_Control_Point, int Total_Element ){
	int StepMax=1, StepNo=1;

	//AVS用のinpファイルの制作
	fp = fopen( "AVS/1_1_force_120_0122.inp", "w");
	fprintf(fp,"# AVS field file\n");					//注釈文（必ず先頭に「#」）
	fprintf(fp,"%d\n", StepMax);						//ステップ数の設定
	fprintf(fp,"data\n" );								//データの繰り返しタイプ	data,geom,data_geom

	fprintf(fp,"step%d\n", StepNo);					//ステップ番号
	AVS_inputInp_Quad_4( Total_Element, Total_Control_Point);	//節点座標と要素の取得
	AVS_inputAns_2D( Total_Control_Point, Total_Element );			//各種計算結果の取得
	printf("Finish Make_AVS_Step%d\n",StepNo);

	fclose(fp);
}
*/

/*
////////////////////////////////////////////////////////////////
//////////////////J積分/////////////////////////////////////////
////////////////////////////////////////////////////////////////



//B_xマトリックスを求める関数
int Make_B_x_Matrix_Quad_4(double B_x[DIMENSION][KIEL_SIZE], double Local_coord[DIMENSION], double X[No_Control_point_ON_ELEMENT][DIMENSION], double *J ){
	double a[DIMENSION][DIMENSION], b[DIMENSION][No_Control_point_ON_ELEMENT];
	int i,j,k;

	Jacobian_Quad_4( a, Local_coord, X );

	*J = InverseMatrix_2D( a );
	if( *J <= 0 )return -999;

	for( i = 0; i < DIMENSION; i++ ){
		for( j = 0; j < No_Control_point_ON_ELEMENT; j++ ){
			b[i][j] = 0.0;
			for( k = 0; k < DIMENSION; k++ ){
				b[i][j] += a[k][i] * dN_Quad_4( j, Local_coord, k);
			}
		}
	}
	for( i = 0; i < No_Control_point_ON_ELEMENT; i++ ){
		B_x[0][2*i] = b[0][i];	B_x[0][2*i+1] = 0.0;
		B_x[1][2*i] = 0.0;		B_x[1][2*i+1] = b[0][i];
	}
	return 0;
}







void Make_Strain_x_Quad_4(double E, double nu, int Total_Element){
	static double U[KIEL_SIZE];
	static double B_x[DIMENSION][KIEL_SIZE],X[No_Control_point_ON_ELEMENT][DIMENSION],J;
	double w[POW_Ng] = {1.0,1.0,1.0,1.0};
	double G = 1/pow(3,0.5);
	double Gxi[Total_Element][POW_Ng][DIMENSION] = { {{(1+G)/2,(1-G)/2},{(1+G)/2,(1+G)/2},{(1-G)/2,(1+G)/2},{(1-G)/2,(1-G)/2}},
	 {{(1+G)/2,(1-G)/2},{(1+G)/2,(1+G)/2},{(1-G)/2,(1+G)/2},{(1-G)/2,(1-G)/2}}};

	int N,e,i,j;

	for( e = 0; e < Total_Element; e++ ){
		for( N = 0; N < POW_Ng; N++)
			for( i = 0; i < DIMENSION; i ++ )
				Strain_x[e][N][i] = 0.0;
		//Bマトリックスと各要素の変位を取得
		for( i = 0; i < No_Control_point_ON_ELEMENT; i++ ){
			for( j = 0; j < DIMENSION; j++ ){
				U[ i*DIMENSION +j ] = Displacement[ Controlpoint_of_Element[e][i]*DIMENSION + j ];
				X[i][j] = Node_Coordinate[ Controlpoint_of_Element[e][i] ][j];
			}
		}
		//歪
		for( N = 0; N < POW_Ng; N++ ){
			Make_B_x_Matrix_Quad_4( B_x, Gxi[N], X ,&J );
			for( i = 0; i < DIMENSION; i++ )
				for( j = 0; j < KIEL_SIZE; j++ )
					 Strain_x[e][N][i] += B_x[i][j] * U[j] * w[N];
		}
	}
}












//エネルギーモーメンタムテンソルを求める関数
void Make_EMT(double E, double nu, int Total_Element){

	int i, j, k;
	double W;
	double W_x[DIMENSION];

	int K_D[DIMENSION] = {1, 0};
	double W_K_D[DIMENSION];
	double P_1j[DIMENSION];

	Make_Stress_2D(E, nu, Total_Element);
	Make_Strain_Quad_4(E, nu, Total_Element);
	Make_Strain_x_Quad_4(E, nu, Total_Element);

	W = 0.0;


	for( i = 0; i < DIMENSION; i++){
		W_x[i] = 0.0;
	}


	for( k = 0; k < POW_Ng; k++ ){
		for( i = 0; i < Total_Element; i++ ){
			for( j = 0; j < DIMENSION+1; j++ ){

				W += (1.0/2.0)*Stress[i][k][j]*Strain[i][k][j];
			}
		}
	}
	printf("\nW = %lf\n", W);


	for( i = 0; i < DIMENSION; i++ ){
		W_K_D[i] = W*K_D[i];

		printf("\nW_K_D[%d] = %lf\n", i, W_K_D[i]);
	}


	for( k = 0; k < POW_Ng; k++ ){
		for( i = 0; i < Total_Element; i++ ){
			for( j = 0; j < DIMENSION; j++ ){

				if(j == 0){
					W_x[j] += Stress[i][k][j]*Strain_x[i][k][j]+Stress[i][k][j+2]*Strain_x[i][k][j+1];
				}

				else if(j == 1){
					W_x[j] += Stress[i][k][j+1]*Strain_x[i][k][j-1]+Stress[i][k][j]*Strain_x[i][k][j];
				}
			}
		}
	}


	for( j = 0; j < DIMENSION; j++ ){
		printf("\nW_x[%d] = %le\n", j, W_x[j]);
	}


	for( i = 0; i < DIMENSION; i++ ){
		P_1j[i] = W_K_D[i] - W_x[i];

		printf("\nP_1j[%d] = %lf\n", i, P_1j[i]);
	}
}
*/

__host__ int SerchForElement(int mesh_n, int iPatch, int Total_Element, int iX, int iY)
{
	int iii = 0;

	printf("iii%d\n", iii);
	printf("Total_Element%d\n", Total_Element);
	for (iii = 0; iii < Total_Element; iii++)
	{
		// printf("%d\n",iii+Total_Element_to_mesh[mesh_n]);
		// printf("ipatch%d\n",iPatch);
		if (Element_patch[iii + Total_Element_to_mesh[mesh_n]] == iPatch)
		{
			// printf("Check SerchForElement 1 iii = %d\n", iii+Total_Element_to_mesh[mesh_n]);
			// printf("ENC[iPatch][iii][0] = %d ENC[iPatch][iii][1] = %d  iX = %d  iY = %d\n",
			// 		ENC[iPatch][iii+Total_Element_to_mesh[mesh_n]][0],
			// 		ENC[iPatch][iii+Total_Element_to_mesh[mesh_n]][1],
			// 		iX, iY);
			if (iX == ENC[iPatch][iii + Total_Element_to_mesh[mesh_n]][0] && iY == ENC[iPatch][iii + Total_Element_to_mesh[mesh_n]][1])
				goto loopend;
			/* iii --; */

			// printf("Check SerchForElement 2 iii = %d\n", iii);
		}
	}
loopend:

	return (iii);
}

__host__ void Setting_Dist_Load_2D(int mesh_n, int iPatch, int Total_Element, int iCoord, double val_Coord,
								   double Range_Coord[2], int type_load, double Coeff_Dist_Load[3])
{
	int iii, jjj;
	// int iDir_Element[MAX_N_KNOT], jDir_Element;
	int N_Seg_Load_Element_iDir = 0, jCoord;
	int iRange_ele[2] /*, jRange_ele*/;
	int iPos[2] = {-10000, -10000}, jPos[2] = {-10000, -10000};
	// double Coord_Seg_Load_Element_iDir[MAX_N_KNOT][2], Coord_Seg_Load_Element_jDir[2];
	int No_Element_for_Integration[MAX_N_KNOT], No_Element_For_Dist_Load;
	int iX, iY;
	// int Element_Integration;
	int iControlpoint[MAX_NO_CCpoint_ON_ELEMENT], ic, ig, NNG = 3;
	double val_jCoord_Local = 0.0;
	double GaussPt[3], Weight[3];
	double Gg = pow(3.0 / 5.0, 0.5);

	/* type_load: 0: Dist load in x direction
	 * 	              1:              y direction
	 * 	              2:              normal direciton */

	GaussPt[0] = -Gg;
	GaussPt[1] = 0.0;
	GaussPt[2] = Gg;
	Weight[0] = 5.0 / 9.0;
	Weight[1] = 8.0 / 9.0;
	Weight[2] = 5.0 / 9.0;

	/* iCoord=0: Load on Eta=Constant
	   iCoord=1: Load on Xi=Constant */
	if (iCoord == 0)
		jCoord = 1;
	if (iCoord == 1)
		jCoord = 0;

	/* val_Coord: Value of Eta or Xi of the line or surface to give the distributed load */

	/* Setting elements needed to computed the distributed load */

	for (iii = Order[iPatch][iCoord]; iii < No_knot[iPatch][iCoord] - Order[iPatch][iCoord] - 1; iii++)
	{
		double epsi = 0.00000000001;
		/* iPos[0] = -10000; iPos[1] = -10000; jPos[0] = -10000; jPos[1] = -10000;*/
		printf("Check1 iii = %d\n", iii);
		printf("Check2 Position_Knots[iCoord][iii]= %f  Range_Coord[0] =%f Position_Knots[iCoord][iii+1] = %f\n", Position_Knots[iPatch][iCoord][iii], Range_Coord[0], Position_Knots[iPatch][iCoord][iii + 1]);
		/*

		if(Position_Knots[iCoord][iii]-epsi <= Range_Coord[0] &&
			Position_Knots[iCoord][iii+1]+epsi > Range_Coord[0]) iPos[0] = iii;

		if(Position_Knots[iCoord][iii]-epsi <= Range_Coord[1] &&
						Position_Knots[iCoord][iii+1]+epsi > Range_Coord[1]) iPos[1] = iii+1;
	*/
		if (Position_Knots[iPatch][iCoord][iii] - epsi <= Range_Coord[0])
			iPos[0] = iii;
		if (Position_Knots[iPatch][iCoord][iii + 1] - epsi <= Range_Coord[1])
			iPos[1] = iii + 1;
	}
	iRange_ele[0] = iPos[0] - Order[iPatch][iCoord];
	iRange_ele[1] = iPos[1] - Order[iPatch][iCoord] - 1;
	printf("iPos[0] = %d iPos[1] = %d\n", iPos[0], iPos[1]);
	printf("iRange_ele[0] = %d  iRange_ele[1] = %d\n", iRange_ele[0], iRange_ele[1]);

	if (iPos[0] < 0 || iPos[1] < 0)
	{
		printf("Error (Stop) iPos[0] = %d iPos[1] = %d\n", iPos[0], iPos[1]);
		exit(0);
	}

	for (jjj = Order[iPatch][jCoord]; jjj < No_knot[iPatch][jCoord] - Order[iPatch][jCoord] - 1; jjj++)
	{
		double epsi = 0.00000000001;
		/* jjj=Order[jCoord]; */
		if (Position_Knots[iPatch][jCoord][jjj] - epsi <= val_Coord &&
			Position_Knots[iPatch][jCoord][jjj + 1] + epsi > val_Coord)
		{
			jPos[0] = jjj;
			jPos[1] = jjj + 1;
			val_jCoord_Local = -1.0 + 2.0 * (val_Coord - Position_Knots[iPatch][jCoord][jjj]) /
										  (Position_Knots[iPatch][jCoord][jjj + 1] - Position_Knots[iPatch][jCoord][jjj]);
		}
		//(2019_06_13)printf("Check jjj count: jjj =  %d\n",jjj);
	}
	// jRange_ele = jPos[0] - Order[iPatch][jCoord];
	//(2019_06_13)printf("jPos[0] = %d jPos[1] = %d  jRange_ele = %d val_jCoord_Local = %f\n",jPos[0],jPos[1],jRange_ele, val_jCoord_Local);

	if (jPos[0] < 0 || jPos[1] < 0)
	{
		printf("Error (Stop) jPos[0] = %d jPos[1] = %d\n", jPos[0], jPos[1]);
		exit(0);
	}

	for (iii = iPos[0]; iii < iPos[1]; iii++)
	{
		// Coord_Seg_Load_Element_iDir[iii][0] = Position_Knots[iPatch][iCoord][iii + iPos[0]];
		// Coord_Seg_Load_Element_iDir[iii][1] = Position_Knots[iPatch][iCoord][iii + iPos[0] + 1];
		// iDir_Element[N_Seg_Load_Element_iDir] = iii - Order[iPatch][iCoord];
		N_Seg_Load_Element_iDir++;
	}

	// Coord_Seg_Load_Element_jDir[0] = Position_Knots[iPatch][iCoord][jPos[0]];
	// Coord_Seg_Load_Element_jDir[1] = Position_Knots[iPatch][iCoord][jPos[1]];
	// jDir_Element = jPos[0] - Order[iPatch][iCoord];
	iii = 0;
	if (iCoord == 1)
	{
		// int iX, iY;
		iX = jPos[0] - Order[iPatch][0];
		//(2019_06_13)printf("Check iPos[0] = %d  iPos[1] = %d\n",iPos[0],iPos[1]);
		for (iY = iPos[0] - Order[iPatch][1]; iY < iPos[1] - Order[iPatch][1]; iY++)
		{
			//(2019_06_13)printf("Check iY = %d\n",iY);
			No_Element_for_Integration[iii] = SerchForElement(mesh_n, iPatch, Total_Element, iX, iY);
			printf("Check No_Element_for_Integration[%d] = %d\n", iii, No_Element_for_Integration[iii]); //(2019_06_13)
			iii++;
		}
	}

	if (iCoord == 0)
	{
		// int iX, iY;
		iY = jPos[0] - Order[iPatch][1];
		//(2019_06_13)printf("Check iPos[0] = %d  iPos[1] = %d\n",iPos[0],iPos[1]);
		for (iX = iPos[0] - Order[iPatch][0]; iX < iPos[1] - Order[iPatch][0]; iX++)
		{
			//(2019_06_13)printf("Check iX = %d\n",iX);
			No_Element_for_Integration[iii] = SerchForElement(mesh_n, iPatch, Total_Element, iX, iY);
			printf("Check No_Element_for_Integration[%d] = %d\n", iii, No_Element_for_Integration[iii]); //(2019_06_13)
			iii++;
		}
	}
	No_Element_For_Dist_Load = iii;
	//(2019_06_13)printf("No_Element_For_Dist_Load = %d\n",No_Element_For_Dist_Load);

	/* Book keeping finished */

	for (iii = 0; iii < No_Element_For_Dist_Load; iii++)
	{ // B
		//(2019_06_13)printf("Check3 iii = %d\n",iii);
		//(2019_06_13)printf("Total_element_all_ID[No_Element_for_Integration[iii]] = %d\n No_Element_for_Integration[iii] = %d  iii = %d\n",
		// Total_element_all_ID[No_Element_for_Integration[iii]],No_Element_for_Integration[iii],iii);
		if (Total_element_all_ID[No_Element_for_Integration[iii]] == 1)
		{ // A
			iX = ENC[iPatch][No_Element_for_Integration[iii]][0];
			iY = ENC[iPatch][No_Element_for_Integration[iii]][1];
			//(2019_06_13)printf("iX = %d  iY = %d\n",iX, iY);

			for (ic = 0; ic < (Order[iPatch][0] + 1) * (Order[iPatch][1] + 1); ic++)
				iControlpoint[ic] = Controlpoint_of_Element[No_Element_for_Integration[iii]][ic];

			for (ig = 0; ig < NNG; ig++)
			{
				double Local_Coord[2], sfc, dxyzdge[3], detJ, XiEtaCoordParen, valDistLoad;
				int icc;
				Local_Coord[jCoord] = val_jCoord_Local;
				Local_Coord[iCoord] = GaussPt[ig];
				printf("ig = %d   Local_Coord[jCoord] = %f Local_Coord[iCoord] = %f\n", ig, Local_Coord[jCoord], Local_Coord[iCoord]);

				ShapeFunc_from_paren(Local_Coord, iCoord, No_Element_for_Integration[iii]);
				XiEtaCoordParen = Position_Data_param[iCoord];
				printf("Check  Coeff_Dist_Load[0] = %f Coeff_Dist_Load[1] = %f  Coeff_Dist_Load[2] = %f  Position_Data_param[iCoord] = %f\n", Coeff_Dist_Load[0], Coeff_Dist_Load[1], Coeff_Dist_Load[2], Position_Data_param[iCoord]);
				valDistLoad = Coeff_Dist_Load[0] + Coeff_Dist_Load[1] * XiEtaCoordParen + Coeff_Dist_Load[2] * XiEtaCoordParen * XiEtaCoordParen;

				//き裂の分布荷重を与える時のために...
				// printf("XiEtaCoordParen=%lf\n",XiEtaCoordParen);
				// double theta;
				// theta = XiEtaCoordParen*PI/2;
				// theta = XiEtaCoordParen*2*PI/line_No_real_element[0][1];
				// printf("theta=%lf\n",theta*180/PI);
				// valDistLoad = cos(theta);
				// printf("valDistLoad=%2.10lf\n",valDistLoad);

				dxyzdge[0] = 0.0;
				dxyzdge[1] = 0.0;
				dxyzdge[2] = 0.0;
				for (icc = 0; icc < (Order[iPatch][0] + 1) * (Order[iPatch][1] + 1); icc++)
				{
					// printf("dxy0:%lf\tdxy1:%lf\n",dxyzdge[0],dxyzdge[1]);
					// printf("Node%lf\tNode%lf\n",Node_Coordinate[iControlpoint[icc]][0],Node_Coordinate[iControlpoint[icc]][1]);
					dxyzdge[0] += dShape_func(icc, iCoord, Local_Coord, No_Element_for_Integration[iii]) * Node_Coordinate[iControlpoint[icc]][0];
					dxyzdge[1] += dShape_func(icc, iCoord, Local_Coord, No_Element_for_Integration[iii]) * Node_Coordinate[iControlpoint[icc]][1];
					// printf("dxy0:%lf\tdxy1:%lf\n",dxyzdge[0],dxyzdge[1]);
				}

				// printf("dxy")

				detJ = sqrt(dxyzdge[0] * dxyzdge[0] + dxyzdge[1] * dxyzdge[1]);
				//(2019_06_13)
				printf("Check the value of detJ etc: detJ = %f dxyzdge[0] = %f dxyzdge[1] = %f\n", detJ, dxyzdge[0], dxyzdge[1]);
				if (type_load < 2)
				{
					for (ic = 0; ic < (Order[iPatch][0] + 1) * (Order[iPatch][1] + 1); ic++)
					{
						// printf("Order[%d][0];%d,Order[%d][1]:%d\n",iPatch,Order[iPatch][0],iPatch,Order[iPatch][1]);
						sfc = Shape_func(ic, Local_Coord, No_Element_for_Integration[iii]);
						Equivalent_Nodal_Force[iControlpoint[ic]][type_load] += valDistLoad * sfc * detJ * Weight[ig];
						//(2019_06_13)printf("Check ic = %d sfc = %f   Weight[ig] = %f  valDistLoad = %f\n",ic,sfc,Weight[ig],valDistLoad);
						//(2019_06_13)printf("Equivalent_Nodal_Force[%d][%d]:%le\n",iControlpoint[ic],type_load,Equivalent_Nodal_Force[iControlpoint[ic]][type_load] );
					}
				}

				if (type_load == 2)
				{
					double LoadDir[2];
					LoadDir[0] = dxyzdge[1] / detJ;
					LoadDir[1] = -dxyzdge[0] / detJ;
					// printf("%lf\t%lf\n",LoadDir[0],LoadDir[1]);
					for (ic = 0; ic < (Order[iPatch][0] + 1) * (Order[iPatch][1] + 1); ic++)
					{
						sfc = Shape_func(ic, Local_Coord, No_Element_for_Integration[iii]);
						// printf("sfc=%lf\n",sfc);
						Equivalent_Nodal_Force[iControlpoint[ic]][0] +=
							LoadDir[0] * valDistLoad * sfc * detJ * Weight[ig];
						Equivalent_Nodal_Force[iControlpoint[ic]][1] +=
							LoadDir[1] * valDistLoad * sfc * detJ * Weight[ig];
						printf("Equivalent_Nodal_Force[%d][0]=%lf\nEquivalent_Nodal_Force[%d][1]=%lf\n",
							   iControlpoint[ic], Equivalent_Nodal_Force[iControlpoint[ic]][0],
							   iControlpoint[ic], Equivalent_Nodal_Force[iControlpoint[ic]][1]);
						// printf("LoadDir[0]*(0.5-0.5*cos(2*theta))*sfc*detJ*Weight[%d]=%lf*%lf*%lf*%lf*%lf=%lf\n",ig,LoadDir[0],(0.5-0.5*cos(2*theta)),sfc,detJ,Weight[ig],LoadDir[0] * (0.5-0.5*cos(2*theta)) * sfc * detJ * Weight[ig]);
						// printf("LoadDir[1]*(0.5-0.5*cos(2*theta))*sfc*detJ*Weight[%d]=%lf*%lf*%lf*%lf*%lf=%lf\n",ig,LoadDir[1],(0.5-0.5*cos(2*theta)),sfc,detJ,Weight[ig],LoadDir[1] * (0.5-0.5*cos(2*theta)) * sfc * detJ * Weight[ig]);
					}
				}
				if (type_load == 3)
				{
					double LoadDir[2];
					LoadDir[0] = dxyzdge[0] / detJ;
					LoadDir[1] = dxyzdge[1] / detJ;
					for (ic = 0; ic < (Order[iPatch][0] + 1) * (Order[iPatch][1] + 1); ic++)
					{
						sfc = Shape_func(ic, Local_Coord,
										 No_Element_for_Integration[iii]);
						Equivalent_Nodal_Force[iControlpoint[ic]][0] +=
							LoadDir[0] * valDistLoad * sfc * detJ * Weight[ig];
						Equivalent_Nodal_Force[iControlpoint[ic]][1] +=
							LoadDir[1] * valDistLoad * sfc * detJ * Weight[ig];
						// printf("Equivalent_Nodal_Force[%d][0]=%lf\nEquivalent_Nodal_Force[%d][1]=%lf\n",iControlpoint[ic],Equivalent_Nodal_Force[iControlpoint[ic]][0],iControlpoint[ic],Equivalent_Nodal_Force[iControlpoint[ic]][1]);
						// printf("LoadDir[0]*sin(theta)*cos(theta)*sfc*detJ*Weight[%d]=%lf*%lf*%lf*%lf*%lf=%lf\n",ig,LoadDir[0],sin(theta)*cos(theta),sfc,detJ,Weight[ig],LoadDir[0] * sin(theta)*cos(theta) * sfc * detJ * Weight[ig]);
						// printf("LoadDir[1]*sin(theta)*cos(theta)*sfc*detJ*Weight[%d]=%lf*%lf*%lf*%lf*%lf=%lf\n",ig,LoadDir[1],sin(theta)*cos(theta),sfc,detJ,Weight[ig],LoadDir[1] * sin(theta)*cos(theta) * sfc * detJ * Weight[ig]);
					}
				}
			}
		} // A
	}	  // B
}

//重ね合わせた結果の出力(NURBS_input_for_s-IGA)
__host__ void GetLocData()
{
	double temp;

	//必要なのはローカルのパッチ数とコントロールポイント数
	printf("Start Get Local Data\n\n");
	fp = fopen("input_local.txt", "r");

	fscanf(fp, "%lf%lf", &temp, &temp);
	fscanf(fp, "\n");
	fscanf(fp, "%d", &n_patch_loc);
	fscanf(fp, "\n");
	fscanf(fp, "%d", &loc_cntl_p_n);
	fscanf(fp, "\n");
	fclose(fp);
	printf("patches(in local):% d\n", n_patch_loc);
	printf("control points(in local):% d\n", loc_cntl_p_n);
	printf("\nFinish Get Local Data\n\n");
}

__host__ void ReadFile()
{
	int i, j;
	double temp1, temp2, temp3;
	int temp_int;
	printf("Start Reading input\n\n");
	fp = fopen("input_for_NURBS.txt", "r");

	fscanf(fp, "%lf%lf", &E, &nu);
	fscanf(fp, "\n");
	printf("E nu: % 1.4e % 1.4e\n", E, nu);

	fscanf(fp, "%d", &patch_n);
	fscanf(fp, "\n");
	printf("patches: %d \n", patch_n);
	if (patch_n > MAX_PATCHES)
	{
		printf("Error!!\n");
		printf("Too many patches!\n"
			   "Maximum of patches is %d (Now %d)\n"
			   "\n",
			   MAX_PATCHES, patch_n);
		exit(1);
	}

	fscanf(fp, "%d", &cntl_p_n);
	fscanf(fp, "\n");
	printf("total control points:%d \n", cntl_p_n);
	if (cntl_p_n > MAX_CNRL_P)
	{
		printf("Error!!\n");
		printf("Too many control points!\n"
			   "Maximum of control points is %d (Now %d)\n"
			   "\n",
			   MAX_CNRL_P, cntl_p_n);
		exit(1);
	}

	for (i = 0; i < patch_n; i++)
	{
		fscanf(fp, "%d%d", &order_xi[i], &order_eta[i]);
		fscanf(fp, "\n");
		printf("order %d: %d %d\n", i, order_xi[i], order_eta[i]);
		if (order_xi[i] > MAX_ORDER)
		{
			printf("Error!!\n");
			printf("Order too big at xi!\n"
				   "Maximum of order is %d (Now %d at patch %d)\n"
				   "\n",
				   MAX_ORDER, order_xi[i], i);
			exit(1);
		}
		if (order_eta[i] > MAX_ORDER)
		{
			printf("Error!!\n");
			printf("Order too big at eta!\n"
				   "Maximum of order is %d (Now %d at patch %d)\n"
				   "\n",
				   MAX_ORDER, order_eta[i], i);
			exit(1);
		}
	}

	for (i = 0; i < patch_n; i++)
	{
		fscanf(fp, "%d%d", &knot_n_xi[i], &knot_n_eta[i]);
		fscanf(fp, "\n");
		printf("knots %d: %d %d\n", i, knot_n_xi[i], knot_n_eta[i]);
		if (knot_n_xi[i] > MAX_KNOTS)
		{
			printf("Error!!\n");
			printf("Knot vector too long at xi!\n"
				   "Maximum of knot vector is %d (Now %d at patch %d)\n"
				   "\n",
				   MAX_KNOTS, knot_n_xi[i], i);
			exit(1);
		}
		if (knot_n_eta[i] > MAX_KNOTS)
		{
			printf("Error!!\n");
			printf("Knot vector too long at eta!\n"
				   "Maximum of knot vector is %d (Now %d at patch %d)\n"
				   "\n",
				   MAX_KNOTS, knot_n_eta[i], i);
			exit(1);
		}
	}

	for (i = 0; i < patch_n; i++)
	{
		fscanf(fp, "%d%d", &cntl_p_n_xi[i], &cntl_p_n_eta[i]);
		printf("control points %d: %d %d\n",
			   i, cntl_p_n_xi[i], cntl_p_n_eta[i]);
		fscanf(fp, "\n");
	}
	printf("\n");

	for (i = 0; i < patch_n; i++)
	{
		for (j = 0; j < cntl_p_n_xi[i] * cntl_p_n_eta[i]; j++)
		{
			fscanf(fp, "%d", &temp_index[i][j]);
			printf("%d ", temp_index[i][j]);
		}
		fscanf(fp, "\n");
		printf("\n");
	}
	printf("\n");

	fscanf(fp, "%lf%lf%lf", &temp1, &temp2, &temp3);
	fscanf(fp, "\n");

	for (i = 0; i < patch_n; i++)
	{
		for (j = 0; j < knot_n_xi[i]; j++)
		{
			fscanf(fp, "%le", &knot_vec_xi[i][j]);
			printf("%f\t", knot_vec_xi[i][j]);
		}
		printf("\n");
		for (j = 0; j < knot_n_eta[i]; j++)
		{
			fscanf(fp, "%le", &knot_vec_eta[i][j]);
			printf("%f\t", knot_vec_eta[i][j]);
		}
		printf("\n");
	}
	printf("\n");

	for (i = 0; i < cntl_p_n; i++)
	{
		fscanf(fp, "%d%le%le%le",
			   &temp_int,
			   &temp_cntl_px[i], &temp_cntl_py[i], &temp_weight[i]);
		printf("%d\t%f\t%f\t%f\n",
			   temp_int,
			   temp_cntl_px[i], temp_cntl_py[i], temp_weight[i]);
	}
	printf("\n");

	for (i = 0; i < patch_n; i++)
	{
		for (j = 0; j < cntl_p_n_xi[i] * cntl_p_n_eta[i]; j++)
		{
			cntl_px[i][j] = temp_cntl_px[temp_index[i][j]];
			cntl_py[i][j] = temp_cntl_py[temp_index[i][j]];
			weight[i][j] = temp_weight[temp_index[i][j]];
			printf("%d\t%f\t%f\t%f\n",
				   temp_index[i][j],
				   cntl_px[i][j], cntl_py[i][j], weight[i][j]);
		}
		printf("\n");
	}
	fclose(fp);
	printf("End Reading input\n\n");

	if (fields_flag)
	{
		printf("Start Reading displacement\n\n");
		fp = fopen("Displacement.dat", "r");
		char buff[256];

		fscanf(fp, "%s", buff);
		fscanf(fp, "%s", buff);

		for (i = 0; i < cntl_p_n; i++)
		{
			fscanf(fp, "%d:%le%le",
				   &temp_int, &temp_disp_x[i], &temp_disp_y[i]);
			printf("%d\t%1.6e\t%1.6e\n",
				   temp_int, temp_disp_x[i], temp_disp_y[i]);
		}
		printf("\n");

		for (i = 0; i < patch_n; i++)
		{
			for (j = 0; j < cntl_p_n_xi[i] * cntl_p_n_eta[i]; j++)
			{
				disp_cntl_px[i][j] = temp_disp_x[temp_index[i][j]];
				disp_cntl_py[i][j] = temp_disp_y[temp_index[i][j]];
				printf("%d\t%f\t%f\t%f\n",
					   temp_index[i][j], cntl_px[i][j], cntl_py[i][j], weight[i][j]);
			}
			printf("\n");
		}
		fclose(fp); // ファイルを閉じる
		printf("End Reading displpacement\n\n");
	}

	fp = fopen("Displacement_loc.dat", "w");
	glo_cntl_p_n = cntl_p_n - loc_cntl_p_n;
	fprintf(fp, "label=Displacement\n"
				"num_items=%d\n\n",
			loc_cntl_p_n);
	for (i = 0; i < loc_cntl_p_n; i++)
	{
		// fprintf(fp, "%d:	%le %le \n", i, temp_disp_x[i + glo_cntl_p_n], temp_disp_y[i + glo_cntl_p_n]);
		fprintf(fp, "%d:	%.16e %.16e \n", i, temp_disp_x[i + glo_cntl_p_n], temp_disp_y[i + glo_cntl_p_n]);
	}
	fclose(fp);
}

__host__ int CalcXiEtaByNR(double px, double py,
						   double *input_knot_vec_xi, double *input_knot_vec_eta,
						   double *cntl_px, double *cntl_py,
						   double *disp_cntl_px, double *disp_cntl_py,
						   int cntl_p_n_xi, int cntl_p_n_eta,
						   double *weight, int order_xi, int order_eta,
						   double *output_xi, double *output_eta,
						   double *disp_x_glo, double *disp_y_glo,
						   double *strain_xx_glo, double *strain_yy_glo, double *strain_xy_glo)
{
	double temp_xi, temp_eta;
	double temp_x, temp_y;
	double temp_matrix[2][2];
	double temp_dxi, temp_deta;
	double temp_tol_x = DBL_MAX;
	double temp_tol_y = DBL_MAX;

	(*output_xi) = 0;
	(*output_eta) = 0;

	int i;
	// double tol = 10e-8;
	// int repeat = 10000;
	int repeat = 100;
	double tol = 10e-14;

	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	// printf("% 1.8e % 1.8e\n", temp_xi, temp_eta);

	for (i = 0; i < repeat; i++)
	{
		rNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
					   cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
					   weight, order_xi, order_eta,
					   temp_xi, temp_eta,
					   &temp_x, &temp_y,
					   &temp_matrix[0][0], &temp_matrix[0][1],
					   &temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol) {
		if (temp_tol_x + temp_tol_y < tol)
		{
			// printf("rNURBS\n");
			// printf("repeat = %d\n", i);
			if (temp_xi == input_knot_vec_xi[0] || temp_eta == input_knot_vec_eta[0])
			{
				break;
			}
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			int knot_n_xi = cntl_p_n_xi + order_xi + 1;
			int knot_n_eta = cntl_p_n_eta + order_eta + 1;
			double dtilda_xi = 0.0;
			double dtilda_eta = 0.0;
			double disp_x, disp_y;
			double dxi_x, deta_x, dxi_y, deta_y;
			double dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y;
			double temp_matrix2[2][2];
			double temp;
			double strain_xx, strain_yy, strain_xy;
			// double stress_xx, stress_yy, stress_xy;

			for (i = 0; i < knot_n_xi; i++)
			{
				if (input_knot_vec_xi[i] < temp_xi && temp_xi <= input_knot_vec_xi[i + 1])
				{
					dtilda_xi = (input_knot_vec_xi[i + 1] - input_knot_vec_xi[i]) / 2.0;
					// printf("xi%f\n", dtilda_xi);
					break;
				}
			}
			for (i = 0; i < knot_n_eta; i++)
			{
				if (input_knot_vec_eta[i] < temp_eta && temp_eta <= input_knot_vec_eta[i + 1])
				{
					dtilda_eta = (input_knot_vec_eta[i + 1] - input_knot_vec_eta[i]) / 2.0;
					// printf("eta%f\n", dtilda_eta);
					break;
				}
			}

			rNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
						   cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
						   weight, order_xi, order_eta,
						   temp_xi, temp_eta,
						   &temp, &temp,
						   &dxi_x, &deta_x,
						   &dxi_y, &deta_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			// 	   dxi_x, deta_x, dxi_y, deta_y);

			rNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
						   disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
						   weight, order_xi, order_eta,
						   temp_xi, temp_eta,
						   &disp_x, &disp_y,
						   &dxi_disp_x, &deta_disp_x,
						   &dxi_disp_y, &deta_disp_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			// 	   dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y);

			temp_matrix2[0][0] = dxi_x * dtilda_xi;
			temp_matrix2[0][1] = dxi_y * dtilda_xi;
			temp_matrix2[1][0] = deta_x * dtilda_eta;
			temp_matrix2[1][1] = deta_y * dtilda_eta;

			InverseMatrix_2D(temp_matrix2);

			strain_xx = temp_matrix2[0][0] * dtilda_xi * dxi_disp_x + temp_matrix2[0][1] * dtilda_eta * deta_disp_x;
			strain_yy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_y + temp_matrix2[1][1] * dtilda_eta * deta_disp_y;
			strain_xy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_x + temp_matrix2[1][1] * dtilda_eta * deta_disp_x + temp_matrix2[0][0] * dtilda_xi * dxi_disp_y + temp_matrix2[0][1] * dtilda_eta * deta_disp_y;

			// 応力の計算を行わないのでコメントアウト
			// double D_matrix[3][3] = {{0.0}};
			// int DM = 1;
			// if (DM == 0) { //平面応力状態
			// 	temp = E * (1.0 - nu * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu * temp;
			// 	D_matrix[1][0] = nu * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - nu) / 2.0 * temp;
			// } else if (DM == 1) { //平面ひずみ状態(2Dの場合はこっち)
			// 	temp = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][0] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp;
			// }

			// strainしか使わないのでコメントアウト
			// stress_xx = D_matrix[0][0] * strain_xx + D_matrix[0][1] * strain_yy;
			// stress_yy = D_matrix[1][0] * strain_xx + D_matrix[1][1] * strain_yy;
			// stress_xy = D_matrix[2][2] * strain_xy;

			temp = sqrt(disp_x * disp_x + disp_y * disp_y);
			// printf("x:   % 1.8e\n", px);
			// printf("y:   % 1.8e\n", py);
			// printf("xi:  % 1.8e\n", temp_xi);
			// printf("eta: % 1.8e\n", temp_eta);
			// printf("Displacement x: % 1.8e\n", disp_x);
			// printf("Displacement y: % 1.8e\n", disp_y);
			// printf("Displacement  : % 1.8e\n", temp);
			// printf("Strain xx: % 1.8e\n", strain_xx);
			// printf("Strain yy: % 1.8e\n", strain_yy);
			// printf("Strain xy: % 1.8e\n", strain_xy);
			// printf("Stress xx: % 1.8e\n", stress_xx);
			// printf("Stress yy: % 1.8e\n", stress_yy);
			// printf("Stress xy: % 1.8e\n", stress_xy);

			(*disp_x_glo) = disp_x;
			(*disp_y_glo) = disp_y;

			(*strain_xx_glo) = strain_xx;
			(*strain_yy_glo) = strain_yy;
			(*strain_xy_glo) = strain_xy;

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	for (i = 0; i < repeat; i++)
	{
		lNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
					   cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
					   weight, order_xi, order_eta,
					   temp_xi, temp_eta,
					   &temp_x, &temp_y,
					   &temp_matrix[0][0], &temp_matrix[0][1],
					   &temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol) {
		if (temp_tol_x + temp_tol_y < tol)
		{
			// printf("lNURBS\n");
			// printf("repeat = %d\n", i);
			if (temp_xi == input_knot_vec_xi[cntl_p_n_xi + order_xi] || temp_eta == input_knot_vec_eta[cntl_p_n_eta + order_eta])
			{
				break;
			}
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			int knot_n_xi = cntl_p_n_xi + order_xi + 1;
			int knot_n_eta = cntl_p_n_eta + order_eta + 1;
			double dtilda_xi = 0.0;
			double dtilda_eta = 0.0;
			double disp_x, disp_y;
			double dxi_x, deta_x, dxi_y, deta_y;
			double dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y;
			double temp_matrix2[2][2];
			double temp;
			double strain_xx, strain_yy, strain_xy;
			// double stress_xx, stress_yy, stress_xy;

			for (i = 0; i < knot_n_xi; i++)
			{
				if (input_knot_vec_xi[i] <= temp_xi && temp_xi < input_knot_vec_xi[i + 1])
				{
					dtilda_xi = (input_knot_vec_xi[i + 1] - input_knot_vec_xi[i]) / 2.0;
					// printf("%f\n", dtilda_xi);
					break;
				}
			}
			for (i = 0; i < knot_n_eta; i++)
			{
				if (input_knot_vec_eta[i] <= temp_eta && temp_eta < input_knot_vec_eta[i + 1])
				{
					dtilda_eta = (input_knot_vec_eta[i + 1] - input_knot_vec_eta[i]) / 2.0;
					// printf("%f\n", dtilda_eta);
					break;
				}
			}

			lNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
						   cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
						   weight, order_xi, order_eta,
						   temp_xi, temp_eta,
						   &temp, &temp,
						   &dxi_x, &deta_x,
						   &dxi_y, &deta_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			// 	   dxi_x, deta_x, dxi_y, deta_y);

			lNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
						   disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
						   weight, order_xi, order_eta,
						   temp_xi, temp_eta,
						   &disp_x, &disp_y,
						   &dxi_disp_x, &deta_disp_x,
						   &dxi_disp_y, &deta_disp_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			// 	   dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y);

			temp_matrix2[0][0] = dxi_x * dtilda_xi;
			temp_matrix2[0][1] = dxi_y * dtilda_xi;
			temp_matrix2[1][0] = deta_x * dtilda_eta;
			temp_matrix2[1][1] = deta_y * dtilda_eta;

			InverseMatrix_2D(temp_matrix2);

			strain_xx = temp_matrix2[0][0] * dtilda_xi * dxi_disp_x + temp_matrix2[0][1] * dtilda_eta * deta_disp_x;
			strain_yy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_y + temp_matrix2[1][1] * dtilda_eta * deta_disp_y;
			strain_xy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_x + temp_matrix2[1][1] * dtilda_eta * deta_disp_x + temp_matrix2[0][0] * dtilda_xi * dxi_disp_y + temp_matrix2[0][1] * dtilda_eta * deta_disp_y;

			// 応力の計算を行わないのでコメントアウト
			// double D_matrix[3][3] = {{0.0}};
			// int DM = 1;
			// if (DM == 0) { //平面応力状態
			// 	temp = E * (1.0 - nu * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu * temp;
			// 	D_matrix[1][0] = nu * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - nu) / 2.0 * temp;
			// } else if (DM == 1) { //平面ひずみ状態(2Dの場合はこっち)
			// 	temp = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][0] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp;
			// }

			// strainしか使わないのでコメントアウト
			// stress_xx = D_matrix[0][0] * strain_xx + D_matrix[0][1] * strain_yy;
			// stress_yy = D_matrix[1][0] * strain_xx + D_matrix[1][1] * strain_yy;
			// stress_xy = D_matrix[2][2] * strain_xy;

			temp = sqrt(disp_x * disp_x + disp_y * disp_y);
			// printf("x:   % 1.8e\n", px);
			// printf("y:   % 1.8e\n", py);
			// printf("xi:  % 1.8e\n", temp_xi);
			// printf("eta: % 1.8e\n", temp_eta);
			// printf("Displacement x: % 1.8e\n", disp_x);
			// printf("Displacement y: % 1.8e\n", disp_y);
			// printf("Displacement  : % 1.8e\n", temp);
			// printf("Strain xx: % 1.8e\n", strain_xx);
			// printf("Strain yy: % 1.8e\n", strain_yy);
			// printf("Strain xy: % 1.8e\n", strain_xy);
			// printf("Stress xx: % 1.8e\n", stress_xx);
			// printf("Stress yy: % 1.8e\n", stress_yy);
			// printf("Stress xy: % 1.8e\n", stress_xy);

			(*disp_x_glo) = disp_x;
			(*disp_y_glo) = disp_y;

			(*strain_xx_glo) = strain_xx;
			(*strain_yy_glo) = strain_yy;
			(*strain_xy_glo) = strain_xy;

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	for (i = 0; i < repeat; i++)
	{
		rlNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
						cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
						weight, order_xi, order_eta,
						temp_xi, temp_eta,
						&temp_x, &temp_y,
						&temp_matrix[0][0], &temp_matrix[0][1],
						&temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol) {
		if (temp_tol_x + temp_tol_y < tol)
		{
			// printf("rlNURBS\n");
			if (temp_xi == input_knot_vec_xi[0])
			{
				break;
			}
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			int knot_n_xi = cntl_p_n_xi + order_xi + 1;
			int knot_n_eta = cntl_p_n_eta + order_eta + 1;
			double dtilda_xi = 0.0;
			double dtilda_eta = 0.0;
			double disp_x, disp_y;
			double dxi_x, deta_x, dxi_y, deta_y;
			double dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y;
			double temp_matrix2[2][2];
			double temp;
			double strain_xx, strain_yy, strain_xy;
			// double stress_xx, stress_yy, stress_xy;

			for (i = 0; i < knot_n_xi; i++)
			{
				if (input_knot_vec_xi[i] < temp_xi && temp_xi <= input_knot_vec_xi[i + 1])
				{
					dtilda_xi = (input_knot_vec_xi[i + 1] - input_knot_vec_xi[i]) / 2.0;
					// printf("%f\n", dtilda_xi);
					break;
				}
			}
			for (i = 0; i < knot_n_eta; i++)
			{
				if (input_knot_vec_eta[i] <= temp_eta && temp_eta < input_knot_vec_eta[i + 1])
				{
					dtilda_eta = (input_knot_vec_eta[i + 1] - input_knot_vec_eta[i]) / 2.0;
					// printf("%f\n", dtilda_eta);
					break;
				}
			}

			rlNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
							cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
							weight, order_xi, order_eta,
							temp_xi, temp_eta,
							&temp, &temp,
							&dxi_x, &deta_x,
							&dxi_y, &deta_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			//	   dxi_x, deta_x, dxi_y, deta_y);

			rlNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
							disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
							weight, order_xi, order_eta,
							temp_xi, temp_eta,
							&disp_x, &disp_y,
							&dxi_disp_x, &deta_disp_x,
							&dxi_disp_y, &deta_disp_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			//	   dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y);

			temp_matrix2[0][0] = dxi_x * dtilda_xi;
			temp_matrix2[0][1] = dxi_y * dtilda_xi;
			temp_matrix2[1][0] = deta_x * dtilda_eta;
			temp_matrix2[1][1] = deta_y * dtilda_eta;

			InverseMatrix_2D(temp_matrix2);

			strain_xx = temp_matrix2[0][0] * dtilda_xi * dxi_disp_x + temp_matrix2[0][1] * dtilda_eta * deta_disp_x;
			strain_yy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_y + temp_matrix2[1][1] * dtilda_eta * deta_disp_y;
			strain_xy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_x + temp_matrix2[1][1] * dtilda_eta * deta_disp_x + temp_matrix2[0][0] * dtilda_xi * dxi_disp_y + temp_matrix2[0][1] * dtilda_eta * deta_disp_y;

			// 応力の計算を行わないのでコメントアウト
			// double D_matrix[3][3] = {{0.0}};
			// int DM = 1;
			// if (DM == 0) { //平面応力状態
			// 	temp = E * (1.0 - nu * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu * temp;
			// 	D_matrix[1][0] = nu * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - nu) / 2.0 * temp;
			// } else if (DM == 1) { //平面ひずみ状態(2Dの場合はこっち)
			// 	temp = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][0] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp;
			// }

			// strainしか使わないのでコメントアウト
			// stress_xx = D_matrix[0][0] * strain_xx + D_matrix[0][1] * strain_yy;
			// stress_yy = D_matrix[1][0] * strain_xx + D_matrix[1][1] * strain_yy;
			// stress_xy = D_matrix[2][2] * strain_xy;

			temp = sqrt(disp_x * disp_x + disp_y * disp_y);
			// printf("x:   % 1.8e\n", px);
			// printf("y:   % 1.8e\n", py);
			// printf("xi:  % 1.8e\n", temp_xi);
			// printf("eta: % 1.8e\n", temp_eta);
			// printf("Displacement x: % 1.8e\n", disp_x);
			// printf("Displacement y: % 1.8e\n", disp_y);
			// printf("Displacement  : % 1.8e\n", temp);
			// printf("Strain xx: % 1.8e\n", strain_xx);
			// printf("Strain yy: % 1.8e\n", strain_yy);
			// printf("Strain xy: % 1.8e\n", strain_xy);
			// printf("Stress xx: % 1.8e\n", stress_xx);
			// printf("Stress yy: % 1.8e\n", stress_yy);
			// printf("Stress xy: % 1.8e\n", stress_xy);

			(*disp_x_glo) = disp_x;
			(*disp_y_glo) = disp_y;

			(*strain_xx_glo) = strain_xx;
			(*strain_yy_glo) = strain_yy;
			(*strain_xy_glo) = strain_xy;

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;
	for (i = 0; i < repeat; i++)
	{
		lrNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
						cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
						weight, order_xi, order_eta,
						temp_xi, temp_eta,
						&temp_x, &temp_y,
						&temp_matrix[0][0], &temp_matrix[0][1],
						&temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		//収束した場合////////////////////////////////////////////////////////////////
		// if (temp_tol_x < tol && temp_tol_y < tol) {
		if (temp_tol_x + temp_tol_y < tol)
		{
			// printf("lrNURBS\n");
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;

			int knot_n_xi = cntl_p_n_xi + order_xi + 1;
			int knot_n_eta = cntl_p_n_eta + order_eta + 1;
			double dtilda_xi = 0.0;
			double dtilda_eta = 0.0;
			double disp_x, disp_y;
			double dxi_x, deta_x, dxi_y, deta_y;
			double dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y;
			double temp_matrix2[2][2];
			double temp;
			double strain_xx, strain_yy, strain_xy;
			// double stress_xx, stress_yy, stress_xy;

			for (i = 0; i < knot_n_xi; i++)
			{
				if (input_knot_vec_xi[i] <= temp_xi && temp_xi < input_knot_vec_xi[i + 1])
				{
					dtilda_xi = (input_knot_vec_xi[i + 1] - input_knot_vec_xi[i]) / 2.0;
					// printf("%f\n", dtilda_xi);
					break;
				}
			}
			for (i = 0; i < knot_n_eta; i++)
			{
				if (input_knot_vec_eta[i] < temp_eta && temp_eta <= input_knot_vec_eta[i + 1])
				{
					dtilda_eta = (input_knot_vec_eta[i + 1] - input_knot_vec_eta[i]) / 2.0;
					// printf("%f\n", dtilda_eta);
					break;
				}
			}

			lrNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
							cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
							weight, order_xi, order_eta,
							temp_xi, temp_eta,
							&temp, &temp,
							&dxi_x, &deta_x,
							&dxi_y, &deta_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			//	   dxi_x, deta_x, dxi_y, deta_y);

			lrNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
							disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
							weight, order_xi, order_eta,
							temp_xi, temp_eta,
							&disp_x, &disp_y,
							&dxi_disp_x, &deta_disp_x,
							&dxi_disp_y, &deta_disp_y);
			// printf("% 1.4e % 1.4e % 1.4e % 1.4e\n",
			//	   dxi_disp_x, deta_disp_x, dxi_disp_y, deta_disp_y);

			temp_matrix2[0][0] = dxi_x * dtilda_xi;
			temp_matrix2[0][1] = dxi_y * dtilda_xi;
			temp_matrix2[1][0] = deta_x * dtilda_eta;
			temp_matrix2[1][1] = deta_y * dtilda_eta;

			InverseMatrix_2D(temp_matrix2);

			strain_xx = temp_matrix2[0][0] * dtilda_xi * dxi_disp_x + temp_matrix2[0][1] * dtilda_eta * deta_disp_x;
			strain_yy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_y + temp_matrix2[1][1] * dtilda_eta * deta_disp_y;
			strain_xy = temp_matrix2[1][0] * dtilda_xi * dxi_disp_x + temp_matrix2[1][1] * dtilda_eta * deta_disp_x + temp_matrix2[0][0] * dtilda_xi * dxi_disp_y + temp_matrix2[0][1] * dtilda_eta * deta_disp_y;

			// 応力の計算を行わないのでコメントアウト
			// double D_matrix[3][3] = {{0.0}};
			// int DM = 1;
			// if (DM == 0) { //平面応力状態
			// 	temp = E * (1.0 - nu * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu * temp;
			// 	D_matrix[1][0] = nu * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - nu) / 2.0 * temp;
			// } else if (DM == 1) { //平面ひずみ状態(2Dの場合はこっち)
			// 	temp = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
			// 	D_matrix[0][0] = temp;
			// 	D_matrix[0][1] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][0] = nu / (1.0 - nu) * temp;
			// 	D_matrix[1][1] = temp;
			// 	D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp;
			// }

			// strainしか使わないのでコメントアウト
			// stress_xx = D_matrix[0][0] * strain_xx + D_matrix[0][1] * strain_yy;
			// stress_yy = D_matrix[1][0] * strain_xx + D_matrix[1][1] * strain_yy;
			// stress_xy = D_matrix[2][2] * strain_xy;

			temp = sqrt(disp_x * disp_x + disp_y * disp_y);
			// printf("x:   % 1.8e\n", px);
			// printf("y:   % 1.8e\n", py);
			// printf("xi:  % 1.8e\n", temp_xi);
			// printf("eta: % 1.8e\n", temp_eta);
			// printf("Displacement x: % 1.8e\n", disp_x);
			// printf("Displacement y: % 1.8e\n", disp_y);
			// printf("Displacement  : % 1.8e\n", temp);
			// printf("Strain xx: % 1.8e\n", strain_xx);
			// printf("Strain yy: % 1.8e\n", strain_yy);
			// printf("Strain xy: % 1.8e\n", strain_xy);
			// printf("Stress xx: % 1.8e\n", stress_xx);
			// printf("Stress yy: % 1.8e\n", stress_yy);
			// printf("Stress xy: % 1.8e\n", stress_xy);

			(*disp_x_glo) = disp_x;
			(*disp_y_glo) = disp_y;

			(*strain_xx_glo) = strain_xx;
			(*strain_yy_glo) = strain_yy;
			(*strain_xy_glo) = strain_xy;

			return i;
		}

		InverseMatrix_2D(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];

		// temp_tol = sqrt(temp_dxi * temp_dxi + temp_deta * temp_deta);
		// printf("% 1.15e % 1.15e % 1.15e\n", temp_xi, temp_eta, temp_tol);
	}
	// printf("% 1.8e % 1.8e\n", temp_x, temp_y);
	return 0;
}

__host__ static void Calculation(int order_xi, int order_eta,
								 int knot_n_xi, int knot_n_eta,
								 int cntl_p_n_xi, int cntl_p_n_eta,
								 double *input_knot_vec_xi, double *input_knot_vec_eta,
								 double *cntl_px, double *cntl_py,
								 double *disp_cntl_px, double *disp_cntl_py,
								 double *weight)
{
	int i, j, k, l;
	double temp1, temp2, temp3;
	double temp_matrix[2][2];

	//計算するξ,ηの値決定と ∂ξ/∂チルダξ, ∂η/∂チルダη の計算
	double calc_xi[MAX_POINTS];	  //計算するξの値
	double calc_eta[MAX_POINTS];  //計算するηの値
	double dtilda_xi[MAX_KNOTS];  // ∂ξ/∂チルダξ
	double dtilda_eta[MAX_KNOTS]; // ∂η/∂チルダη
	k = 0;
	l = 0;
	for (i = 0; i < knot_n_xi - 1; i++)
	{
		if (input_knot_vec_xi[i] != input_knot_vec_xi[i + 1])
		{
			calc_xi[k] = input_knot_vec_xi[i];
			printf("%d\t%f\n", k, calc_xi[k]);
			dtilda_xi[l] = (input_knot_vec_xi[i + 1] - input_knot_vec_xi[i]) / 2.0;
			printf("%d\t%f\n", k, dtilda_xi[k]);
			k++;
			l++;
			if (division_ele_xi > 1)
			{
				temp1 = (input_knot_vec_xi[i + 1] - input_knot_vec_xi[i]) / (double)division_ele_xi;
				for (j = 1; j < division_ele_xi; j++)
				{
					calc_xi[k] = calc_xi[k - 1] + temp1;
					printf("%d\t%f\n", k, calc_xi[k]);
					k++;
				}
			}
		}
	}
	calc_xi[k] = input_knot_vec_xi[knot_n_xi - 1];
	printf("%d\t%f\n", k, calc_xi[k]);
	// printf("\n");
	division_n_xi = k + 1;
	element_n_xi = l;

	k = 0;
	l = 0;
	for (i = 0; i < knot_n_eta - 1; i++)
	{
		if (input_knot_vec_eta[i] != input_knot_vec_eta[i + 1])
		{
			calc_eta[k] = input_knot_vec_eta[i];
			// printf("%d\t%f\n", k, calc_eta[k]);
			dtilda_eta[l] = (input_knot_vec_eta[i + 1] - input_knot_vec_eta[i]) / 2.0;
			// printf("%d\t%f\n", k, dtilda_eta[k]);
			k++;
			l++;
			if (division_ele_eta > 1)
			{
				temp1 = (input_knot_vec_eta[i + 1] - input_knot_vec_eta[i]) / (double)division_ele_eta;
				for (j = 1; j < division_ele_eta; j++)
				{
					calc_eta[k] = calc_eta[k - 1] + temp1;
					// printf("%d\t%f\n", k, calc_eta[k]);
					k++;
				}
			}
		}
	}
	calc_eta[k] = input_knot_vec_eta[knot_n_eta - 1];
	// printf("%d\t%f\n", k, calc_eta[k]);
	// printf("\n");
	division_n_eta = k + 1;
	element_n_eta = l;

	if (element_n_xi > MAX_ELEMENTS)
	{
		printf("Error!!\n");
		printf("Too many elements at xi!\n"
			   "Maximum of elements is %d (Now %d)\n"
			   "\n",
			   MAX_ELEMENTS, element_n_xi);
		exit(1);
	}
	if (element_n_eta > MAX_ELEMENTS)
	{
		printf("Error!!\n");
		printf("Too many elements at eta!\n"
			   "Maximum of elements is %d (Now %d)\n"
			   "\n",
			   MAX_ELEMENTS, element_n_eta);
		exit(1);
	}

	int ii, jj, kk, ll;

	//メッシュ座標計算
	printf("Start Calculation mesh\n\n");
	for (i = 0; i < division_n_xi; i++)
	{
		ii = i / division_ele_xi;
		kk = i % division_ele_xi;
		for (j = 0; j < division_n_eta; j++)
		{
			jj = j / division_ele_eta;
			ll = j % division_ele_eta;
			lNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
						   cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
						   weight, order_xi, order_eta,
						   calc_xi[i], calc_eta[j],
						   &coord_x[i][j], &coord_y[i][j],
						   &dxi_x[ii][jj][kk][ll], &deta_x[ii][jj][kk][ll],
						   &dxi_y[ii][jj][kk][ll], &deta_y[ii][jj][kk][ll]);
			// printf("[%d][%d] [%d][%d][%d][%d]"
			// 	   "% 1.4e % 1.4e "
			// 	   "% 1.4e % 1.4e\n",
			// 	   i, j, ii, jj, kk, ll,
			// 	   calc_xi[i], calc_eta[j],
			// 	   coord_x[i][j], coord_y[i][j]);
		}
		// printf("\n");
	}
	printf("\n");
	printf("End Calculation mesh\n\n");

	if (fields_flag)
	{
		//変位計算
		printf("Start Calculation displpacement\n\n");
		for (i = 0; i < division_n_xi; i++)
		{
			ii = i / division_ele_xi;
			kk = i % division_ele_xi;
			for (j = 0; j < division_n_eta; j++)
			{
				jj = j / division_ele_eta;
				ll = j % division_ele_eta;
				lNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
							   disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
							   weight, order_xi, order_eta,
							   calc_xi[i], calc_eta[j],
							   &disp_x[i][j], &disp_y[i][j],
							   &dxi_disp_x[ii][jj][kk][ll], &deta_disp_x[ii][jj][kk][ll],
							   &dxi_disp_y[ii][jj][kk][ll], &deta_disp_y[ii][jj][kk][ll]);
				// printf("[%d][%d] [%d][%d][%d][%d]"
				// 	   "% 1.4e % 1.4e "
				// 	   "% 1.4e % 1.4e\n",
				// 	   i, j, ii, jj, kk, ll,
				// 	   calc_xi[i], calc_eta[j],
				// 	   disp_x[i][j], disp_y[i][j]);
			}
			// printf("\n");
		}
		printf("\n");
		printf("End Calculation displpacement\n\n");

		//足りない微分値計算
		for (ii = 0; ii < element_n_xi; ii++)
		{
			for (jj = 0; jj < element_n_eta; jj++)
			{
				kk = division_ele_xi;
				i = (ii + 1) * division_ele_xi;
				j = jj * division_ele_eta;
				for (ll = 1; ll < division_ele_eta; ll++)
				{
					j++;
					rNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								   cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								   weight, order_xi, order_eta,
								   calc_xi[i], calc_eta[j],
								   &coord_x[i][j], &coord_y[i][j],
								   &dxi_x[ii][jj][kk][ll], &deta_x[ii][jj][kk][ll],
								   &dxi_y[ii][jj][kk][ll], &deta_y[ii][jj][kk][ll]);
					rNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								   disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								   weight, order_xi, order_eta,
								   calc_xi[i], calc_eta[j],
								   &disp_x[i][j], &disp_y[i][j],
								   &dxi_disp_x[ii][jj][kk][ll], &deta_disp_x[ii][jj][kk][ll],
								   &dxi_disp_y[ii][jj][kk][ll], &deta_disp_y[ii][jj][kk][ll]);
					/*
					printf("[%d][%d] [%d][%d][%d][%d]"
						   "% 1.4e % 1.4e % 1.4e % 1.4e"
						   "% 1.4e % 1.4e % 1.4e % 1.4e\n",
						   i, j, ii, jj, kk, ll,
						   dxi_x[ii][jj][kk][ll], deta_x[ii][jj][kk][ll],
						   dxi_y[ii][jj][kk][ll], deta_y[ii][jj][kk][ll],
						   dxi_disp_x[ii][jj][kk][ll], deta_disp_x[ii][jj][kk][ll],
						   dxi_disp_y[ii][jj][kk][ll], deta_disp_y[ii][jj][kk][ll]);
					*/
				}

				ll = division_ele_eta;
				i = ii * division_ele_xi;
				j = (jj + 1) * division_ele_eta;
				for (kk = 1; kk <= division_ele_xi; kk++)
				{
					i++;
					rNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								   cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								   weight, order_xi, order_eta,
								   calc_xi[i], calc_eta[j],
								   &coord_x[i][j], &coord_y[i][j],
								   &dxi_x[ii][jj][kk][ll], &deta_x[ii][jj][kk][ll],
								   &dxi_y[ii][jj][kk][ll], &deta_y[ii][jj][kk][ll]);
					rNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								   disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								   weight, order_xi, order_eta,
								   calc_xi[i], calc_eta[j],
								   &disp_x[i][j], &disp_y[i][j],
								   &dxi_disp_x[ii][jj][kk][ll], &deta_disp_x[ii][jj][kk][ll],
								   &dxi_disp_y[ii][jj][kk][ll], &deta_disp_y[ii][jj][kk][ll]);
					/*
					printf("[%d][%d] [%d][%d][%d][%d]"
						   "% 1.4e % 1.4e % 1.4e % 1.4e"
						   "% 1.4e % 1.4e % 1.4e % 1.4e\n",
						   i, j, ii, jj, kk, ll,
						   dxi_x[ii][jj][kk][ll], deta_x[ii][jj][kk][ll],
						   dxi_y[ii][jj][kk][ll], deta_y[ii][jj][kk][ll],
						   dxi_disp_x[ii][jj][kk][ll], deta_disp_x[ii][jj][kk][ll],
						   dxi_disp_y[ii][jj][kk][ll], deta_disp_y[ii][jj][kk][ll]);
					*/
				}

				kk = division_ele_xi;
				ll = 0;
				i = (ii + 1) * division_ele_xi;
				j = jj * division_ele_eta;
				rlNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								weight, order_xi, order_eta,
								calc_xi[i], calc_eta[j],
								&coord_x[i][j], &coord_y[i][j],
								&dxi_x[ii][jj][kk][ll], &deta_x[ii][jj][kk][ll],
								&dxi_y[ii][jj][kk][ll], &deta_y[ii][jj][kk][ll]);
				rlNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								weight, order_xi, order_eta,
								calc_xi[i], calc_eta[j],
								&disp_x[i][j], &disp_y[i][j],
								&dxi_disp_x[ii][jj][kk][ll], &deta_disp_x[ii][jj][kk][ll],
								&dxi_disp_y[ii][jj][kk][ll], &deta_disp_y[ii][jj][kk][ll]);
				/*
					printf("[%d][%d] [%d][%d][%d][%d]"
						   "% 1.4e % 1.4e % 1.4e % 1.4e"
						   "% 1.4e % 1.4e % 1.4e % 1.4e\n",
						   i, j, ii, jj, kk, ll,
						   dxi_x[ii][jj][kk][ll], deta_x[ii][jj][kk][ll],
						   dxi_y[ii][jj][kk][ll], deta_y[ii][jj][kk][ll],
						   dxi_disp_x[ii][jj][kk][ll], deta_disp_x[ii][jj][kk][ll],
						   dxi_disp_y[ii][jj][kk][ll], deta_disp_y[ii][jj][kk][ll]);
				*/

				kk = 0;
				ll = division_ele_eta;
				i = ii * division_ele_xi;
				j = (jj + 1) * division_ele_eta;
				lrNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								cntl_px, cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								weight, order_xi, order_eta,
								calc_xi[i], calc_eta[j],
								&coord_x[i][j], &coord_y[i][j],
								&dxi_x[ii][jj][kk][ll], &deta_x[ii][jj][kk][ll],
								&dxi_y[ii][jj][kk][ll], &deta_y[ii][jj][kk][ll]);
				lrNURBS_surface(input_knot_vec_xi, input_knot_vec_eta,
								disp_cntl_px, disp_cntl_py, cntl_p_n_xi, cntl_p_n_eta,
								weight, order_xi, order_eta,
								calc_xi[i], calc_eta[j],
								&disp_x[i][j], &disp_y[i][j],
								&dxi_disp_x[ii][jj][kk][ll], &deta_disp_x[ii][jj][kk][ll],
								&dxi_disp_y[ii][jj][kk][ll], &deta_disp_y[ii][jj][kk][ll]);
				/*
					printf("[%d][%d] [%d][%d][%d][%d]"
						   "% 1.4e % 1.4e % 1.4e % 1.4e"
						   "% 1.4e % 1.4e % 1.4e % 1.4e\n",
						   i, j, ii, jj, kk, ll,
						   dxi_x[ii][jj][kk][ll], deta_x[ii][jj][kk][ll],
						   dxi_y[ii][jj][kk][ll], deta_y[ii][jj][kk][ll],
						   dxi_disp_x[ii][jj][kk][ll], deta_disp_x[ii][jj][kk][ll],
						   dxi_disp_y[ii][jj][kk][ll], deta_disp_y[ii][jj][kk][ll]);
				printf("\n");
				*/
			}
		}

		/*
		for (ii = 0; ii < element_n_xi; ii++) {
			for (jj = 0; jj < element_n_eta; jj++) {
				for (kk = 0; kk <= division_ele_xi; kk++) {
					for (ll = 0; ll <= division_ele_eta; ll++) {
						printf("[%d][%d][%d][%d]"
							   "% 1.4e % 1.4e % 1.4e % 1.4e"
							   "% 1.4e % 1.4e % 1.4e % 1.4e\n",
							   ii, jj, kk, ll,
							   dxi_x[ii][jj][kk][ll], deta_x[ii][jj][kk][ll],
							   dxi_y[ii][jj][kk][ll], deta_y[ii][jj][kk][ll],
							   dxi_disp_x[ii][jj][kk][ll], deta_disp_x[ii][jj][kk][ll],
							   dxi_disp_y[ii][jj][kk][ll], deta_disp_y[ii][jj][kk][ll]);
					}
				}
			}
		}
		printf("\n");
		*/

		//ひずみ計算
		printf("Start Calculation Strain\n\n");
		for (i = 0; i < element_n_xi; i++)
		{
			for (j = 0; j < element_n_eta; j++)
			{
				temp1 = dtilda_xi[i];
				temp2 = dtilda_eta[j];
				for (k = 0; k < division_ele_xi + 1; k++)
				{
					for (l = 0; l < division_ele_eta + 1; l++)
					{
						temp_matrix[0][0] = dxi_x[i][j][k][l] * temp1;
						temp_matrix[0][1] = dxi_y[i][j][k][l] * temp1;
						temp_matrix[1][0] = deta_x[i][j][k][l] * temp2;
						temp_matrix[1][1] = deta_y[i][j][k][l] * temp2;

						InverseMatrix_2D(temp_matrix);

						strain_xx[i][j][k][l] = temp_matrix[0][0] * temp1 * dxi_disp_x[i][j][k][l] + temp_matrix[0][1] * temp2 * deta_disp_x[i][j][k][l];
						strain_yy[i][j][k][l] = temp_matrix[1][0] * temp1 * dxi_disp_y[i][j][k][l] + temp_matrix[1][1] * temp2 * deta_disp_y[i][j][k][l];
						strain_xy[i][j][k][l] = temp_matrix[1][0] * temp1 * dxi_disp_x[i][j][k][l] + temp_matrix[1][1] * temp2 * deta_disp_x[i][j][k][l] + temp_matrix[0][0] * temp1 * dxi_disp_y[i][j][k][l] + temp_matrix[0][1] * temp2 * deta_disp_y[i][j][k][l];

						// printf("[%d][%d][%d][%d]\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\n",
						// 	   i, j, k, l, temp1, temp2,
						// 	   strain_xx[i][j][k][l], strain_yy[i][j][k][l], strain_xy[i][j][k][l]);
					}
				}
			}
			// printf("\n");
		}
		printf("End Calculation Strain\n\n");

		// Dマトリクスの計算
		double D_matrix[3][3] = {{0.0}};
		if (DM == 0)
		{ //平面応力状態
			temp1 = E * (1.0 - nu * nu);
			D_matrix[0][0] = temp1;
			D_matrix[0][1] = nu * temp1;
			D_matrix[1][0] = nu * temp1;
			D_matrix[1][1] = temp1;
			D_matrix[2][2] = (1.0 - nu) / 2.0 * temp1;
		}
		else if (DM == 1)
		{ //平面ひずみ状態(2Dの場合はこっち)
			temp1 = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
			D_matrix[0][0] = temp1;
			D_matrix[0][1] = nu / (1.0 - nu) * temp1;
			D_matrix[1][0] = nu / (1.0 - nu) * temp1;
			D_matrix[1][1] = temp1;
			D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp1;
		}

		printf("Start Calculation Stress\n\n");
		for (i = 0; i < element_n_xi; i++)
		{
			for (j = 0; j < element_n_eta; j++)
			{
				for (k = 0; k < division_ele_xi + 1; k++)
				{
					for (l = 0; l < division_ele_eta + 1; l++)
					{
						stress_xx[i][j][k][l] = D_matrix[0][0] * strain_xx[i][j][k][l] + D_matrix[0][1] * strain_yy[i][j][k][l];
						stress_yy[i][j][k][l] = D_matrix[1][0] * strain_xx[i][j][k][l] + D_matrix[1][1] * strain_yy[i][j][k][l];
						stress_xy[i][j][k][l] = D_matrix[2][2] * strain_xy[i][j][k][l];
						// printf("[%d][%d][%d][%d]\t% 1.4e\t% 1.4e\t% 1.4e\n", i, j, k, l,
						// 	   stress_xx[i][j][k][l], stress_yy[i][j][k][l], stress_xy[i][j][k][l]);
					}
				}
				// printf("\n");
			}
		}
		printf("End Calculation Stress\n\n");
	}

	//書き込み
	fp = fopen("view.dat", "a");
	if (fields_flag)
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				temp1 = disp_x[i][j];
				temp2 = disp_y[i][j];
				temp3 = sqrt(temp1 * temp1 + temp2 * temp2);
				fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j],
						disp_x[i][j], disp_y[i][j], temp3);
			}
		}
		for (i = 0; i < element_n_xi; i++)
		{
			for (j = 0; j < element_n_eta; j++)
			{
				for (k = 0; k < division_ele_xi + 1; k++)
				{
					for (l = 0; l < division_ele_eta + 1; l++)
					{
						fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\n",
								strain_xx[i][j][k][l], strain_yy[i][j][k][l], strain_xy[i][j][k][l],
								stress_xx[i][j][k][l], stress_yy[i][j][k][l], stress_xy[i][j][k][l]);
					}
				}
			}
		}
	}
	else
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				fprintf(fp, "% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j]);
			}
		}
	}
	fclose(fp);
	// machino
	fp = fopen("view_r_theta.dat", "a");
	if (fields_flag)
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				temp1 = disp_x[i][j];
				temp2 = disp_y[i][j];
				temp3 = sqrt(temp1 * temp1 + temp2 * temp2);
				fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j],
						disp_x[i][j], disp_y[i][j], temp3);
			}
		}
		for (i = 0; i < element_n_xi; i++)
		{
			for (j = 0; j < element_n_eta; j++)
			{
				for (k = 0; k < division_ele_xi + 1; k++)
				{
					for (l = 0; l < division_ele_eta + 1; l++)
					{
						double stress_rr, stress_theta;
						double sum = stress_xx[i][j][k][l] + stress_yy[i][j][k][l];
						double dif = stress_xx[i][j][k][l] - stress_yy[i][j][k][l];
						double tau2 = stress_xy[i][j][k][l] * stress_xy[i][j][k][l];
						stress_rr = sum * 0.5 + sqrt(dif * dif + 4 * tau2) * 0.5;
						stress_theta = sum * 0.5 - sqrt(dif * dif + 4 * tau2) * 0.5;
						fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t0.0\n",
								strain_xx[i][j][k][l], strain_yy[i][j][k][l], strain_xy[i][j][k][l],
								stress_rr, stress_theta);
					}
				}
			}
		}
	}
	else
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				fprintf(fp, "% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j]);
			}
		}
	}
	fclose(fp);
	//グラフ用ファイル書き込み
	fp = fopen("disp_graph.txt", "a");
	for (i = 0; i < division_n_xi; i++)
	{
		for (j = 0; j < division_n_eta; j++)
		{
			temp1 = disp_x[i][j];
			temp2 = disp_y[i][j];
			// temp3 = sqrt(temp1 * temp1 + temp2 * temp2);
			fprintf(fp, "%d\t% 1.15e\t% 1.15e\t% 1.15e\t% 1.15e\n",
					graph_patch_n,
					coord_x[i][j], coord_y[i][j],
					disp_x[i][j], disp_y[i][j]);
		}
	}
	fclose(fp);

	fp = fopen("stress_y_graph.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					fprintf(fp, "%d\t% 1.15e\t% 1.15e\t% 1.15e\n",
							graph_patch_n,
							coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
							coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
							stress_yy[i][j][k][l]);
				}
			}
		}
	}
	fclose(fp);

	fp = fopen("stress_y_graph_0.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					if (coord_y[i * division_ele_xi + k][j * division_ele_eta + l] == 0.000000000000000e+00)
					{
						fprintf(fp, "%d\t% 1.15e\t% 1.15e\t% 1.15e\n",
								graph_patch_n,
								coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
								coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
								stress_yy[i][j][k][l]);
					}
				}
			}
		}
	}
	fclose(fp);

	fp = fopen("stress_vm_graph.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					double stress_vm;
					double sum = stress_xx[i][j][k][l] + stress_yy[i][j][k][l];
					double dif = stress_xx[i][j][k][l] - stress_yy[i][j][k][l];
					double tau2 = stress_xy[i][j][k][l] * stress_xy[i][j][k][l];
					double temp1, temp2;
					double temp3;
					temp1 = 0.5 * sum;
					temp2 = 0.5 * sqrt(dif * dif + 4 * tau2);
					stress_vm = sqrt(temp1 * temp1 + 3 * temp2 * temp2);
					temp3 = sqrt(calc_xi[i * division_ele_xi + k] * calc_xi[i * division_ele_xi + k] + calc_eta[j * division_ele_eta + l] * calc_eta[j * division_ele_eta + l]);
					fprintf(fp, "% 1.10e\t% 1.10e\t% 1.10e\t% 1.10e\t"
								"% 1.10e\t% 1.10e\n",
							calc_xi[i * division_ele_xi + k],
							calc_eta[j * division_ele_eta + l],
							coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
							coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
							stress_vm, temp3);
				}
			}
		}
	}
	fclose(fp);
}

// for s-IGA
//重ね合わせた結果の出力
__host__ static void Calculation_overlay(int order_xi_loc, int order_eta_loc,
										 int knot_n_xi_loc, int knot_n_eta_loc,
										 int cntl_p_n_xi_loc, int cntl_p_n_eta_loc,
										 double *knot_vec_xi_loc, double *knot_vec_eta_loc,
										 double *cntl_px_loc, double *cntl_py_loc,
										 double *weight_loc,
										 int order_xi_glo, int order_eta_glo,
										 int cntl_p_n_xi_glo, int cntl_p_n_eta_glo,
										 double *knot_vec_xi_glo, double *knot_vec_eta_glo,
										 double *cntl_px_glo, double *cntl_py_glo,
										 double *disp_cntl_px_glo, double *disp_cntl_py_glo,
										 double *weight_glo)
{
	int i, j, k, l;
	double temp1, temp2, temp3;
	// double temp_matrix[2][2];

	double output_xi, output_eta;
	double disp_x_glo;
	double disp_y_glo;
	double strain_xx_glo = 0;
	double strain_yy_glo = 0;
	double strain_xy_glo = 0;
	//, strain_yy_glo, strain_xy_glo;

	//計算するξ,ηの値決定と ∂ξ/∂チルダξ, ∂η/∂チルダη の計算
	double calc_xi_loc[MAX_POINTS];	 //計算するξの値local
	double calc_eta_loc[MAX_POINTS]; //計算するηの値local
	// double dtilda_xi[MAX_KNOTS];		// ∂ξ/∂チルダξ
	// double dtilda_eta[MAX_KNOTS];	// ∂η/∂チルダη
	k = 0;
	l = 0;
	for (i = 0; i < knot_n_xi_loc - 1; i++)
	{
		if (knot_vec_xi_loc[i] != knot_vec_xi_loc[i + 1])
		{
			calc_xi_loc[k] = knot_vec_xi_loc[i];
			printf("%d\t%f\n", k, calc_xi_loc[k]);
			// dtilda_xi[l] = ( knot_vec_xi_loc[i + 1] - knot_vec_xi_loc[i] ) / 2.0;
			// printf("%d\t%f\n", k, dtilda_xi[k]);
			k++;
			l++;
			if (division_ele_xi > 1)
			{
				temp1 = (knot_vec_xi_loc[i + 1] - knot_vec_xi_loc[i]) / (double)division_ele_xi;
				for (j = 1; j < division_ele_xi; j++)
				{
					calc_xi_loc[k] = calc_xi_loc[k - 1] + temp1;
					printf("%d\t%f\n", k, calc_xi_loc[k]);
					k++;
				}
			}
		}
	}
	calc_xi_loc[k] = knot_vec_xi_loc[knot_n_xi_loc - 1];
	printf("%d\t%f\n", k, calc_xi_loc[k]);
	// printf("\n");
	division_n_xi = k + 1;
	element_n_xi = l;

	k = 0;
	l = 0;
	for (i = 0; i < knot_n_eta_loc - 1; i++)
	{
		if (knot_vec_eta_loc[i] != knot_vec_eta_loc[i + 1])
		{
			calc_eta_loc[k] = knot_vec_eta_loc[i];
			// printf("%d\t%f\n", k, calc_eta[k]);
			// dtilda_eta[l] = ( knot_vec_eta_loc[i + 1] - knot_vec_eta_loc[i] ) / 2.0;
			// printf("%d\t%f\n", k, dtilda_eta[k]);
			k++;
			l++;
			if (division_ele_eta > 1)
			{
				temp1 = (knot_vec_eta_loc[i + 1] - knot_vec_eta_loc[i]) / (double)division_ele_eta;
				for (j = 1; j < division_ele_eta; j++)
				{
					calc_eta_loc[k] = calc_eta_loc[k - 1] + temp1;
					// printf("%d\t%f\n", k, calc_eta[k]);
					k++;
				}
			}
		}
	}
	calc_eta_loc[k] = knot_vec_eta_loc[knot_n_eta_loc - 1];
	// printf("%d\t%f\n", k, calc_eta[k]);
	// printf("\n");
	division_n_eta = k + 1;
	element_n_eta = l;

	if (element_n_xi > MAX_ELEMENTS)
	{
		printf("Error!!\n");
		printf("Too many elements at xi!\n"
			   "Maximum of elements is %d (Now %d)\n"
			   "\n",
			   MAX_ELEMENTS, element_n_xi);
		exit(1);
	}
	if (element_n_eta > MAX_ELEMENTS)
	{
		printf("Error!!\n");
		printf("Too many elements at eta!\n"
			   "Maximum of elements is %d (Now %d)\n"
			   "\n",
			   MAX_ELEMENTS, element_n_eta);
		exit(1);
	}

	int ii, jj, kk, ll;

	//メッシュ座標計算
	printf("Start Calculation overlay mesh\n\n");
	printf("Start Calculation overlay displpacement\n\n");
	printf("Start Calculation overlay Strain\n\n");
	for (i = 0; i < division_n_xi; i++)
	{
		ii = i / division_ele_xi;
		kk = i % division_ele_xi;
		for (j = 0; j < division_n_eta; j++)
		{
			jj = j / division_ele_eta;
			ll = j % division_ele_eta;
			lNURBS_surface(knot_vec_xi_loc, knot_vec_eta_loc,
						   cntl_px_loc, cntl_py_loc, cntl_p_n_xi_loc, cntl_p_n_eta_loc,
						   weight_loc, order_xi_loc, order_eta_loc,
						   calc_xi_loc[i], calc_eta_loc[j],
						   &coord_x[i][j], &coord_y[i][j],
						   &dxi_x[ii][jj][kk][ll], &deta_x[ii][jj][kk][ll],
						   &dxi_y[ii][jj][kk][ll], &deta_y[ii][jj][kk][ll]);
			// printf("[%d][%d] [%d][%d][%d][%d]"
			// 	   "% 1.4e % 1.4e "
			// 	   "% 1.4e % 1.4e\n",
			// 	   i, j, ii, jj, kk, ll,
			// 	   calc_xi_loc[i], calc_eta_loc[j],
			// 	   coord_x[i][j], coord_y[i][j]);

			int itr_n = CalcXiEtaByNR(coord_x[i][j], coord_y[i][j],
									  knot_vec_xi_glo, knot_vec_eta_glo,
									  cntl_px_glo, cntl_py_glo,
									  disp_cntl_px_glo, disp_cntl_py_glo,
									  cntl_p_n_xi_glo, cntl_p_n_eta_glo,
									  weight_glo, order_xi_glo, order_eta_glo,
									  &output_xi, &output_eta,
									  &disp_x_glo, &disp_y_glo,
									  &strain_xx_glo, &strain_yy_glo, &strain_xy_glo);
			if (itr_n == 0)
			{
				// printf("itr=0\n");
			}
			// printf("iteration : %d\n",itr_n);

			//ローカル内の表示点上のグローバル変位
			// printf("disp_x_glo =% 1.4e\tdisp_y_glo =% 1.4e\n", disp_x_glo, disp_y_glo);
			// printf("%1.4e\t%1.4e\n",disp_x[i][j],disp_y[i][j]);
			disp_x[i][j] += disp_x_glo;
			disp_y[i][j] += disp_y_glo;
			// printf("% 1.4e\t% 1.4e\n",disp_x[i][j],disp_y[i][j]);

			//ローカル内の表示点上のグローバルひずみ
			// printf("strain_xx_glo =% 1.4e\n"
			//        "strain_yy_glo =% 1.4e\n"
			//        "strain_xy_glo =% 1.4e\n",
			//        strain_xx_glo, strain_yy_glo, strain_xy_glo);
			strain_xx[ii][jj][kk][ll] += strain_xx_glo;
			strain_yy[ii][jj][kk][ll] += strain_yy_glo;
			strain_xy[ii][jj][kk][ll] += strain_xy_glo;
			// printf("test[%d][%d][%d][%d]\n",ii,jj,kk,ll);
			if (jj > 0 && ll == 0)
			{
				strain_xx[ii][jj - 1][kk][division_ele_eta] += strain_xx_glo;
				strain_yy[ii][jj - 1][kk][division_ele_eta] += strain_yy_glo;
				strain_xy[ii][jj - 1][kk][division_ele_eta] += strain_xy_glo;
				// printf("test[%d][%d][%d][%d]\n",ii,jj-1,kk,division_ele_eta);
			}
			if (ii > 0 && kk == 0)
			{
				strain_xx[ii - 1][jj][division_ele_xi][ll] += strain_xx_glo;
				strain_yy[ii - 1][jj][division_ele_xi][ll] += strain_yy_glo;
				strain_xy[ii - 1][jj][division_ele_xi][ll] += strain_xy_glo;
				// printf("test[%d][%d][%d][%d]\n",ii-1,jj,division_ele_xi,ll);
			}
			if (ii > 0 && jj > 0 && kk == 0 && ll == 0)
			{
				strain_xx[ii - 1][jj - 1][division_ele_xi][division_ele_eta] += strain_xx_glo;
				strain_yy[ii - 1][jj - 1][division_ele_xi][division_ele_eta] += strain_yy_glo;
				strain_xy[ii - 1][jj - 1][division_ele_xi][division_ele_eta] += strain_xy_glo;
				// printf("test[%d][%d][%d][%d]\n",ii-1,jj-1,division_ele_xi,division_ele_eta);
			}
			// printf("% 1.4e\t% 1.4e\t% 1.4e\n",
			// 		strain_xx[ii][jj][kk][ll],
			// 		strain_yy[ii][jj][kk][ll],
			// 		strain_xy[ii][jj][kk][ll]);
		}
		// printf("\n");
	}
	printf("\n");
	printf("End Calculation overlay mesh\n\n");
	printf("End Calculation overlay displpacement\n\n");
	printf("End Calculation overlay Strain\n\n");

	// Dマトリクスの計算
	double D_matrix[3][3] = {{0.0}};
	if (DM == 0)
	{ //平面応力状態
		temp1 = E * (1.0 - nu * nu);
		D_matrix[0][0] = temp1;
		D_matrix[0][1] = nu * temp1;
		D_matrix[1][0] = nu * temp1;
		D_matrix[1][1] = temp1;
		D_matrix[2][2] = (1.0 - nu) / 2.0 * temp1;
	}
	else if (DM == 1)
	{ //平面ひずみ状態(2Dの場合はこっち)
		temp1 = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
		D_matrix[0][0] = temp1;
		D_matrix[0][1] = nu / (1.0 - nu) * temp1;
		D_matrix[1][0] = nu / (1.0 - nu) * temp1;
		D_matrix[1][1] = temp1;
		D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp1;
	}

	printf("Start Calculation overlay Stress\n\n");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					stress_xx[i][j][k][l] = D_matrix[0][0] * strain_xx[i][j][k][l] + D_matrix[0][1] * strain_yy[i][j][k][l];
					stress_yy[i][j][k][l] = D_matrix[1][0] * strain_xx[i][j][k][l] + D_matrix[1][1] * strain_yy[i][j][k][l];
					stress_xy[i][j][k][l] = D_matrix[2][2] * strain_xy[i][j][k][l];
					// printf("[%d][%d][%d][%d]\t% 1.4e\t% 1.4e\t% 1.4e\n", i, j, k, l,
					// 	   stress_xx[i][j][k][l], stress_yy[i][j][k][l], stress_xy[i][j][k][l]);
				}
			}
			// printf("\n");
		}
	}
	printf("End Calculation overlay Stress\n\n");

	//書き込み
	fp = fopen("overlay_view.dat", "a");
	if (fields_flag)
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				temp1 = disp_x[i][j];
				temp2 = disp_y[i][j];
				temp3 = sqrt(temp1 * temp1 + temp2 * temp2);
				fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j],
						disp_x[i][j], disp_y[i][j], temp3);
			}
		}
		for (i = 0; i < element_n_xi; i++)
		{
			for (j = 0; j < element_n_eta; j++)
			{
				for (k = 0; k < division_ele_xi + 1; k++)
				{
					for (l = 0; l < division_ele_eta + 1; l++)
					{
						fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\n",
								strain_xx[i][j][k][l], strain_yy[i][j][k][l], strain_xy[i][j][k][l],
								stress_xx[i][j][k][l], stress_yy[i][j][k][l], stress_xy[i][j][k][l]);
					}
				}
			}
		}
	}
	else
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				fprintf(fp, "% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j]);
			}
		}
	}
	fclose(fp);
	// machino
	fp = fopen("overlay_view_r_theta.dat", "a");
	if (fields_flag)
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				temp1 = disp_x[i][j];
				temp2 = disp_y[i][j];
				temp3 = sqrt(temp1 * temp1 + temp2 * temp2);
				fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j],
						disp_x[i][j], disp_y[i][j], temp3);
			}
		}
		for (i = 0; i < element_n_xi; i++)
		{
			for (j = 0; j < element_n_eta; j++)
			{
				for (k = 0; k < division_ele_xi + 1; k++)
				{
					for (l = 0; l < division_ele_eta + 1; l++)
					{
						double stress_rr, stress_theta;
						double sum = stress_xx[i][j][k][l] + stress_yy[i][j][k][l];
						double dif = stress_xx[i][j][k][l] - stress_yy[i][j][k][l];
						double tau2 = stress_xy[i][j][k][l] * stress_xy[i][j][k][l];
						stress_rr = sum * 0.5 + sqrt(dif * dif + 4 * tau2) * 0.5;
						stress_theta = sum * 0.5 - sqrt(dif * dif + 4 * tau2) * 0.5;
						fprintf(fp, "% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t% 1.4e\t0.0\n",
								strain_xx[i][j][k][l], strain_yy[i][j][k][l], strain_xy[i][j][k][l],
								stress_rr, stress_theta);
					}
				}
			}
		}
	}
	else
	{
		fprintf(fp, "%d\t%d\t%d\t%d\n",
				division_n_xi, division_n_eta,
				element_n_xi, element_n_eta);
		for (i = 0; i < division_n_xi; i++)
		{
			for (j = 0; j < division_n_eta; j++)
			{
				fprintf(fp, "% 1.4e\t% 1.4e\n",
						coord_x[i][j], coord_y[i][j]);
			}
		}
	}
	fclose(fp);
	//グラフ用ファイル書き込み
	fp = fopen("over_disp_graph.txt", "a");
	for (i = 0; i < division_n_xi; i++)
	{
		for (j = 0; j < division_n_eta; j++)
		{
			temp1 = disp_x[i][j];
			temp2 = disp_y[i][j];
			// temp3 = sqrt(temp1 * temp1 + temp2 * temp2);
			fprintf(fp, "%d\t% 1.15e\t% 1.15e\t% 1.15e\t% 1.15e\n",
					graph_patch_n,
					coord_x[i][j], coord_y[i][j],
					disp_x[i][j], disp_y[i][j]);
		}
	}
	fclose(fp);

	fp = fopen("over_stress_x_graph.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					fprintf(fp, "% 1.10e\t% 1.10e\t% 1.10e\n",
							coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
							coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
							stress_xx[i][j][k][l]);
				}
			}
		}
	}
	fclose(fp);

	fp = fopen("over_stress_y_graph.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					fprintf(fp, "% 1.15e\t% 1.15e\t% 1.15e\n",
							coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
							coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
							stress_yy[i][j][k][l]);
				}
			}
		}
	}
	fclose(fp);

	fp = fopen("over_stress_y_graph_0.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					if (coord_y[i * division_ele_xi + k][j * division_ele_eta + l] == 0.000000000000000e+00)
					{
						fprintf(fp, "% 1.15e\t% 1.15e\t% 1.15e\n",
								coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
								coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
								stress_yy[i][j][k][l]);
					}
				}
			}
		}
	}
	fclose(fp);

	fp = fopen("over_stress_r_theta_graph.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					double stress_rr, stress_theta;
					double sum = stress_xx[i][j][k][l] + stress_yy[i][j][k][l];
					double dif = stress_xx[i][j][k][l] - stress_yy[i][j][k][l];
					double tau2 = stress_xy[i][j][k][l] * stress_xy[i][j][k][l];
					stress_rr = sum * 0.5 + sqrt(dif * dif + 4 * tau2) * 0.5;
					stress_theta = sum * 0.5 - sqrt(dif * dif + 4 * tau2) * 0.5;
					fprintf(fp, "% 1.10e\t% 1.10e\t% 1.10e\t% 1.10e\t% 1.10e\t% 1.10e\n",
							calc_xi_loc[i * division_ele_xi + k],
							calc_eta_loc[j * division_ele_eta + l],
							coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
							coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
							stress_rr, stress_theta);
				}
			}
		}
	}
	fclose(fp);

	fp = fopen("over_stress_vm_graph.txt", "a");
	for (i = 0; i < element_n_xi; i++)
	{
		for (j = 0; j < element_n_eta; j++)
		{
			for (k = 0; k < division_ele_xi + 1; k++)
			{
				for (l = 0; l < division_ele_eta + 1; l++)
				{
					double stress_vm;
					double sum = stress_xx[i][j][k][l] + stress_yy[i][j][k][l];
					double dif = stress_xx[i][j][k][l] - stress_yy[i][j][k][l];
					double tau2 = stress_xy[i][j][k][l] * stress_xy[i][j][k][l];
					double temp1, temp2;
					temp1 = 0.5 * sum;
					temp2 = 0.5 * sqrt(dif * dif + 4 * tau2);
					stress_vm = sqrt(temp1 * temp1 + 3 * temp2 * temp2);
					fprintf(fp, "% 1.10e\t% 1.10e\t% 1.10e\t% 1.10e\t% 1.10e\n",
							calc_xi_loc[i * division_ele_xi + k],
							calc_eta_loc[j * division_ele_eta + l],
							coord_x[i * division_ele_xi + k][j * division_ele_eta + l],
							coord_y[i * division_ele_xi + k][j * division_ele_eta + l],
							stress_vm);
				}
			}
		}
	}
	fclose(fp);
}

/*
static void Calculation_at_GP(double E, double nu)
{
	//通常IGAでのガウス点での値
	int i, j, k, e;

	Make_gauss_array(1);

	//メッシュ座標計算
	int ele_glo_n = real_Total_Element_on_mesh[0];// グローバルメッシュの要素数

	double U_temp[MAX_KIEL_SIZE];
	double B[D_MATRIX_SIZE][MAX_KIEL_SIZE];
	double J;

	for (e = 0; e < ele_glo_n; e++)
	{
		double X_temp[No_Control_point_ON_ELEMENT[Element_patch[e]]][DIMENSION];

		// printf("ele = %d\n\n", e);

		//strain_GPの初期化
		for (i = 0; i < GP_2D; i++)
		{
			for (j = 0; j < 3; j++)
			{
				strain_GP[e][i][j] = 0.0;
			}
		}

		// printf("x\ty\n");
		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[e]]; j++)
		{
			for (k = 0; k < DIMENSION; k++)
			{
				U_temp[j * DIMENSION + k] = Displacement[Controlpoint_of_Element[e][j] * DIMENSION + k];
				X_temp[j][k] = Node_Coordinate[Controlpoint_of_Element[e][j]][k];
			}
			// printf("%.15e\t%.15e\n", X_temp[j][0], X_temp[j][1]);
		}
		// printf("\n");

		for (i = 0; i < GP_2D; i++)	//ガウス点のループ
		{
			double data_result_shape[2] = {0.0, 0.0};
			double R_shape_func;

			// printf("Gxi_x\t_Gxi_y\tR_shape_func\n");
			for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[e]]; j++)
			{
				R_shape_func = Shape_func(j, Total_Control_Point_to_mesh[Total_mesh], Gxi[i], e);
				for (k = 0; k < DIMENSION; k++)
				{
					data_result_shape[k] += R_shape_func * X_temp[j][k];
				}
			}

			//物理座標[要素番号(ローカル内で0から始まる)][ガウス点番号][DIMENSION]
			for (j = 0; j < DIMENSION; j++)
			{
				coordinate_GP[e][i][j] = data_result_shape[j];
			}
		}

		for (i = 0; i < GP_2D; i++)	//ガウス点のループ
		{
			Make_B_Matrix(e, B, Gxi[i], X_temp, &J, Total_Control_Point_to_mesh[Total_mesh]);
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				for (k = 0; k < KIEL_SIZE; k++)
				{
					strain_GP[e][i][j] += B[j][k] * U_temp[k];
				}
			}
			Jac[e][i] = J;
		}
	}

	//Dマトリクスの計算
	double temp1;
	double D_matrix[3][3] = {{0.0}};
	if (DM == 0) { //平面応力状態
		temp1 = E * (1.0 - nu * nu);
		D_matrix[0][0] = temp1;
		D_matrix[0][1] = nu * temp1;
		D_matrix[1][0] = nu * temp1;
		D_matrix[1][1] = temp1;
		D_matrix[2][2] = (1.0 - nu) / 2.0 * temp1;
	} else if (DM == 1) { //平面ひずみ状態(2Dの場合はこっち)
		temp1 = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
		D_matrix[0][0] = temp1;
		D_matrix[0][1] = nu / (1.0 - nu) * temp1;
		D_matrix[1][0] = nu / (1.0 - nu) * temp1;
		D_matrix[1][1] = temp1;
		D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp1;
	}

	for (e = 0; e < ele_glo_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			stress_GP[e][i][0] = D_matrix[0][0] * strain_GP[e][i][0] + D_matrix[0][1] * strain_GP[e][i][1];
			stress_GP[e][i][1] = D_matrix[1][0] * strain_GP[e][i][0] + D_matrix[1][1] * strain_GP[e][i][1];
			stress_GP[e][i][2] = D_matrix[2][2] * strain_GP[e][i][2];
		}
	}

	//座標変換
	double theta = 0.0;

	for (e = 0; e < ele_glo_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			theta = atan2(coordinate_GP[e][i][1], coordinate_GP[e][i][0]);

			stress_r_theta_GP[e][i][0] = stress_GP[e][i][0] * pow(cos(theta), 2.0)
									   + stress_GP[e][i][1] * pow(sin(theta), 2.0)
									   + 2.0 * stress_GP[e][i][2] * sin(theta) * cos(theta);
			stress_r_theta_GP[e][i][1] = stress_GP[e][i][0] * pow(sin(theta), 2.0)
									   + stress_GP[e][i][1] * pow(cos(theta), 2.0)
									   - 2.0 * stress_GP[e][i][2] * cos(theta) * sin(theta);
			stress_r_theta_GP[e][i][2] = (stress_GP[e][i][1] - stress_GP[e][i][0])
									   * sin(theta) * cos(theta) + stress_GP[e][i][2]
									   * (pow(cos(theta), 2.0) - pow(sin(theta), 2.0));
		}
	}

	//厚肉円筒の理論解
	double r_t = 0.0;

	for (e = 0; e < ele_glo_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			r_t = sqrt(pow(coordinate_GP[e][i][0], 2.0) + pow(coordinate_GP[e][i][1], 2.0));

			stress_theory_r_theta[e][i][0] = (pow(r_t, 2.0) - 4.0) / (pow(r_t, 2.0) * 3.0);
			stress_theory_r_theta[e][i][1] = (pow(r_t, 2.0) + 4.0) / (pow(r_t, 2.0) * 3.0);
		}
	}

	//書き込み
	fp = fopen("at_GP_overlay_data.txt", "w");
	fprintf(fp, "e\tガウス番号\tx\ty\tstress_xx\tstress_yy\tstress_r\tstress_theta\n");
	for (e = 0; e < ele_glo_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			fprintf(fp, "%d\t%d\t%.15e\t%.15e\t%.15e\t%.15e\t%.15e\t%.15e\n", e, i, coordinate_GP[e][i][0], coordinate_GP[e][i][1], stress_GP[e][i][0], stress_GP[e][i][1], stress_r_theta_GP[e][i][0], stress_r_theta_GP[e][i][1]);
		}
	}
	fclose(fp);

	fp = fopen("at_GP_overlay_for_error_norm.txt", "w");
	fprintf(fp, "e\tガウス番号\tx\ty\tstress_r-theory\tstress_theta-theory\n");
	for (e = 0; e < ele_glo_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			fprintf(fp, "%d\t%d\t%.15e\t%.15e\t%.15e\t%.15e\n", e, i, coordinate_GP[e][i][0], coordinate_GP[e][i][1], stress_r_theta_GP[e][i][0] - stress_theory_r_theta[e][i][0], stress_r_theta_GP[e][i][1] - stress_theory_r_theta[e][i][1]);
		}
	}
	fclose(fp);


	//error normを計算
	//ガウス点で出したtheoryとの差の二乗を面積分
	double temp2 = 0.0, temp3 = 0.0, temp4 = 0.0, temp5 = 0.0;
	for (e = 0; e < ele_glo_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			temp2 += w[i] * pow((stress_r_theta_GP[e][i][0] - stress_theory_r_theta[e][i][0]), 2.0) * Jac[e][i];
			temp3 += w[i] * pow((stress_r_theta_GP[e][i][1] - stress_theory_r_theta[e][i][1]), 2.0) * Jac[e][i];
			temp4 += w[i] * pow(stress_r_theta_GP[e][i][0], 2.0) * Jac[e][i];
			temp5 += w[i] * pow(stress_r_theta_GP[e][i][1], 2.0) * Jac[e][i];
		}
	}
	fp = fopen("at_GP_overlay_for_error_norm_surface_integral.txt", "w");
	fprintf(fp, "(stress_r-theory)^2_surface_integral\t(stress_theta-thory)^2_surface_integral\tstress_r^2_surface_integral\n");
	fprintf(fp, "%.15e\t%.15e\t%.15e\t%.15e\n", temp2, temp3, temp4, temp5);
	fclose(fp);
}


void Calculation_overlay_at_GP(double E, double nu,
							   int order_xi_glo, int order_eta_glo,
							   int knot_n_xi_glo, int knot_n_eta_glo,
							   int cntl_p_n_xi_glo, int cntl_p_n_eta_glo,
							   double *knot_vec_xi_glo, double *knot_vec_eta_glo,
							   double *cntl_px_glo, double *cntl_py_glo,
							   double *disp_cntl_px_glo, double *disp_cntl_py_glo,
							   double *weight_glo)
{
	//s-IGAでのローカル上のガウス点での重ね合わせた値
	int i, j, k, e;

	double disp_glo[DIMENSION];
	double strain_glo[D_MATRIX_SIZE];

	Make_gauss_array(1);

	double G_GP_knot[GP_2D][DIMENSION];

	//メッシュ座標計算
	int ele_glo_n = real_Total_Element_on_mesh[0];// グローバルメッシュの要素数
	int ele_loc_n = real_Total_Element_to_mesh[Total_mesh] - real_Total_Element_on_mesh[0];// ローカルメッシュの要素数

	double U_temp[MAX_KIEL_SIZE];
	double B[D_MATRIX_SIZE][MAX_KIEL_SIZE];
	double J;

	for (e = 0; e < ele_loc_n; e++)
	{
		int El_No_loc = ele_glo_n + e;
		double X_temp[No_Control_point_ON_ELEMENT[Element_patch[El_No_loc]]][DIMENSION];

		//strain_GPの初期化
		for (i = 0; i < GP_2D; i++)
		{
			for (j = 0; j < 3; j++)
			{
				strain_GP[e][i][j] = 0.0;
			}
		}

		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No_loc]]; j++)
		{
			for (k = 0; k < DIMENSION; k++)
			{
				U_temp[j * DIMENSION + k] = Displacement[Controlpoint_of_Element[El_No_loc][j] * DIMENSION + k];
				X_temp[j][k] = Node_Coordinate[Controlpoint_of_Element[El_No_loc][j]][k];
			}
		}

		for (i = 0; i < GP_2D; i++)	//ガウス点のループ
		{
			double data_result_shape[2] = {0.0, 0.0};
			double R_shape_func;

			for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No_loc]]; j++)
			{
				R_shape_func = Shape_func(j, Total_Control_Point_to_mesh[Total_mesh], Gxi[i], El_No_loc);
				for (k = 0; k < DIMENSION; k++)
				{
					data_result_shape[k] += R_shape_func * X_temp[j][k];
				}
			}

			for (j = 0; j < DIMENSION; j++)
			{
				coordinate_GP[e][i][j] = data_result_shape[j];
			}
		}

		for (i = 0; i < GP_2D; i++)	//ガウス点のループ
		{
			Make_B_Matrix(El_No_loc, B, Gxi[i], X_temp, &J, Total_Control_Point_to_mesh[Total_mesh]);
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				for (k = 0; k < KIEL_SIZE; k++)
				{
					strain_GP[e][i][j] += B[j][k] * U_temp[k];
				}
			}
			Jac[e][i] = J;
		}

		for (i = 0; i < GP_2D; i++)	//ガウス点のループ
		{
			CalcXiEtaByNR(coordinate_GP[e][i][0], coordinate_GP[e][i][1],
						  knot_vec_xi_glo, knot_vec_eta_glo,
						  cntl_px_glo, cntl_py_glo,
						  disp_cntl_px_glo, disp_cntl_py_glo,
						  cntl_p_n_xi_glo, cntl_p_n_eta_glo,
						  weight_glo, order_xi_glo, order_eta_glo,
						  &G_GP_knot[i][0], &G_GP_knot[i][1],
						  &disp_glo[0], &disp_glo[1],
						  &strain_glo[0], &strain_glo[1], &strain_glo[2]);

			//重ね合わせ
			for (j = 0; j < 3; j++) //xx, yy, xyを重ね合わせる
			{
				strain_GP[e][i][j] += strain_glo[j];
			}
		}
	}

	//Dマトリクスの計算
	double temp1;
	double D_matrix[3][3] = {{0.0}};
	if (DM == 0) { //平面応力状態
		temp1 = E * (1.0 - nu * nu);
		D_matrix[0][0] = temp1;
		D_matrix[0][1] = nu * temp1;
		D_matrix[1][0] = nu * temp1;
		D_matrix[1][1] = temp1;
		D_matrix[2][2] = (1.0 - nu) / 2.0 * temp1;
	} else if (DM == 1) { //平面ひずみ状態(2Dの場合はこっち)
		temp1 = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
		D_matrix[0][0] = temp1;
		D_matrix[0][1] = nu / (1.0 - nu) * temp1;
		D_matrix[1][0] = nu / (1.0 - nu) * temp1;
		D_matrix[1][1] = temp1;
		D_matrix[2][2] = (1.0 - 2.0 * nu) / 2.0 / (1.0 - nu) * temp1;
	}

	for (e = 0; e < ele_loc_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			stress_GP[e][i][0] = D_matrix[0][0] * strain_GP[e][i][0] + D_matrix[0][1] * strain_GP[e][i][1];
			stress_GP[e][i][1] = D_matrix[1][0] * strain_GP[e][i][0] + D_matrix[1][1] * strain_GP[e][i][1];
			stress_GP[e][i][2] = D_matrix[2][2] * strain_GP[e][i][2];
		}
	}

	//座標変換
	double theta = 0.0;

	for (e = 0; e < ele_loc_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			theta = atan2(coordinate_GP[e][i][1], coordinate_GP[e][i][0]);

			stress_r_theta_GP[e][i][0] = stress_GP[e][i][0] * pow(cos(theta), 2.0)
									   + stress_GP[e][i][1] * pow(sin(theta), 2.0)
									   + 2.0 * stress_GP[e][i][2] * sin(theta) * cos(theta);
			stress_r_theta_GP[e][i][1] = stress_GP[e][i][0] * pow(sin(theta), 2.0)
									   + stress_GP[e][i][1] * pow(cos(theta), 2.0)
									   - 2.0 * stress_GP[e][i][2] * cos(theta) * sin(theta);
			stress_r_theta_GP[e][i][2] = (stress_GP[e][i][1] - stress_GP[e][i][0])
									   * sin(theta) * cos(theta) + stress_GP[e][i][2]
									   * (pow(cos(theta), 2.0) - pow(sin(theta), 2.0));
		}
	}

	//円孔を有する無限平板の理論解
	double r_t = 0.0, theta_t = 0.0;

	for (e = 0; e < ele_loc_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			r_t = sqrt(pow(coordinate_GP[e][i][0], 2.0) + pow(coordinate_GP[e][i][1], 2.0));
			theta_t = atan2(coordinate_GP[e][i][1], coordinate_GP[e][i][0]);

			// stress_theory_r_theta[e][i][0] = (10.0 / 2.0) * (1.0 - pow((1.0 / r_t), 2.0))
			// 							   + (10.0 / 2.0) * (1.0 + 3.0 * pow((1.0 / r_t), 4.0)
			// 							   - 4.0 * pow((1.0 / r_t), 2.0)) * cos(2.0 * theta_t);
			// stress_theory_r_theta[e][i][1] = (10.0 / 2.0) * (1.0 + pow((1.0 / r_t), 2.0))
			// 							   - (10.0 / 2.0) * (1.0 + 3.0 * pow((1.0 / r_t), 4.0))
			// 							   * cos(2.0 * theta_t);
			// stress_theory_r_theta[e][i][2] = - (10.0 / 2.0) * (1.0 - 3.0 * pow((1.0 / r_t), 4.0)
			// 							   + 2.0 * pow((1.0 / r_t), 2.0)) * sin(2.0 * theta_t);
			stress_theory_r_theta[e][i][0] = (10.0 / 2.0) * (1.0 - pow((1.0 / r_t), 2.0))
										   - (10.0 / 2.0) * (1.0 + 3.0 * pow((1.0 / r_t), 4.0)
										   - 4.0 * pow((1.0 / r_t), 2.0)) * cos(2.0 * theta_t);
			stress_theory_r_theta[e][i][1] = (10.0 / 2.0) * (1.0 + pow((1.0 / r_t), 2.0))
										   + (10.0 / 2.0) * (1.0 + 3.0 * pow((1.0 / r_t), 4.0))
										   * cos(2.0 * theta_t);
			stress_theory_r_theta[e][i][2] = (10.0 / 2.0) * (1.0 - 3.0 * pow((1.0 / r_t), 4.0)
										   + 2.0 * pow((1.0 / r_t), 2.0)) * sin(2.0 * theta_t);
		}
	}

	//書き込み
	fp = fopen("at_GP_overlay_data.txt", "w");
	fprintf(fp, "e\tガウス番号\tx\ty\tstress_xx\tstress_yy\tstress_r\tstress_theta\n");
	for (e = 0; e < ele_loc_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			fprintf(fp, "%d\t%d\t%.15e\t%.15e\t%.15e\t%.15e\t%.15e\t%.15e\n", e, i, coordinate_GP[e][i][0], coordinate_GP[e][i][1], stress_GP[e][i][0], stress_GP[e][i][1], stress_r_theta_GP[e][i][0], stress_r_theta_GP[e][i][1]);
		}
	}
	fclose(fp);

	fp = fopen("at_GP_overlay_for_error_norm.txt", "w");
	fprintf(fp, "e\tガウス番号\tx\ty\tstress_r-theory\tstress_theta-theory\n");
	for (e = 0; e < ele_loc_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			fprintf(fp, "%d\t%d\t%.15e\t%.15e\t%.15e\t%.15e\n", e, i, coordinate_GP[e][i][0], coordinate_GP[e][i][1], stress_r_theta_GP[e][i][0] - stress_theory_r_theta[e][i][0], stress_r_theta_GP[e][i][1] - stress_theory_r_theta[e][i][1]);
		}
	}
	fclose(fp);


	//error normを計算
	//ガウス点で出したtheoryとの差の二乗を面積分
	double temp2 = 0.0, temp3 = 0.0, temp4 = 0.0, temp5 = 0.0, temp6 = 0.0;
	for (e = 0; e < ele_loc_n; e++)
	{
		for (i = 0; i < GP_2D; i++)
		{
			temp2 += w[i] * pow((stress_r_theta_GP[e][i][0] - stress_theory_r_theta[e][i][0]), 2.0) * Jac[e][i];
			temp3 += w[i] * pow((stress_r_theta_GP[e][i][1] - stress_theory_r_theta[e][i][1]), 2.0) * Jac[e][i];
			temp4 += w[i] * pow(stress_r_theta_GP[e][i][0], 2.0) * Jac[e][i];
			temp5 += w[i] * pow(stress_r_theta_GP[e][i][1], 2.0) * Jac[e][i];
			temp6 += w[i] * Jac[e][i];
		}
	}
	fp = fopen("at_GP_overlay_for_error_norm_surface_integral.txt", "w");
	fprintf(fp, "(stress_r-theory)^2_surface_integral\t(stress_theta-thory)^2_surface_integral\tstress_r^2_surface_integral\tstress_theta^2_surface_integral\t面積(analysis)\n");
	fprintf(fp, "%.15e\t%.15e\t%.15e\t%.15e\t%.15e\n", temp2, temp3, temp4, temp5, temp6);
	fclose(fp);
}
*/

__host__ void K_output_svg(int ndof)
{
	// [K] = [[K^G, K^GL], [K^GL, K^L]]

	int i, j;

	char color_vec[2][10] = {"#f5f5f5", "#ee82ee"};
	// 0	whitesmoke
	// 1	violet
	// https://www.colordic.org/

	double space = 3.0, scale = 1000.0 / (((double)ndof) + 2.0 * space);

	double width = (((double)ndof) + 2.0 * space) * scale;
	double height = width;

	char str[256] = "K_matrix.svg";
	fp = fopen(str, "w");

	fprintf(fp, "<?xml version='1.0'?>\n");
	// fprintf(fp, "<svg width='%lept' height='%lept' viewBox='0 0 %le %le' style = 'background: #eee' xmlns='http://www.w3.org/2000/svg' xmlns:xlink='http://www.w3.org/1999/xlink'>\n", width, height, width, height);
	fprintf(fp, "<svg width='%le' height='%le' version='1.1' style='background: #eee' xmlns='http://www.w3.org/2000/svg' xmlns:xlink='http://www.w3.org/1999/xlink'>\n", width, height);

	double xx = space * scale;
	double yy = space * scale;
	double ww = ndof * scale;
	double hh = ndof * scale;
	fprintf(fp, "<rect x='%le' y='%le' width='%le' height='%le' fill='%s' />\n", xx, yy, ww, hh, color_vec[0]);

	// 各行の成分を抽出
	for (i = 0; i < ndof; i++)
	{
		int K_bool[ndof]; //一行分保存する
		for (j = 0; j < ndof; j++)
		{
			K_bool[j] = 0;
		}

		for (j = 0; j < ndof; j++)
		{
			int temp_count;
			if (i <= j)
			{
				temp_count = RowCol_to_icount(i, j);
			}
			else if (i > j)
			{
				temp_count = RowCol_to_icount(j, i);
			}

			if (temp_count != -1)
			{
				K_bool[j] = 1;
			}
		}

		for (j = 0; j < ndof; j++)
		{
			double x = (((double)j) + space) * scale;
			double y = (((double)i) + space) * scale;
			// if (K_bool[j] == 0)
			// {
			// 	fprintf(fp, "<rect x='%le' y='%le' width='%le' height='%le' fill='%s' />\n", x, y, scale, scale, color_vec[0]);
			// }
			if (K_bool[j] == 1)
			{
				fprintf(fp, "<rect x='%le' y='%le' width='%le' height='%le' fill='%s' />\n", x, y, scale, scale, color_vec[1]);
			}
		}
	}

	fprintf(fp, "</svg>");
	fclose(fp);
}

__host__ void Make_gauss_array(int select_GP)
{
	int i, j;

	if (select_GP == 0)
	{
		GP_1dir = Ng;
	}
	else if (select_GP == 1)
	{
		GP_1dir = Ng_extended;
	}

	GP_2D = GP_1dir * GP_1dir;

	if (GP_1dir == 3)
	{
		double G1 = pow((3.0 / 5.0), 0.5);
		double G_vec[3] = {-G1, 0.0, G1};
		double w1 = 8.0 / 9.0;
		double w2 = 5.0 / 9.0;
		double w_vec[3] = {w2, w1, w2};

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
	else if (GP_1dir == 4)
	{
		double A = pow((6.0 / 5.0), 0.5);
		double G1 = pow(((3.0 - 2.0 * A) / 7.0), 0.5);
		double G2 = pow(((3.0 + 2.0 * A) / 7.0), 0.5);
		double G_vec[4] = {-G2, -G1, G1, G2};
		double B = pow(30.0, 0.5);
		double w1 = (18.0 + B) / 36.0;
		double w2 = (18.0 - B) / 36.0;
		double w_vec[4] = {w2, w1, w1, w2};

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
	else if (GP_1dir == 5)
	{
		double A = pow((10.0 / 7.0), 0.5);
		double G1 = pow((5.0 - 2.0 * A), 0.5) / 3.0;
		double G2 = pow((5.0 + 2.0 * A), 0.5) / 3.0;
		double G_vec[5] = {-G2, -G1, 0.0, G1, G2};
		double B = pow(70.0, 0.5);
		double w1 = 128.0 / 225.0;
		double w2 = (322.0 + 13.0 * B) / 900.0;
		double w3 = (322.0 - 13.0 * B) / 900.0;
		double w_vec[5] = {w3, w2, w1, w2, w3};

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
	else if (GP_1dir == 10)
	{
		double G_vec[10];
		double w_vec[10];

		G_vec[0] = -0.9739065285171717;
		G_vec[1] = -0.8650633666889845;
		G_vec[2] = -0.6794095682990244;
		G_vec[3] = -0.4333953941292472;
		G_vec[4] = -0.1488743389816312;
		G_vec[5] = 0.1488743389816312;
		G_vec[6] = 0.4333953941292472;
		G_vec[7] = 0.6794095682990244;
		G_vec[8] = 0.8650633666889845;
		G_vec[9] = 0.9739065285171717;

		w_vec[0] = 0.0666713443086881;
		w_vec[1] = 0.1494513491505804;
		w_vec[2] = 0.2190863625159820;
		w_vec[3] = 0.2692667193099965;
		w_vec[4] = 0.2955242247147530;
		w_vec[5] = 0.2955242247147530;
		w_vec[6] = 0.2692667193099965;
		w_vec[7] = 0.2190863625159820;
		w_vec[8] = 0.1494513491505804;
		w_vec[9] = 0.0666713443086881;

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
}

__host__ void Make_Displacement_grad_glo_check(int Total_Element)
{
	int i, j, j1, j2;
	int re;

	for (re = 0; re < Total_Element; re++)
	{
		i = real_element[re];
		// printf("El_No;i=%d\n", real_element[re]);
		double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION];
		// printf("Total_Element=%d\tre=%d\tEl_No=%d\n", Total_Element, re, i);

		for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; j1++)
		{
			for (j2 = 0; j2 < DIMENSION; j2++)
			{
				X[j1][j2] = Node_Coordinate[Controlpoint_of_Element[i][j1]][j2];
				// printf("X[%d][%d] = %.10e\n", j1, j2, Node_Coordinate[Controlpoint_of_Element[i][j1]][j2]);
			}
		}

		if (Element_mesh[i] > 0) //ローカルメッシュ上の要素について
		{
			if (NNLOVER[i] > 0) //重なっている要素が存在するとき
			{
				for (j = 0; j < NNLOVER[i]; j++)
				{
					double XG[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION];
					for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[NELOVER[i][j]]]; j1++)
					{
						for (j2 = 0; j2 < DIMENSION; j2++)
						{
							XG[j1][j2] = Node_Coordinate[Controlpoint_of_Element[NELOVER[i][j]][j1]][j2];
							//重なっている要素の物理座標取得
							// printf("XG[%d][%d] = %.10e\n", j1, j2, Node_Coordinate[Controlpoint_of_Element[NELOVER[i][j]][j1]][j2]);
						}
					}
					Make_Displacement_grad_glo(i, NELOVER[i][j], X, XG);
				}
			}
		}
	}
}

__host__ int Make_Displacement_grad_glo(int El_No_loc, int El_No_glo,
										double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION],
										double XG[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION])
{
	int i, j, jj, k;
	int BDBJ_flag;

	static double U[MAX_KIEL_SIZE];
	double B_glo[D_MATRIX_SIZE][MAX_KIEL_SIZE], b_grad_glo[DIMENSION * DIMENSION][2 * MAX_NO_CCpoint_ON_ELEMENT];
	// double X_glo[No_Control_point_ON_ELEMENT[Element_patch[El_No_glo]]][DIMENSION];
	double J = 0.0;

	Make_gauss_array(0);

	double G_Gxi[GP_2D][DIMENSION]; //グローバルパッチ上での親要素内座標xi_bar,eta_bar

	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No_glo]]; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			U[i * DIMENSION + j] = Displacement[Controlpoint_of_Element[El_No_glo][i] * DIMENSION + j];
			// printf("Controlpoint_of_Element[%d][%d] = %d\n", El_No_glo, i, Controlpoint_of_Element[El_No_glo][i]);
			// printf("Displacement[Controlpoint_of_Element[%d][%d]] = %.10e\n", El_No_glo, i, Displacement[Controlpoint_of_Element[El_No_glo][i] * DIMENSION + j]);
		}
	}
	for (i = 0; i < GP_2D; i++) //ガウス点のループ(local)
	{
		// printf("gauss point number:%d\n", i);
		////ローカルガウス点がグローバル要素に含まれているかの判定
		//ローカル要素ガウス点の物理座標算出
		double data_result_shape[2] = {0.0};
		double output_xi, output_eta;
		int patch_n = 0;
		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No_loc]]; j++)
		{
			double R_shape_func = Shape_func(j, Gxi[i], El_No_loc);
			for (jj = 0; jj < DIMENSION; jj++)
			{
				data_result_shape[jj] += R_shape_func * X[j][jj]; // Node_coordinate[controlpoint_of_element[element_n_loc][j]][jj]
			}
		}

		//ローカル要素ガウス点のグローバルパッチ上のパラメータ空間座標算出
		for (j = 0; j < Total_Patch_on_mesh[0]; j++) //グローバルメッシュ[0]上
		{
			Calc_xi_eta(data_result_shape[0], data_result_shape[1],
						Position_Knots[j][0], Position_Knots[j][1],
						No_Control_point[j][0], No_Control_point[j][1], Order[j][0], Order[j][1],
						&output_xi, &output_eta);
			// printf("  x: % 1.8e\n", data_result_shape[0]);
			// printf("  y: % 1.8e\n", data_result_shape[1]);
			// printf(" xi: % 1.8e\n", output_xi);
			// printf("eta: % 1.8e\n", output_eta);
			// printf("patch_n: %d\n", j);
			patch_n = j;
		}
		//要素内外判定

		if (output_xi >= Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0]] &&
			output_xi < Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0] + 1] &&
			output_eta >= Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1]] &&
			output_eta < Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1] + 1]) //要素内であるとき
		{
			BDBJ_flag = 1;
			printf("BDBJ_flag\n");

			//親要素座標の算出
			G_Gxi[i][0] = -1 + 2 * (output_xi - Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0]]) /
								   (Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0] + 1] - Position_Knots[patch_n][0][Order[patch_n][0] + ENC[patch_n][El_No_glo][0]]);
			G_Gxi[i][1] = -1 + 2 * (output_eta - Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1]]) /
								   (Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1] + 1] - Position_Knots[patch_n][1][Order[patch_n][1] + ENC[patch_n][El_No_glo][1]]);
			// printf("G_Gxi[][]=\n");
		}
		else //要素外であるとき
		{
			BDBJ_flag = 0;
		}

		// printf("i=%d\n",i );

		////結合要素剛性マトリックス計算
		//要素内であるとき、次を計算
		if (BDBJ_flag)
		{
			// printf("BDBJ_flag\ti=%d\n",i );
			//重なるグローバル要素のBマトリックス
			//  Make_B_Matrix(El_No_glo, BG, G_Gxi[i], XG, &J,
			//  			  Total_Control_Point_to_mesh[Total_mesh]);
			Make_B_Matrix(El_No_glo, B_glo, G_Gxi[i], XG, &J);
			Make_b_grad_Matrix(El_No_glo, b_grad_glo, G_Gxi[i], XG, &J);
			for (j = 0; j < DIMENSION * DIMENSION; j++)
			{
				for (k = 0; k < 2 * No_Control_point_ON_ELEMENT[Element_patch[El_No_glo]]; k++)
				{
					Disp_grad_glo[El_No_loc][i][j] += b_grad_glo[j][k] * U[k];
					// printf("b_grad_glo[%d][%d] = %.10e\n", j, k, b_grad_glo[j][k]);
				}
				// printf("U[%d] = %.14e\tDisp_grad_glo[%d][%d][%d] = %.14e\n", k, U[k], El_No_loc, i, j, Disp_grad_glo[El_No_loc][i][j]);
			}
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				for (k = 0; k < KIEL_SIZE; k++)
				{
					Strain_glo[El_No_loc][i][j] += B_glo[j][k] * U[k];
				}
				// printf("Strain_glo[%d][%d][%d] = %.10e\n", El_No_loc, i, j, Strain_glo[El_No_loc][i][j]);
			}
		}
	}

	return 0;
}

__host__ void Make_auxiliary_mode1(int e, double E, double nu, int DM, double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double crack_front_coordinates_x, double crack_front_coordinates_y)
{
	int i, j, N;
	static double Dinv[D_MATRIX_SIZE][D_MATRIX_SIZE];
	double unit_basis_local[DIMENSION] = {0.0};
	double r_tip = sqrt(crack_front_coordinates_x * crack_front_coordinates_x + crack_front_coordinates_y * crack_front_coordinates_y);
	double crack_front_coordinates_x_local, crack_front_coordinates_y_local;
	double data_result_shape_local[DIMENSION] = {0.0};
	double r, rad;
	double mu = E / (2.0 * (1.0 + nu));
	double Pi = 4.0 * atan(1.0);

	Make_gauss_array(0);

	Make_D_Matrix_2D(Dinv, E, nu, DM);

	InverseMatrix_3X3(Dinv);

	// printf("x_crackfront : % 1.8e\n", crack_front_coordinates_x);
	// printf("y_crackfront : % 1.8e\n", crack_front_coordinates_y);

	// x'-y'（き裂先端）座標における単位基底ベクトル
	unit_basis_local[0] = crack_front_coordinates_x / r_tip;
	unit_basis_local[1] = crack_front_coordinates_y / r_tip;
	// printf("unit_basis[0] : % 1.8e\n", unit_basis_local[0]);
	// printf("unit_basis[1] : % 1.8e\n", unit_basis_local[1]);
	crack_front_coordinates_x_local = unit_basis_local[0] * crack_front_coordinates_x + unit_basis_local[1] * crack_front_coordinates_y;
	crack_front_coordinates_y_local = -unit_basis_local[1] * crack_front_coordinates_x + unit_basis_local[0] * crack_front_coordinates_y;
	// printf("x_crackfront_local : % 1.8e\n", crack_front_coordinates_x_local);
	// printf("y_crackfront_local : % 1.8e\n", crack_front_coordinates_y_local);

	for (N = 0; N < GP_2D; N++)
	{
		//各要素でガウス点での物理座標の算出する
		double data_result_shape[2] = {0.0};
		for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
		{
			double R_shape_func = Shape_func(i, Gxi[N], e);
			for (j = 0; j < DIMENSION; j++)
			{
				data_result_shape[j] += R_shape_func * X[i][j];
			}
		}
		printf("x_gauss : % 1.8e\n", data_result_shape[0]);
		printf("y_gauss : % 1.8e\n", data_result_shape[1]);

		data_result_shape_local[0] = unit_basis_local[0] * data_result_shape[0] + unit_basis_local[1] * data_result_shape[1];
		data_result_shape_local[1] = -unit_basis_local[1] * data_result_shape[0] + unit_basis_local[0] * data_result_shape[1];
		printf("x_gauss_local : % 1.8e\n", data_result_shape_local[0]);
		printf("y_gauss_local : % 1.8e\n", data_result_shape_local[1]);

		//ガウス点でのx'-y'座標をき裂先端での極座標に変換する
		r = sqrt((data_result_shape_local[0] - crack_front_coordinates_x_local) * (data_result_shape_local[0] - crack_front_coordinates_x_local) + (data_result_shape_local[1] - crack_front_coordinates_y_local) * (data_result_shape_local[1] - crack_front_coordinates_y_local));
		rad = atan2(data_result_shape_local[1] - crack_front_coordinates_y_local, data_result_shape_local[0] - crack_front_coordinates_x_local);
		printf("r_gauss : % 1.8e\n", r);
		printf("rad_gauss : % 1.8e\n", rad);
		printf("degree_gauss : % 1.8e\n", rad * 180.0 / Pi);

		// x'-y'座標での変位勾配を算出する
		Disp_grad_aux_mode1_local[e][N][0] = sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * (1.0 - 2.0 * nu + sin(rad / 2.0) * sin(rad / 2.0) - 2.0 * sin(rad) * sin(rad / 2.0) * cos(rad / 2.0)) / (2.0 * mu);
		Disp_grad_aux_mode1_local[e][N][1] = sqrt(1.0 / (2.0 * Pi * r)) * sin(rad / 2.0) * (1.0 - 2.0 * nu + sin(rad / 2.0) * sin(rad / 2.0) + 2.0 * cos(rad / 2.0) * cos(rad / 2.0) * cos(rad)) / (2.0 * mu);
		Disp_grad_aux_mode1_local[e][N][2] = -sqrt(1.0 / (2.0 * Pi * r)) * sin(rad / 2.0) * (2.0 - 2.0 * nu - cos(rad / 2.0) * cos(rad / 2.0) + 2.0 * sin(rad) * sin(rad / 2.0) * cos(rad / 2.0)) / (2.0 * mu);
		Disp_grad_aux_mode1_local[e][N][3] = sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * (2.0 - 2.0 * nu - cos(rad / 2.0) * cos(rad / 2.0) + 2.0 * sin(rad / 2.0) * sin(rad / 2.0) * cos(rad)) / (2.0 * mu);
		printf("Disp_grad_aux_mode1_local[%d][%d][0] = %1.10e\n", e, N, Disp_grad_aux_mode1_local[e][N][0]);
		printf("Disp_grad_aux_mode1_local[%d][%d][1] = %1.10e\n", e, N, Disp_grad_aux_mode1_local[e][N][1]);
		printf("Disp_grad_aux_mode1_local[%d][%d][2] = %1.10e\n", e, N, Disp_grad_aux_mode1_local[e][N][2]);
		printf("Disp_grad_aux_mode1_local[%d][%d][3] = %1.10e\n", e, N, Disp_grad_aux_mode1_local[e][N][3]);

		//変位勾配をx'-y'座標からx-y座標に変換する
		Disp_grad_aux_mode1[e][N][0] = T[0][0] * T[0][0] * Disp_grad_aux_mode1_local[e][N][0] + T[0][0] * T[1][0] * (Disp_grad_aux_mode1_local[e][N][1] + Disp_grad_aux_mode1_local[e][N][2]) + T[1][0] * T[1][0] * Disp_grad_aux_mode1_local[e][N][3];
		Disp_grad_aux_mode1[e][N][1] = T[0][0] * T[0][1] * Disp_grad_aux_mode1_local[e][N][0] + T[0][0] * T[1][1] * Disp_grad_aux_mode1_local[e][N][1] + T[0][1] * T[1][0] * Disp_grad_aux_mode1_local[e][N][2] + T[1][0] * T[1][1] * Disp_grad_aux_mode1_local[e][N][3];
		Disp_grad_aux_mode1[e][N][2] = T[0][0] * T[0][1] * Disp_grad_aux_mode1_local[e][N][0] + T[0][0] * T[1][1] * Disp_grad_aux_mode1_local[e][N][2] + T[0][1] * T[1][0] * Disp_grad_aux_mode1_local[e][N][1] + T[1][0] * T[1][1] * Disp_grad_aux_mode1_local[e][N][3];
		Disp_grad_aux_mode1[e][N][3] = T[0][1] * T[0][1] * Disp_grad_aux_mode1_local[e][N][0] + T[0][1] * T[1][1] * (Disp_grad_aux_mode1_local[e][N][1] + Disp_grad_aux_mode1_local[e][N][2]) + T[1][1] * T[1][1] * Disp_grad_aux_mode1_local[e][N][3];
		printf("Disp_grad_aux_mode1[%d][%d][0] = %1.10e\n", e, N, Disp_grad_aux_mode1[e][N][0]);
		printf("Disp_grad_aux_mode1[%d][%d][1] = %1.10e\n", e, N, Disp_grad_aux_mode1[e][N][1]);
		printf("Disp_grad_aux_mode1[%d][%d][2] = %1.10e\n", e, N, Disp_grad_aux_mode1[e][N][2]);
		printf("Disp_grad_aux_mode1[%d][%d][3] = %1.10e\n", e, N, Disp_grad_aux_mode1[e][N][3]);

		printf("strain_Disp_grad_aux_mode1[%d][%d][0] = %1.10e\n", e, N, Disp_grad_aux_mode1[e][N][0]);
		printf("strain_Disp_grad_aux_mode1[%d][%d][1] = %1.10e\n", e, N, Disp_grad_aux_mode1[e][N][3]);
		printf("strain_Disp_grad_aux_mode1[%d][%d][2] = %1.10e\n", e, N, Disp_grad_aux_mode1[e][N][1] + Disp_grad_aux_mode1[e][N][2]);

		// x'-y'座標での応力を算出する
		Stress_aux_mode1_local[e][N][0] = sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * (1.0 - sin(rad / 2.0) * sin(3.0 * rad / 2.0));
		Stress_aux_mode1_local[e][N][1] = sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * (1.0 + sin(rad / 2.0) * sin(3.0 * rad / 2.0));
		Stress_aux_mode1_local[e][N][2] = sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * sin(rad / 2.0) * cos(3.0 * rad / 2.0);
		printf("Stress_aux_mode1_local[%d][%d][0] = %1.10e\n", e, N, Stress_aux_mode1_local[e][N][0]);
		printf("Stress_aux_mode1_local[%d][%d][1] = %1.10e\n", e, N, Stress_aux_mode1_local[e][N][1]);
		printf("Stress_aux_mode1_local[%d][%d][2] = %1.10e\n", e, N, Stress_aux_mode1_local[e][N][2]);

		//応力をx'-y'座標からx-y座標に変換する
		Stress_aux_mode1[e][N][0] = T[0][0] * T[0][0] * Stress_aux_mode1_local[e][N][0] + 2.0 * T[0][0] * T[1][0] * Stress_aux_mode1_local[e][N][2] + T[1][0] * T[1][0] * Stress_aux_mode1_local[e][N][1];
		Stress_aux_mode1[e][N][1] = T[0][1] * T[0][1] * Stress_aux_mode1_local[e][N][0] + 2.0 * T[0][1] * T[1][1] * Stress_aux_mode1_local[e][N][2] + T[1][1] * T[1][1] * Stress_aux_mode1_local[e][N][1];
		Stress_aux_mode1[e][N][2] = T[0][0] * T[0][1] * Stress_aux_mode1_local[e][N][0] + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * Stress_aux_mode1_local[e][N][2] + T[1][0] * T[1][1] * Stress_aux_mode1_local[e][N][1];
		printf("Stress_aux_mode1[%d][%d][0] = %1.10e\n", e, N, Stress_aux_mode1[e][N][0]);
		printf("Stress_aux_mode1[%d][%d][1] = %1.10e\n", e, N, Stress_aux_mode1[e][N][1]);
		printf("Stress_aux_mode1[%d][%d][2] = %1.10e\n", e, N, Stress_aux_mode1[e][N][2]);

		//ひずみを算出する
		for (i = 0; i < D_MATRIX_SIZE; i++)
		{
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				Strain_aux_mode1[e][N][i] += Dinv[i][j] * Stress_aux_mode1[e][N][j];
			}
			printf("Strain_aux_mode1[%d][%d][%d] = %1.10e\n", e, N, i, Strain_aux_mode1[e][N][i]);
		}

		//相互ポテンシャルエネルギ密度Wを算出する
		StrainEnergyDensity_aux_mode1[e][N] = Stress_overlay[e][N][0] * Strain_aux_mode1[e][N][0] + Stress_overlay[e][N][1] * Strain_aux_mode1[e][N][1] + Stress_overlay[e][N][2] * Strain_aux_mode1[e][N][2];
		printf("StrainEnergyDensity_aux_mode1[%d][%d] = %1.10e\n", e, N, StrainEnergyDensity_aux_mode1[e][N]);
	}
}

__host__ void Make_auxiliary_mode2(int e, double E, double nu, int DM, double X[MAX_NO_CCpoint_ON_ELEMENT][DIMENSION], double crack_front_coordinates_x, double crack_front_coordinates_y)
{
	int i, j, N;
	static double Dinv[D_MATRIX_SIZE][D_MATRIX_SIZE];
	double unit_basis_local[DIMENSION] = {0.0};
	double r_tip = sqrt(crack_front_coordinates_x * crack_front_coordinates_x + crack_front_coordinates_y * crack_front_coordinates_y);
	double crack_front_coordinates_x_local, crack_front_coordinates_y_local;
	double data_result_shape_local[DIMENSION] = {0.0};
	double r, rad;
	double mu = E / (2.0 * (1 + nu));
	double Pi = 4.0 * atan(1.0);

	Make_gauss_array(0);

	Make_D_Matrix_2D(Dinv, E, nu, DM);

	InverseMatrix_3X3(Dinv);

	// printf("x_crackfront : % 1.8e\n", crack_front_coordinates_x);
	// printf("y_crackfront : % 1.8e\n", crack_front_coordinates_y);

	// x'-y'（き裂先端）座標における単位基底ベクトル
	unit_basis_local[0] = crack_front_coordinates_x / r_tip;
	unit_basis_local[1] = crack_front_coordinates_y / r_tip;
	// printf("unit_basis[0] : % 1.8e\n", unit_basis_local[0]);
	// printf("unit_basis[1] : % 1.8e\n", unit_basis_local[1]);
	crack_front_coordinates_x_local = unit_basis_local[0] * crack_front_coordinates_x + unit_basis_local[1] * crack_front_coordinates_y;
	crack_front_coordinates_y_local = -unit_basis_local[1] * crack_front_coordinates_x + unit_basis_local[0] * crack_front_coordinates_y;
	// printf("x_crackfront_local : % 1.8e\n", crack_front_coordinates_x_local);
	// printf("y_crackfront_local : % 1.8e\n", crack_front_coordinates_y_local);

	for (N = 0; N < GP_2D; N++)
	{
		//各要素でガウス点での物理座標の算出する
		double data_result_shape[2] = {0.0};
		for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[e]]; i++)
		{
			double R_shape_func = Shape_func(i, Gxi[N], e);
			for (j = 0; j < DIMENSION; j++)
			{
				X[i][j] = Node_Coordinate[Controlpoint_of_Element[e][i]][j];
				data_result_shape[j] += R_shape_func * X[i][j];
			}
		}
		// printf("x_gauss : % 1.8e\n", data_result_shape[0]);
		// printf("y_gauss : % 1.8e\n", data_result_shape[1]);

		data_result_shape_local[0] = unit_basis_local[0] * data_result_shape[0] + unit_basis_local[1] * data_result_shape[1];
		data_result_shape_local[1] = -unit_basis_local[1] * data_result_shape[0] + unit_basis_local[0] * data_result_shape[1];
		// printf("x_gauss_local : % 1.8e\n", data_result_shape_local[0]);
		// printf("y_gauss_local : % 1.8e\n", data_result_shape_local[1]);

		//ガウス点での物理座標をき裂先端での円筒座標に変換する
		r = sqrt((data_result_shape_local[0] - crack_front_coordinates_x_local) * (data_result_shape_local[0] - crack_front_coordinates_x_local) + (data_result_shape_local[1] - crack_front_coordinates_y_local) * (data_result_shape_local[1] - crack_front_coordinates_y_local));
		rad = atan2(data_result_shape_local[1] - crack_front_coordinates_y_local, data_result_shape_local[0] - crack_front_coordinates_x_local);
		// printf("r_gauss : % 1.8e\n", r);
		// printf("rad_gauss : % 1.8e\n", rad);
		// printf("degree_gauss : % 1.8e\n", rad * 180.0 / Pi);

		// x'-y'座標での変位勾配を算出する
		Disp_grad_aux_mode2_local[e][N][0] = -sqrt(1.0 / (2.0 * Pi * r)) * sin(rad / 2.0) * (2.0 - 2.0 * nu + cos(rad / 2.0) * cos(rad / 2.0) - 2.0 * sin(rad) * sin(rad / 2.0) * cos(rad / 2.0)) / (2.0 * mu);
		Disp_grad_aux_mode2_local[e][N][1] = sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * (2.0 - 2.0 * nu + cos(rad / 2.0) * cos(rad / 2.0) - 2.0 * sin(rad / 2.0) * sin(rad / 2.0) * cos(rad)) / (2.0 * mu);
		Disp_grad_aux_mode2_local[e][N][2] = -sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * (1.0 - 2.0 * nu - sin(rad / 2.0) * sin(rad / 2.0) + 2.0 * sin(rad) * sin(rad / 2.0) * cos(rad / 2.0)) / (2.0 * mu);
		Disp_grad_aux_mode2_local[e][N][3] = -sqrt(1.0 / (2.0 * Pi * r)) * sin(rad / 2.0) * (1.0 - 2.0 * nu - sin(rad / 2.0) * sin(rad / 2.0) + 2.0 * cos(rad / 2.0) * cos(rad / 2.0) * cos(rad)) / (2.0 * mu);
		// printf("Disp_grad_aux_mode2_local[%d][%d][0] = %1.10e\n", e, N, Disp_grad_aux_mode2_local[e][N][0]);
		// printf("Disp_grad_aux_mode2_local[%d][%d][1] = %1.10e\n", e, N, Disp_grad_aux_mode2_local[e][N][1]);
		// printf("Disp_grad_aux_mode2_local[%d][%d][2] = %1.10e\n", e, N, Disp_grad_aux_mode2_local[e][N][2]);
		// printf("Disp_grad_aux_mode2_local[%d][%d][3] = %1.10e\n", e, N, Disp_grad_aux_mode2_local[e][N][3]);

		//変位勾配をx'-y'座標からx-y座標に変換する
		Disp_grad_aux_mode2[e][N][0] = T[0][0] * T[0][0] * Disp_grad_aux_mode2_local[e][N][0] + T[0][0] * T[1][0] * (Disp_grad_aux_mode2_local[e][N][1] + Disp_grad_aux_mode2_local[e][N][2]) + T[1][0] * T[1][0] * Disp_grad_aux_mode2_local[e][N][3];
		Disp_grad_aux_mode2[e][N][1] = T[0][0] * T[0][1] * Disp_grad_aux_mode2_local[e][N][0] + T[0][0] * T[1][1] * Disp_grad_aux_mode2_local[e][N][1] + T[0][1] * T[1][0] * Disp_grad_aux_mode2_local[e][N][2] + T[1][0] * T[1][1] * Disp_grad_aux_mode2_local[e][N][3];
		Disp_grad_aux_mode2[e][N][2] = T[0][0] * T[0][1] * Disp_grad_aux_mode2_local[e][N][0] + T[0][0] * T[1][1] * Disp_grad_aux_mode2_local[e][N][2] + T[0][1] * T[1][0] * Disp_grad_aux_mode2_local[e][N][1] + T[1][0] * T[1][1] * Disp_grad_aux_mode2_local[e][N][3];
		Disp_grad_aux_mode2[e][N][3] = T[0][1] * T[0][1] * Disp_grad_aux_mode2_local[e][N][0] + T[0][1] * T[1][1] * (Disp_grad_aux_mode2_local[e][N][1] + Disp_grad_aux_mode2_local[e][N][2]) + T[1][1] * T[1][1] * Disp_grad_aux_mode2_local[e][N][3];
		// printf("Disp_grad_aux_mode2[%d][%d][0] = %1.10e\n", e, N, Disp_grad_aux_mode2[e][N][0]);
		// printf("Disp_grad_aux_mode2[%d][%d][1] = %1.10e\n", e, N, Disp_grad_aux_mode2[e][N][1]);
		// printf("Disp_grad_aux_mode2[%d][%d][2] = %1.10e\n", e, N, Disp_grad_aux_mode2[e][N][2]);
		// printf("Disp_grad_aux_mode2[%d][%d][3] = %1.10e\n", e, N, Disp_grad_aux_mode2[e][N][3]);

		// printf("strain_Disp_grad_aux_mode2[%d][%d][0] = %1.10e\n", e, N, Disp_grad_aux_mode2[e][N][0]);
		// printf("strain_Disp_grad_aux_mode2[%d][%d][1] = %1.10e\n", e, N, Disp_grad_aux_mode2[e][N][3]);
		// printf("strain_Disp_grad_aux_mode2[%d][%d][2] = %1.10e\n", e, N, Disp_grad_aux_mode2[e][N][1] + Disp_grad_aux_mode2[e][N][2]);

		// x'-y'座標での応力を算出する
		Stress_aux_mode2_local[e][N][0] = -sqrt(1.0 / (2.0 * Pi * r)) * sin(rad / 2.0) * (2.0 + cos(rad / 2.0) * cos(3.0 * rad / 2.0));
		Stress_aux_mode2_local[e][N][1] = sqrt(1.0 / (2.0 * Pi * r)) * sin(rad / 2.0) * cos(rad / 2.0) * cos(3.0 * rad / 2.0);
		Stress_aux_mode2_local[e][N][2] = sqrt(1.0 / (2.0 * Pi * r)) * cos(rad / 2.0) * (1.0 - sin(rad / 2.0) * sin(3.0 * rad / 2.0));
		// printf("Stress_aux_mode2_local[%d][%d][0] = %1.10e\n", e, N, Stress_aux_mode2_local[e][N][0]);
		// printf("Stress_aux_mode2_local[%d][%d][1] = %1.10e\n", e, N, Stress_aux_mode2_local[e][N][1]);
		// printf("Stress_aux_mode2_local[%d][%d][2] = %1.10e\n", e, N, Stress_aux_mode2_local[e][N][2]);

		//応力をx'-y'座標からx-y座標に変換する
		Stress_aux_mode2[e][N][0] = T[0][0] * T[0][0] * Stress_aux_mode2_local[e][N][0] + 2.0 * T[0][0] * T[1][0] * Stress_aux_mode2_local[e][N][2] + T[1][0] * T[1][0] * Stress_aux_mode2_local[e][N][1];
		Stress_aux_mode2[e][N][1] = T[0][1] * T[0][1] * Stress_aux_mode2_local[e][N][0] + 2.0 * T[0][1] * T[1][1] * Stress_aux_mode2_local[e][N][2] + T[1][1] * T[1][1] * Stress_aux_mode2_local[e][N][1];
		Stress_aux_mode2[e][N][2] = T[0][0] * T[0][1] * Stress_aux_mode2_local[e][N][0] + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * Stress_aux_mode2_local[e][N][2] + T[1][0] * T[1][1] * Stress_aux_mode2_local[e][N][1];
		// printf("Stress_aux_mode2[%d][%d][0] = %1.10e\n", e, N, Stress_aux_mode2[e][N][0]);
		// printf("Stress_aux_mode2[%d][%d][1] = %1.10e\n", e, N, Stress_aux_mode2[e][N][1]);
		// printf("Stress_aux_mode2[%d][%d][2] = %1.10e\n", e, N, Stress_aux_mode2[e][N][2]);

		//ひずみを算出する
		for (i = 0; i < D_MATRIX_SIZE; i++)
		{
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				Strain_aux_mode2[e][N][i] += Dinv[i][j] * Stress_aux_mode2[e][N][j];
			}
			// printf("Strain_aux_mode2[%d][%d][%d] = %1.10e\n", e, N, i, Strain_aux_mode2[e][N][i]);
		}

		//相互ポテンシャルエネルギ密度Wを算出する
		StrainEnergyDensity_aux_mode2[e][N] = Stress_overlay[e][N][0] * Strain_aux_mode2[e][N][0] + Stress_overlay[e][N][1] * Strain_aux_mode2[e][N][1] + Stress_overlay[e][N][2] * Strain_aux_mode2[e][N][2];
		// printf("StrainEnergyDensity_aux_mode2[%d][%d] = %1.10e\n", e, N, StrainEnergyDensity_aux_mode2[e][N]);
	}
}