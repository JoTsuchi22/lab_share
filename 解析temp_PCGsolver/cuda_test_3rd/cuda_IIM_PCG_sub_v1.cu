#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <time.h>

// header
#include "s_IGA_header.h"
#include "s_IGA_sub.h"


// ファイル読み込み1回目
void Get_Input_1(int tm, int *Total_Knot_to_mesh,
				 int *Total_Patch_on_mesh, int *Total_Patch_to_mesh,
				 int *Total_Control_Point_on_mesh, int *Total_Control_Point_to_mesh,
				 int *Total_Constraint_to_mesh, int *Total_Load_to_mesh, int *Total_DistributeForce_to_mesh,
				 char **argv)
{
	char s[256];
	int temp_i;
	double temp_d;

	int i, j;

	fp = fopen(argv[tm + 1], "r");

	// 材料定数
	fscanf(fp, "%lf %lf", &E, &nu);
	fgets(s, 256, fp);
	printf("E: %le, nu: %le\n", E, nu);

	// パッチ数
	fscanf(fp, "%d", &temp_i);
	fgets(s, 256, fp);
	int No_Patch = temp_i;
	int *CP = (int *)malloc(sizeof(int) * temp_i * DIMENSION);
	printf("No_Patch: %d\n", temp_i);
	Total_Patch_on_mesh[tm] = temp_i;
	Total_Patch_to_mesh[tm + 1] = Total_Patch_to_mesh[tm] + temp_i;
	printf("Total_Patch_to_mesh[%d] = %d\n", tm, Total_Patch_to_mesh[tm]);

	// コントロールポイント数
	fscanf(fp, "%d", &temp_i);
	fgets(s, 256, fp);
	int Total_Control_Point = temp_i;
	printf("Total_Control_Point: %d\n", temp_i);
	Total_Control_Point_on_mesh[tm] = temp_i;
	Total_Control_Point_to_mesh[tm + 1] = Total_Control_Point_to_mesh[tm] + temp_i;
	printf("Total_Control_Point_to_mesh[%d] = %d\n", tm, Total_Control_Point_to_mesh[tm]);

	// 各方向の次数(スキップ)
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &temp_i);
		}
	}
	fgets(s, 256, fp);

	// ノット数
	Total_Knot_to_mesh[tm + 1] = Total_Knot_to_mesh[tm];
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &temp_i);
			Total_Knot_to_mesh[tm + 1] += temp_i;
		}
	}

	// 各パッチ各方向のコントロールポイント数(スキップ)
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &temp_i);
			CP[i * DIMENSION + j] = temp_i;
		}
	}
	fgets(s, 256, fp);

	// パッチコネクティビティ(スキップ)
	for (i = 0; i < Total_Patch_on_mesh[tm]; i++)
	{
		for (j = 0; j < CP[i * DIMENSION + 0] * CP[i * DIMENSION + 1]; j++)
		{
		fscanf(fp, "%d", &temp_i);
		}
	}
	fgets(s, 256, fp);

	// 境界条件
	int Total_Constraint, Total_Load, Total_DistributeForce;
	fscanf(fp, "%d %d %d", &Total_Constraint, &Total_Load, &Total_DistributeForce);
	Total_Constraint_to_mesh[tm + 1] = Total_Constraint_to_mesh[tm] + Total_Constraint;
	Total_Load_to_mesh[tm + 1] = Total_Load_to_mesh[tm] + Total_Load;
	Total_DistributeForce_to_mesh[tm + 1] = Total_DistributeForce_to_mesh[tm] + Total_DistributeForce;

	printf("Total_Constraint = %d\n", Total_Constraint);
	printf("Total_Load = %d\n", Total_Load);
	printf("Total_DistributedForce = %d\n", Total_DistributeForce);

	fclose(fp);
	free(CP);
}


// ファイル読み込み2回目
void Get_Input_2(int tm, int *Total_Knot_to_mesh,
				 int *Total_Patch_to_mesh, int *Total_Control_Point_to_mesh,
				 int *Total_Element_on_mesh, int *Total_Element_to_mesh,
				 int *Total_Constraint_to_mesh, int *Total_Load_to_mesh, int *Total_DistributeForce_to_mesh,
				 int *Order, int *No_knot, int *No_Control_point, int *No_Control_point_in_patch,
				 int *Patch_Control_point, double *Position_Knots, int *No_Control_point_ON_ELEMENT,
				 double *Node_Coordinate, double *Control_Coord_x, double *Control_Coord_y, double *Control_Weight,
				 int *Constraint_Node_Dir, double *Value_of_Constraint,
				 int *Load_Node_Dir, double *Value_of_Load,
				 int *type_load_array, int *iPatch_array, int *iCoord_array,
				 double *val_Coord_array, double *Range_Coord_array, double *Coeff_Dist_Load_array,
				 int *Total_Control_Point_to_patch, int *Total_Knot_to_patch_dim,
				 char **argv)
{
	char s[256];
	int temp_i;
	double temp_d;

	int i, j, k;

	fp = fopen(argv[tm + 1], "r");

	// 材料定数(スキップ)
	fscanf(fp, "%lf %lf", &temp_d, &temp_d);
	fgets(s, 256, fp);

	// パッチ数(スキップ)
	fscanf(fp, "%d", &temp_i);
	fgets(s, 256, fp);
	int No_Patch = temp_i;

	// コントロールポイント数(スキップ)
	fscanf(fp, "%d", &temp_i);
	fgets(s, 256, fp);
	int Total_Control_Point = temp_i;

	// 各方向の次数
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &temp_i);
			Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] = temp_i;
			printf("Order[%d] = %d\n", (i + Total_Patch_to_mesh[tm]) * DIMENSION + j, Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j]);
		}
	}
	fgets(s, 256, fp);

	// ノット数
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &temp_i);
			No_knot[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] = temp_i;
			Total_Knot_to_patch_dim[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j + 1] = Total_Knot_to_patch_dim[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] + temp_i;
			printf("No_knot[%d] = %d\n", (i + Total_Patch_to_mesh[tm]) * DIMENSION + j, No_knot[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j]);
		}
	}
	fgets(s, 256, fp);

	// 各パッチ各方向のコントロールポイント数
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			fscanf(fp, "%d", &temp_i);
			No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] = temp_i;
			printf("No_Control_point[%d] = %d\n", (i + Total_Patch_to_mesh[tm]) * DIMENSION + j, No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j]);
		}
	}
	fgets(s, 256, fp);

	for (i = 0; i < No_Patch; i++)
	{
		No_Control_point_in_patch[i + Total_Patch_to_mesh[tm]] = 1;
	}

	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			No_Control_point_in_patch[i + Total_Patch_to_mesh[tm]] *= No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j];
		}
	}

	for (i = 0; i < No_Patch; i++)
	{
		Total_Control_Point_to_patch[i + Total_Patch_to_mesh[tm] + 1] = Total_Control_Point_to_patch[i + Total_Patch_to_mesh[tm]] + No_Control_point_in_patch[i + Total_Patch_to_mesh[tm]];
	}

	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			if (No_knot[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] != No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] + Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] + 1)
			{
				printf("wrong relationship between the number of knot vector and the number of control_point \n");
				printf("in mesh_No.%d in patch_No.%d direction:%d\n", tm, i, j);
			}
		}
	}

	for (i = 0; i < No_Patch; i++)
	{
		printf("No_Control_point_in_patch[%d] = %d\t", i + Total_Patch_to_mesh[tm], No_Control_point_in_patch[i + Total_Patch_to_mesh[tm]]);
	}
	printf("\n");

	// パッチコネクティビティ
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < No_Control_point_in_patch[i + Total_Patch_to_mesh[tm]]; j++)
		{
			fscanf(fp, "%d", &temp_i);
			Patch_Control_point[Total_Control_Point_to_patch[i + Total_Patch_to_mesh[tm]] + j] = temp_i;
			if (tm > 0)
			{
				Patch_Control_point[Total_Control_Point_to_patch[i + Total_Patch_to_mesh[tm]] + j] += Total_Control_Point_to_mesh[tm];
			}
		}
	}
	fgets(s, 256, fp);

	// 境界条件(スキップ)
	int Total_Constraint, Total_Load, Total_DistributeForce;
	fscanf(fp, "%d %d %d", &Total_Constraint, &Total_Load, &Total_DistributeForce);
	fgets(s, 256, fp);

	// ノットベクトルの読み込み
	for (i = 0; i < No_Patch; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			for (k = 0; k < No_knot[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j]; k++)
			{
				fscanf(fp, "%lf", &temp_d);
				Position_Knots[Total_Knot_to_patch_dim[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] + k] = temp_d;
				printf("%le\t", Position_Knots[Total_Knot_to_patch_dim[(i + Total_Patch_to_mesh[tm]) * DIMENSION + j] + k]);
			}
			printf("\n");
		}
	}
	fgets(s, 256, fp);

	int Total_Element = 0;
	for (i = 0; i < No_Patch; i++)
	{
		if (DIMENSION == 2)
		{
			Total_Element += (No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 0] - Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 0])
						   * (No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 1] - Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 1]);
			No_Control_point_ON_ELEMENT[i + Total_Patch_to_mesh[tm]] = (Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 0] + 1) * (Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 1] + 1);
		}
		else if (DIMENSION == 3)
		{
			Total_Element += (No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 0] - Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 0])
						   * (No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 1] - Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 1])
						   * (No_Control_point[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 2] - Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 2]);
			No_Control_point_ON_ELEMENT[i + Total_Patch_to_mesh[tm]] = (Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 0] + 1) * (Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 1] + 1) * (Order[(i + Total_Patch_to_mesh[tm]) * DIMENSION + 2] + 1);
		}
	}
	printf("Total_Element = %d\n", Total_Element);
	Total_Element_on_mesh[tm] = Total_Element;
	Total_Element_to_mesh[tm + 1] = Total_Element_to_mesh[tm] + Total_Element;
	printf("Total_Element_on_mesh[%d] = %d\n", tm, Total_Element_on_mesh[tm]);

	for (i = 0; i < No_Patch; i++)
	{
		printf("No_Control_point_ON_ELEMENT[%d] = %d\n", i + Total_Patch_to_mesh[tm], No_Control_point_ON_ELEMENT[i + Total_Patch_to_mesh[tm]]);
	}

	// 節点座標
	for (i = 0; i < Total_Control_Point; i++)
	{
		fscanf(fp, "%d", &temp_i);
		for (j = 0; j < DIMENSION + 1; j++)
		{
			fscanf(fp, "%lf", &temp_d);
			Node_Coordinate[(temp_i + Total_Control_Point_to_mesh[tm]) * (DIMENSION + 1) + j] = temp_d;
		}
	}
	for (i = 0; i < Total_Control_Point; i++)
	{
		for (j = 0; j < DIMENSION + 1; j++)
		{
			// コントロールポイント座標・重みの新たな配列(for s-IGA/NewtonLaphson) DIMENSION == 2 の場合のみ記述
			if (j == 0)
			{
				Control_Coord_x[i + Total_Control_Point_to_mesh[tm]] = Node_Coordinate[(i + Total_Control_Point_to_mesh[tm]) * (DIMENSION + 1) + j];
			}
			else if (j == 1)
			{
				Control_Coord_y[i + Total_Control_Point_to_mesh[tm]] = Node_Coordinate[(i + Total_Control_Point_to_mesh[tm]) * (DIMENSION + 1) + j];
			}
			else if (j == DIMENSION)
			{
				Control_Weight[i + Total_Control_Point_to_mesh[tm]] = Node_Coordinate[(i + Total_Control_Point_to_mesh[tm]) * (DIMENSION + 1) + j];
			}
		}
	}
	fgets(s, 256, fp);

	// 拘束
	for (i = 0; i < Total_Constraint; i++)
	{
		fscanf(fp, "%d %d %lf",
			   &Constraint_Node_Dir[(i + Total_Constraint_to_mesh[tm]) * 2 + 0],
			   &Constraint_Node_Dir[(i + Total_Constraint_to_mesh[tm]) * 2 + 1],
			   &Value_of_Constraint[i + Total_Constraint_to_mesh[tm]]);
		Constraint_Node_Dir[(i + Total_Constraint_to_mesh[tm]) * 2 + 0] = Constraint_Node_Dir[(i + Total_Constraint_to_mesh[tm]) * 2 + 0] + Total_Control_Point_to_mesh[tm];

		printf("Constraint_Node_Dir[%d] = %d Constraint_Node_Dir[%d] = %d Value_of_Constraint[%d] = %le \n",
			   (i + Total_Constraint_to_mesh[tm]) * 2 + 0, Constraint_Node_Dir[(i + Total_Constraint_to_mesh[tm]) * 2 + 0],
			   (i + Total_Constraint_to_mesh[tm]) * 2 + 1, Constraint_Node_Dir[(i + Total_Constraint_to_mesh[tm]) * 2 + 1],
			   i + Total_Constraint_to_mesh[tm], Value_of_Constraint[i + Total_Constraint_to_mesh[tm]]);
	}
	fgets(s, 256, fp);

	// 荷重
	for (i = 0; i < Total_Load; i++)
	{
		fscanf(fp, "%d %d %lf",
			   &Load_Node_Dir[(i + Total_Load_to_mesh[tm]) * 2 + 0],
			   &Load_Node_Dir[(i + Total_Load_to_mesh[tm]) * 2 + 1],
			   &Value_of_Load[i + Total_Load_to_mesh[tm]]);
		Load_Node_Dir[(i + Total_Load_to_mesh[tm]) * 2 + 0] = Load_Node_Dir[(i + Total_Load_to_mesh[tm]) * 2 + 0] + Total_Control_Point_to_mesh[tm];

		printf("Load_Node_Dir[%d]= %d Load_Node_Dir[%d]= %d Value_of_Load[%d] = %le\n",
			   (i + Total_Load_to_mesh[tm]) * 2 + 0, Load_Node_Dir[(i + Total_Load_to_mesh[tm]) * 2 + 0],
			   (i + Total_Load_to_mesh[tm]) * 2 + 1, Load_Node_Dir[(i + Total_Load_to_mesh[tm]) * 2 + 1],
			   i + Total_Load_to_mesh[tm], Value_of_Load[i + Total_Load_to_mesh[tm]]);
	}
	fgets(s, 256, fp);

	int type_load, iPatch, iCoord;
	double val_Coord, Range_Coord[2], Coeff_Dist_Load[3];

	for (i = 0; i < Total_DistributeForce; i++)
	{
		fscanf(fp, "%d %d %d %lf %lf %lf %lf %lf %lf", &type_load, &iPatch, &iCoord, &val_Coord, &Range_Coord[0], &Range_Coord[1], &Coeff_Dist_Load[0], &Coeff_Dist_Load[1], &Coeff_Dist_Load[2]);
		printf("Distibuted load nober: %d\n", i);
		printf("type_load: %d  iPatch: %d iCoord: %d  val_Coord: %.15e  Range_Coord: %.15e  %.15e\n Coef_Dist_Load: %.15e %.15e %.15e\n",
			   type_load, iPatch, iCoord, val_Coord, Range_Coord[0], Range_Coord[1], Coeff_Dist_Load[0], Coeff_Dist_Load[1], Coeff_Dist_Load[2]);

		// for s-IGA
		type_load_array[i + Total_DistributeForce_to_mesh[tm]] = type_load;
		iPatch_array[i + Total_DistributeForce_to_mesh[tm]] = iPatch;
		iCoord_array[i + Total_DistributeForce_to_mesh[tm]] = iCoord;
		val_Coord_array[i + Total_DistributeForce_to_mesh[tm]] = val_Coord;
		Range_Coord_array[(i + Total_DistributeForce_to_mesh[tm]) * 2 + 0] = Range_Coord[0];
		Range_Coord_array[(i + Total_DistributeForce_to_mesh[tm]) * 2 + 1] = Range_Coord[1];
		Coeff_Dist_Load_array[(i + Total_DistributeForce_to_mesh[tm]) * 3 + 0] = Coeff_Dist_Load[0];
		Coeff_Dist_Load_array[(i + Total_DistributeForce_to_mesh[tm]) * 3 + 1] = Coeff_Dist_Load[1];
		Coeff_Dist_Load_array[(i + Total_DistributeForce_to_mesh[tm]) * 3 + 2] = Coeff_Dist_Load[2];
	}
	fclose(fp);
}


// INC 等の作成
void Make_INC(int tm, int *Total_Patch_on_mesh, int *Total_Patch_to_mesh,
			  int *Total_Element_on_mesh, int *Total_Element_to_mesh,
			  int *Total_Control_Point_on_mesh, int *Total_Control_Point_to_mesh,
			  int *Total_DistributeForce_to_mesh,
			  int *INC, int *Patch_Control_point, int *Total_Control_Point_to_patch,
			  int *No_Control_point, int *Controlpoint_of_Element,
			  int *Element_patch, int *Element_mesh,
			  double *difference, int *Total_Knot_to_patch_dim,
			  int *Total_element_all_ID, int *ENC,
			  int *real_element_line, int *line_No_real_element,
			  int *line_No_Total_element, int *real_element,
			  int *real_El_No_on_mesh, int *real_Total_Element_on_mesh,
			  int *real_Total_Element_to_mesh, double *Equivalent_Nodal_Force,
			  int *type_load_array, int *iPatch_array, int *iCoord_array,
			  double *val_Coord_array, double *Range_Coord_array, double *Coeff_Dist_Load_array,
			  int *Order, int *No_knot, int *Total_Knot_to_mesh, double *Node_Coordinate,
			  double *Position_Knots, int *No_Control_point_ON_ELEMENT)
{
	// INC の計算(節点番号をξ, ηの番号で表す為の配列)
	for (tm = 0; tm < Total_mesh; tm++)
	{
		int b, B, e, h, i, j, k, l, n, p, q, x, y, ii, jj, kk, iii, kkk, iiloc, jjloc, kkloc, r = 0;
		int type_load, iPatch, iCoord;
		double val_Coord, Range_Coord[2], Coeff_Dist_Load[3];
		int No_Patch = Total_Patch_on_mesh[tm];
		int Total_Patch_to_Now = Total_Patch_to_mesh[tm];
		int Total_Element = Total_Element_on_mesh[tm];
		int Total_Element_to_Now = Total_Element_to_mesh[tm];
		int Total_Control_Point = Total_Control_Point_on_mesh[tm];
		int Total_DistributeForce = Total_DistributeForce_to_mesh[tm + 1] - Total_DistributeForce_to_mesh[tm];
		if (DIMENSION == 2) // for s-IGA
		{
			e = 0;
			for (l = 0; l < No_Patch; l++)
			{
				i = 0;
				for (jj = 0; jj < No_Control_point[(l + Total_Patch_to_Now) * DIMENSION + 1]; jj++)
				{
					for (ii = 0; ii < No_Control_point[(l + Total_Patch_to_Now) * DIMENSION + 0]; ii++)
					{
						INC[Patch_Control_point[Total_Control_Point_to_patch[l + Total_Patch_to_Now] + i] * DIMENSION + 0] = ii;
						INC[Patch_Control_point[Total_Control_Point_to_patch[l + Total_Patch_to_Now] + i] * DIMENSION + 1] = jj;

						if (ii >= Order[(l + Total_Patch_to_Now) * DIMENSION + 0] && jj >= Order[(l + Total_Patch_to_Now) * DIMENSION + 1])
						{

							for (jjloc = 0; jjloc <= Order[(l + Total_Patch_to_Now) * DIMENSION + 1]; jjloc++)
							{
								for (iiloc = 0; iiloc <= Order[(l + Total_Patch_to_Now) * DIMENSION + 0]; iiloc++)
								{
									B = Patch_Control_point[Total_Control_Point_to_patch[l + Total_Patch_to_Now] + i - jjloc * No_Control_point[(l + Total_Patch_to_Now) * DIMENSION + 0] - iiloc];
									b = jjloc * (Order[(l + Total_Patch_to_Now) * DIMENSION + 0] + 1) + iiloc;
									Controlpoint_of_Element[(e + Total_Element_to_Now) * MAX_NO_CCpoint_ON_ELEMENT + b] = B;
								}
							}
							Element_patch[e + Total_Element_to_Now] = l + Total_Patch_to_Now;
							Element_mesh[e + Total_Element_to_Now] = tm;
							e++;
						}
						i++;
					}
				}
			}
		}

		// 2次元のデバッグのための一時的なコメントアウト，3次元のプログラム作成時にはコメントアウトをはずす	
		// if (DIMENSION == 3)
		// {
		// 	e = 0;
		// 	for (l = 0; l < No_Patch; l++)
		// 	{
		// 		i = 0;
		// 		for (kk = 0; kk < No_Control_point[l][2]; kk++)
		// 		{
		// 			for (jj = 0; jj < No_Control_point[l][1]; jj++)
		// 			{
		// 				for (ii = 0; ii < No_Control_point[l][0]; ii++)
		// 				{
		// 					INC[l][Patch_Control_point[l][i]][0] = ii;
		// 					INC[l][Patch_Control_point[l][i]][1] = jj;
		// 					INC[l][Patch_Control_point[l][i]][2] = kk;
		// 					if (ii >= Order[l][0] && jj >= Order[l][1] && kk >= Order[l][2])
		// 					{
		// 						for (kkloc = 0; kkloc < Order[l][2]; kkloc++)
		// 						{
		// 							for (jjloc = 0; jjloc <= Order[l][1]; jjloc++)
		// 							{
		// 								for (iiloc = 0; iiloc <= Order[l][0]; iiloc++)
		// 								{
		// 									B = Patch_Control_point[l][i - jjloc * No_Control_point[l][0] - iiloc];
		// 									b = jjloc * (Order[l][0] + 1) + iiloc;
		// 									Controlpoint_of_Element[e * MAX_NO_CCpoint_ON_ELEMENT + b] = B;
		// 								}
		// 							}
		// 						}
		// 						Element_patch[e] = l + Total_Patch_to_Now;
		// 						e++;
		// 					}
		// 					i++;
		// 				}
		// 			}
		// 		}
		// 	}
		// }

		// for s-IGA line_No_real_elementの初期化
		for (l = 0; l < No_Patch; l++)
		{
			for (j = 0; j < DIMENSION; j++)
			{
				line_No_real_element[(l + Total_Patch_to_Now) * DIMENSION + j] = 0;
			}
		}

		for (l = 0; l < No_Patch; l++)
		{
			for (j = 0; j < DIMENSION; j++)
			{
				line_No_Total_element[(l + Total_Patch_to_Now) * DIMENSION + j] = No_knot[(l + Total_Patch_to_Now) * DIMENSION + j] - 2 * Order[(l + Total_Patch_to_Now) * DIMENSION + j] - 1;

				for (kkk = Order[(l + Total_Patch_to_Now) * DIMENSION + j]; kkk < No_knot[(l + Total_Patch_to_Now) * DIMENSION + j] - Order[(l + Total_Patch_to_Now) * DIMENSION + j] - 1; kkk++)
				{
					difference[Total_Knot_to_patch_dim[(l + Total_Patch_to_Now) * DIMENSION + j] + kkk - Order[(l + Total_Patch_to_Now) * DIMENSION + j]]
						= Position_Knots[Total_Knot_to_patch_dim[(l + Total_Patch_to_Now) * DIMENSION + j] + kkk + 1] - Position_Knots[Total_Knot_to_patch_dim[(l + Total_Patch_to_Now) * DIMENSION + j] + kkk];
					if (difference[Total_Knot_to_patch_dim[(l + Total_Patch_to_Now) * DIMENSION + j] + kkk - Order[(l + Total_Patch_to_Now) * DIMENSION + j]] != 0)
					{
						line_No_real_element[(l + Total_Patch_to_Now) * DIMENSION + j]++;
					}
				}
			}
		}

		// 要素に行番号, 列番号をつける
		if (DIMENSION == 2)
		{
			for (h = 0; h < Total_Element; h++)
			{
				Total_element_all_ID[h] = 0;
			}

			i = 0;
			for (l = 0; l < No_Patch; l++)
			{
				for (y = 0; y < line_No_Total_element[(l + Total_Patch_to_Now) * DIMENSION + 1]; y++)
				{
					for (x = 0; x < line_No_Total_element[(l + Total_Patch_to_Now) * DIMENSION + 0]; x++)
					{
						ENC[(i + Total_Element_to_mesh[tm]) * DIMENSION + 0] = x;
						ENC[(i + Total_Element_to_mesh[tm]) * DIMENSION + 1] = y;
						i++;
					}
				}
			}
		}

		// 必要な要素の行と列の番号を求める
		for (j = 0; j < DIMENSION; j++)
		{
			for (l = 0; l < No_Patch; l++)
			{
				e = 0;

				for (k = 0; k < line_No_Total_element[(l + Total_Patch_to_Now) * DIMENSION + j]; k++)
				{
					if (difference[Total_Knot_to_patch_dim[(l + Total_Patch_to_Now) * DIMENSION + j] + k] != 0)
					{
						real_element_line[(l + Total_Patch_to_Now) * (Total_Element_to_mesh[Total_mesh] * DIMENSION) + e * DIMENSION + j] = k;
						e++;
					}
				}
			}
		}

		// 必要な要素列上の要素のIDを1にする
		if (DIMENSION == 2)
		{
			for (n = 0; n < Total_Element; n++)
			{
				for (p = 0; p < line_No_real_element[(Element_patch[n + Total_Element_to_Now]) * DIMENSION + 0]; p++)
				{
					if (ENC[(n + Total_Element_to_mesh[tm]) * DIMENSION + 0] == real_element_line[(Element_patch[n + Total_Element_to_Now]) * (Total_Element_to_mesh[Total_mesh] * DIMENSION) + p * DIMENSION + 0])
					{
						for (q = 0; q < line_No_real_element[(Element_patch[n + Total_Element_to_Now]) * DIMENSION + 1]; q++)
						{
							if (ENC[(n + Total_Element_to_mesh[tm]) * DIMENSION + 1] == real_element_line[(Element_patch[n + Total_Element_to_Now]) * (Total_Element_to_mesh[Total_mesh] * DIMENSION) + q * DIMENSION + 1])
							{
								Total_element_all_ID[n]++;
							}
						}
					}
				}

				// IDが1の要素に番号を振る
				if (Total_element_all_ID[n] == 1)
				{
					real_element[r + real_Total_Element_to_mesh[tm]] = n + Total_Element_to_Now;
					real_El_No_on_mesh[tm * Total_Element_to_mesh[Total_mesh] + r] = n + Total_Element_to_Now;
					r++;
				}
			}

			// for s-IGA real_Total_Elementの初期化
			int real_Total_Element = 0;

			for (l = 0; l < No_Patch; l++)
			{
				real_Total_Element += line_No_real_element[(l + Total_Patch_to_Now) * DIMENSION + 0] * line_No_real_element[(l + Total_Patch_to_Now) * DIMENSION + 1];
			}
			real_Total_Element_on_mesh[tm] = real_Total_Element;
			real_Total_Element_to_mesh[tm + 1] = real_Total_Element_to_mesh[tm] + real_Total_Element;
		}

		for (i = 0; i < Total_DistributeForce; i++)
		{
			type_load = type_load_array[i + Total_DistributeForce_to_mesh[tm]];
			iPatch = iPatch_array[i + Total_DistributeForce_to_mesh[tm]];
			iCoord = iCoord_array[i + Total_DistributeForce_to_mesh[tm]];
			val_Coord = val_Coord_array[i + Total_DistributeForce_to_mesh[tm]];
			Range_Coord[0] = Range_Coord_array[(i + Total_DistributeForce_to_mesh[tm]) * 2 + 0];
			Range_Coord[1] = Range_Coord_array[(i + Total_DistributeForce_to_mesh[tm]) * 2 + 1];
			Coeff_Dist_Load[0] = Coeff_Dist_Load_array[(i + Total_DistributeForce_to_mesh[tm]) * 3 + 0];
			Coeff_Dist_Load[1] = Coeff_Dist_Load_array[(i + Total_DistributeForce_to_mesh[tm]) * 3 + 1];
			Coeff_Dist_Load[2] = Coeff_Dist_Load_array[(i + Total_DistributeForce_to_mesh[tm]) * 3 + 2];

			printf("type_load:%d\tiPatch:%d\tiCoord:%d\tval_coord:%lf\t", type_load, iPatch, iCoord, val_Coord);
			printf("Range0:%lf\tRange1:%lf\t", Range_Coord[0], Range_Coord[1]);
			printf("Coeff0:%lf\n", Coeff_Dist_Load[0]);
			Setting_Dist_Load_2D(tm, iPatch, Total_Element_to_mesh[tm + 1], iCoord, val_Coord,
								 Range_Coord, type_load, Coeff_Dist_Load, Total_Knot_to_mesh,
						  		 Controlpoint_of_Element, Order, No_knot, Total_element_all_ID,
						  		 Total_Knot_to_patch_dim, Position_Knots, Equivalent_Nodal_Force,
						  		 Total_Element_on_mesh, Total_Element_to_mesh, Element_patch, ENC,
						  		 Node_Coordinate, INC, No_Control_point_ON_ELEMENT, Total_Control_Point_to_mesh,
						  		 No_Control_point);
		}
	}
}


// Distributed Load
void Setting_Dist_Load_2D(int mesh_n, int iPatch, int Total_Element, int iCoord, double val_Coord,
						  double Range_Coord[2], int type_load, double Coeff_Dist_Load[3], int *Total_Knot_to_mesh,
						  int *Controlpoint_of_Element, int *Order, int *No_knot, int *Total_element_all_ID,
						  int *Total_Knot_to_patch_dim, double *Position_Knots, double *Equivalent_Nodal_Force,
						  int *Total_Element_on_mesh, int *Total_Element_to_mesh, int *Element_patch, int *ENC,
						  double *Node_Coordinate, int *INC, int *No_Control_point_ON_ELEMENT, int *Total_Control_Point_to_mesh,
						  int *No_Control_point)
{
	int iii, jjj;
	int N_Seg_Load_Element_iDir = 0, jCoord;
	int iRange_ele[2];
	int iPos[2] = {-10000, -10000}, jPos[2] = {-10000, -10000};
	int No_Element_For_Dist_Load;
	int iX, iY;
	int iControlpoint[MAX_NO_CCpoint_ON_ELEMENT], ic, ig, NNG = 3;
	double val_jCoord_Local = 0.0;
	double GaussPt[3], Weight[3];
	double Gg = pow(3.0 / 5.0, 0.5);

	double Position_Data_param[DIMENSION];
	int *No_Element_for_Integration = (int *)malloc(sizeof(int) * Total_Knot_to_mesh[Total_mesh]); // No_Element_for_Integration[MAX_N_KNOT]
 
	GaussPt[0] = -Gg;
	GaussPt[1] = 0.0;
	GaussPt[2] = Gg;
	Weight[0] = 5.0 / 9.0;
	Weight[1] = 8.0 / 9.0;
	Weight[2] = 5.0 / 9.0;

	// iCoord=0: Load on Eta=Constant
	// iCoord=1: Load on Xi=Constant
	if (iCoord == 0)
		jCoord = 1;
	if (iCoord == 1)
		jCoord = 0;

	// val_Coord: Value of Eta or Xi of the line or surface to give the distributed load
	// Setting elements needed to computed the distributed load

	for (iii = Order[iPatch * DIMENSION + iCoord]; iii < No_knot[iPatch * DIMENSION + iCoord] - Order[iPatch * DIMENSION + iCoord] - 1; iii++)
	{
		double epsi = 0.00000000001;

		if (Position_Knots[Total_Knot_to_patch_dim[iPatch * DIMENSION + iCoord] + iii] - epsi <= Range_Coord[0])
			iPos[0] = iii;
		if (Position_Knots[Total_Knot_to_patch_dim[iPatch * DIMENSION + iCoord] + iii + 1] - epsi <= Range_Coord[1])
			iPos[1] = iii + 1;
	}
	iRange_ele[0] = iPos[0] - Order[iPatch * DIMENSION + iCoord];
	iRange_ele[1] = iPos[1] - Order[iPatch * DIMENSION + iCoord] - 1;
	printf("iPos[0] = %d iPos[1] = %d\n", iPos[0], iPos[1]);
	printf("iRange_ele[0] = %d  iRange_ele[1] = %d\n", iRange_ele[0], iRange_ele[1]);

	if (iPos[0] < 0 || iPos[1] < 0)
	{
		printf("Error (Stop) iPos[0] = %d iPos[1] = %d\n", iPos[0], iPos[1]);
		exit(0);
	}

	for (jjj = Order[iPatch * DIMENSION + jCoord]; jjj < No_knot[iPatch * DIMENSION + jCoord] - Order[iPatch * DIMENSION + jCoord] - 1; jjj++)
	{
		double epsi = 0.00000000001;

		if (Position_Knots[Total_Knot_to_patch_dim[iPatch * DIMENSION + jCoord] + jjj] - epsi <= val_Coord
			&& Position_Knots[Total_Knot_to_patch_dim[iPatch * DIMENSION + jCoord] + jjj + 1] + epsi > val_Coord)
		{
			jPos[0] = jjj;
			jPos[1] = jjj + 1;
			val_jCoord_Local = -1.0 + 2.0 * (val_Coord - Position_Knots[Total_Knot_to_patch_dim[iPatch * DIMENSION + jCoord] + jjj])
							 / (Position_Knots[Total_Knot_to_patch_dim[iPatch * DIMENSION + jCoord] + jjj + 1] - Position_Knots[Total_Knot_to_patch_dim[iPatch * DIMENSION + jCoord] + jjj]);
		}
	}

	if (jPos[0] < 0 || jPos[1] < 0)
	{
		printf("Error (Stop) jPos[0] = %d jPos[1] = %d\n", jPos[0], jPos[1]);
		exit(0);
	}

	for (iii = iPos[0]; iii < iPos[1]; iii++)
	{
		N_Seg_Load_Element_iDir++;
	}

	iii = 0;
	if (iCoord == 1)
	{
		iX = jPos[0] - Order[iPatch * DIMENSION + 0];
		for (iY = iPos[0] - Order[iPatch * DIMENSION + 1]; iY < iPos[1] - Order[iPatch * DIMENSION + 1]; iY++)
		{
			No_Element_for_Integration[iii] = SearchForElement(mesh_n, iPatch, iX, iY, Total_Element_on_mesh, Total_Element_to_mesh, Element_patch, ENC);
			printf("Check No_Element_for_Integration[%d] = %d\n", iii, No_Element_for_Integration[iii]);
			iii++;
		}
	}

	if (iCoord == 0)
	{
		iY = jPos[0] - Order[iPatch * DIMENSION + 1];
		for (iX = iPos[0] - Order[iPatch * DIMENSION + 0]; iX < iPos[1] - Order[iPatch * DIMENSION + 0]; iX++)
		{
			No_Element_for_Integration[iii] = SearchForElement(mesh_n, iPatch, iX, iY, Total_Element_on_mesh, Total_Element_to_mesh, Element_patch, ENC);
			printf("Check No_Element_for_Integration[%d] = %d\n", iii, No_Element_for_Integration[iii]);
			iii++;
		}
	}
	No_Element_For_Dist_Load = iii;

	// Book keeping finished

	for (iii = 0; iii < No_Element_For_Dist_Load; iii++)
	{
		if (Total_element_all_ID[No_Element_for_Integration[iii]] == 1)
		{
			iX = ENC[No_Element_for_Integration[iii] * DIMENSION + 0];
			iY = ENC[No_Element_for_Integration[iii] * DIMENSION + 1];

			for (ic = 0; ic < (Order[iPatch * DIMENSION + 0] + 1) * (Order[iPatch * DIMENSION + 1] + 1); ic++)
				iControlpoint[ic] = Controlpoint_of_Element[No_Element_for_Integration[iii] * MAX_NO_CCpoint_ON_ELEMENT + ic];

			for (ig = 0; ig < NNG; ig++)
			{
				double Local_Coord[2], sfc, dxyzdge[3], detJ, XiEtaCoordParen, valDistLoad;
				int icc;
				Local_Coord[jCoord] = val_jCoord_Local;
				Local_Coord[iCoord] = GaussPt[ig];
				printf("ig = %d   Local_Coord[jCoord] = %f Local_Coord[iCoord] = %f\n", ig, Local_Coord[jCoord], Local_Coord[iCoord]);

				ShapeFunc_from_paren(Position_Data_param, Local_Coord, iCoord, No_Element_for_Integration[iii], INC, Position_Knots, Total_Knot_to_patch_dim, Controlpoint_of_Element, Element_patch);
				XiEtaCoordParen = Position_Data_param[iCoord];
				printf("Check  Coeff_Dist_Load[0] = %f Coeff_Dist_Load[1] = %f  Coeff_Dist_Load[2] = %f  Position_Data_param[iCoord] = %f\n", Coeff_Dist_Load[0], Coeff_Dist_Load[1], Coeff_Dist_Load[2], Position_Data_param[iCoord]);
				valDistLoad = Coeff_Dist_Load[0] + Coeff_Dist_Load[1] * XiEtaCoordParen + Coeff_Dist_Load[2] * XiEtaCoordParen * XiEtaCoordParen;

				dxyzdge[0] = 0.0;
				dxyzdge[1] = 0.0;
				dxyzdge[2] = 0.0;
				for (icc = 0; icc < (Order[iPatch * DIMENSION + 0] + 1) * (Order[iPatch * DIMENSION + 1] + 1); icc++)
				{
					dxyzdge[0] += dShape_func(icc, iCoord, Local_Coord, No_Element_for_Integration[iii], Controlpoint_of_Element, No_Control_point_ON_ELEMENT, Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC, Order, Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point)
								* Node_Coordinate[iControlpoint[icc] * (DIMENSION + 1) + 0];
					dxyzdge[1] += dShape_func(icc, iCoord, Local_Coord, No_Element_for_Integration[iii], Controlpoint_of_Element, No_Control_point_ON_ELEMENT, Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC, Order, Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point)
								* Node_Coordinate[iControlpoint[icc] * (DIMENSION + 1) + 1];
				}

				detJ = sqrt(dxyzdge[0] * dxyzdge[0] + dxyzdge[1] * dxyzdge[1]);
				printf("Check the value of detJ etc: detJ = %f dxyzdge[0] = %f dxyzdge[1] = %f\n", detJ, dxyzdge[0], dxyzdge[1]);
				if (type_load < 2)
				{
					for (ic = 0; ic < (Order[iPatch * DIMENSION + 0] + 1) * (Order[iPatch * DIMENSION + 1] + 1); ic++)
					{
						sfc = Shape_func(ic, Local_Coord, No_Element_for_Integration[iii], Node_Coordinate, Total_Control_Point_to_mesh, Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT, Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);
						Equivalent_Nodal_Force[iControlpoint[ic] * DIMENSION + type_load] += valDistLoad * sfc * detJ * Weight[ig];
					}
				}

				if (type_load == 2)
				{
					double LoadDir[2];
					LoadDir[0] = dxyzdge[1] / detJ;
					LoadDir[1] = -dxyzdge[0] / detJ;
					for (ic = 0; ic < (Order[iPatch * DIMENSION + 0] + 1) * (Order[iPatch * DIMENSION + 1] + 1); ic++)
					{
						sfc = Shape_func(ic, Local_Coord, No_Element_for_Integration[iii], Node_Coordinate, Total_Control_Point_to_mesh, Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT, Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);
						Equivalent_Nodal_Force[iControlpoint[ic] * DIMENSION + 0] += LoadDir[0] * valDistLoad * sfc * detJ * Weight[ig];
						Equivalent_Nodal_Force[iControlpoint[ic] * DIMENSION + 1] += LoadDir[1] * valDistLoad * sfc * detJ * Weight[ig];
						printf("Equivalent_Nodal_Force[%d][0]=%le\nEquivalent_Nodal_Force[%d][1]=%le\n",
							   iControlpoint[ic], Equivalent_Nodal_Force[iControlpoint[ic] * DIMENSION + 0],
							   iControlpoint[ic], Equivalent_Nodal_Force[iControlpoint[ic] * DIMENSION + 1]);
					}
				}
				if (type_load == 3)
				{
					double LoadDir[2];
					LoadDir[0] = dxyzdge[0] / detJ;
					LoadDir[1] = dxyzdge[1] / detJ;
					for (ic = 0; ic < (Order[iPatch * DIMENSION + 0] + 1) * (Order[iPatch * DIMENSION + 1] + 1); ic++)
					{
						sfc = Shape_func(ic, Local_Coord, No_Element_for_Integration[iii], Node_Coordinate, Total_Control_Point_to_mesh, Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT, Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);
						Equivalent_Nodal_Force[iControlpoint[ic] * DIMENSION + 0] += LoadDir[0] * valDistLoad * sfc * detJ * Weight[ig];
						Equivalent_Nodal_Force[iControlpoint[ic] * DIMENSION + 1] += LoadDir[1] * valDistLoad * sfc * detJ * Weight[ig];
					}
				}
			}
		}
	}
	free(No_Element_for_Integration);
}


int SearchForElement(int mesh_n, int iPatch, int iX, int iY, int *Total_Element_on_mesh, int *Total_Element_to_mesh, int *Element_patch, int *ENC)
{
	int iii;

	for (iii = 0; iii < Total_Element_on_mesh[mesh_n]; iii++)
	{
		if (Element_patch[iii + Total_Element_to_mesh[mesh_n]] == iPatch)
		{
			if (iX == ENC[(iii + Total_Element_to_mesh[mesh_n]) * DIMENSION + 0] && iY == ENC[(iii + Total_Element_to_mesh[mesh_n]) * DIMENSION + 1])
				goto loopend;
		}
	}
	loopend:

	return (iii);
}


// for s_IGA, coupled matrix を求める, 要素の重なりを要素のガウス点から求める
void Check_coupled_Glo_Loc_element_for_Gauss(int mesh_n_over, int mesh_n_org, int *NNLOVER, int *NELOVER,
											 double *Gauss_Coordinate, double *Gauss_Coordinate_ex, double *Jac, double *Jac_ex, double *B_Matrix, double *B_Matrix_ex, double *Loc_parameter_on_Glo, double *Loc_parameter_on_Glo_ex,
											 int *real_Total_Element_to_mesh, double *Node_Coordinate, int *Total_Control_Point_to_mesh, int *Controlpoint_of_Element,
											 int *INC, int *Element_patch, int *Order, int *No_Control_point_ON_ELEMENT, double *Position_Knots,
											 int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point,
											 double *Control_Coord_x, double *Control_Coord_y, double *Control_Weight,
											 int *real_Total_Element_on_mesh, int *real_element, int *Total_Patch_on_mesh, int *line_No_Total_element)
{
	int re, e;
	int i, j, k, m;
	int b, l, ll;
	int n_elements_over_point[POW_Ng_extended];
	int MAX_NNLOVER = 0;

	int *temp_element_n = (int *)malloc(sizeof(int) * MAX_N_ELEMENT_OVER_POINT);
	int *element_n_point = (int *)malloc(sizeof(int) * MAX_N_ELEMENT_OVER_ELEMENT);
	int *Check_coupled_No = (int *)malloc(sizeof(int) * MAX_N_ELEMENT_OVER);

	for (i = 0; i < MAX_N_ELEMENT_OVER; i++)
	{
		Check_coupled_No[i] = 0;
	}

	for (m = 0; m < 2; m++) // 最初 Ng 個のガウス点で重なりを求め, NNLOVER[e] >= 2 の e に対して, 再度 Ng_extended 個のガウス点で重なりを求める
	{
		Make_gauss_array(m);

		// グローバルパッチの Preprocessing 作成
		if (m == 0)
		{
			double AAA = 0.0;
			for (re = 0; re < real_Total_Element_on_mesh[mesh_n_org]; re++)
			{
				e = real_element[re + real_Total_Element_to_mesh[mesh_n_org]];
				Preprocessing(m, e, Gauss_Coordinate, Gauss_Coordinate_ex, B_Matrix, B_Matrix_ex,
							  Jac, Jac_ex, real_Total_Element_to_mesh, Node_Coordinate, Total_Control_Point_to_mesh,
							  Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT,
							  Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);
			}
		}

		// ローカルパッチ(mesh_n_over)各要素の頂点の物理座標算出
		for (re = 0; re < real_Total_Element_on_mesh[mesh_n_over]; re++)
		{
			int i_gg, i_ee;
			double output_para[DIMENSION];
			int Total_n_elements;

			e = real_element[re + real_Total_Element_to_mesh[mesh_n_over]];

			if (m == 0 || (m == 1 && NNLOVER[e] >= 2))
			{
				Preprocessing(m, e, Gauss_Coordinate, Gauss_Coordinate_ex, B_Matrix, B_Matrix_ex,
							  Jac, Jac_ex, real_Total_Element_to_mesh, Node_Coordinate, Total_Control_Point_to_mesh,
							  Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT,
							  Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);

				if (m == 1)
				{
					NNLOVER[e] = 0;
					for (i = 0; i < NNLOVER[e]; i++)
					{
						NELOVER[e * MAX_N_ELEMENT_OVER + i] = 0;
					}
				}

				Total_n_elements = 0;
				k = 0;
				ll = 0;

				// ローカルパッチ各要素のガウス点の物理座標のグローバルパッチでの(xi, eta)算出
				for (i = 0; i < Total_Patch_on_mesh[mesh_n_org]; i++)
				{
					// グローバルパッチ i での各方向ノットベクトル
					double *temp_Position_Knots_xi = (double *)malloc(sizeof(double) * No_knot[i * DIMENSION + 0]);
					double *temp_Position_Knots_eta = (double *)malloc(sizeof(double) * No_knot[i * DIMENSION + 1]);

					for (j = 0; j < No_knot[i * DIMENSION + 0]; j++)
					{
						temp_Position_Knots_xi[j] = Position_Knots[Total_Knot_to_patch_dim[i * DIMENSION + 0] + j];
					}
					for (j = 0; j < No_knot[i * DIMENSION + 1]; j++)
					{
						temp_Position_Knots_eta[j] = Position_Knots[Total_Knot_to_patch_dim[i * DIMENSION + 1] + j];
					}

					for (i_ee = 0; i_ee < GP_1dir; i_ee++)
					{
						for (i_gg = 0; i_gg < GP_1dir; i_gg++)
						{
							int g_n = i_ee * GP_1dir + i_gg;
							double data_result_shape[DIMENSION] = {0.0};

							for (l = 0; l < DIMENSION; l++)
							{
								if (m == 0)
								{
									data_result_shape[l] = Gauss_Coordinate[e * GP_2D * DIMENSION + g_n * DIMENSION + l];
								}
								else if (m == 1)
								{
									data_result_shape[l] = Gauss_Coordinate_ex[e * GP_2D * DIMENSION + g_n * DIMENSION + l];
								}
							}
							int itr_n = Calc_xi_eta(data_result_shape[0], data_result_shape[1],
													temp_Position_Knots_xi, temp_Position_Knots_eta,
													No_Control_point[i * DIMENSION + 0], No_Control_point[i * DIMENSION + 1], Order[i * DIMENSION + 0], Order[i * DIMENSION + 1],
													&output_para[0], &output_para[1],
													Position_Knots, Total_Knot_to_patch_dim,
													No_Control_point, Order, Control_Coord_x, Control_Coord_y, Control_Weight, No_knot);
							
							if (m == 0)
							{
								Loc_parameter_on_Glo[e * GP_2D * DIMENSION + g_n * DIMENSION + 0] = output_para[0];
								Loc_parameter_on_Glo[e * GP_2D * DIMENSION + g_n * DIMENSION + 1] = output_para[1];
							}
							else if (m == 1)
							{
								Loc_parameter_on_Glo_ex[e * GP_2D * DIMENSION + g_n * DIMENSION + 0] = output_para[0];
								Loc_parameter_on_Glo_ex[e * GP_2D * DIMENSION + g_n * DIMENSION + 1] = output_para[1];
							}

							// Newton Laphsonによって出力されたxi,etaから重なる要素を求める
							n_elements_over_point[k] = ele_check(i, output_para, No_Control_point, Position_Knots, Total_Knot_to_patch_dim, No_knot, Order, temp_element_n, line_No_Total_element);
							if (itr_n == 0) // data_result_shapeがグローバルメッシュ上にないとき
							{
								n_elements_over_point[k] = 0;
							}
							Total_n_elements += n_elements_over_point[k];
							for (l = 0; l < n_elements_over_point[k]; l++)
							{
								element_n_point[ll] = temp_element_n[l];
								ll++;
							}
							k++;
						}
					}
					free(temp_Position_Knots_xi), free(temp_Position_Knots_eta);
				}
				// 昇順ソート
				sort(Total_n_elements, element_n_point);
				// 重複削除
				NNLOVER[e] = duplicate_delete(Total_n_elements, e, NELOVER, element_n_point); // NNLOVER: 要素 e に重なる要素の総数
			}
		}
	}

	for (re = 0; re < real_Total_Element_on_mesh[mesh_n_over]; re++)
	{
		e = real_element[re + real_Total_Element_to_mesh[mesh_n_over]];

		Check_coupled_No[NNLOVER[e]]++;

		if (MAX_NNLOVER < NNLOVER[e])
		{
			MAX_NNLOVER = NNLOVER[e];
		}
	}

	// 重なっている要素の割合
	printf("MAX_NNLOVER = %d\n", MAX_NNLOVER);
	for (i = 0; i <= MAX_NNLOVER; i++)
	{
		double Percent_Check_coupled_No = (double)Check_coupled_No[i] * 100.0 / (double)real_Total_Element_on_mesh[mesh_n_over];
		printf("Check_coupled_No[%d] = %d\t\t%.2lf %%\n", i, Check_coupled_No[i], Percent_Check_coupled_No);
	}
	
	free(element_n_point), free(temp_element_n), free(Check_coupled_No);
}


void Make_Loc_Glo(int *real_Total_Element_on_mesh, int *real_Total_Element_to_mesh, int *real_element, int *NNLOVER, int *NELOVER)
{
	int i, j, k;
	int jj;
	int e;
	int count;

	int j_n = real_Total_Element_to_mesh[Total_mesh] - real_Total_Element_on_mesh[0];

	for (i = 0; i < real_Total_Element_on_mesh[0]; i++)
	{
		e = real_element[i];
		count = 0;

		for (j = 0; j < j_n; j++)
		{
			jj = real_element[real_Total_Element_to_mesh[1] + j]; //ローカルメッシュ上のreal element番号

			if (NNLOVER[jj] > 0)
			{
				for (k = 0; k < NNLOVER[jj]; k++)
				{
					if (NELOVER[jj * MAX_N_ELEMENT_OVER + k] == e)
					{
						NELOVER[e * MAX_N_ELEMENT_OVER + count] = jj;
						count++;
					}
				}
			}
		}
		NNLOVER[e] = count;
	}
}


// Newton Raphsonによって出力されたxi,etaから重なる要素を求める
int ele_check(int patch_n, double para_coord[DIMENSION], int *No_Control_point, double *Position_Knots, int *Total_Knot_to_patch_dim, int *No_knot, int *Order, int *temp_element_n, int *line_No_Total_element)
{
	int i, j, k, l;
	int kk, ll;
	int RangeCheck_flag;					// 要素を求め終えたら立てるフラグ
	int temp_ad[DIMENSION][MAX_ORDER + 1];	// 要素の位置を求めるための値
	int No_line[DIMENSION];					// xi, etaが含まれている要素の列数
	int n = 1;

	for (j = 0; j < DIMENSION; j++)
	{
		// 初期化
		l = 0;
		No_line[j] = 0;
		for (i = 0; i < MAX_ORDER + 1; i++)
		{
			temp_ad[j][i] = 0;
		}
		RangeCheck_flag = 0;

		for (k = 0; k < No_Control_point[patch_n * DIMENSION + j] - 1; k++)
		{
			if (RangeCheck_flag == 1)
				break;

			// Local要素の頂点がGlobalパッチ内にない場合
			if (para_coord[j] < Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + 0] || para_coord[j] > Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + No_knot[patch_n * DIMENSION + j] - 1])
			{
				RangeCheck_flag++;
			}

			// Local要素の頂点がGlobal要素内部にある場合
			if (para_coord[j] < Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + Order[patch_n * DIMENSION + j] + k])
			{
				int kk = 0;
				for (kk = 0; kk < k + 1; kk++)
				{
					if (para_coord[j] > Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + Order[patch_n * DIMENSION + j] + k - kk])
					{
						temp_ad[j][l] = k - kk;
						l++;
						RangeCheck_flag++;
						break;
					}
				}
			}

			// Local要素の頂点がGlobal要素境界上にある場合
			if (para_coord[j] == Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + Order[patch_n * DIMENSION + j] + k])
			{
				//頂点の座標がGlobalパッチの始点上にある場合
				if (para_coord[j] == Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + 0])
				{
					temp_ad[j][l] = k;
					l++;
					break;
				}
				//頂点の座標がGlobalパッチの終点上にある場合
				if (para_coord[j] == Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + No_knot[patch_n * DIMENSION + j] - 1])
				{
					temp_ad[j][l] = k - 1;
					l++;
					break;
				}
				//頂点の座標がGlobal要素境界上にある場合
				else
				{
					temp_ad[j][l] = k - 1;
					l++;
					temp_ad[j][l] = k;
					l++;
				}
				for (kk = 0; kk < Order[patch_n * DIMENSION + j]; kk++)
				{
					if (para_coord[j] == Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + Order[patch_n * DIMENSION + j] + k + kk + 1])
					// 多重ノット(次数分ループ)
					{
						printf("C0 continuity\n");
						temp_ad[j][l] = k + kk;
						l++;
					}
					if (para_coord[j] != Position_Knots[Total_Knot_to_patch_dim[patch_n * DIMENSION + j] + Order[patch_n * DIMENSION + j] + k + kk + 1])
						break;
				}
				RangeCheck_flag++;
			}
		}
		No_line[j] = l;
		
		// 各方向のNo_lineを掛け合わせる
		n *= l;
	}

	if (DIMENSION == 2)
	{
		for (l = 0; l < No_line[1]; l++)
		{
			for (ll = 0; ll < No_line[0]; ll++)
			{
				temp_element_n[l * No_line[0] + ll] = temp_ad[0][ll] + temp_ad[1][l] * line_No_Total_element[patch_n * DIMENSION + 0];
			}
		}
	}
	return n;
}


//昇順ソート
void sort(int total, int *element_n_point)
{
	int i, j;
	int temp;

	for (i = 0; i < total; i++)
	{
		for (j = i + 1; j < total; j++)
		{
			if (element_n_point[i] > element_n_point[j])
			{
				temp = element_n_point[i];
				element_n_point[i] = element_n_point[j];
				element_n_point[j] = temp;
			}
		}
	}
}


//重複削除
int duplicate_delete(int total, int element_n, int *NELOVER, int *element_n_point)
{
	int i, j;

	j = 0;
	NELOVER[element_n * MAX_N_ELEMENT_OVER + j] = element_n_point[0];
	j++;
	for (i = 1; i < total; i++)
	{
		if (element_n_point[i] != element_n_point[i - 1])
		{
			NELOVER[element_n * MAX_N_ELEMENT_OVER + j] = element_n_point[i];
			j++;
		}
	}
	// j = 要素element_nに重なる要素の総数
	return j;
}


// Preprocessing
void Preprocessing(int m, int e, double *Gauss_Coordinate, double *Gauss_Coordinate_ex, double *B_Matrix, double *B_Matrix_ex,
				   double *Jac, double *Jac_ex, int *real_Total_Element_to_mesh, double *Node_Coordinate, int *Total_Control_Point_to_mesh,
				   int *Controlpoint_of_Element, int *INC, int *Element_patch, int *Order, int *No_Control_point_ON_ELEMENT,
				   double *Position_Knots, int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point)
{
	double *a_matrix = (double *)malloc(sizeof(double) * POW_Ng_extended * DIMENSION * DIMENSION); // a_matrix[POW_Ng_extended * DIMENSION * DIMENSION]
	double *dSF = (double *)malloc(sizeof(double) * POW_Ng * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION); // dSF[POW_Ng * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION]
	double *dSF_ex = (double *)malloc(sizeof(double) * POW_Ng_extended * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION); // dSF_ex[POW_Ng_extended * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION]

	// ガウス点の物理座標を計算
	Make_Gauss_Coordinate(m, e, Gauss_Coordinate, Gauss_Coordinate_ex, Node_Coordinate, Total_Control_Point_to_mesh,
						  Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT,
						  Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);

	// ガウス点でのヤコビアン, Bマトリックスを計算
	Make_dSF(m, e, dSF, dSF_ex, Element_patch, No_Control_point_ON_ELEMENT,
			 Controlpoint_of_Element, Total_Control_Point_to_mesh, Node_Coordinate, INC, Order,
			 Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);
	Make_Jac(m, e, Jac, Jac_ex, dSF, dSF_ex, a_matrix, Node_Coordinate, Controlpoint_of_Element, No_Control_point_ON_ELEMENT, Element_patch);
	Make_B_Matrix(m, e, B_Matrix, B_Matrix_ex, dSF, dSF_ex, a_matrix, No_Control_point_ON_ELEMENT, Element_patch);

	free(a_matrix), free(dSF), free(dSF_ex);
}


void Make_Gauss_Coordinate(int m, int e, double *Gauss_Coordinate, double *Gauss_Coordinate_ex, double *Node_Coordinate, int *Total_Control_Point_to_mesh,
						   int *Controlpoint_of_Element, int *INC, int *Element_patch, int *Order, int *No_Control_point_ON_ELEMENT,
						   double *Position_Knots, int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point)
{
	int i, j, k;
	double temp_coordinate[DIMENSION];
	double R;

	for (i = 0; i < GP_2D; i++)
	{
		temp_coordinate[0] = Gxi[i][0];
		temp_coordinate[1] = Gxi[i][1];
		
		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[e]]; j++)
		{
			R = Shape_func(j, temp_coordinate, e, Node_Coordinate, Total_Control_Point_to_mesh,
						   Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT,
						   Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point);

			for (k = 0; k < DIMENSION; k++)
			{
				if (m == 0)
				{
					Gauss_Coordinate[e * GP_2D * DIMENSION + i * DIMENSION + k] += R * Node_Coordinate[Controlpoint_of_Element[e * MAX_NO_CCpoint_ON_ELEMENT + j] * (DIMENSION + 1) + k];
				}
				else if (m == 1)
				{
					Gauss_Coordinate_ex[e * GP_2D * DIMENSION + i * DIMENSION + k] += R * Node_Coordinate[Controlpoint_of_Element[e * MAX_NO_CCpoint_ON_ELEMENT + j] * (DIMENSION + 1) + k];
				}
			}
		}
	}
}


void Make_dSF(int m, int e, double *dSF, double *dSF_ex, int *Element_patch, int *No_Control_point_ON_ELEMENT,
			  int *Controlpoint_of_Element, int *Total_Control_Point_to_mesh, double *Node_Coordinate, int *INC, int *Order,
			  double *Position_Knots, int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point)
{
	int i, j, k;
	double temp_coordinate[DIMENSION];

	for (i = 0; i < GP_2D; i++)
	{
		temp_coordinate[0] = Gxi[i][0];
		temp_coordinate[1] = Gxi[i][1];

		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[e]]; j++)
		{
			for (k = 0; k < DIMENSION; k++)
			{
				if (m == 0)
				{
					dSF[i * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION + j * DIMENSION + k] = dShape_func(j, k, temp_coordinate, e, Controlpoint_of_Element, No_Control_point_ON_ELEMENT,
				   																					 Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC, Order, Position_Knots,
				   																					 Total_Knot_to_patch_dim, No_knot, No_Control_point);
				}
				else if (m == 1)
				{
					dSF_ex[i * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION + j * DIMENSION + k] = dShape_func(j, k, temp_coordinate, e, Controlpoint_of_Element, No_Control_point_ON_ELEMENT,
				   																						Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC, Order, Position_Knots,
				   																						Total_Knot_to_patch_dim, No_knot, No_Control_point);
				}
			}
		}
	}
}


void Make_Jac(int m, int e, double *Jac, double *Jac_ex, double *dSF, double *dSF_ex, double *a_matrix, double *Node_Coordinate, int *Controlpoint_of_Element, int *No_Control_point_ON_ELEMENT, int *Element_patch)
{
	int i, j, k, l;
	double J;
	double a[DIMENSION][DIMENSION];

	for (i = 0; i < GP_2D; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			for (k = 0; k < DIMENSION; k++)
			{
				a[j][k] = 0.0;
				for (l = 0; l < No_Control_point_ON_ELEMENT[Element_patch[e]]; l++)
				{
					if (m == 0)
					{
						a[j][k] += dSF[i * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION + l * DIMENSION + k] * Node_Coordinate[Controlpoint_of_Element[e * MAX_NO_CCpoint_ON_ELEMENT + l] * (DIMENSION + 1) + j];
					}
					else if (m == 1)
					{
						a[j][k] += dSF_ex[i * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION + l * DIMENSION + k] * Node_Coordinate[Controlpoint_of_Element[e * MAX_NO_CCpoint_ON_ELEMENT + l] * (DIMENSION + 1) + j];
					}
				}
			}
		}

		if (DIMENSION == 2)
		{
			J = InverseMatrix_2x2(a);
			
		}
		else if (DIMENSION == 3)
		{
			J = InverseMatrix_3x3(a);
		}

		if (m == 0)
		{
			Jac[e * GP_2D + i] = J;
		}
		else if (m == 1)
		{
			Jac_ex[e * GP_2D + i] = J;
		}

		for (j = 0; j < DIMENSION; j++)
		{
			for (k = 0; k < DIMENSION; k++)
			{
				a_matrix[i * DIMENSION * DIMENSION + j * DIMENSION + k] = a[j][k];
			}
		}

		if (J <= 0)
		{
			printf("Error, J <= 0\n");
			exit(1);
		}
	}
}


void Make_B_Matrix(int m, int e, double *B_Matrix, double *B_Matrix_ex, double *dSF, double *dSF_ex, double *a_matrix, int *No_Control_point_ON_ELEMENT, int *Element_patch)
{
	int i, j, k, l;
	double *b = (double *)malloc(sizeof(double) * DIMENSION * MAX_NO_CCpoint_ON_ELEMENT);	// b[DIMENSION][MAX_NO_CCpoint_ON_ELEMENT]

	for (i = 0; i < GP_2D; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			for (k = 0; k < No_Control_point_ON_ELEMENT[Element_patch[e]]; k++)
			{
				b[j * MAX_NO_CCpoint_ON_ELEMENT + k] = 0.0;
				for (l = 0; l < DIMENSION; l++)
				{
					if (m == 0)
					{
						b[j * MAX_NO_CCpoint_ON_ELEMENT + k] += a_matrix[i * DIMENSION * DIMENSION + l * DIMENSION + j] * dSF[i * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION + k * DIMENSION + l];
					}
					else if (m == 1)
					{
						b[j * MAX_NO_CCpoint_ON_ELEMENT + k] += a_matrix[i * DIMENSION * DIMENSION + l * DIMENSION + j] * dSF_ex[i * MAX_NO_CCpoint_ON_ELEMENT * DIMENSION + k * DIMENSION + l];
					}
				}
			}
		}

		// 2次元
		if (m == 0)
		{
			for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[e]]; j++)
			{
				B_Matrix[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 0 * MAX_KIEL_SIZE + (2 * j)] = b[0 * MAX_NO_CCpoint_ON_ELEMENT + j];
				B_Matrix[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 0 * MAX_KIEL_SIZE + (2 * j + 1)] = 0.0;
				B_Matrix[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 1 * MAX_KIEL_SIZE + (2 * j)] = 0.0;
				B_Matrix[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 1 * MAX_KIEL_SIZE + (2 * j + 1)] = b[1 * MAX_NO_CCpoint_ON_ELEMENT + j];
				B_Matrix[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 2 * MAX_KIEL_SIZE + (2 * j)] = b[1 * MAX_NO_CCpoint_ON_ELEMENT + j];
				B_Matrix[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 2 * MAX_KIEL_SIZE + (2 * j + 1)] = b[0 * MAX_NO_CCpoint_ON_ELEMENT + j];
			}
		}
		else if (m == 1)
		{
			for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[e]]; j++)
			{
				B_Matrix_ex[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 0 * MAX_KIEL_SIZE + (2 * j)] = b[0 * MAX_NO_CCpoint_ON_ELEMENT + j];
				B_Matrix_ex[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 0 * MAX_KIEL_SIZE + (2 * j + 1)] = 0.0;
				B_Matrix_ex[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 1 * MAX_KIEL_SIZE + (2 * j)] = 0.0;
				B_Matrix_ex[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 1 * MAX_KIEL_SIZE + (2 * j + 1)] = b[1 * MAX_NO_CCpoint_ON_ELEMENT + j];
				B_Matrix_ex[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 2 * MAX_KIEL_SIZE + (2 * j)] = b[1 * MAX_NO_CCpoint_ON_ELEMENT + j];
				B_Matrix_ex[e * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + 2 * MAX_KIEL_SIZE + (2 * j + 1)] = b[0 * MAX_NO_CCpoint_ON_ELEMENT + j];
			}
		}
	}
	free(b);
}


void Make_gauss_array(int select_GP)
{
	int i, j;

	if (select_GP == 0)
	{
		GP_1dir = Ng;
	}
	else if (select_GP == 1)
	{
		GP_1dir = Ng_extended;
	}

	GP_2D = GP_1dir * GP_1dir;

	if (GP_1dir == 3)
	{
		double G1 = pow((3.0 / 5.0), 0.5);
		double G_vec[3] = {-G1, 0.0, G1};
		double w1 = 8.0 / 9.0;
		double w2 = 5.0 / 9.0;
		double w_vec[3] = {w2, w1, w2};

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
	else if (GP_1dir == 4)
	{
		double A = pow((6.0 / 5.0), 0.5);
		double G1 = pow(((3.0 - 2.0 * A) / 7.0), 0.5);
		double G2 = pow(((3.0 + 2.0 * A) / 7.0), 0.5);
		double G_vec[4] = {-G2, -G1, G1, G2};
		double B = pow(30.0, 0.5);
		double w1 = (18.0 + B) / 36.0;
		double w2 = (18.0 - B) / 36.0;
		double w_vec[4] = {w2, w1, w1, w2};

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
	else if (GP_1dir == 5)
	{
		double A = pow((10.0 / 7.0), 0.5);
		double G1 = pow((5.0 - 2.0 * A), 0.5) / 3.0;
		double G2 = pow((5.0 + 2.0 * A), 0.5) / 3.0;
		double G_vec[5] = {-G2, -G1, 0.0, G1, G2};
		double B = pow(70.0, 0.5);
		double w1 = 128.0 / 225.0;
		double w2 = (322.0 + 13.0 * B) / 900.0;
		double w3 = (322.0 - 13.0 * B) / 900.0;
		double w_vec[5] = {w3, w2, w1, w2, w3};

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
	else if (GP_1dir == 10)
	{
		double G_vec[10];
		double w_vec[10];

		G_vec[0] = -0.9739065285171717;
		G_vec[1] = -0.8650633666889845;
		G_vec[2] = -0.6794095682990244;
		G_vec[3] = -0.4333953941292472;
		G_vec[4] = -0.1488743389816312;
		G_vec[5] = 0.1488743389816312;
		G_vec[6] = 0.4333953941292472;
		G_vec[7] = 0.6794095682990244;
		G_vec[8] = 0.8650633666889845;
		G_vec[9] = 0.9739065285171717;

		w_vec[0] = 0.0666713443086881;
		w_vec[1] = 0.1494513491505804;
		w_vec[2] = 0.2190863625159820;
		w_vec[3] = 0.2692667193099965;
		w_vec[4] = 0.2955242247147530;
		w_vec[5] = 0.2955242247147530;
		w_vec[6] = 0.2692667193099965;
		w_vec[7] = 0.2190863625159820;
		w_vec[8] = 0.1494513491505804;
		w_vec[9] = 0.0666713443086881;

		for (i = 0; i < GP_1dir; i++)
		{
			for (j = 0; j < GP_1dir; j++)
			{
				w[j + (GP_1dir * i)] = w_vec[i] * w_vec[j];
				Gxi[j + (GP_1dir * i)][0] = G_vec[j];
				Gxi[j + (GP_1dir * i)][1] = G_vec[i];
			}
		}
	}
}


// K matrix
// 応力歪マトリックス
void Make_D_Matrix(double *D)
{
	int i, j;

	// 2次元
	if (DIMENSION == 2)
	{
		if (DM == 0) // 平面応力状態
		{
			double Eone = E / (1.0 - nu * nu);
			double D1[3][3] = {{Eone, nu * Eone, 0}, {nu * Eone, Eone, 0}, {0, 0, (1 - nu) / 2 * Eone}};

			for (i = 0; i < D_MATRIX_SIZE; i++)
				for (j = 0; j < D_MATRIX_SIZE; j++)
					D[i * D_MATRIX_SIZE + j] = D1[i][j];
		}
		else if (DM == 1) // 平面ひずみ状態(2Dの場合はこっち)
		{
			double Eone = E * (1.0 - nu) / (1.0 + nu) / (1.0 - 2.0 * nu);
			double D1[3][3] = {{Eone, nu / (1.0 - nu) * Eone, 0}, {nu / (1.0 - nu) * Eone, Eone, 0}, {0, 0, (1 - 2 * nu) / 2 / (1.0 - nu) * Eone}};

			for (i = 0; i < D_MATRIX_SIZE; i++)
				for (j = 0; j < D_MATRIX_SIZE; j++)
					D[i * D_MATRIX_SIZE + j] = D1[i][j];
		}
	}
}


// 拘束されている行数を省いた行列の番号の制作
void Make_Index_Dof(int *Total_Control_Point_to_mesh, int *Total_Constraint_to_mesh, int *Constraint_Node_Dir, int *Index_Dof)
{
	int i, k = 0;

	// 拘束されている自由度(Degree Of free)をERRORにする
	for (i = 0; i < Total_Constraint_to_mesh[Total_mesh]; i++)
	{
		Index_Dof[Constraint_Node_Dir[i * 2 + 0] * DIMENSION + Constraint_Node_Dir[i * 2 + 1]] = ERROR;
	}
	// ERROR以外に番号を付ける
	for (i = 0; i < Total_Control_Point_to_mesh[Total_mesh] * DIMENSION; i++)
	{
		if (Index_Dof[i] != ERROR)
		{
			Index_Dof[i] = k;
			k++;
		}
	}

	K_Whole_Size = k;
	printf("K_Whole_Size = %d\n", k);
}


void Make_K_Whole_Ptr_Col(int *Total_Element_to_mesh, int *Total_Control_Point_to_mesh, int *Total_Control_Point_To_Node,
						  int *No_Control_point_ON_ELEMENT, int *Element_patch, int *Controlpoint_of_Element, int *Node_To_Node, int *NNLOVER,
						  int *NELOVER, int *Index_Dof, int *K_Whole_Ptr, int *K_Whole_Col)
{
	int i, j, k, ii, jj;
	int N, NE, i_index, j_index;

	// 初期化
	for (i = 0; i < K_Whole_Size + 1; i++)
		K_Whole_Ptr[i] = 0;

	// 大きく分割するためのループ
	for (N = 0; N < Total_Control_Point_to_mesh[Total_mesh]; N += K_DIVISION_LENGE)
	{
		// 各節点に接する節点を取得
		for (i = 0; i < K_DIVISION_LENGE; i++)
		{
			Total_Control_Point_To_Node[i] = 0;
		}
		for (i = 0; i < Total_Element_to_mesh[Total_mesh]; i++)
		{
			for (ii = 0; ii < No_Control_point_ON_ELEMENT[Element_patch[i]]; ii++)
			{
				NE = Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + ii] - N;
				if (0 <= NE && NE < K_DIVISION_LENGE)
				{
					for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[i]]; j++) //ローカル要素
					{
						// 数字がない時
						if (Total_Control_Point_To_Node[NE] == 0)
						{
							// 節点番号を取得
							Node_To_Node[NE * Total_Control_Point_to_mesh[Total_mesh] + 0] = Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + j];
							Total_Control_Point_To_Node[NE]++;
						}
						// 同じものがあったら
						// k > 0 以降の取得
						// kのカウント
						for (k = 0; k < Total_Control_Point_To_Node[NE]; k++)
						{
							if (Node_To_Node[NE * Total_Control_Point_to_mesh[Total_mesh] + k] == Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + j])
							{
								break;
							}
						}
						// 未設定のNode_To_Node取得
						if (k == Total_Control_Point_To_Node[NE])
						{
							Node_To_Node[NE * Total_Control_Point_to_mesh[Total_mesh] + k] = Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + j];
							Total_Control_Point_To_Node[NE]++;
						}
					}
					// 別メッシュとの重なりを考慮
					if (NNLOVER[i] > 0)
					{
						for (jj = 0; jj < NNLOVER[i]; jj++)
						{
							for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[NELOVER[i * MAX_N_ELEMENT_OVER + jj]]]; j++) //ローカル要素
							{
								// 数字がない時
								if (Total_Control_Point_To_Node[NE] == 0)
								{
									// 節点番号を取得
									Node_To_Node[NE * Total_Control_Point_to_mesh[Total_mesh] + 0] = Controlpoint_of_Element[NELOVER[i * MAX_N_ELEMENT_OVER + jj] * MAX_NO_CCpoint_ON_ELEMENT + j];
									Total_Control_Point_To_Node[NE]++;
								}

								// 同じものがあったら
								// k > 0 以降の取得
								// kのカウント
								for (k = 0; k < Total_Control_Point_To_Node[NE]; k++)
								{
									if (Node_To_Node[NE * Total_Control_Point_to_mesh[Total_mesh] + k] == Controlpoint_of_Element[NELOVER[i * MAX_N_ELEMENT_OVER + jj] * MAX_NO_CCpoint_ON_ELEMENT + j])
									{
										break;
									}
								}
								// 未設定のNode_To_Node取得
								if (k == Total_Control_Point_To_Node[NE])
								{
									Node_To_Node[NE * Total_Control_Point_to_mesh[Total_mesh] + k] = Controlpoint_of_Element[NELOVER[i * MAX_N_ELEMENT_OVER + jj] * MAX_NO_CCpoint_ON_ELEMENT + j];
									Total_Control_Point_To_Node[NE]++;
								}
							}
						}
					}
				}
			}
		}
		// 順番に並び替える
		for (i = 0; i < K_DIVISION_LENGE; i++)
		{
			if (N + i < Total_Control_Point_to_mesh[Total_mesh])
			{
				for (j = 0; j < Total_Control_Point_To_Node[i]; j++)
				{
					int Min = Node_To_Node[i * Total_Control_Point_to_mesh[Total_mesh] + j], No = j;
					for (k = j; k < Total_Control_Point_To_Node[i]; k++)
					{
						if (Min > Node_To_Node[i * Total_Control_Point_to_mesh[Total_mesh] + k])
						{
							Min = Node_To_Node[i * Total_Control_Point_to_mesh[Total_mesh] + k];
							No = k;
						}
					}
					for (k = No; k > j; k--)
					{
						Node_To_Node[i * Total_Control_Point_to_mesh[Total_mesh] + k] = Node_To_Node[i * Total_Control_Point_to_mesh[Total_mesh] + k - 1];
					}
					Node_To_Node[i * Total_Control_Point_to_mesh[Total_mesh] + j] = Min;
				}
			}
		}

		// 節点からcol ptrを求める
		ii = 0;
		k = 0;
		for (i = 0; i < K_DIVISION_LENGE; i++)
		{
			for (ii = 0; ii < DIMENSION; ii++)
			{
				if (N + i < Total_Control_Point_to_mesh[Total_mesh])
				{
					i_index = Index_Dof[(N + i) * DIMENSION + ii];
					k = 0;
					if (i_index >= 0)
					{
						K_Whole_Ptr[i_index + 1] = K_Whole_Ptr[i_index];
						for (j = 0; j < Total_Control_Point_To_Node[i]; j++)
						{
							for (jj = 0; jj < DIMENSION; jj++)
							{
								j_index = Index_Dof[Node_To_Node[i * Total_Control_Point_to_mesh[Total_mesh] + j] * DIMENSION + jj];
								if (j_index >= 0 && j_index >= i_index)
								{
									K_Whole_Ptr[i_index + 1]++;
									K_Whole_Col[K_Whole_Ptr[i_index] + k] = j_index;
									k++;
								}
							}
						}
					}
				}
			}
		}
	}
}


// valを求める
void Make_K_Whole_Val(int *real_Total_Element_to_mesh, int *real_element, int *Element_mesh, int *NNLOVER,
					  int *No_Control_point_ON_ELEMENT, int *Element_patch, double *Jac, double *Jac_ex, double *B_Matrix, double *B_Matrix_ex,
					  double *D, int *Index_Dof, int *Controlpoint_of_Element, int *K_Whole_Ptr, int *K_Whole_Col, double *K_Whole_Val, int *NELOVER,
					  double *Loc_parameter_on_Glo, double *Loc_parameter_on_Glo_ex, double *Position_Knots, int *Total_Knot_to_patch_dim,
					  int *Order, int *ENC, int *Total_Control_Point_to_mesh, double *Node_Coordinate, int *INC, int *No_knot, int *No_Control_point)
{
	int i, j, j1, j2, k1, k2, l;
	int a, b, re;

	double *K_EL = (double *)malloc(sizeof(double) * MAX_KIEL_SIZE * MAX_KIEL_SIZE);			// K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE]
	double *coupled_K_EL= (double *)malloc(sizeof(double) * MAX_KIEL_SIZE * MAX_KIEL_SIZE);		// coupled_K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE]

	for (re = 0; re < real_Total_Element_to_mesh[Total_mesh]; re++)
	{
		i = real_element[re];
		printf("element : %d\n\n", i);

		if (Element_mesh[i] == 0 && re == 0) // 2つめの条件は効率化のため
		{
			// GP_2D = POW_Ng;
			Make_gauss_array(0);
		}
		else if (Element_mesh[i] > 0)
		{
			if (NNLOVER[i] == 1 && (NNLOVER[real_element[re - 1]] != 1 || Element_mesh[real_element[re - 1]] == 0)) // 2つめ以降の条件は効率化のため
			{
				// GP_2D = POW_Ng;
				Make_gauss_array(0);
			}
			else if (NNLOVER[i] >= 2 && (NNLOVER[real_element[re - 1]] == 1 || Element_mesh[real_element[re - 1]] == 0)) // 2つめ以降の条件は効率化のため
			{
				// GP_2D = POW_Ng_extended;
				Make_gauss_array(1);
			}
		}

		// 各要素のK_ELを求める
		Make_K_EL(i, K_EL, No_Control_point_ON_ELEMENT, Element_patch, Jac, Jac_ex, B_Matrix, B_Matrix_ex, D);

		// Valを求める
		for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; j1++)
		{
			for (j2 = 0; j2 < DIMENSION; j2++)
			{
				a = Index_Dof[Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + j1] * DIMENSION + j2];
				if (a >= 0)
				{
					for (k1 = 0; k1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; k1++)
					{
						for (k2 = 0; k2 < DIMENSION; k2++)
						{
							b = Index_Dof[Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + k1] * DIMENSION + k2];
							if (b >= 0 && b >= a)
							{
								for (l = K_Whole_Ptr[a]; l < K_Whole_Ptr[a + 1]; l++)
								{
									if (K_Whole_Col[l] == b)
									{
										K_Whole_Val[l] += K_EL[(j1 * DIMENSION + j2) * MAX_KIEL_SIZE + k1 * DIMENSION + k2];
										break;
									}
								}
							}
						}
					}
				}
			}
		}

		if (Element_mesh[i] > 0 && NNLOVER[i] > 0) // ローカルメッシュ上の要素について, 重なっている要素が存在するとき
		{
			for (j = 0; j < NNLOVER[i]; j++)
			{
				// 各要素のcoupled_K_ELを求める
				Make_coupled_K_EL(i, NELOVER[i * MAX_N_ELEMENT_OVER + j], coupled_K_EL, No_Control_point_ON_ELEMENT, 
								  Jac, Jac_ex, B_Matrix, B_Matrix_ex, D,
								  Loc_parameter_on_Glo, Loc_parameter_on_Glo_ex, Position_Knots,
								  Total_Knot_to_patch_dim, Order, ENC, Controlpoint_of_Element,
								  Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC,
								  No_knot, No_Control_point);

				// Valを求める
				for (j1 = 0; j1 < No_Control_point_ON_ELEMENT[Element_patch[NELOVER[i * MAX_N_ELEMENT_OVER + j]]]; j1++)
				{
					for (j2 = 0; j2 < DIMENSION; j2++)
					{
						a = Index_Dof[Controlpoint_of_Element[NELOVER[i * MAX_N_ELEMENT_OVER + j] * MAX_NO_CCpoint_ON_ELEMENT + j1] * DIMENSION + j2];
						if (a >= 0)
						{
							for (k1 = 0; k1 < No_Control_point_ON_ELEMENT[Element_patch[i]]; k1++)
							{
								for (k2 = 0; k2 < DIMENSION; k2++)
								{
									b = Index_Dof[Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + k1] * DIMENSION + k2];
									if (b >= 0 && b >= a)
									{
										for (l = K_Whole_Ptr[a]; l < K_Whole_Ptr[a + 1]; l++)
										{
											if (K_Whole_Col[l] == b)
											{
												K_Whole_Val[l] += coupled_K_EL[(j1 * DIMENSION + j2) * MAX_KIEL_SIZE + k1 * DIMENSION + k2];
												break;
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}

	free(K_EL), free(coupled_K_EL);
}


// 要素合成マトリックス
void Make_K_EL(int El_No, double *K_EL, int *No_Control_point_ON_ELEMENT, int *Element_patch,
			   double *Jac, double *Jac_ex, double *B_Matrix, double *B_Matrix_ex, double *D)
{
	int i, j, k, l;
	
	int KIEL_SIZE = No_Control_point_ON_ELEMENT[Element_patch[El_No]] * DIMENSION;

	double *B = (double *)malloc(sizeof(double) * D_MATRIX_SIZE * MAX_KIEL_SIZE); // B[D_MATRIX_SIZE][MAX_KIEL_SIZE];
	double *K1 = (double *)malloc(sizeof(double) * MAX_KIEL_SIZE * MAX_KIEL_SIZE); // K1[MAX_KIEL_SIZE][MAX_KIEL_SIZE];
	double J;

	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			K_EL[i * MAX_KIEL_SIZE + j] = 0.0;
		}
	}

	for (i = 0; i < GP_2D; i++)
	{
		// J, B の作成
		if (GP_2D == POW_Ng)
		{
			J = Jac[El_No * GP_2D + i];
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				for (k = 0; k < MAX_KIEL_SIZE; k++)
				{
					B[j * MAX_KIEL_SIZE + k] = B_Matrix[El_No * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + j * MAX_KIEL_SIZE + k];
				}
			}
		}
		else if (GP_2D == POW_Ng_extended)
		{
			J = Jac_ex[El_No * GP_2D + i];
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				for (k = 0; k < MAX_KIEL_SIZE; k++)
				{
					B[j * MAX_KIEL_SIZE + k] = B_Matrix_ex[El_No * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + j * MAX_KIEL_SIZE + k];
				}
			}
		}

		BDBJ(KIEL_SIZE, B, D, J, K1);
		for (k = 0; k < KIEL_SIZE; k++)
		{
			for (l = 0; l < KIEL_SIZE; l++)
			{
				K_EL[k * MAX_KIEL_SIZE + l] += w[i] * K1[k * MAX_KIEL_SIZE + l];
			}
		}
	}

	free(B), free(K1);
}


// 結合要素剛性マトリックス
void Make_coupled_K_EL(int El_No_loc, int El_No_glo, double *coupled_K_EL, int *No_Control_point_ON_ELEMENT, 
					   double *Jac, double *Jac_ex, double *B_Matrix, double *B_Matrix_ex, double *D,
					   double *Loc_parameter_on_Glo, double *Loc_parameter_on_Glo_ex, double *Position_Knots,
					   int *Total_Knot_to_patch_dim, int *Order, int *ENC, int *Controlpoint_of_Element,
					   int *Total_Control_Point_to_mesh, int *Element_patch, double *Node_Coordinate, int *INC,
					   int *No_knot, int *No_Control_point)
{
	int i, j, jj, k, l;

	int BDBJ_flag;
	int KIEL_SIZE = No_Control_point_ON_ELEMENT[Element_patch[El_No_glo]] * DIMENSION;

	double *B = (double *)malloc(sizeof(double) * D_MATRIX_SIZE * MAX_KIEL_SIZE); // B[D_MATRIX_SIZE][MAX_KIEL_SIZE];
	double *BG = (double *)malloc(sizeof(double) * D_MATRIX_SIZE * MAX_KIEL_SIZE); // BG[D_MATRIX_SIZE][MAX_KIEL_SIZE];
	double *K1 = (double *)malloc(sizeof(double) * MAX_KIEL_SIZE * MAX_KIEL_SIZE); // K1[MAX_KIEL_SIZE][MAX_KIEL_SIZE];
	double J;

	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			coupled_K_EL[i * MAX_KIEL_SIZE + j] = 0.0;
		}
	}

	for (i = 0; i < GP_2D; i++)
	{
		// J, B, BG の作成
		double para[DIMENSION];
		double G_Gxi[DIMENSION];

		if (GP_2D == POW_Ng)
		{
			J = Jac[El_No_loc * GP_2D + i];
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				for (k = 0; k < MAX_KIEL_SIZE; k ++)
				{
					B[j * MAX_KIEL_SIZE + k] = B_Matrix[El_No_loc * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + j * MAX_KIEL_SIZE + k];
				}
			}
			para[0] = Loc_parameter_on_Glo[El_No_loc * GP_2D * DIMENSION + i * DIMENSION + 0];
			para[1] = Loc_parameter_on_Glo[El_No_loc * GP_2D * DIMENSION + i * DIMENSION + 1];
		}
		else if (GP_2D == POW_Ng_extended)
		{
			J = Jac_ex[El_No_loc * GP_2D + i];
			for (j = 0; j < D_MATRIX_SIZE; j++)
			{
				for (k = 0; k < MAX_KIEL_SIZE; k ++)
				{
					B[j * MAX_KIEL_SIZE + k] = B_Matrix_ex[El_No_loc * GP_2D * D_MATRIX_SIZE * MAX_KIEL_SIZE + i * D_MATRIX_SIZE * MAX_KIEL_SIZE + j * MAX_KIEL_SIZE + k];
				}
			}
			para[0] = Loc_parameter_on_Glo_ex[El_No_loc * GP_2D * DIMENSION + i * DIMENSION + 0];
			para[1] = Loc_parameter_on_Glo_ex[El_No_loc * GP_2D * DIMENSION + i * DIMENSION + 1];
		}

		// 要素内外判定
		if (para[0] >= Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 0] + Order[0 * DIMENSION + 0] + ENC[El_No_glo * DIMENSION + 0]] &&
			para[0] <  Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 0] + Order[0 * DIMENSION + 0] + ENC[El_No_glo * DIMENSION + 0] + 1] &&
			para[1] >= Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 1] + Order[0 * DIMENSION + 1] + ENC[El_No_glo * DIMENSION + 1]] &&
			para[1] <  Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 1] + Order[0 * DIMENSION + 1] + ENC[El_No_glo * DIMENSION + 1] + 1])
		{
			BDBJ_flag = 1;

			// 親要素座標の算出
			G_Gxi[0] = -1.0 + 2.0 * (para[0] - Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 0] + Order[0 * DIMENSION + 0] + ENC[El_No_glo * DIMENSION + 0]])
						/ (Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 0] + Order[0 * DIMENSION + 0] + ENC[El_No_glo * DIMENSION + 0] + 1] - Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 0] + Order[0 * DIMENSION + 0] + ENC[El_No_glo * DIMENSION + 0]]);
			G_Gxi[1] = -1.0 + 2.0 * (para[1] - Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 1] + Order[0 * DIMENSION + 1] + ENC[El_No_glo * DIMENSION + 1]])
						/ (Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 1] + Order[0 * DIMENSION + 1] + ENC[El_No_glo * DIMENSION + 1] + 1] - Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 1] + Order[0 * DIMENSION + 1] + ENC[El_No_glo * DIMENSION + 1]]);
		}
		else
		{
			BDBJ_flag = 0;
		}

		// 要素内であるとき, 結合要素剛性マトリックス計算
		if (BDBJ_flag)
		{
			// 重なるグローバル要素のBマトリックス
			Make_BG_Matrix(El_No_glo, BG, G_Gxi, Controlpoint_of_Element, No_Control_point_ON_ELEMENT,
						   Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC, Order, Position_Knots,
						   Total_Knot_to_patch_dim, No_knot, No_Control_point);
			// BGTDBLJの計算
			coupled_BDBJ(KIEL_SIZE, B, D, BG, J, K1);
			for (k = 0; k < KIEL_SIZE; k++)
			{
				for (l = 0; l < KIEL_SIZE; l++)
				{
					coupled_K_EL[k * MAX_KIEL_SIZE + l] += w[i] * K1[k * MAX_KIEL_SIZE + l];
				}
			}
		}
	}

	free(B), free(BG), free(K1);
}


// BGマトリックスを求める関数
void Make_BG_Matrix(int El_No, double *BG, double Local_coord[DIMENSION], int *Controlpoint_of_Element, int *No_Control_point_ON_ELEMENT,
					int *Total_Control_Point_to_mesh, int *Element_patch, double *Node_Coordinate, int *INC, int *Order, double *Position_Knots,
					int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point)
{
	double a[DIMENSION][DIMENSION];
	double *b = (double *)malloc(sizeof(double) * DIMENSION * MAX_NO_CCpoint_ON_ELEMENT); // b[DIMENSION][MAX_NO_CCpoint_ON_ELEMENT]

	int i, j, k;

	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			a[i][j] = 0.0;
			for (k = 0; k < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; k++)
			{
				a[i][j] += dShape_func(k, j, Local_coord, El_No, Controlpoint_of_Element, No_Control_point_ON_ELEMENT,
									   Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC, Order, Position_Knots,
									   Total_Knot_to_patch_dim, No_knot, No_Control_point)
						 * Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + k] * (DIMENSION + 1) + i];
			}
		}
	}

	InverseMatrix_2x2(a);

	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; j++)
		{
			b[i * MAX_NO_CCpoint_ON_ELEMENT + j] = 0.0;
			for (k = 0; k < DIMENSION; k++)
			{
				b[i * MAX_NO_CCpoint_ON_ELEMENT + j] += a[k][i]
													  * dShape_func(j, k, Local_coord, El_No, Controlpoint_of_Element, No_Control_point_ON_ELEMENT,
																	Total_Control_Point_to_mesh, Element_patch, Node_Coordinate, INC, Order, Position_Knots,
																	Total_Knot_to_patch_dim, No_knot, No_Control_point);
			}
		}
	}

	// 2次元
	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		BG[0 * MAX_KIEL_SIZE + 2 * i] = b[0 * MAX_NO_CCpoint_ON_ELEMENT + i];
		BG[0 * MAX_KIEL_SIZE + 2 * i + 1] = 0.0;
		BG[1 * MAX_KIEL_SIZE + 2 * i] = 0.0;
		BG[1 * MAX_KIEL_SIZE + 2 * i + 1] = b[1 * MAX_NO_CCpoint_ON_ELEMENT + i];
		BG[2 * MAX_KIEL_SIZE + 2 * i] = b[1 * MAX_NO_CCpoint_ON_ELEMENT + i];
		BG[2 * MAX_KIEL_SIZE + 2 * i + 1] = b[0 * MAX_NO_CCpoint_ON_ELEMENT + i];
	}

	free(b);
}


// ガウスの数値積分
void BDBJ(int KIEL_SIZE, double *B, double *D, double J, double *K_EL)
{
	int i, j, k;
	double *BD = (double *)calloc(MAX_KIEL_SIZE * D_MATRIX_SIZE, sizeof(double)); // BD[MAX_KIEL_SIZE][D_MATRIX_SIZE];

	// [B]T[D][B]Jの計算
	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < D_MATRIX_SIZE; j++)
		{
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				BD[i * D_MATRIX_SIZE + j] += B[k * MAX_KIEL_SIZE + i] * D[k * D_MATRIX_SIZE + j];
			}
		}
	}

	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			K_EL[i * MAX_KIEL_SIZE + j] = 0.0;
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				K_EL[i * MAX_KIEL_SIZE + j] += BD[i * D_MATRIX_SIZE + k] * B[k * MAX_KIEL_SIZE + j];
			}
			K_EL[i * MAX_KIEL_SIZE + j] *= J;
		}
	}

	free(BD);
}


// 結合ガウスの数値積分
void coupled_BDBJ(int KIEL_SIZE, double *B, double *D, double *BG, double J, double *K_EL)
{
	int i, j, k;
	double *BD = (double *)calloc(MAX_KIEL_SIZE * D_MATRIX_SIZE, sizeof(double)); // BD[MAX_KIEL_SIZE][D_MATRIX_SIZE];

	//[B]T[D][B]Jの計算
	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < D_MATRIX_SIZE; j++)
		{
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				BD[i * D_MATRIX_SIZE + j] += BG[k * MAX_KIEL_SIZE + i] * D[k * D_MATRIX_SIZE + j];
			}
		}
	}

	for (i = 0; i < KIEL_SIZE; i++)
	{
		for (j = 0; j < KIEL_SIZE; j++)
		{
			K_EL[i * MAX_KIEL_SIZE + j] = 0.0;
			for (k = 0; k < D_MATRIX_SIZE; k++)
			{
				K_EL[i * MAX_KIEL_SIZE + j] += BD[i * D_MATRIX_SIZE + k] * B[k * MAX_KIEL_SIZE + j];
			}
			K_EL[i * MAX_KIEL_SIZE + j] *= J;
		}
	}

	free(BD);
}


// F vector
// 荷重の行列を作る
void Make_F_Vec(double *rhs_vec, int *Total_Load_to_mesh, int *Index_Dof, int *Load_Node_Dir, double *Value_of_Load)
{
	int i, index;
	for (i = 0; i < K_Whole_Size; i++)
		rhs_vec[i] = 0.0;
	
	for (i = 0; i < Total_Load_to_mesh[Total_mesh]; i++)
	{
		index = Index_Dof[Load_Node_Dir[i * 2 + 0] * DIMENSION + Load_Node_Dir[i * 2 + 1]];
		if (index >= 0)
			rhs_vec[index] += Value_of_Load[i];
	}
}


// 強制変位対策
void Make_F_Vec_disp_const(int *Total_Constraint_to_mesh, int *real_Total_Element_to_mesh, int *No_Control_point_ON_ELEMENT, int *Element_patch,
						   int *Index_Dof, int *Controlpoint_of_Element, int *real_El_No_on_mesh, int *Total_Element_to_mesh, 
						   int *Constraint_Node_Dir, double *Value_of_Constraint, double *rhs_vec, int *real_element,
						   double *Jac, double *Jac_ex, double *B_Matrix, double *B_Matrix_ex, double *D)
{
	int ie, idir, inode, jdir, jnode, kk_const;
	int ii, iii, b, bb, jj, j1, j2, ii_local, jj_local;

	int i;

	double *K_EL = (double *)malloc(sizeof(double) * MAX_KIEL_SIZE * MAX_KIEL_SIZE);			// K_EL[MAX_KIEL_SIZE][MAX_KIEL_SIZE]

	Make_gauss_array(0);

	for (ie = 0; ie < real_Total_Element_to_mesh[Total_mesh]; ie++)
	{
		i = real_element[ie];
		int KIEL_SIZE = No_Control_point_ON_ELEMENT[Element_patch[i]] * DIMENSION;

		iii = 0;
		for (idir = 0; idir < DIMENSION; idir++)
		{
			for (inode = 0; inode < No_Control_point_ON_ELEMENT[Element_patch[i]]; inode++)
			{
				b = Index_Dof[Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + inode] * DIMENSION + idir];
				if (b < 0)
					iii++;
			}
		}

		if (iii > 0)
		{
			Make_K_EL(i, K_EL, No_Control_point_ON_ELEMENT, Element_patch, Jac, Jac_ex, B_Matrix, B_Matrix_ex, D);
			for (idir = 0; idir < DIMENSION; idir++)
			{
				for (inode = 0; inode < No_Control_point_ON_ELEMENT[Element_patch[i]]; inode++)
				{
					// ii = Controlpoint_of_Element[real_El_No_on_mesh[Total_mesh * Total_Element_to_mesh[Total_mesh] + ie] * MAX_NO_CCpoint_ON_ELEMENT + inode] * DIMENSION + idir;
					ii = Controlpoint_of_Element[0 + inode] * DIMENSION + idir;
					b = Index_Dof[ii];
					if (b >= 0)
					{
						ii_local = inode * DIMENSION + idir;
						for (jdir = 0; jdir < DIMENSION; jdir++)
						{
							for (jnode = 0; jnode < No_Control_point_ON_ELEMENT[Element_patch[i]]; jnode++)
							{
								jj = Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + jnode] * DIMENSION + jdir;
								bb = Index_Dof[jj];
								if (bb < 0)
								{
									jj_local = jnode * DIMENSION + jdir;
									for (kk_const = 0; kk_const < Total_Constraint_to_mesh[Total_mesh]; kk_const++)
									{
										if (Controlpoint_of_Element[i * MAX_NO_CCpoint_ON_ELEMENT + jnode] == Constraint_Node_Dir[kk_const * 2 + 0] && jdir == Constraint_Node_Dir[kk_const * 2 + 1])
										{
											rhs_vec[b] -= K_EL[ii_local * MAX_KIEL_SIZE + jj_local] * Value_of_Constraint[kk_const];
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
	free(K_EL);
}


// 分布荷重の等価節点力を足す
void Add_Equivalent_Nodal_Force_to_F_Vec(int *Total_Control_Point_to_mesh, int *Index_Dof, double *rhs_vec, double *Equivalent_Nodal_Force)
{
	int i, j, index;
	for (j = 0; j < DIMENSION; j++)
	{
		for (i = 0; i < Total_Control_Point_to_mesh[Total_mesh]; i++)
		{
			index = Index_Dof[i * DIMENSION + j];
			if (index >= 0)
			{
				rhs_vec[index] += Equivalent_Nodal_Force[i * DIMENSION + j];
			}
		}
	}
}



// PCG solver
// 前処理付共役勾配法により[K]{d}={f}を解く
void PCG_Solver(int max_itetarion, double eps, double *K_Whole_Val, int *K_Whole_Ptr, int *K_Whole_Col, double *sol_vec, double *rhs_vec, int *Total_Control_Point_on_mesh, int *Index_Dof)
{
	int i, j, k;
	int ndof = K_Whole_Size;

	double *r = (double *)malloc(sizeof(double) * ndof);
	double *p = (double *)calloc(ndof, sizeof(double));
	double *y = (double *)malloc(sizeof(double) * ndof);
	double *r2 = (double *)calloc(ndof, sizeof(double));

	// 初期化
	for (i = 0; i < ndof; i++)
		sol_vec[i] = 0.0;

	// 前処理行列作成
	double *M = (double *)calloc(MAX_NON_ZERO, sizeof(double));
	int *M_Ptr = (int *)calloc(MAX_K_WHOLE_SIZE + 1, sizeof(int));
	int *M_Col = (int *)calloc(MAX_NON_ZERO, sizeof(int));
	Make_M(M, M_Ptr, M_Col, ndof, Total_Control_Point_on_mesh, K_Whole_Val, K_Whole_Ptr, K_Whole_Col, Index_Dof);

	// 第0近似解に対する残差の計算
	double *ax = (double *)calloc(ndof, sizeof(double));
	for (i = 0; i < ndof; i++)
	{
		for (j = K_Whole_Ptr[i]; j < K_Whole_Ptr[i + 1]; j++)
		{
			ax[i] += K_Whole_Val[j] * sol_vec[K_Whole_Col[j]];
			if (i != K_Whole_Col[j])
			{
				ax[K_Whole_Col[j]] += K_Whole_Val[j] * sol_vec[i];
			}
		}
	}
	for (i = 0; i < ndof; i++)
	{
		r[i] = rhs_vec[i] - ax[i];
	}
	free(ax);

	// 第0近似解に対する残差の計算
	// for (i = 0; i < ndof; i++)
	// {
	// 	r[i] = rhs_vec[i];
	// }

	// p_0 = (LDL^T)^-1 r_0 の計算 <- CG法で M = [[K^G, 0], [0, K^L]] とし,p_0 = (LDL^T)^-1 r_0 = M^-1 r_0
	CG(ndof, p, M, M_Ptr, M_Col, r);

	// double rr0 = inner_product(ndof, r, p), rr1;
	double rr0;
	double alpha, beta;

	double e = 0.0;
	for (k = 0; k < max_itetarion; k++)
	{
		// rr0 の計算
		rr0 = inner_product(ndof, r, p);

		// y = AP の計算
		for (i = 0; i < ndof; i++)
		{
			double *temp_array_K = (double *)calloc(ndof, sizeof(double));
			for (j = 0; j < ndof; j++)
			{
				int temp1;
				if (i <= j)
				{
					temp1 = RowCol_to_icount(i, j, K_Whole_Ptr, K_Whole_Col); // temp_array_K[i][j]
				}
				else if (i > j)
				{
					temp1 = RowCol_to_icount(j, i, K_Whole_Ptr, K_Whole_Col); // temp_array_K[i][j] = temp_array_K[j][i]
				}

				if (temp1 != -1)
				{
					temp_array_K[j] = K_Whole_Val[temp1];
				}
			}
			y[i] = inner_product(ndof, temp_array_K, p);
			free(temp_array_K);
		}

		// alpha = r*r/(P*AP)の計算
		double temp_scaler = inner_product(ndof, p, y);
		alpha = rr0 / temp_scaler;
		// printf("alpha %le\n", alpha);

		// 解x, 残差rの更新
		for (i = 0; i < ndof; i++)
		{
			sol_vec[i] += alpha * p[i];
			r[i] -= alpha * y[i];
		}

		// (r*r)_(k+1)の計算
		CG(ndof, r2, M, M_Ptr, M_Col, r);

		// rr1 = inner_product(ndof, r, r2); // 旧
		// rr1 = inner_product(ndof, y, r2); // 新
		// printf("rr1 %le\n", rr1);

		// 収束判定 (||r||<=eps)
		// double rr1 = inner_product(ndof, y, r2);
		// e = sqrt(fabs(rr1));
		// if(e < eps)
		// {
		//     k++;
		//     break;
		// }

		// 収束判定 (CG法と同じ)
		double e1 = 0.0, e2 = 0.0;
		for (i = 0; i < ndof; i++)
		{
			e1 += p[i] * p[i];
			e2 += sol_vec[i] * sol_vec[i];
		}
		e = fabs(alpha) * sqrt(e1 / e2);
		if (e < eps)
		{
			k++;
			break;
		}

		// βの計算とPの更新
		// beta = rr1 / rr0; //旧
		// beta = - rr1 / temp_scaler; // 新
		beta = -inner_product(ndof, y, r2) / temp_scaler;

		for (i = 0; i < ndof; i++)
		{
			// p[i] = r2[i] - beta * p[i];
			p[i] = r2[i] + beta * p[i];
		}
		// printf("beta %le\n", beta);

		// (r*r)_(k+1)を次のステップのために確保しておく
		// rr0 = rr1;

		printf("itr %d\t", k);
		printf("eps %.15e", e);
		// if (rr1 < 0)
		// {
		// 	printf("\t rr1 < 0");
		// }
		printf("\n");
	}

	int max_itr_result = k;
	double eps_result = e;

	printf("\nndof = %d\n", ndof);
	printf("itr_result = %d\n", max_itr_result);
	printf("eps_result = %.15e\n", eps_result);

	free(r), free(p), free(y), free(r2);
	free(M), free(M_Ptr), free(M_Col);
}


void Make_M(double *M, int *M_Ptr, int *M_Col, int ndof, int *Total_Control_Point_on_mesh, double *K_Whole_Val, int *K_Whole_Ptr, int *K_Whole_Col, int *Index_Dof)
{
	int i, j;
	int ndof_glo = 0;

	// グローバルパッチのdofを求める
	for (i = 0; i < Total_Control_Point_on_mesh[0] * DIMENSION; i++)
	{
		if (Index_Dof[i] != ERROR)
		{
			ndof_glo++;
		}
	}
	printf("ndof		%d\n", ndof);
	printf("ndof_glo	%d\n", ndof_glo);

	int counter = 0;

	// M = [[K^G, 0], [0, K^L]] を作成
	M_Ptr[0] = 0;
	for (i = 0; i < ndof; i++)
	{
		M_Ptr[i + 1] = M_Ptr[i];

		for (j = K_Whole_Ptr[i]; j < K_Whole_Ptr[i + 1]; j++)
		{
			if (i < ndof_glo && K_Whole_Col[j] < ndof_glo)
			{
				M[counter] = K_Whole_Val[j];
				M_Col[counter] = K_Whole_Col[j];
				counter++;
				M_Ptr[i + 1]++;
			}
			else if (i >= ndof_glo)
			{
				M[counter] = K_Whole_Val[j];
				M_Col[counter] = K_Whole_Col[j];
				counter++;
				M_Ptr[i + 1]++;
			}
		}
	}
}


void CG(int ndof, double *solution_vec, double *M, int *M_Ptr, int *M_Col, double *right_vec)
{
	// CG solver
	double *gg = (double *)malloc(sizeof(double) * MAX_K_WHOLE_SIZE); // gg[MAX_K_WHOLE_SIZE]
	double *dd = (double *)malloc(sizeof(double) * MAX_K_WHOLE_SIZE); // dd[MAX_K_WHOLE_SIZE]
	double *pp = (double *)malloc(sizeof(double) * MAX_K_WHOLE_SIZE); // pp[MAX_K_WHOLE_SIZE]
	double qqq, ppp, rrr;
	double alphak, betak;
	int i, ii, itr, istop;
	int max_itr = ndof;
	double eps = 1.0e-13;

	for (i = 0; i < ndof; i++)
	{
		solution_vec[i] = 0.0;
	}
	M_mat_vec_crs(M, M_Ptr, M_Col, dd, solution_vec, ndof);
	for (i = 0; i < ndof; i++)
	{
		gg[i] = right_vec[i] - dd[i];
		pp[i] = gg[i];
	}
	for (itr = 0; itr < max_itr; itr++)
	{
		ppp = inner_product(ndof, gg, gg);
		M_mat_vec_crs(M, M_Ptr, M_Col, dd, pp, ndof);
		rrr = inner_product(ndof, dd, pp);
		alphak = ppp / rrr;
		for (ii = 0; ii < ndof; ii++)
		{
			solution_vec[ii] += alphak * pp[ii];
			gg[ii] -= alphak * dd[ii];
		}
		qqq = inner_product(ndof, gg, dd);
		betak = qqq / rrr;
		for (ii = 0; ii < ndof; ii++)
			pp[ii] = gg[ii] - betak * pp[ii];
		istop = M_check_conv_CG(ndof, alphak, pp, eps, solution_vec);
		if (istop == 1)
			break;
	}
	printf("\titr %d\n", itr);
	free(gg), free(dd), free(pp);
}


void M_mat_vec_crs(double *M, int *M_Ptr, int *M_Col, double *vec_result, double *vec, const int ndof)
{
	int i, j, icount = 0;

	for (i = 0; i < ndof; i++)
		vec_result[i] = 0;
	for (i = 0; i < ndof; i++)
	{
		for (j = M_Ptr[i]; j < M_Ptr[i + 1]; j++)
		{
			vec_result[i] += M[icount] * vec[M_Col[j]];
			if (i != M_Col[j])
				vec_result[M_Col[j]] += M[icount] * vec[i];
			icount++;
		}
	}
}


double inner_product(int ndof, double *vec1, double *vec2)
{
	double rrr = 0.0;
	int i;
	for (i = 0; i < ndof; i++)
	{
		rrr += vec1[i] * vec2[i];
	}
	return (rrr);
}


int M_check_conv_CG(int ndof, double alphak, double *pp, double eps, double *solution_vec)
{
	double rrr1 = 0.0, rrr2 = 0.0, rrr3;
	int i, istop = 0;
	for (i = 0; i < ndof; i++)
	{
		rrr1 += pp[i] * pp[i];
		rrr2 += solution_vec[i] * solution_vec[i];
	}
	rrr3 = fabs(alphak) * sqrt(rrr1 / rrr2);
	if (rrr3 < eps)
		istop = 1;
	return (istop);
}


int RowCol_to_icount(int row, int col, int *K_Whole_Ptr, int *K_Whole_Col)
{
	for (int j = K_Whole_Ptr[row]; j < K_Whole_Ptr[row + 1]; j++)
	{
		if (K_Whole_Col[j] == col)
		{
			return j;
		}
		else if (K_Whole_Col[j] > col)
		{
			return -1;
		}
	}
	return -1;
}


// tool
// 逆行列を元の行列に代入
double InverseMatrix_2x2(double M[DIMENSION][DIMENSION])
{
	int i, j;
	double a[2][2];
	double det = M[0][0] * M[1][1] - M[0][1] * M[1][0];

	if (det == 0)
		return ERROR;

	for (i = 0; i < 2; i++)
	{
		for (j = 0; j < 2; j++)
			a[i][j] = M[i][j];
	}
	M[0][0] = a[1][1] / det;
	M[0][1] = a[0][1] * (-1) / det;
	M[1][0] = a[1][0] * (-1) / det;
	M[1][1] = a[0][0] / det;

	return det;
}


double InverseMatrix_3x3(double M[DIMENSION][DIMENSION])
{
	int i, j;
	double a[3][3];
	double det = M[0][0] * M[1][1] * M[2][2] + M[1][0] * M[2][1] * M[0][2] + M[2][0] * M[0][1] * M[1][2] - M[0][0] * M[2][1] * M[1][2] - M[2][0] * M[1][1] * M[0][2] - M[1][0] * M[0][1] * M[2][2];

	if (det == 0)
		return ERROR;

	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
			a[i][j] = M[i][j];
	}
	M[0][0] = (a[1][1] * a[2][2] - a[1][2] * a[2][1]) / det;
	M[0][1] = (a[0][2] * a[2][1] - a[0][1] * a[2][2]) / det;
	M[0][2] = (a[0][1] * a[1][2] - a[0][2] * a[1][1]) / det;
	M[1][0] = (a[1][2] * a[2][0] - a[1][0] * a[2][2]) / det;
	M[1][1] = (a[0][0] * a[2][2] - a[0][2] * a[2][0]) / det;
	M[1][2] = (a[0][2] * a[1][0] - a[0][0] * a[1][2]) / det;
	M[2][0] = (a[1][0] * a[2][1] - a[1][1] * a[2][0]) / det;
	M[2][1] = (a[0][1] * a[2][0] - a[0][0] * a[2][1]) / det;
	M[2][2] = (a[0][0] * a[1][1] - a[0][1] * a[1][0]) / det;

	return det;
}


// Shape Function
double Shape_func(int I_No, double Local_coord[DIMENSION], int El_No, double *Node_Coordinate, int *Total_Control_Point_to_mesh,
				  int *Controlpoint_of_Element, int *INC, int *Element_patch, int *Order, int *No_Control_point_ON_ELEMENT,
				  double *Position_Knots, int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point)
{
	int i, j;
	double R, weight_func = 0.0;

	double Position_Data_param[DIMENSION];

	double *shape_func = (double *)malloc(sizeof(double) * MAX_NO_CCpoint_ON_ELEMENT);
	double *Shape = (double *)malloc(sizeof(double) * DIMENSION * Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER); // Shape[DIMENSION][MAX_N_NODE][10]
	double *dShape = (double *)malloc(sizeof(double) * DIMENSION * Total_Control_Point_to_mesh[Total_mesh]); // dShape[DIMENSION][MAX_N_NODE]

	for (i = 0; i < MAX_NO_CCpoint_ON_ELEMENT; i++)
	{
		shape_func[i] = 1.0;
	}

	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			ShapeFunc_from_paren(Position_Data_param, Local_coord, j, El_No, INC, Position_Knots, Total_Knot_to_patch_dim, Controlpoint_of_Element, Element_patch);
			ShapeFunction1D(Position_Data_param, j, El_No, Shape, dShape, No_knot, Total_Control_Point_to_mesh, Position_Knots, Total_Knot_to_patch_dim, Element_patch, Order, No_Control_point);
			shape_func[i] *= Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + j] * MAX_ORDER + Order[Element_patch[El_No] * DIMENSION + j]];
		}
		weight_func += shape_func[i] * Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * (DIMENSION + 1) + DIMENSION];
	}

	free(Shape), free(dShape);

	if (I_No < No_Control_point_ON_ELEMENT[Element_patch[El_No]])
		R = shape_func[I_No] * Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + I_No] * (DIMENSION + 1) + DIMENSION] / weight_func;

	else
		R = ERROR;

	free(shape_func);
	return R;
}


void ShapeFunc_from_paren(double Position_Data_param[DIMENSION], double Local_coord[DIMENSION], int j, int e, int *INC,
						  double *Position_Knots, int *Total_Knot_to_patch_dim, int *Controlpoint_of_Element, int *Element_patch)
{
	int i = 0;

	i = INC[Controlpoint_of_Element[e * MAX_NO_CCpoint_ON_ELEMENT + 0] * DIMENSION + j];
	Position_Data_param[j] = ((Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + i + 1] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + i]) * Local_coord[j] + (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + i + 1] + Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + i])) / 2.0;
}


void ShapeFunction1D(double Position_Data_param[DIMENSION], int j, int e, double *Shape, double *dShape, int *No_knot, int *Total_Control_Point_to_mesh,
					 double *Position_Knots, int *Total_Knot_to_patch_dim,int *Element_patch, int *Order, int *No_Control_point)
{
	int ii;
	int p;

	for (ii = 0; ii < No_knot[Element_patch[e] * DIMENSION + j]; ii++)
	{
		if (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii] == Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1])
		{
			Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + 0] = 0.0;
		}
		else if (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii] != Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1] && Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii] <= Position_Data_param[j] && Position_Data_param[j] < Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1])
		{
			Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + 0] = 1.0;
		}
		else if (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii] != Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1] && Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1] == Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + (No_knot[Element_patch[e] * DIMENSION + j] - 1)] && Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii] <= Position_Data_param[j] && Position_Data_param[j] <= Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1])
		{
			Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + 0] = 1.0;
		}
		else
		{
			Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + 0] = 0.0;
		}
	}

	for (ii = 0; ii < No_knot[Element_patch[e] * DIMENSION + j]; ii++)
	{
		for (p = 1; p <= Order[Element_patch[e] * DIMENSION + j]; p++)
		{
			Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + p] = 0.0;
		}
	}

	double left_term, right_term;
	for (p = 1; p <= Order[Element_patch[e] * DIMENSION + j]; p++)
	{
		for (ii = 0; ii < No_knot[Element_patch[e] * DIMENSION + j]; ii++)
		{
			left_term = 0.0;
			right_term = 0.0;

			if ((Position_Data_param[j] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii]) * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + p - 1] == 0 && Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + p] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii] == 0)
				left_term = 0.0;
			else
			{
				left_term = (Position_Data_param[j] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii]) / (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + p] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii]) * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + p - 1];
			}
			if ((Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + p + 1] - Position_Data_param[j]) * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + (ii + 1) * MAX_ORDER + p - 1] == 0 && Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + p + 1] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1] == 0)
				right_term = 0.0;
			else
			{
				right_term = (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + p + 1] - Position_Data_param[j]) / (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + p + 1] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1]) * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + (ii + 1) * MAX_ORDER + p - 1];
			}
			Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + p] = left_term + right_term;
		}
	}

	double dleft_term, dright_term;
	for (ii = 0; ii < No_Control_point[Element_patch[e] * DIMENSION + j] + 1; ii++)
	{
		dleft_term = 0.0;
		dright_term = 0.0;

		if (Order[Element_patch[e] * DIMENSION + j] * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + Order[Element_patch[e] * DIMENSION + j] - 1] == 0 && Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + Order[Element_patch[e] * DIMENSION + j]] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii] == 0)
			dleft_term = 0.0;
		else
			dleft_term = Order[Element_patch[e] * DIMENSION + j] / (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + Order[Element_patch[e] * DIMENSION + j]] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii]) * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + ii * MAX_ORDER + Order[Element_patch[e] * DIMENSION + j] - 1];

		if (Order[Element_patch[e] * DIMENSION + j] * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + (ii + 1) * MAX_ORDER + Order[Element_patch[e] * DIMENSION + j] - 1] == 0 && Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + Order[Element_patch[e] * DIMENSION + j] + 1] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1] == 0)
			dright_term = 0.0;
		else
			dright_term = Order[Element_patch[e] * DIMENSION + j] / (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + Order[Element_patch[e] * DIMENSION + j] + 1] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + ii + 1]) * Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + (ii + 1) * MAX_ORDER + Order[Element_patch[e] * DIMENSION + j] - 1];

		dShape[j * Total_Control_Point_to_mesh[Total_mesh] + ii] = dleft_term - dright_term;
	}
}


double dShape_func(int I_No, int xez, double Local_coord[DIMENSION], int El_No, int *Controlpoint_of_Element, int *No_Control_point_ON_ELEMENT,
				   int *Total_Control_Point_to_mesh, int *Element_patch, double *Node_Coordinate, int *INC, int *Order, double *Position_Knots,
				   int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point)
{
	double dR;

	double *dShape_func1 = (double *)malloc(sizeof(double) * Total_Control_Point_to_mesh[Total_mesh]); // dShape_func1[MAX_N_NODE];
	double *dShape_func2 = (double *)malloc(sizeof(double) * Total_Control_Point_to_mesh[Total_mesh]); // dShape_func2[MAX_N_NODE];

	NURBS_deriv(Local_coord, El_No, Node_Coordinate, Total_Control_Point_to_mesh, Controlpoint_of_Element, INC, Element_patch, Order, No_Control_point_ON_ELEMENT,
				Position_Knots, Total_Knot_to_patch_dim, No_knot, No_Control_point, dShape_func1, dShape_func2);

	if (xez != 0 && xez != 1)
		dR = ERROR;

	else if (I_No < No_Control_point_ON_ELEMENT[Element_patch[El_No]])
	{
		if (xez == 0)
		{
			dR = dShape_func1[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + I_No]]
			   * dShapeFunc_from_paren(xez, El_No, INC, Controlpoint_of_Element, Position_Knots, Total_Knot_to_patch_dim, Element_patch);
		}
		else if (xez == 1)
		{
			dR = dShape_func2[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + I_No]]
			   * dShapeFunc_from_paren(xez, El_No, INC, Controlpoint_of_Element, Position_Knots, Total_Knot_to_patch_dim, Element_patch);
		}
	}
	else
		dR = ERROR;

	free(dShape_func1), free(dShape_func2);
	return dR;
}


void NURBS_deriv(double Local_coord[DIMENSION], int El_No, double *Node_Coordinate, int *Total_Control_Point_to_mesh,
				 int *Controlpoint_of_Element, int *INC, int *Element_patch, int *Order, int *No_Control_point_ON_ELEMENT,
				 double *Position_Knots, int *Total_Knot_to_patch_dim, int *No_knot, int *No_Control_point,
				 double *dShape_func1, double *dShape_func2)
{
	int i, j;
	double weight_func = 0.0;
	double dWeight_func1 = 0.0;
	double dWeight_func2 = 0.0;

	double Position_Data_param[DIMENSION];

	double *shape_func = (double *)malloc(sizeof(double) * MAX_NO_CCpoint_ON_ELEMENT);
	double *Shape = (double *)malloc(sizeof(double) * DIMENSION * Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER); // Shape[DIMENSION][MAX_N_NODE][10]
	double *dShape = (double *)malloc(sizeof(double) * DIMENSION * Total_Control_Point_to_mesh[Total_mesh]); // dShape[DIMENSION][MAX_N_NODE]

	for (i = 0; i < MAX_NO_CCpoint_ON_ELEMENT; i++)
	{
		shape_func[i] = 1.0;
	}

	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			ShapeFunc_from_paren(Position_Data_param, Local_coord, j, El_No, INC, Position_Knots, Total_Knot_to_patch_dim, Controlpoint_of_Element, Element_patch);
			ShapeFunction1D(Position_Data_param, j, El_No, Shape, dShape, No_knot, Total_Control_Point_to_mesh, Position_Knots, Total_Knot_to_patch_dim, Element_patch, Order, No_Control_point);
			shape_func[i] *= Shape[j * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + j] * MAX_ORDER + Order[Element_patch[El_No] * DIMENSION + j]];
		}
		weight_func += shape_func[i] * Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * (DIMENSION + 1) + DIMENSION];
	}
	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		dWeight_func1 += dShape[0 * Total_Control_Point_to_mesh[Total_mesh] + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 0]] * Shape[1 * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 1] * MAX_ORDER + Order[Element_patch[El_No] * DIMENSION + 1]] * Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * (DIMENSION + 1) + DIMENSION];
		dWeight_func2 += Shape[0 * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 0] * MAX_ORDER + Order[Element_patch[El_No] * DIMENSION + 0]] * dShape[1 * Total_Control_Point_to_mesh[Total_mesh] + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 1]] * Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * (DIMENSION + 1) + DIMENSION];
	}
	for (i = 0; i < No_Control_point_ON_ELEMENT[Element_patch[El_No]]; i++)
	{
		dShape_func1[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i]] = Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * (DIMENSION + 1) + DIMENSION] * (weight_func * dShape[0 * Total_Control_Point_to_mesh[Total_mesh] + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 0]] * Shape[1 * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 1] * MAX_ORDER + Order[Element_patch[El_No] * DIMENSION + 1]] - dWeight_func1 * shape_func[i]) / (weight_func * weight_func);
		dShape_func2[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i]] = Node_Coordinate[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * (DIMENSION + 1) + DIMENSION] * (weight_func * Shape[0 * (Total_Control_Point_to_mesh[Total_mesh] * MAX_ORDER) + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 0] * MAX_ORDER + Order[Element_patch[El_No] * DIMENSION + 0]] * dShape[1 * Total_Control_Point_to_mesh[Total_mesh] + INC[Controlpoint_of_Element[El_No * MAX_NO_CCpoint_ON_ELEMENT + i] * DIMENSION + 1]] - dWeight_func2 * shape_func[i]) / (weight_func * weight_func);
	}

	free(Shape), free(dShape), free(shape_func);
}


double dShapeFunc_from_paren(int j, int e, int *INC, int *Controlpoint_of_Element, double *Position_Knots, int *Total_Knot_to_patch_dim, int *Element_patch)
{
	int i;
	double dPosition_Data_param;

	i = INC[Controlpoint_of_Element[e * MAX_NO_CCpoint_ON_ELEMENT + 0] * DIMENSION + j];

	dPosition_Data_param = (Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + i + 1] - Position_Knots[Total_Knot_to_patch_dim[Element_patch[e] * DIMENSION + j] + i]) / 2.0;
	return dPosition_Data_param;
}


// Newton-Raphson法, from NURBSviewer
double BasisFunc(double *knot_vec, int knot_index, int order, double xi, double *output, double *d_output)
{
	int p, j;
	double sum1 = 0.0;
	double sum2 = 0.0;
	double temp_basis[MAX_ORDER][MAX_ORDER];
	(*output) = 0.0;
	(*d_output) = 0.0;

	if (knot_vec[knot_index] <= xi && knot_vec[knot_index + order + 1] >= xi)
	{
		for (j = 0; j <= order; j++)
		{
			if ((knot_vec[knot_index + j] <= xi) && (xi <= knot_vec[knot_index + j + 1]))
			{
				temp_basis[j][0] = 1.0;
			}
			else
			{
				temp_basis[j][0] = 0.0;
			}
		}

		if (order > 0)
		{
			for (p = 1; p <= order; p++)
			{
				for (j = 0; j <= order - p; j++)
				{
					sum1 = 0.0;
					sum2 = 0.0;
					if ((knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) != 0.0)
					{
						sum1 = (xi - knot_vec[knot_index + j]) / (knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) * temp_basis[j][p - 1];
					}
					if ((knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) != 0.0)
					{
						sum2 = (knot_vec[knot_index + j + p + 1] - xi) / (knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) * temp_basis[j + 1][p - 1];
					}
					temp_basis[j][p] = sum1 + sum2;
				}
			}
			sum1 = 0.0;
			sum2 = 0.0;
			if ((knot_vec[knot_index + order] - knot_vec[knot_index]) != 0.0)
			{
				sum1 = order / (knot_vec[knot_index + order] - knot_vec[knot_index]) * temp_basis[0][order - 1];
			}
			if ((knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) != 0.0)
			{
				sum2 = order / (knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) * temp_basis[1][order - 1];
			}
		}
		(*output) = temp_basis[0][order];
		(*d_output) = sum1 - sum2;
	}
	return (*output);
}


double rBasisFunc(double *knot_vec, int knot_index, int order, double xi, double *output, double *d_output)
{
	int p, j;
	double sum1 = 0.0;
	double sum2 = 0.0;
	double temp_basis[MAX_ORDER][MAX_ORDER];
	(*output) = 0.0;
	(*d_output) = 0.0;

	if (knot_vec[knot_index] <= xi && xi <= knot_vec[knot_index + order + 1])
	{
		if (knot_index == 0)
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[j] <= xi) && (xi <= knot_vec[j + 1]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}
		else
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[knot_index + j] < xi) && (xi <= knot_vec[knot_index + j + 1]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}

		if (order > 0)
		{
			for (p = 1; p <= order; p++)
			{
				for (j = 0; j <= order - p; j++)
				{
					sum1 = 0.0;
					sum2 = 0.0;
					if ((knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) != 0.0)
					{
						sum1 = (xi - knot_vec[knot_index + j]) / (knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) * temp_basis[j][p - 1];
					}
					if ((knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) != 0.0)
					{
						sum2 = (knot_vec[knot_index + j + p + 1] - xi) / (knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) * temp_basis[j + 1][p - 1];
					}
					temp_basis[j][p] = sum1 + sum2;
				}
			}
			sum1 = 0.0;
			sum2 = 0.0;

			// for (int temp_i = 0; temp_i < No_knot[0][0]; temp_i++)
			// {
			// 	printf("knot_vec[%d] = %f\n", temp_i, knot_vec[temp_i]);
			// }

			if ((knot_vec[knot_index + order] - knot_vec[knot_index]) != 0.0)
			{
				sum1 = order / (knot_vec[knot_index + order] - knot_vec[knot_index]) * temp_basis[0][order - 1];
			}
			if ((knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) != 0.0)
			{
				sum2 = order / (knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) * temp_basis[1][order - 1];
			}
		}
		(*output) = temp_basis[0][order];
		(*d_output) = sum1 - sum2;
	}
	return (*output);
}


double lBasisFunc(double *knot_vec, int knot_index, int cntl_p_n, int order, double xi, double *output, double *d_output)
{
	int p, j;
	double sum1 = 0.0;
	double sum2 = 0.0;
	double temp_basis[MAX_ORDER][MAX_ORDER];
	(*output) = 0.0;
	(*d_output) = 0.0;

	if (knot_vec[knot_index] <= xi && xi <= knot_vec[knot_index + order + 1])
	{
		if (knot_index == cntl_p_n - 1)
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[cntl_p_n - 1 + j] <= xi) && (xi <= knot_vec[cntl_p_n + j]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}
		else
		{
			for (j = 0; j <= order; j++)
			{
				if ((knot_vec[knot_index + j] <= xi) && (xi < knot_vec[knot_index + j + 1]))
				{
					temp_basis[j][0] = 1.0;
				}
				else
				{
					temp_basis[j][0] = 0.0;
				}
			}
		}

		if (order > 0)
		{
			for (p = 1; p <= order; p++)
			{
				for (j = 0; j <= order - p; j++)
				{
					sum1 = 0.0;
					sum2 = 0.0;
					if ((knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) != 0.0)
					{
						sum1 = (xi - knot_vec[knot_index + j]) / (knot_vec[knot_index + j + p] - knot_vec[knot_index + j]) * temp_basis[j][p - 1];
					}
					if ((knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) != 0.0)
					{
						sum2 = (knot_vec[knot_index + j + p + 1] - xi) / (knot_vec[knot_index + j + p + 1] - knot_vec[knot_index + j + 1]) * temp_basis[j + 1][p - 1];
					}
					temp_basis[j][p] = sum1 + sum2;
				}
			}
			sum1 = 0.0;
			sum2 = 0.0;
			if ((knot_vec[knot_index + order] - knot_vec[knot_index]) != 0.0)
			{
				sum1 = order / (knot_vec[knot_index + order] - knot_vec[knot_index]) * temp_basis[0][order - 1];
			}
			if ((knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) != 0.0)
			{
				sum2 = order / (knot_vec[knot_index + order + 1] - knot_vec[knot_index + 1]) * temp_basis[1][order - 1];
			}
		}
		(*output) = temp_basis[0][order];
		(*d_output) = sum1 - sum2;
	}
	return (*output);
}


double NURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
					 double *cntl_px, double *cntl_py,
					 int cntl_p_n_xi, int cntl_p_n_eta,
					 double *weight, int order_xi, int order_eta,
					 double xi, double eta,
					 double *output_x, double *output_y,
					 double *output_dxi_x, double *output_deta_x,
					 double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	// int index_max_xi = cntl_p_n_xi; //2020_09_12
	int index_max_xi = cntl_p_n_xi - 1; // 2020_09_12
	int index_min_eta = 0;
	// int index_max_eta = cntl_p_n_eta; //2020_09_12
	int index_max_eta = cntl_p_n_eta - 1; // 2020_09_12

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] > xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi; // 2020_09_12

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] > eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta; // 2020_09_12

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		BasisFunc(input_knot_vec_xi, i, order_xi, xi,
				  &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			BasisFunc(input_knot_vec_eta, j, order_eta, eta,
					  &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}


double rNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
					  double *cntl_px, double *cntl_py,
					  int cntl_p_n_xi, int cntl_p_n_eta,
					  double *weight, int order_xi, int order_eta,
					  double xi, double eta,
					  double *output_x, double *output_y,
					  double *output_dxi_x, double *output_deta_x,
					  double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		rBasisFunc(input_knot_vec_xi, i, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			rBasisFunc(input_knot_vec_eta, j, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}


double lNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
					  double *cntl_px, double *cntl_py,
					  int cntl_p_n_xi, int cntl_p_n_eta,
					  double *weight, int order_xi, int order_eta,
					  double xi, double eta,
					  double *output_x, double *output_y,
					  double *output_dxi_x, double *output_deta_x,
					  double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		lBasisFunc(input_knot_vec_xi, i,
				   cntl_p_n_xi, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			lBasisFunc(input_knot_vec_eta, j,
					   cntl_p_n_eta, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}


double rlNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
					   double *cntl_px, double *cntl_py,
					   int cntl_p_n_xi, int cntl_p_n_eta,
					   double *weight, int order_xi, int order_eta,
					   double xi, double eta,
					   double *output_x, double *output_y,
					   double *output_dxi_x, double *output_deta_x,
					   double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		rBasisFunc(input_knot_vec_xi, i, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			lBasisFunc(input_knot_vec_eta, j,
					   cntl_p_n_eta, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}


double lrNURBS_surface(double *input_knot_vec_xi, double *input_knot_vec_eta,
					   double *cntl_px, double *cntl_py,
					   int cntl_p_n_xi, int cntl_p_n_eta,
					   double *weight, int order_xi, int order_eta,
					   double xi, double eta,
					   double *output_x, double *output_y,
					   double *output_dxi_x, double *output_deta_x,
					   double *output_dxi_y, double *output_deta_y)
{
	int i, j, temp_index;
	double temp1, temp2, temp3;
	double molecule_x, molecule_y;
	double dxi_molecule_x, dxi_molecule_y;
	double deta_molecule_x, deta_molecule_y;
	double denominator, dxi_denominator, deta_denominator;
	double temp_output_xi, temp_output_eta;
	double temp_d_output_xi, temp_d_output_eta;
	molecule_x = 0.0;
	molecule_y = 0.0;
	denominator = 0.0;
	dxi_molecule_x = 0.0;
	dxi_molecule_y = 0.0;
	dxi_denominator = 0.0;
	deta_molecule_x = 0.0;
	deta_molecule_y = 0.0;
	deta_denominator = 0.0;

	int index_min_xi = 0;
	int index_max_xi = cntl_p_n_xi - 1;
	int index_min_eta = 0;
	int index_max_eta = cntl_p_n_eta - 1;

	for (i = 0; i < cntl_p_n_xi; i++)
	{
		if (input_knot_vec_xi[i + 1] >= xi)
		{
			index_min_xi = i - order_xi;
			index_max_xi = i + 1;
			break;
		}
	}
	if (index_min_xi < 0)
		index_min_xi = 0;
	if (index_max_xi > cntl_p_n_xi)
		index_max_xi = cntl_p_n_xi;

	for (i = 0; i < cntl_p_n_eta; i++)
	{
		if (input_knot_vec_eta[i + 1] >= eta)
		{
			index_min_eta = i - order_eta;
			index_max_eta = i + 1;
			break;
		}
	}
	if (index_min_eta < 0)
		index_min_eta = 0;
	if (index_max_eta > cntl_p_n_eta)
		index_max_eta = cntl_p_n_eta;

	for (i = index_min_xi; i <= index_max_xi; i++)
	{
		lBasisFunc(input_knot_vec_xi, i,
				   cntl_p_n_xi, order_xi, xi,
				   &temp_output_xi, &temp_d_output_xi);
		for (j = index_min_eta; j <= index_max_eta; j++)
		{
			rBasisFunc(input_knot_vec_eta, j, order_eta, eta,
					   &temp_output_eta, &temp_d_output_eta);
			temp_index = i + j * cntl_p_n_xi;
			temp1 = temp_output_xi * temp_output_eta * weight[temp_index];
			temp2 = temp_d_output_xi * temp_output_eta * weight[temp_index];
			temp3 = temp_output_xi * temp_d_output_eta * weight[temp_index];
			molecule_x += temp1 * cntl_px[temp_index];
			molecule_y += temp1 * cntl_py[temp_index];
			denominator += temp1;
			dxi_molecule_x += temp2 * cntl_px[temp_index];
			dxi_molecule_y += temp2 * cntl_py[temp_index];
			dxi_denominator += temp2;
			deta_molecule_x += temp3 * cntl_px[temp_index];
			deta_molecule_y += temp3 * cntl_py[temp_index];
			deta_denominator += temp3;
		}
	}
	(*output_x) = molecule_x / denominator;
	(*output_y) = molecule_y / denominator;

	temp1 = denominator * denominator;
	(*output_dxi_x) = (dxi_molecule_x * denominator - molecule_x * dxi_denominator) / temp1;
	(*output_dxi_y) = (dxi_molecule_y * denominator - molecule_y * dxi_denominator) / temp1;
	(*output_deta_x) = (deta_molecule_x * denominator - molecule_x * deta_denominator) / temp1;
	(*output_deta_y) = (deta_molecule_y * denominator - molecule_y * deta_denominator) / temp1;
	return denominator;
}


//算出したローカルパッチ各要素の頂点の物理座標のグローバルパッチでの(xi,eta)算出
int Calc_xi_eta(double px, double py,
				double *input_knot_vec_xi, double *input_knot_vec_eta,
				int cntl_p_n_xi, int cntl_p_n_eta, int order_xi, int order_eta,
				double *output_xi, double *output_eta,
				double *Position_Knots, int *Total_Knot_to_patch_dim, int *No_Control_point, int *Order,
				double *Control_Coord_x, double *Control_Coord_y, double *Control_Weight, int *No_knot)
{
	double temp_xi, temp_eta;
	double temp_x, temp_y;
	double temp_matrix[2][2];
	double temp_dxi, temp_deta;
	double temp_tol_x, temp_tol_y;

	(*output_xi) = 0;
	(*output_eta) = 0;

	int i;
	// int repeat = 1000;
	// double tol = 10e-8;
	int repeat = 100;
	double tol = 10e-14;

	double *temp_Position_Knots_xi = (double *)malloc(sizeof(double) * No_knot[0 * DIMENSION + 0]);
	double *temp_Position_Knots_eta = (double *)malloc(sizeof(double) * No_knot[0 * DIMENSION + 1]);
	for (i = 0; i < No_knot[0 * DIMENSION + 0]; i++)
	{
		temp_Position_Knots_xi[i] = Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 0] + i];
	}
	for (i = 0; i < No_knot[0 * DIMENSION + 1]; i++)
	{
		temp_Position_Knots_eta[i] = Position_Knots[Total_Knot_to_patch_dim[0 * DIMENSION + 1] + i];
	}

	// 初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;

	for (i = 0; i < repeat; i++)
	{
		rNURBS_surface(temp_Position_Knots_xi, temp_Position_Knots_eta,
					   Control_Coord_x, Control_Coord_y,
					   No_Control_point[0 * DIMENSION + 0], No_Control_point[0 * DIMENSION + 1],
					   Control_Weight, Order[0 * DIMENSION + 0], Order[0 * DIMENSION + 1],
					   temp_xi, temp_eta,
					   &temp_x, &temp_y,
					   &temp_matrix[0][0], &temp_matrix[0][1],
					   &temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		// 収束した場合
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;
			return i;
		}

		InverseMatrix_2x2(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];
	}

	// 初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;

	for (i = 0; i < repeat; i++)
	{
		lNURBS_surface(temp_Position_Knots_xi, temp_Position_Knots_eta,
					   Control_Coord_x, Control_Coord_y,
					   No_Control_point[0 * DIMENSION + 0], No_Control_point[0 * DIMENSION + 1],
					   Control_Weight, Order[0 * DIMENSION + 0], Order[0 * DIMENSION + 1],
					   temp_xi, temp_eta,
					   &temp_x, &temp_y,
					   &temp_matrix[0][0], &temp_matrix[0][1],
					   &temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		// 収束した場合
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;
			return i;
		}

		InverseMatrix_2x2(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];
	}

	// 初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;

	for (i = 0; i < repeat; i++)
	{
		rlNURBS_surface(temp_Position_Knots_xi, temp_Position_Knots_eta,
						Control_Coord_x, Control_Coord_y,
						No_Control_point[0 * DIMENSION + 0], No_Control_point[0 * DIMENSION + 1],
						Control_Weight, Order[0 * DIMENSION + 0], Order[0 * DIMENSION + 1],
						temp_xi, temp_eta,
						&temp_x, &temp_y,
						&temp_matrix[0][0], &temp_matrix[0][1],
						&temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		// 収束した場合
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;
			return i;
		}

		InverseMatrix_2x2(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];
	}

	// 初期値の設定
	temp_xi = input_knot_vec_xi[0] + input_knot_vec_xi[cntl_p_n_xi + order_xi];
	temp_xi *= 0.5;
	temp_eta = input_knot_vec_eta[0] + input_knot_vec_eta[cntl_p_n_eta + order_eta];
	temp_eta *= 0.5;

	for (i = 0; i < repeat; i++)
	{
		lrNURBS_surface(temp_Position_Knots_xi, temp_Position_Knots_eta,
						Control_Coord_x, Control_Coord_y,
						No_Control_point[0 * DIMENSION + 0], No_Control_point[0 * DIMENSION + 1],
						Control_Weight, Order[0 * DIMENSION + 0], Order[0 * DIMENSION + 1],
						temp_xi, temp_eta,
						&temp_x, &temp_y,
						&temp_matrix[0][0], &temp_matrix[0][1],
						&temp_matrix[1][0], &temp_matrix[1][1]);

		temp_tol_x = px - temp_x;
		temp_tol_x *= temp_tol_x;
		temp_tol_y = py - temp_y;
		temp_tol_y *= temp_tol_y;

		// 収束した場合
		if (temp_tol_x + temp_tol_y < tol)
		{
			(*output_xi) = temp_xi;
			(*output_eta) = temp_eta;
			return i;
		}

		InverseMatrix_2x2(temp_matrix);

		temp_dxi = temp_matrix[0][0] * (px - temp_x) + temp_matrix[0][1] * (py - temp_y);
		temp_deta = temp_matrix[1][0] * (px - temp_x) + temp_matrix[1][1] * (py - temp_y);
		temp_xi = temp_xi + temp_dxi;
		temp_eta = temp_eta + temp_deta;
		if (temp_xi < input_knot_vec_xi[0])
			temp_xi = input_knot_vec_xi[0];
		if (temp_xi > input_knot_vec_xi[cntl_p_n_xi + order_xi])
			temp_xi = input_knot_vec_xi[cntl_p_n_xi + order_xi];
		if (temp_eta < input_knot_vec_eta[0])
			temp_eta = input_knot_vec_eta[0];
		if (temp_eta > input_knot_vec_eta[cntl_p_n_eta + order_eta])
			temp_eta = input_knot_vec_eta[cntl_p_n_eta + order_eta];
	}

	free(temp_Position_Knots_xi), free(temp_Position_Knots_eta);

	return 0;
}


// for s_IGA overlay


// output
void K_output_svg(int *K_Whole_Ptr, int *K_Whole_Col)
{
	// [K] = [[K^G, K^GL], [K^GL, K^L]]

	int i, j;
	int ndof = K_Whole_Size;

	char color_vec[2][10] = {"#f5f5f5", "#ee82ee"};
	// 0	whitesmoke
	// 1	violet
	// https://www.colordic.org/

	double space = 3.0, scale = 1000.0 / (((double)ndof) + 2.0 * space);

	double width = (((double)ndof) + 2.0 * space) * scale;
	double height = width;

	char str[256] = "K_matrix.svg";
	fp = fopen(str, "w");

	fprintf(fp, "<?xml version='1.0'?>\n");
	fprintf(fp, "<svg width='%le' height='%le' version='1.1' style='background: #eee' xmlns='http://www.w3.org/2000/svg' xmlns:xlink='http://www.w3.org/1999/xlink'>\n", width, height);

	double xx = space * scale;
	double yy = space * scale;
	double ww = ndof * scale;
	double hh = ndof * scale;
	fprintf(fp, "<rect x='%le' y='%le' width='%le' height='%le' fill='%s' />\n", xx, yy, ww, hh, color_vec[0]);

	// 各行の成分を抽出
	for (i = 0; i < ndof; i++)
	{
		int *K_bool = (int *)malloc(sizeof(int) * ndof); // 一行分保存する
		for (j = 0; j < ndof; j++)
		{
			K_bool[j] = 0;
		}

		for (j = 0; j < ndof; j++)
		{
			int temp_count;
			if (i <= j)
			{
				temp_count = RowCol_to_icount(i, j, K_Whole_Ptr, K_Whole_Col);
			}
			else if (i > j)
			{
				temp_count = RowCol_to_icount(j, i, K_Whole_Ptr, K_Whole_Col);
			}

			if (temp_count != -1)
			{
				K_bool[j] = 1;
			}
		}

		for (j = 0; j < ndof; j++)
		{
			double x = (((double)j) + space) * scale;
			double y = (((double)i) + space) * scale;
			// if (K_bool[j] == 0)
			// {
			// 	fprintf(fp, "<rect x='%le' y='%le' width='%le' height='%le' fill='%s' />\n", x, y, scale, scale, color_vec[0]);
			// }
			if (K_bool[j] == 1)
			{
				fprintf(fp, "<rect x='%le' y='%le' width='%le' height='%le' fill='%s' />\n", x, y, scale, scale, color_vec[1]);
			}
		}
		free(K_bool);
	}

	fprintf(fp, "</svg>");
	fclose(fp);
}

