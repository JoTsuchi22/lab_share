#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "myheader.h"
extern double ZETAWIN[K][K];

extern void test()
{
	int j;
	printf("%d\n", K);
	for (j = 0; j < K; j++)
	{
		for (int k = 0; k < K; k++)
		{
			ZETAWIN[j][k] = j * K + k;
			if (j == 3)
			printf("ZETAWIN[%d][%d] = %le\n", j, k, ZETAWIN[j][k]);
		}
	}
}